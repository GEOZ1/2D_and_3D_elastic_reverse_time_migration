#include "hip/hip_runtime.h"
__constant__ const int BDIMX3=32;
__constant__ const int BDIMY3=16;
__constant__ const int radius3=6;
__constant__ const float pai1=3.1415926;

__global__ void caculate_ex_amp_time(float *p_d,float *ex_amp_d,float *ex_time_d,int it,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;
			/*if(ex_amp_d[in_idx]<p_d[in_idx]) 
				{			
					ex_time_d[in_idx]=it;
					ex_amp_d[in_idx]=p_d[in_idx];///////emphasize  fabs(p_d[in_idx]);
				}*/
			if(fabs(ex_amp_d[in_idx])<fabs(p_d[in_idx]))	
				{
					ex_time_d[in_idx]=it;
					ex_amp_d[in_idx]=fabs(p_d[in_idx]);
				}
		}
}

__global__ void caculate_ex_amp_time_new(float *vxp_d,float *vzp_d,float *ex_amp_d,float *ex_time_d,int it,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

			if(fabs(ex_amp_d[in_idx])<sqrt(vxp_d[in_idx]*vxp_d[in_idx]+vzp_d[in_idx]*vzp_d[in_idx]))	
				{
					ex_time_d[in_idx]=it;
					ex_amp_d[in_idx]=sqrt(vxp_d[in_idx]*vxp_d[in_idx]+vzp_d[in_idx]*vzp_d[in_idx]);
				}
		}
}

__global__ void caculate_ex_tp_time_new(float *tp1_d,float *ex_amp_d,float *ex_tp_time_d,int it,int dimx,int dimz)
//caculate_ex_tp_time_new<<<dimGrid,dimBlock>>>(tp1_d,ex_amp_tp_d,ex_tp_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

			if(fabs(ex_amp_d[in_idx])<fabs(tp1_d[in_idx]))	
				{
					ex_tp_time_d[in_idx]=it;
					ex_amp_d[in_idx]=tp1_d[in_idx];
				}
		}
}

__global__ void caculate_ex_x_z(float *ex_amp_x_d,float *ex_amp_z_d,float *vxp_d,float *vzp_d,float *ex_time_d,int it,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				if(it==ex_time_d[in_idx])
				{
					ex_amp_x_d[in_idx]=vxp_d[in_idx];
					ex_amp_z_d[in_idx]=vzp_d[in_idx];
				}
		}
}

__global__ void caculate_ex_x_z_new(float *ex_amp_x_d,float *ex_amp_z_d,float *vxp_d,float *vzp_d,float *ex_time_d,int it,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				if(it==ex_time_d[in_idx])
				{
					ex_amp_x_d[in_idx]=vxp_d[in_idx];
					ex_amp_z_d[in_idx]=1.0*(vzp_d[in_idx+1]+vzp_d[in_idx-1]+vzp_d[in_idx+dimz]+vzp_d[in_idx-dimz])/4.0;
				}
		}
}

__global__ void caculate_ex_angle(float *ex_angle1_d,float *angle_pp_d,float *ex_time_d,int it,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				if(it==ex_time_d[in_idx])
					ex_angle1_d[in_idx]=angle_pp_d[in_idx];
					/*{	
						if(angle_pp_d[in_idx]>0)	ex_angle1_d[in_idx]=int(angle_pp_d[in_idx]+0.5);
						if(angle_pp_d[in_idx]<=0)	ex_angle1_d[in_idx]=int(angle_pp_d[in_idx]-0.5);
					}*/
		}
}

__global__ void caculate_ex_angle_pp_only_RTM(float *angle_pp_d,float *poyn_px_d,float *poyn_pz_d,float *ex_time_d,int it,int dimx,int dimz)
{
		__shared__ float s_data1[BDIMY3+2*radius3][BDIMX3+2*radius3];
		__shared__ float s_data2[BDIMY3+2*radius3][BDIMX3+2*radius3];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx=0.0,sumz=0.0;
		int m,n;
		
		int tx = threadIdx.x+radius3;
		int tz = threadIdx.y+radius3;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				__syncthreads();				

				s_data1[tz][tx]=poyn_px_d[in_idx];
				s_data2[tz][tx]=poyn_pz_d[in_idx];

				if(threadIdx.y<radius3)
				{
					s_data1[threadIdx.y][threadIdx.x]=poyn_px_d[in_idx-radius3-radius3*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius3]=poyn_px_d[in_idx-radius3+radius3*dimz];//up
					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_px_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_px_d[in_idx+BDIMY3+radius3*dimz];//down


					s_data2[threadIdx.y][threadIdx.x]=poyn_pz_d[in_idx-radius3-radius3*dimz];//up
					s_data2[threadIdx.y][threadIdx.x+2*radius3]=poyn_pz_d[in_idx-radius3+radius3*dimz];//up
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_pz_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_pz_d[in_idx+BDIMY3+radius3*dimz];//down

				}
				if(threadIdx.x<radius3)
				{
					s_data1[tz][threadIdx.x]=poyn_px_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data1[tz][threadIdx.x+BDIMX3+radius3]=poyn_px_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				
					s_data2[tz][threadIdx.x]=poyn_pz_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data2[tz][threadIdx.x+BDIMX3+radius3]=poyn_pz_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				}

				__syncthreads();
///note that  x/z
//least_square size of scale	 
				if(it==ex_time_d[in_idx])
				{ 
				
					for(m=-2;m<=+2;m++)
						for(n=-2;n<=+2;n++)
							{
								sumx=sumx+1.0*s_data1[tz+m][tx+n]*s_data2[tz+m][tx+n];

								sumz=sumz+1.0*s_data2[tz+m][tx+n]*s_data2[tz+m][tx+n];
							}
					if(sumz!=0)	angle_pp_d[in_idx]=float(atan(double(sumx*1.0/sumz)))*180/pai1;
				}
		}
}

__global__ void caculate_ex_angle_rp_only_RTM(float *angle_pp_d,float *poyn_px_d,float *poyn_pz_d,float *ex_time_d,int it,int dimx,int dimz)
//caculate_ex_angle_rp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_rpp_d,poyn_rpx_d,poyn_rpz_d,ex_time_d,it,nx_append_radius,nz_append_radius);
{
		__shared__ float s_data1[BDIMY3+2*radius3][BDIMX3+2*radius3];
		__shared__ float s_data2[BDIMY3+2*radius3][BDIMX3+2*radius3];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx=0.0,sumz=0.0;
		int m=0,n=0;
		
		int tx = threadIdx.x+radius3;
		int tz = threadIdx.y+radius3;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				__syncthreads();				

				s_data1[tz][tx]=poyn_px_d[in_idx];
				s_data2[tz][tx]=poyn_pz_d[in_idx];

				if(threadIdx.y<radius3)
				{
					s_data1[threadIdx.y][threadIdx.x]=poyn_px_d[in_idx-radius3-radius3*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius3]=poyn_px_d[in_idx-radius3+radius3*dimz];//up
					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_px_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_px_d[in_idx+BDIMY3+radius3*dimz];//down

					s_data2[threadIdx.y][threadIdx.x]=poyn_pz_d[in_idx-radius3-radius3*dimz];//up
					s_data2[threadIdx.y][threadIdx.x+2*radius3]=poyn_pz_d[in_idx-radius3+radius3*dimz];//up
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_pz_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_pz_d[in_idx+BDIMY3+radius3*dimz];//down

				}
				if(threadIdx.x<radius3)
				{
					s_data1[tz][threadIdx.x]=poyn_px_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data1[tz][threadIdx.x+BDIMX3+radius3]=poyn_px_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				
					s_data2[tz][threadIdx.x]=poyn_pz_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data2[tz][threadIdx.x+BDIMX3+radius3]=poyn_pz_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				}

				__syncthreads();
///note that  x/z
//least_square size of scale	 
				if(it==ex_time_d[in_idx])
				{ 
				
					for(m=-4;m<=+4;m++)
						for(n=-4;n<=+4;n++)
							{
								sumx=sumx+1.0*s_data1[tz+m][tx+n]*s_data2[tz+m][tx+n];

								sumz=sumz+1.0*s_data2[tz+m][tx+n]*s_data2[tz+m][tx+n];
							}
					if(sumz!=0)	angle_pp_d[in_idx]=float(atan(double(sumx*1.0/sumz)))*180/pai1;
				}
		}
}

__global__ void caculate_ex_angle_new(float *ex_angle_d,float *angle_pp_d,float *normal_angle_d,float *poyn_px_d,float *poyn_pz_d,float *ex_time_d,int it,int dimx,int dimz)
{
		__shared__ float s_data1[BDIMY3+2*radius3][BDIMX3+2*radius3];
		__shared__ float s_data2[BDIMY3+2*radius3][BDIMX3+2*radius3];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx=0.0,sumz=0.0;
		int m,n;
		
		int tx = threadIdx.x+radius3;
		int tz = threadIdx.y+radius3;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data1[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX3+2*radius3-1-threadIdx.x]=0.0;
		s_data2[BDIMY3+2*radius3-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				__syncthreads();				

				s_data1[tz][tx]=poyn_px_d[in_idx];
				s_data2[tz][tx]=poyn_pz_d[in_idx];

				if(threadIdx.y<radius3)
				{
					s_data1[threadIdx.y][threadIdx.x]=poyn_px_d[in_idx-radius3-radius3*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius3]=poyn_px_d[in_idx-radius3+radius3*dimz];//up

					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_px_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data1[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_px_d[in_idx+BDIMY3+radius3*dimz];//down

					s_data2[threadIdx.y][threadIdx.x]=poyn_pz_d[in_idx-radius3-radius3*dimz];//up
					s_data2[threadIdx.y][threadIdx.x+2*radius3]=poyn_pz_d[in_idx-radius3+radius3*dimz];//up
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x]=poyn_pz_d[in_idx+BDIMY3-radius3*dimz];//down
					s_data2[threadIdx.y+BDIMY3+radius3][threadIdx.x+2*radius3]=poyn_pz_d[in_idx+BDIMY3+radius3*dimz];//down

				}
				if(threadIdx.x<radius3)
				{
					s_data1[tz][threadIdx.x]=poyn_px_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data1[tz][threadIdx.x+BDIMX3+radius3]=poyn_px_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				
					s_data2[tz][threadIdx.x]=poyn_pz_d[in_idx-radius3*dimz];//g_input[in_idx-radius3];//left
					s_data2[tz][threadIdx.x+BDIMX3+radius3]=poyn_pz_d[in_idx+BDIMX3*dimz];//g_input[in_idx+BDIMX3];//right
				}

				__syncthreads();
///note that  x/z
//least_square size of scale	 
				if(it==ex_time_d[in_idx])
				{ 
				
					for(m=-2;m<=+2;m++)
						for(n=-2;n<=+2;n++)
							{
								sumx=sumx+1.0*s_data1[tz+m][tx+n]*s_data2[tz+m][tx+n];

								sumz=sumz+1.0*s_data2[tz+m][tx+n]*s_data2[tz+m][tx+n];
							}
					if(sumz!=0)	angle_pp_d[in_idx]=float(atan(double(sumx*1.0/sumz)))*180/pai1;	
					
					ex_angle_d[in_idx]=angle_pp_d[in_idx]-normal_angle_d[in_idx];

				}
		}
}

__global__ void imaging_ex(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rp_d,int it,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[in_idx])
			rimageup1_d[ix*nz+iz]=rp_d[in_idx]*1.0/fabs(ex_amp_d[in_idx]);
		}
}

__global__ void imaging_ex_for_xxzz(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rvxp1_d,float *rvzp1_d,int it,int source_x_cord,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left,float average)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int sign=1;

		//if(ix<nx&&iz<nz&&iz>50)
		if(ix<nx&&iz<nz&&iz>50&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(ex_amp_d[in_idx]<0)	sign=-1;

			if(it==ex_time_d[in_idx])
			{
				if(fabs(ex_amp_d[in_idx])>average)	rimageup1_d[ix*nz+iz]=(rvxp1_d[in_idx]+rvzp1_d[in_idx])*1.0/ex_amp_d[in_idx];
				
				if(fabs(ex_amp_d[in_idx])<=average)	rimageup1_d[ix*nz+iz]=(rvxp1_d[in_idx]+rvzp1_d[in_idx])*1.0/average*sign;
			}
		}
}	

__global__ void imaging_ex_new(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rp_d,int it,int source_x_cord,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		//if(ix<nx&&iz<nz&&iz>50)
		if(ix<nx&&iz<nz&&iz>50&&iz>1.2*(ix-source_x_cord)&&iz>1.2*(source_x_cord-ix))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[in_idx])
			rimageup1_d[ix*nz+iz]=rp_d[in_idx]*1.0/fabs(ex_amp_d[in_idx]);
		}
}

__global__ void imaging_ex_new_average(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rp_d,int it,int source_x_cord,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left,float average)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int sign=1;

		//if(ix<nx&&iz<nz&&iz>50)
		if(ix<nx&&iz<nz&&iz>50&&iz>1.2*(ix-source_x_cord)&&iz>1.2*(source_x_cord-ix))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(ex_amp_d[in_idx]<0)	sign=-1;

			if(it==ex_time_d[in_idx])
			{
				if(fabs(ex_amp_d[in_idx])>average)	rimageup1_d[ix*nz+iz]=rp_d[in_idx]*1.0/ex_amp_d[in_idx];
				
				if(fabs(ex_amp_d[in_idx])<=average)	rimageup1_d[ix*nz+iz]=rp_d[in_idx]*1.0/average*sign;
			}
		}
}

__global__ void imaging_ex_correlation(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rp_d,int it,int source_x_cord,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left)
//image_npp_xx_d,ex_amp1_x_d,ex_time1_d,rvxp1_d,it,source_x_cord[ishot]-receiver_x_cord[ishot],imaging_size[0],nz,nx_append,nz_append,boundary_up,boundary_left
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		//if(ix<nx&&iz<nz&&iz>50)
		if(ix<nx&&iz<nz&&iz>50&&iz>1.2*(ix-source_x_cord)&&iz>1.2*(source_x_cord-ix))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;

			if(it==ex_time_d[in_idx])
			{
				rimageup1_d[ix*nz+iz]=rp_d[in_idx]*1.0*ex_amp_d[in_idx];
			}
		}
}

__global__ void imaging_ex_ps(float *rimageup1_d,float *ex_amp_d,float *ex_time_d,float *rs_d,float *ex_angle_d,int it,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[in_idx])
				{
					if(ex_angle_d[in_idx]>0)	rimageup1_d[ix*nz+iz]=1.0*rs_d[in_idx]/fabs(ex_amp_d[in_idx]);
					if(ex_angle_d[in_idx]<=0)	rimageup1_d[ix*nz+iz]=-1.0*rs_d[in_idx]/fabs(ex_amp_d[in_idx]);
				}
		}
}

__global__ void imageing_after_correcting(float *rimageup1_d,float *ex_angle_d,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			
			if(ex_angle_d[in_idx]<=0)	rimageup1_d[ix*nz+iz]=-1*rimageup1_d[ix*nz+iz];
		}
}

__global__ void set_adcigs_for_ex(float *r_adcigs_pp_d,float *image_d,float *ex_angle_d,int source_x_cord,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left,int angle_num,int dangle)
//(r_adcigs_pp_d,image_pp_du_d,ex_angle_d,source_x_cord[ishot]-receiver_x_cord[ishot],imaging_size[0],nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int set;
		int r_angle;

		//if(ix<nx&&iz<nz&&iz>50)
		if(ix<nx&&iz<nz&&iz>50&&iz>1.2*(ix-source_x_cord)&&iz>1.2*(source_x_cord-ix))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;	
			
			r_angle=int((fabs(ex_angle_d[in_idx])+0.5)/dangle);

			set=r_angle*nx*nz+ix*nz+iz;

			if(r_angle<angle_num&&ex_angle_d[in_idx]>=0) r_adcigs_pp_d[angle_num*nx*nz+set]+=image_d[ix*nz+iz];//*exp(-(fabs(ex_angle_d[in_idx])-r_angle*dangle)*(fabs(ex_angle_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&ex_angle_d[in_idx]<0) r_adcigs_pp_d[(angle_num-r_angle)*nx*nz+ix*nz+iz]+=image_d[ix*nz+iz];//*exp(-(fabs(ex_angle_d[in_idx])-r_angle*dangle)*(fabs(ex_angle_d[in_idx])-r_angle*dangle)/8);
		}
}


__global__ void set_value_a_to_b(float *angle_pp_d,float *angle_pp1_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			dimx=dimx+2*radius;dimz=dimz+2*radius;
			ix=ix+radius;iz=iz+radius;
			in_idx=ix*dimz+iz;

			angle_pp1_d[in_idx]=angle_pp_d[in_idx];
		}
}

__global__ void sum_image1_and_image2(float *image1_d,float *image2_d,float *image3_d,int nx,int nz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=ix*nz+iz;

			image1_d[in_idx]=image2_d[in_idx]+image3_d[in_idx];
		}
}
__global__ void imaging_correlation_ex_2D(float *ex_resulttp_d,float *ex_amp_d,float *ex_tp_time_d,float *rtp2_d,int nx,int nz,int dimz,int boundary_up,int boundary_left,float *max,int it)
//imaging_correlation_ex_2D<<<dimGrid,dimBlock>>>(ex_result_tp_d,ex_amp_tp_d,ex_tp_time_d,rtp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[0],it);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_tp_time_d[in_idx])
			{
				ex_resulttp_d[ix*nz+iz]=1.0*rtp2_d[in_idx]/ex_amp_d[in_idx];
				//ex_resulttp_d[ix*nz+iz]=1.0*rtp2_d[in_idx]/(ex_amp_d[in_idx]+0.00000000001*(*max));
			}
		}
}
__global__ void imaging_inner_product_ex_2D(float *ex_vresult_pp_d,float *ex_amp_d,float *ex_amp_x_d,float *ex_amp_z_d,float *ex_time_d,float *rvxp2_d,float *rvzp2_d,int nx,int nz,int dimz,int boundary_up,int boundary_left,float *max,int it)
///imaging_inner_product_ex_2D<<<dimGrid,dimBlock>>>(ex_vresultpp_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxp2_d,rvzp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[idx])
			{
				float molecular=ex_amp_x_d[idx]*rvxp2_d[idx]+ex_amp_z_d[idx]*rvzp2_d[idx];

				float denominator=ex_amp_x_d[idx]*ex_amp_x_d[idx]+ex_amp_z_d[idx]*ex_amp_z_d[idx];

				ex_vresult_pp_d[ix*nz+iz]=molecular*1.0/(denominator);
				//ex_vresult_pp_d[ix*nz+iz]=molecular*1.0/(denominator+0.0000000000001*(*max));
				
			}
		}
}

__global__ void imaging_inner_product_ex_2D_new(float *ex_vresult_pp_d,float *ex_amp_d,float *ex_amp_x_d,float *ex_amp_z_d,float *ex_time_d,float *rvxp2_d,float *rvzp2_d,int nx,int nz,int dimz,int boundary_up,int boundary_left,float *max,int it)
///imaging_inner_product_ex_2D<<<dimGrid,dimBlock>>>(ex_vresultpp_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxp2_d,rvzp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[idx])
			{
				float change_rpz=1.0*(rvzp2_d[idx+1]+rvzp2_d[idx-1]+rvzp2_d[idx+dimz]+rvzp2_d[idx-dimz])/4.0;

				float molecular=ex_amp_x_d[idx]*rvxp2_d[idx]+ex_amp_z_d[idx]*change_rpz;

				float denominator=ex_amp_x_d[idx]*ex_amp_x_d[idx]+ex_amp_z_d[idx]*ex_amp_z_d[idx];

				ex_vresult_pp_d[ix*nz+iz]=molecular*1.0/(denominator);
				//ex_vresult_pp_d[ix*nz+iz]=molecular*1.0/(denominator+0.0000000000001*(*max));
				
			}
		}
}

__global__ void imaging_correlation_for_xxzz(float *vxp1_d,float *vxs1_d,float *rvxp1_d,float *rvxs1_d,float *resultxx_d,int nx,int nz,int dimz,int boundary_up,int boundary_left)
//imaging_correlation_for_xxzz<<<dimGrid,dimBlock>>>(vxp1_d,vxs1_d,rvxp1_d,rvxs1_d,resultxx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;
			
			resultxx_d[ix*nz+iz]=resultxx_d[ix*nz+iz]+1.0*(vxp1_d[in_idx]+vxs1_d[in_idx])*(rvxp1_d[in_idx]+rvxs1_d[in_idx]);			
		}
}

__global__ void imaging_pp_compensate_dependent_angle_2D(float *ex_angle_pp_d,float *ex_angle_rpp_d,float *com_ex_vresultpp_d,float *ex_vresultpp_d,float *ex_time_d,int nx,int nz,int dimz,int boundary_up,int boundary_left,int it)
//imaging_pp_compensate_dependent_angle_2D<<<dimGrid,dimBlock>>>(ex_angle_pp_d,ex_angle_rpp_d,com_ex_vresultpp_d,ex_vresultpp_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left,it);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;
		float com=0.0;
		float denominrator=0.0;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[idx])
			{
				com=ex_angle_pp_d[idx]-ex_angle_rpp_d[idx];

				if(fabs(com)>=87.5&&fabs(com)<=92.5)	denominrator=0.1;

				else				denominrator=float(cos(1.0*com*pai1/180.0));

				com_ex_vresultpp_d[ix*nz+iz]=ex_vresultpp_d[ix*nz+iz]/(1.0*denominrator+0.0001);

				//ex_angle_rpp_d[idx]=com;
				ex_angle_rpp_d[idx]=ex_angle_pp_d[idx]-ex_angle_rpp_d[idx];	
			}
		}
}

__global__ void imaging_ps_compensate_dependent_angle_2D(float *ex_angle_pp_d,float *ex_angle_rpp_d,float *com_ex_vresultpp_d,float *ex_vresultpp_d,float *ex_time_d,int nx,int nz,int dimz,int boundary_up,int boundary_left,int it)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;
		float com=0.0;
		float denominrator=0.0;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			if(it==ex_time_d[idx])
			{
				com=ex_angle_pp_d[idx]-ex_angle_rpp_d[idx];

				if(fabs(com)<=2.5)	denominrator=0.1;

				else			denominrator=float(sin(fabs(1.0*com*pai1/180.0)));

				com_ex_vresultpp_d[ix*nz+iz]=ex_vresultpp_d[ix*nz+iz]/(1.0*denominrator+0.0001);

				//ex_angle_rpp_d[idx]=com;
				ex_angle_rpp_d[idx]=ex_angle_pp_d[idx]-ex_angle_rpp_d[idx];				
			}
		}
}


__global__ void caculate_ex_open_pp_ps(float *ex_open_pp1_d,float *ex_angle_pp1_d,float *ex_angle_rpp1_d,int nx,int nz,int dimx,int dimz,int boundary_up,int boundary_left,int it,float *ex_time_d)
//caculate_ex_open_pp_ps<<<dimGrid,dimBlock>>>(ex_open_pp_d,ex_angle_pp_d,ex_angle_rpp_d,nx_size,nz,nx_append,nz_append,boundary_up,boundary_left,it,ex_time_d);
{
		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				//dimx=dimx+2*radius;dimz=dimz+2*radius;
				//ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				if(it==ex_time_d[in_idx])
				{
					ex_open_pp1_d[in_idx]=ex_angle_pp1_d[in_idx]-ex_angle_rpp1_d[in_idx];
				} 
		}
}
__global__ void cuda_ex_com_pp_ps_sign(float *ex_com_pp_sign_d,float *ex_open_pp_d,int nx,int nz,int dimx,int dimz,int mark)
//cuda_ex_com_pp_ps_sign<<<dimGrid,dimBlock>>>(ex_com_pp_sign_d,ex_open_pp1_d,nx_size,nz,nx_append,nz_append,0);
{		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				//dimx=dimx+2*radius;dimz=dimz+2*radius;
				//ix=ix+radius;iz=iz+radius;
				in_idx=ix*dimz+iz;

				if(mark==0)	ex_com_pp_sign_d[in_idx]=1.0*cos(1.0*ex_open_pp_d[in_idx]*pai1/180.0);///pp

				if(mark==1)	ex_com_pp_sign_d[in_idx]=fabs(1.0*sin(1.0*ex_open_pp_d[in_idx]*pai1/180.0));///ps
		}
}

__global__ void imaging_pp_compensate_dependent_angle_2D_new(float *ex_open_pp1_d,float *ex_com_pp_sign_d,float *com_ex_vresultpp_d,float *ex_vresultpp_d,float *ex_time_d,int nx,int nz,int dimz,int boundary_up,int boundary_left)
//imaging_pp_compensate_dependent_angle_2D_new<<<dimGrid,dimBlock>>>(ex_open_pp1_d,ex_com_pp_sign_d,com_ex_vresultpp_d,ex_vresultpp_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;
		float denominrator=0.0;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			
			//if(fabs(ex_open_ps1_d[idx])>=87.5&&fabs(ex_open_ps1_d[idx])<=92.5)	denominrator=0.001;			

			denominrator=ex_com_pp_sign_d[idx];//+0.01;

			//if(denominrator!=0)	com_ex_vresultpp_d[ix*nz+iz]=1.0*ex_vresultpp_d[ix*nz+iz]/denominrator;

			if(denominrator<=0)		com_ex_vresultpp_d[ix*nz+iz]=-1.0*ex_vresultpp_d[ix*nz+iz];

			if(denominrator>0)		com_ex_vresultpp_d[ix*nz+iz]=1.0*ex_vresultpp_d[ix*nz+iz];

			com_ex_vresultpp_d[ix*nz+iz]=-1.0*com_ex_vresultpp_d[ix*nz+iz]/(fabs(denominrator)+0.03);
		}
}

__global__ void imaging_ps_compensate_dependent_angle_2D_new(float *ex_open_ps1_d,float *ex_com_ps_sign_d,float *com_ex_vresultps_d,float *ex_vresultps_d,float *ex_time_d,int nx,int nz,int dimz,int boundary_up,int boundary_left)
//imaging_ps_compensate_dependent_angle_2D_new<<<dimGrid,dimBlock>>>(ex_open_pp1_d,ex_com_ps_sign_d,com_ex_vresultps_d,ex_vresultps_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int idx;
		float denominrator=0.0;

		if(ix<nx&&iz<nz)
		{
			idx=(ix+boundary_left)*dimz+iz+boundary_up;
			
			//if(fabs(ex_open_ps1_d[idx])<=2.5)	denominrator=0.001;			

			denominrator=fabs(ex_com_ps_sign_d[idx])+0.1;

			if(denominrator!=0)	com_ex_vresultps_d[ix*nz+iz]=-1.0*ex_vresultps_d[ix*nz+iz]/denominrator;
		}
}
