#include "hip/hip_runtime.h"
///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
	
__global__ void save_and_set_wavefiled(float *vx2_d,float *save_vx_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up,int mark)
//save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx2_d,&save_vx_d[it*nx*nz],nx,nz,nx_append,nz_append,boundary_left,boundary_up,0);
{

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{	
			in_idx = ix*nz+iz;
			in_idx1=(ix+boundary_left)*nz_append+iz+boundary_up;

			if(mark==0)	save_vx_d[in_idx]=vx2_d[in_idx1];

			if(mark==1)	vx2_d[in_idx1]=save_vx_d[in_idx];
		}
}
__global__ void cuda_cal_viscoelastic(float *modul_p_d,float *modul_s_d,float *qp_d,float *qs_d,float *tao_d,float *strain_p_d,float *strain_s_d,float freq,float *velocity_d,float *velocity1_d,float *density_d,int dimx,int dimz)
//cuda_cal_viscoelastic<<<dimGrid,dimBlock>>>(modul_p_d,modul_s_d,qp_d,qs_d,tao_d,strain_p_d,strain_s_d,freq,velocity_d,velocity1_d,density_d,nx_append,nz_append);
{
//////////////we note that  linear solid theroy  L=1
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float w;
		
		w=2*pai*freq;

		if((ix<dimx)&&(iz<dimz))
		{
				//dimx=dimx+2*radius;dimz=dimz+2*radius;
				//ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				//if(qp_d[in_idx]>1088)		qp_d[in_idx]=qp_d[in_idx]*0.015;
				//if(qs_d[in_idx]>1088/sqrt(3.0))	qs_d[in_idx]=qs_d[in_idx]*0.015*sqrt(3.0);
				
				tao_d[in_idx]=1.0/w*(1.0*sqrt(1.0+(1.0/qp_d[in_idx]/qp_d[in_idx]))-1.0/qp_d[in_idx]);
				
				strain_p_d[in_idx]=1.0/(1.0*w*w*tao_d[in_idx]);

				strain_s_d[in_idx]=1.0*(1+1.0*w*tao_d[in_idx]*qs_d[in_idx])/(1.0*w*qs_d[in_idx]-1.0*w*w*tao_d[in_idx]);

				//modul_p_d[in_idx]=1.0*velocity_d[in_idx]*velocity_d[in_idx]*density_d[in_idx]*(strain_p_d[in_idx]*1.0/tao_d[in_idx]);

				//modul_s_d[in_idx]=1.0*velocity1_d[in_idx]*velocity1_d[in_idx]*density_d[in_idx]*(strain_s_d[in_idx]*1.0/tao_d[in_idx]);

				///in order to moditfy elastic program at least
				modul_p_d[in_idx]=1.0*velocity_d[in_idx]*velocity_d[in_idx]*(strain_p_d[in_idx]*1.0/tao_d[in_idx]);

				modul_s_d[in_idx]=1.0*velocity1_d[in_idx]*velocity1_d[in_idx]*(strain_s_d[in_idx]*1.0/tao_d[in_idx]);
		}
}


__global__ void fwd_txxzzxzpp_viscoelastic_and_memory(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z)-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z)-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z+dt_real*mem_xx2_d[in_idx]);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*+mem_zz2_d[in_idx]);//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]);
		}
}

__global__ void fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization(float *tp2_d,float *tp1_d,float *txx2_d,float *txx1_d,float *tzz2_d,float *tzz1_d,float *txz2_d,float *txz1_d,float *vx2_d,float *vz2_d,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		float s_attenuation;

		float density,tao;

		float modul_p,modul_s;

		float mem_p1,mem_xx1,mem_zz1,mem_xz1;

		//float tp1;

		float txx1,tzz1,txz1;


		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2_d[in_idx];
				s_data2[tz][tx]=vz2_d[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));

				density=density_d[in_idx];

				tao=tao_d[in_idx];

				modul_p=modul_p_d[in_idx];
				modul_s=modul_s_d[in_idx];
		
				mem_p1=mem_p1_d[in_idx];
				mem_xx1=mem_xx1_d[in_idx];
				mem_zz1=mem_zz1_d[in_idx];
				mem_xz1=mem_xz1_d[in_idx];

				//tp1=tp1_d[in_idx];
				txx1=txx1_d[in_idx];
				tzz1=tzz1_d[in_idx];
				txz1=txz1_d[in_idx];

				s_attenuation=attenuation_d[in_idx];

				__syncthreads();			

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1-1.0*dt_real*1.0/tao*mem_p1+s_velocity*density/tao*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1-1.0*dt_real*1.0/tao*mem_xx1-2.0*s_velocity1*density/tao*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1-1.0*dt_real*1.0/tao*mem_zz1-2.0*s_velocity1*density/tao*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1+modul_p*density*sumx*coe_x+(modul_p-2*modul_s)*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1+(modul_p-2*modul_s)*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				//tp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1+modul_p*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1-1.0*dt_real*1.0/tao*mem_xz1+s_velocity1*density/tao*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1+modul_s*density*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]);
		}
}


__global__ void demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization(float *tp2_d,float *tp1_d,float *txx2_d,float *txx1_d,float *tzz2_d,float *tzz1_d,float *txz2_d,float *txz1_d,float *vx2_d,float *vz2_d,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d,float *dem_p_all_d)
//demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d,float *dem_p_all_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		float s_attenuation;

		float density,tao;

		float modul_p,modul_s;

		float mem_p1,mem_xx1,mem_zz1,mem_xz1;

		//float tp1;

		float txx1,tzz1,txz1;

		float dem_p2,dem_p3,dem_p4,dem_p5,dem_p6,dem_p7;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2_d[in_idx];
				s_data2[tz][tx]=vz2_d[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));

				density=density_d[in_idx];

				tao=tao_d[in_idx];

				modul_p=modul_p_d[in_idx];
				modul_s=modul_s_d[in_idx];
		
				mem_p1=mem_p1_d[in_idx];
				mem_xx1=mem_xx1_d[in_idx];
				mem_zz1=mem_zz1_d[in_idx];
				mem_xz1=mem_xz1_d[in_idx];

				//tp1=tp1_d[in_idx];
				txx1=txx1_d[in_idx];
				tzz1=tzz1_d[in_idx];
				txz1=txz1_d[in_idx];

				dem_p2=dem_p_all_d[2*dimx*dimz+in_idx];
				dem_p3=dem_p_all_d[3*dimx*dimz+in_idx];
				dem_p4=dem_p_all_d[4*dimx*dimz+in_idx];
				dem_p5=dem_p_all_d[5*dimx*dimz+in_idx];
				dem_p6=dem_p_all_d[6*dimx*dimz+in_idx];
				dem_p7=dem_p_all_d[7*dimx*dimz+in_idx];

				s_attenuation=attenuation_d[in_idx];

				__syncthreads();			

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1-1.0*dt_real*1.0/tao*mem_p1+s_velocity*density/tao*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1-1.0*dt_real*1.0/tao*mem_xx1-2.0*s_velocity1*density/tao*ssss*sumz*coe_z+dem_p5);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1-1.0*dt_real*1.0/tao*mem_zz1-2.0*s_velocity1*density/tao*ssss*sumx*coe_x+dem_p6);//s_velocity  and  s_velocity1


				txx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1+modul_p*density*sumx*coe_x+(modul_p-2*modul_s)*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx])+dem_p2);//s_velocity  and  s_velocity1

				tzz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1+(modul_p-2*modul_s)*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx])+dem_p3);//sumx  and  sumz 
				
				//tp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1+modul_p*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1-1.0*dt_real*1.0/tao*mem_xz1+s_velocity1*density/tao*ssss*(sumx1*coe_x+sumz1*coe_z)+dem_p7);//s_velocity  and  s_velocity1

				txz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1+modul_s*density*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]+dem_p4);
		}
}


/*__global__ void demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d,float *dem_p_all_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				//mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z+dem_p_all_d[5*dimx*dimz+in_idx]);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x+dem_p_all_d[6*dimx*dimz+in_idx]);//s_velocity  and  s_velocity1


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx])+dem_p_all_d[2*dimx*dimz+in_idx]);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx])+dem_p_all_d[3*dimx*dimz+in_idx]);//sumx  and  sumz 
				
				//tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z)+dem_p_all_d[7*dimx*dimz+in_idx]);//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]+dem_p_all_d[4*dimx*dimz+in_idx]);
		}
}*/

__global__ void cuda_packaging(float *packaging_d,float dx,float dz,float dt,float coe_x,float coe_z,int dimx,int dimz,float *coe_d)
{
		int ix;
		packaging_d[0]=dx;
		packaging_d[1]=dz;
		packaging_d[2]=dt;
		packaging_d[3]=coe_x;
		packaging_d[4]=coe_z;
		packaging_d[5]=dimx;
		packaging_d[6]=dimz;

		for(ix=0;ix<radius+1;ix++)
		{
			packaging_d[7+ix]=coe_d[ix];
		}
}

//float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,
__global__ void fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *tp2_d,float *tp1_d,float *txx2_d,float *txx1_d,float *tzz2_d,float *tzz1_d,float *txz2_d,float *txz1_d,float *vx2_d,float *vz2_d,float *modul_p_d,float *modul_s_d,float *attenuation_d,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d,float *packaging_d)
//__global__ void fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,packaging_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];

		__shared__ float coe_d[radius+1];

		float dx,dz,dt,coe_x,coe_z;
		int ir,dimx,dimz;
		
		dx=packaging_d[0];
		dz=packaging_d[1];
		dt=packaging_d[2];
		coe_x=packaging_d[3];
		coe_z=packaging_d[4];
		dimx=packaging_d[5];
		dimz=packaging_d[6];

		for(ir=0;ir<radius+1;ir++)
		{
			coe_d[ir]=packaging_d[7+ir];
		}

		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		float s_attenuation;

		float density,tao;

		float modul_p,modul_s;

		float mem_p1,mem_xx1,mem_zz1,mem_xz1;

		float tp1,txx1,tzz1,txz1;


		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2_d[in_idx];
				s_data2[tz][tx]=vz2_d[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));

				density=density_d[in_idx];

				tao=tao_d[in_idx];

				modul_p=modul_p_d[in_idx];
				modul_s=modul_s_d[in_idx];
		
				mem_p1=mem_p1_d[in_idx];
				mem_xx1=mem_xx1_d[in_idx];
				mem_zz1=mem_zz1_d[in_idx];
				mem_xz1=mem_xz1_d[in_idx];

				tp1=tp1_d[in_idx];
				txx1=txx1_d[in_idx];
				tzz1=tzz1_d[in_idx];
				txz1=txz1_d[in_idx];

				s_attenuation=attenuation_d[in_idx];

				__syncthreads();			

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1-1.0*dt_real*1.0/tao*mem_p1+s_velocity*density/tao*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1-1.0*dt_real*1.0/tao*mem_xx1-2.0*s_velocity1*density/tao*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1-1.0*dt_real*1.0/tao*mem_zz1-2.0*s_velocity1*density/tao*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1+modul_p*density*sumx*coe_x+(modul_p-2*modul_s)*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1+(modul_p-2*modul_s)*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1+modul_p*density*sumx*coe_x+modul_p*density*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				/////vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				////vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;

				vz_x_d[in_idx]=sumx1*1.0/dx;

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1-1.0*dt_real*1.0/tao*mem_xz1+s_velocity1*density/tao*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1+modul_s*density*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]);
		}
}


//float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,
/*__global__ void fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d,float *packaging_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float coe_d[radius+1];

		float dx,dz,dt,coe_x,coe_z;
		int ir,dimx,dimz;
		
		dx=packaging_d[0];
		dz=packaging_d[1];
		dt=packaging_d[2];
		coe_x=packaging_d[3];
		coe_z=packaging_d[4];
		dimx=packaging_d[5];
		dimz=packaging_d[6];

		for(ir=0;ir<radius+1;ir++)
		{
			coe_d[ir]=packaging_d[7+ir];
		}
		
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z+dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				/////vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				////vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;

				vz_x_d[in_idx]=sumx1*1.0/dx;

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz2_d[in_idx]);
		}
}*/

__global__ void fwd_txxzzxzpp_viscoelastic(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p1_d,float *mem_xx1_d,float *mem_zz1_d,float *mem_xz1_d)
//fwd_txxzzxzpp_viscoelastic<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p1_d,mem_xx1_d,mem_zz1_d,mem_xz1_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				//s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				s_velocity=velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx];//2017年07月27日 星期四 14时40分31秒 

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p1_d[in_idx]+mem_xx1_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z+dt_real*(mem_p1_d[in_idx]+mem_zz1_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+s_velocity*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z+dt_real*mem_p1_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+dt_real*mem_xz1_d[in_idx]);
		}
}

__global__ void fwd_memory(float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *vx2,float *vz2,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_memory<<<dimGrid,dimBlock>>>(mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,vx2_d,vz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float pppp=0,ssss=0;
		
		float density;

		float tao;

		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));	

				density=density_d[in_idx];

				tao=tao_d[in_idx];
				
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao*mem_p1_d[in_idx]+s_velocity*density/tao*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao*mem_xx1_d[in_idx]-2.0*s_velocity1*density/tao*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao*mem_zz1_d[in_idx]-2.0*s_velocity1*density/tao*ssss*sumx*coe_x);//s_velocity  and  s_velocity1
	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao*mem_xz1_d[in_idx]+s_velocity1*density/tao*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1
		}
}


/*__global__ void fwd_memory(float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *vx2,float *vz2,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_memory<<<dimGrid,dimBlock>>>(mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,vx2_d,vz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));	
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1
	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1
		}
}*/


//////////////source and receiver back propagation based on viscoelastic modeling
__global__ void rfwd_txxzzxzpp_viscoelastic_and_memory_old(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				/*mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1*/

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1

				
				/*txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z)-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]);//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);*/

				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x-(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x-modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]-modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x-modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				/*mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);*/

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);
		}
}

//////////////source and receiver back propagation based on viscoelastic modeling
__global__ void rfwd_txxzzxzpp_viscoelastic_and_memory(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1

				/*mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]+1.0/(1.0-dt_real*1.0/tao_d[in_idx])*(mem_p1_d[in_idx]-s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z)));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]+1.0/(1.0-dt_real*1.0/tao_d[in_idx])*(mem_xx1_d[in_idx]+2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z));//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]+1.0/(1.0-dt_real*1.0/tao_d[in_idx])*(mem_zz1_d[in_idx]+2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x));//s_velocity  and  s_velocity1*/

				
				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z)-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]);//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);

				/*txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x-(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x-modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]-modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x-modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);*/	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);

				/*mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]+1.0/(1.0-dt_real*1.0/tao_d[in_idx])*(mem_xz1_d[in_idx]-s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z)));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);*/
		}
}

__global__ void receiver_fwd_txxzzxzpp_viscoelastic_and_memory(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);
		}
}


__global__ void receiver_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *modul_p_d,float *modul_s_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
	
		float pppp=0,ssss=0;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
				
				pppp=(1.0-(1.0*strain_p_d[in_idx]/tao_d[in_idx]));
				ssss=(1.0-(1.0*strain_s_d[in_idx]/tao_d[in_idx]));
				

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+s_velocity*density_d[in_idx]/tao_d[in_idx]*pppp*(sumx*coe_x+sumz*coe_z));//s_velocity  and  s_velocity1

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumz*coe_z);//s_velocity  and  s_velocity1

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]-2.0*s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*sumx*coe_x);//s_velocity  and  s_velocity1


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_xx2_d[in_idx]));//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+(modul_p_d[in_idx]-2*modul_s_d[in_idx])*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*(mem_p2_d[in_idx]+mem_zz2_d[in_idx]));//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+modul_p_d[in_idx]*density_d[in_idx]*sumx*coe_x+modul_p_d[in_idx]*density_d[in_idx]*sumz*coe_z-dt_real*mem_p2_d[in_idx]);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+s_velocity1*density_d[in_idx]/tao_d[in_idx]*ssss*(sumx1*coe_x+sumz1*coe_z));//s_velocity  and  s_velocity1

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+modul_s_d[in_idx]*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)-dt_real*mem_xz2_d[in_idx]);
		}
}



///////////////2017年07月31日 星期一 20时42分28秒 
__global__ void adjoint_fwd_vx_viscoelastic(float *vx2,float *vx1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *modul_p_d,float *modul_s_d,float *density_d,float *mem_p1_d,float *mem_xx1_d,float *mem_zz1_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//adjoint_fwd_vx_viscoelastic<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,rtzz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,modul_p_d,modul_s_d,s_density_d,rmem_p2_d,rmem_xx2_d,rmem_zz2_d,rmem_xz2_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;

		float tao;
		float modul_p;
		float modul_s;
		
		float s_attenuation;

		float pppp=0,ssss=0;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人

				tao=tao_d[in_idx];

				modul_p=modul_p_d[in_idx];
				modul_s=modul_s_d[in_idx];
//////////////注意伴随状态方程左边存在密度，所以用来反传计算伴随波场跟密度没有关系？？？？？？？？？？

				pppp=1.0/tao*(1.0-(1.0*strain_p_d[in_idx]/tao))*s_velocity;

				ssss=1.0/tao*(1.0-(1.0*strain_s_d[in_idx]/tao))*s_velocity1;

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();


/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumx:the derivation of x direction of txx
				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

////sumxz:the derivation of z direction of txz
				float    sumxz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumxz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumxz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumxz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumxz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumxz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
////sumx1:the derivation of x direction of tzz
				float    sumx1=coe_d[1]*(s_data3[tz][tx]-s_data3[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data3[tz][tx+1]-s_data3[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data3[tz][tx+2]-s_data3[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data3[tz][tx+3]-s_data3[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data3[tz][tx+4]-s_data3[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data3[tz][tx+5]-s_data3[tz][tx-6]);

////sumx:the derivation of x direction of rxx
				float    sumrxx=coe_d[1]*(mem_xx1_d[in_idx]-mem_xx1_d[in_idx-1*dimz]);
					sumrxx+=coe_d[2]*(mem_xx1_d[in_idx+1*dimz]-mem_xx1_d[in_idx-2*dimz]);
					sumrxx+=coe_d[3]*(mem_xx1_d[in_idx+2*dimz]-mem_xx1_d[in_idx-3*dimz]);
					sumrxx+=coe_d[4]*(mem_xx1_d[in_idx+3*dimz]-mem_xx1_d[in_idx-4*dimz]);
					sumrxx+=coe_d[5]*(mem_xx1_d[in_idx+4*dimz]-mem_xx1_d[in_idx-5*dimz]);
					sumrxx+=coe_d[6]*(mem_xx1_d[in_idx+5*dimz]-mem_xx1_d[in_idx-6*dimz]);

////sumx1:the derivation of x direction of rzz
				float    sumrzz=coe_d[1]*(mem_zz1_d[in_idx]-mem_zz1_d[in_idx-1*dimz]);
					sumrzz+=coe_d[2]*(mem_zz1_d[in_idx+1*dimz]-mem_zz1_d[in_idx-2*dimz]);
					sumrzz+=coe_d[3]*(mem_zz1_d[in_idx+2*dimz]-mem_zz1_d[in_idx-3*dimz]);
					sumrzz+=coe_d[4]*(mem_zz1_d[in_idx+3*dimz]-mem_zz1_d[in_idx-4*dimz]);
					sumrzz+=coe_d[5]*(mem_zz1_d[in_idx+4*dimz]-mem_zz1_d[in_idx-5*dimz]);
					sumrzz+=coe_d[6]*(mem_zz1_d[in_idx+5*dimz]-mem_zz1_d[in_idx-6*dimz]);

////sumxz:the derivation of z direction of rxz
				float    sumrxz=coe_d[1]*(mem_xz1_d[in_idx]-mem_xz1_d[in_idx-1]);
					sumrxz+=coe_d[2]*(mem_xz1_d[in_idx+1]-mem_xz1_d[in_idx-2]);
					sumrxz+=coe_d[3]*(mem_xz1_d[in_idx+2]-mem_xz1_d[in_idx-3]);
					sumrxz+=coe_d[4]*(mem_xz1_d[in_idx+3]-mem_xz1_d[in_idx-4]);
					sumrxz+=coe_d[5]*(mem_xz1_d[in_idx+4]-mem_xz1_d[in_idx-5]);
					sumrxz+=coe_d[6]*(mem_xz1_d[in_idx+5]-mem_xz1_d[in_idx-6]);

				//vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+(s_velocity*sumx*coe_x+(s_velocity-2*s_velocity1)*sumx1*coe_x+s_velocity1*sumxz*coe_z));

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+modul_p*sumx*coe_x+(modul_p-2*modul_s)*sumx1*coe_x+modul_s*sumxz*coe_z+pppp*sumrxx*coe_x+(pppp-2.0*ssss)*sumrzz*coe_x+ssss*sumrxz*coe_z);
		}
}


__global__ void adjoint_fwd_vz_viscoelastic(float *vz2,float *vz1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *modul_p_d,float *modul_s_d,float *density_d,float *mem_p1_d,float *mem_xx1_d,float *mem_zz1_d,float *mem_xz1_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//adjoint_fwd_vz_viscoelastic<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtxx2_d,rtxz2_d,rtzz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,modul_p_d,modul_s_d,s_density_d,rmem_p1_d,rmem_xx1_d,rmem_zz1_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		float tao;
		float modul_p;
		float modul_s;

		float pppp=0,ssss=0;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人

				tao=tao_d[in_idx];

				modul_p=modul_p_d[in_idx];
				modul_s=modul_s_d[in_idx];

				pppp=1.0/tao*(1.0-(1.0*strain_p_d[in_idx]/tao))*s_velocity;

				ssss=1.0/tao*(1.0-(1.0*strain_s_d[in_idx]/tao))*s_velocity1;

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

					

/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumz:the derivation of z direction of tzz   ///////data3:tzz1/////// 
				float    sumz=coe_d[1]*(s_data3[tz+1][tx]-s_data3[tz][tx]);
					sumz+=coe_d[2]*(s_data3[tz+2][tx]-s_data3[tz-1][tx]);
					sumz+=coe_d[3]*(s_data3[tz+3][tx]-s_data3[tz-2][tx]);
					sumz+=coe_d[4]*(s_data3[tz+4][tx]-s_data3[tz-3][tx]);
					sumz+=coe_d[5]*(s_data3[tz+5][tx]-s_data3[tz-4][tx]);
					sumz+=coe_d[6]*(s_data3[tz+6][tx]-s_data3[tz-5][tx]);

////sumz1:the derivation of z direction of txx/////data1:txx1////
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

////sumx:the derivation of x direction of txz/////data2:txz1////					
				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

////sumx:the derivation of z direction of rxx
				float    sumrxx=coe_d[1]*(mem_xx1_d[in_idx+1]-mem_xx1_d[in_idx]);
					sumrxx+=coe_d[2]*(mem_xx1_d[in_idx+2]-mem_xx1_d[in_idx-1]);
					sumrxx+=coe_d[3]*(mem_xx1_d[in_idx+3]-mem_xx1_d[in_idx-2]);
					sumrxx+=coe_d[4]*(mem_xx1_d[in_idx+4]-mem_xx1_d[in_idx-3]);
					sumrxx+=coe_d[5]*(mem_xx1_d[in_idx+5]-mem_xx1_d[in_idx-4]);
					sumrxx+=coe_d[6]*(mem_xx1_d[in_idx+6]-mem_xx1_d[in_idx-5]);

////sumx1:the derivation of z direction of rzz
				float    sumrzz=coe_d[1]*(mem_zz1_d[in_idx+1]-mem_zz1_d[in_idx]);
					sumrzz+=coe_d[2]*(mem_zz1_d[in_idx+2]-mem_zz1_d[in_idx-1]);
					sumrzz+=coe_d[3]*(mem_zz1_d[in_idx+3]-mem_zz1_d[in_idx-2]);
					sumrzz+=coe_d[4]*(mem_zz1_d[in_idx+4]-mem_zz1_d[in_idx-3]);
					sumrzz+=coe_d[5]*(mem_zz1_d[in_idx+5]-mem_zz1_d[in_idx-4]);
					sumrzz+=coe_d[6]*(mem_zz1_d[in_idx+6]-mem_zz1_d[in_idx-5]);

////sumxz:the derivation of x direction of rxz
				float    sumrxz=coe_d[1]*(mem_xz1_d[in_idx+1*dimz]-mem_xz1_d[in_idx]);
					sumrxz+=coe_d[2]*(mem_xz1_d[in_idx+2*dimz]-mem_xz1_d[in_idx-1*dimz]);
					sumrxz+=coe_d[3]*(mem_xz1_d[in_idx+3*dimz]-mem_xz1_d[in_idx-2*dimz]);
					sumrxz+=coe_d[4]*(mem_xz1_d[in_idx+4*dimz]-mem_xz1_d[in_idx-3*dimz]);
					sumrxz+=coe_d[5]*(mem_xz1_d[in_idx+5*dimz]-mem_xz1_d[in_idx-4*dimz]);
					sumrxz+=coe_d[6]*(mem_xz1_d[in_idx+6*dimz]-mem_xz1_d[in_idx-5*dimz]);

				//vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+(s_velocity*sumz*coe_z+(s_velocity-2*s_velocity1)*sumz1*coe_z+s_velocity1*sumx*coe_x));

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+modul_p*sumz*coe_z+(modul_p-2*modul_s)*sumz1*coe_z+modul_s*sumx*coe_x+(pppp-2.0*ssss)*sumrxx*coe_z+pppp*sumrzz*coe_z+ssss*sumrxz*coe_x);
		}
}


__global__ void adjoint_fwd_memory(float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *tp1,float *txx1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//adjoint_fwd_memory<<<dimGrid,dimBlock>>>(rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,rtp2_d,rtxx2_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		float s_attenuation;

		float dt_real=dt/1000;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_attenuation=attenuation_d[in_idx];

				__syncthreads();

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_p1_d[in_idx]+1.0*dt_real*tp1[in_idx]);

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xx1_d[in_idx]+1.0*dt_real*txx1[in_idx]);

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_zz1_d[in_idx]+1.0*dt_real*tzz1[in_idx]);
	
				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1_d[in_idx]-1.0*dt_real*1.0/tao_d[in_idx]*mem_xz1_d[in_idx]+1.0*dt_real*txz1[in_idx]);
		}
}

__global__ void adjoint_fwd_memory_new(float *mem_p2_d,float *mem_p1_d,float *mem_xx2_d,float *mem_xx1_d,float *mem_zz2_d,float *mem_zz1_d,float *mem_xz2_d,float *mem_xz1_d,float *tp1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *velocity_d,float *velocity1_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//adjoint_fwd_memory<<<dimGrid,dimBlock>>>(rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,rtp2_d,rtxx2_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
{
		float s_attenuation;

		float dt_real=dt/1000;

		float mem_p1,mem_xx1,mem_zz1,mem_xz1;
		float tp1,txx1,tzz1,txz1;
		float tao;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_attenuation=attenuation_d[in_idx];

				mem_p1=mem_p1_d[in_idx],mem_xx1=mem_xx1_d[in_idx],mem_zz1=mem_zz1_d[in_idx],mem_xz1=mem_xz1_d[in_idx];
				tp1=tp1_d[in_idx],txx1=txx1_d[in_idx],tzz1=tzz1_d[in_idx],txz1=txz1_d[in_idx];
				tao=tao_d[in_idx];

				__syncthreads();

				mem_p2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_p1-1.0*dt_real*1.0/tao*mem_p1+1.0*dt_real*tp1);

				mem_xx2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xx1-1.0*dt_real*1.0/tao*mem_xx1+1.0*dt_real*txx1);

				mem_zz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_zz1-1.0*dt_real*1.0/tao*mem_zz1+1.0*dt_real*tzz1);
	
				mem_xz2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*mem_xz1-1.0*dt_real*1.0/tao*mem_xz1+1.0*dt_real*txz1);
		}
}

__global__ void cal_gradient_in_viscoelastic_media(float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *vx_t_d,float *vz_t_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,float *rvx1_d,float *rvz1_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,float *rmem_xx1_d,float *rmem_xz1_d,float *rmem_zz1_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//cal_gradient_in_viscoelastic_media<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,rmem_xx2_d,rmem_xz2_d,rmem_zz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		float lame1;
		float lame2;
		float mp=0.0;
		float ms=0.0;
		float np=0.0;
		float ns=0.0;
		float A=0.0,B=0.0,C=0.0;
		float D=0.0,E=0.0,F=0.0;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			lame1=s_density_d[in_idx1]*s_velocity_d[in_idx1]*s_velocity_d[in_idx1]-2.0*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];
			lame2=s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];

			mp=1.0*strain_p_d[in_idx1]/tao_d[in_idx1];
			ms=1.0*strain_s_d[in_idx1]/tao_d[in_idx1];

			np=1.0/tao_d[in_idx1]*(1-mp);
			ns=1.0/tao_d[in_idx1]*(1-ms);


			grad_den_d[in_idx]=grad_den_d[in_idx]+1.0*s_density_d[in_idx1]*(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+(-1.0)*lame1*(1.0*mp*(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])+1.0*np*(rmem_xx1_d[in_idx1]+rmem_zz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1]));		
			

			A=(1.0*mp*vx_x_d[in_idx1]+(mp-ms)*vz_z_d[in_idx1])*rtxx1_d[in_idx1];
			B=(1.0*mp*vz_z_d[in_idx1]+(mp-ms)*vx_x_d[in_idx1])*rtzz1_d[in_idx1];
			C=1.0*ms*(vx_z_d[in_idx1]+vz_x_d[in_idx1])*rtxz1_d[in_idx1];

			D=(1.0*np*vx_x_d[in_idx1]+(np-ns)*vz_z_d[in_idx1])*rmem_xx1_d[in_idx1];
			E=(1.0*np*vz_z_d[in_idx1]+(np-ns)*vx_x_d[in_idx1])*rmem_zz1_d[in_idx1];
			F=1.0*ns*(vx_z_d[in_idx1]+vz_x_d[in_idx1])*rmem_xz1_d[in_idx1];

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+(-1.0)*lame2*(2.0*A+2.0*B+C+2.0*D+2.0*E+F);
		}
}

__global__ void cal_gradient_in_viscoelastic_media_new(float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *vx_t_d,float *vz_t_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,float *rvx1_d,float *rvz1_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,float *rmem_xx1_d,float *rmem_xz1_d,float *rmem_zz1_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,float *tao_d,float *strain_p_d,float *strain_s_d)
//cal_gradient_in_viscoelastic_media<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,rmem_xx2_d,rmem_xz2_d,rmem_zz2_d,boundary_left,boundary_up,nx,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		//float lame1;
		//float lame2;
		float mp=0.0;
		float ms=0.0;
		float np=0.0;
		float ns=0.0;
		float A=0.0,B=0.0,C=0.0;
		float D=0.0,E=0.0,F=0.0;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			//lame1=s_density_d[in_idx1]*s_velocity_d[in_idx1]*s_velocity_d[in_idx1]-2.0*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];
			//lame2=s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];

			mp=1.0*strain_p_d[in_idx1]/tao_d[in_idx1];
			ms=1.0*strain_s_d[in_idx1]/tao_d[in_idx1];

			np=1.0/tao_d[in_idx1]*(1-mp);
			ns=1.0/tao_d[in_idx1]*(1-ms);


			grad_den_d[in_idx]=grad_den_d[in_idx]+1.0*(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+(-1.0)*(1.0*mp*(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])+1.0*np*(rmem_xx1_d[in_idx1]+rmem_zz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1]));		
			

			A=(1.0*mp*vx_x_d[in_idx1]+(mp-ms)*vz_z_d[in_idx1])*rtxx1_d[in_idx1];
			B=(1.0*mp*vz_z_d[in_idx1]+(mp-ms)*vx_x_d[in_idx1])*rtzz1_d[in_idx1];
			C=1.0*ms*(vx_z_d[in_idx1]+vz_x_d[in_idx1])*rtxz1_d[in_idx1];

			D=(1.0*np*vx_x_d[in_idx1]+(np-ns)*vz_z_d[in_idx1])*rmem_xx1_d[in_idx1];
			E=(1.0*np*vz_z_d[in_idx1]+(np-ns)*vx_x_d[in_idx1])*rmem_zz1_d[in_idx1];
			F=1.0*ns*(vx_z_d[in_idx1]+vz_x_d[in_idx1])*rmem_xz1_d[in_idx1];

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+(-1.0)*(2.0*A+2.0*B+C+2.0*D+2.0*E+F);
		}

}


//////2017年08月02日 星期三 11时48分46秒 
__global__ void cuda_cal_dem_parameter_viscoelastic_media(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *dem_p6_d,float *dem_p7_d,float *dem_p8_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,float *tmp_perturb_vp_d,float *tmp_perturb_vs_d,float *tmp_perturb_density_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,float *tao_d,float *strain_p_d,float *strain_s_d,float dt,int inversion_para)
//cuda_cal_dem_parameter_viscoelastic_media<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,dem_p6_d,dem_p7_d,dem_p8_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d,dt,inversion_para);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		float dt_real=dt/1000;
		float lame1,lame2;
		float mp=0.0;
		float ms=0.0;
		float np=0.0;
		float ns=0.0;

		float p1,p2,p3;
		float p4,p5,p6;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;
			
				mp=1.0*strain_p_d[in_idx]/tao_d[in_idx];
				ms=1.0*strain_s_d[in_idx]/tao_d[in_idx];

				np=1.0/tao_d[in_idx]*(1-mp);
				ns=1.0/tao_d[in_idx]*(1-ms);
				

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				
				p1=tmp_perturb_lame1_d[in_idx]*lame1*1.0;

				p2=tmp_perturb_lame2_d[in_idx]*lame2*1.0;

				p3=tmp_perturb_den_d[in_idx]*s_density_d[in_idx]*1.0;				

				
				//p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*1.0;

				//p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*1.0;

				//p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;

				p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*s_density_d[in_idx]*1.0;

				p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*s_density_d[in_idx]*1.0;

				p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;

				if(inversion_para==1)
				{
					p1=1.0*p1;

					p2=1.0*p2;

					p3=1.0*p3;
				}

				if(inversion_para==2)
				{
					p1=2.0*s_density_d[in_idx]*s_velocity_d[in_idx]*p4-4.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*(1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;

					//p2=-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;
					p2=2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

				if(inversion_para==3)
				{
					p1=2*s_velocity_d[in_idx]*p4-4*s_velocity1_d[in_idx]*p5+(-1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]+2*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;

					p2=2.0*s_velocity1_d[in_idx]*p5-1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

					dem_p1_d[in_idx]=(-1.0)*dt_real*p3*vx_t_d[in_idx]/s_density_d[in_idx];
					dem_p2_d[in_idx]=(-1.0)*dt_real*p3*vz_t_d[in_idx]/s_density_d[in_idx];

					dem_p3_d[in_idx]=1.0*dt_real*((p1+2*p2)*mp*vx_x_d[in_idx]+((p1+2*p2)*mp-2*p2*ms)*vz_z_d[in_idx]);
		
					dem_p4_d[in_idx]=1.0*dt_real*((p1+2*p2)*mp*vz_z_d[in_idx]+((p1+2*p2)*mp-2*p2*ms)*vx_x_d[in_idx]);

					dem_p5_d[in_idx]=1.0*dt_real*(p2*ms*(vx_z_d[in_idx]+vz_x_d[in_idx]));


					dem_p6_d[in_idx]=1.0*dt_real*((p1+2*p2)*np*vx_x_d[in_idx]+((p1+2*p2)*np-2*p2*ns)*vz_z_d[in_idx]);

					dem_p7_d[in_idx]=1.0*dt_real*((p1+2*p2)*np*vz_z_d[in_idx]+((p1+2*p2)*np-2*p2*ns)*vx_x_d[in_idx]);

					dem_p8_d[in_idx]=1.0*dt_real*(p2*ns*(vx_z_d[in_idx]+vz_x_d[in_idx]));
				
		}
}

__global__ void cuda_cal_dem_parameter_viscoelastic_media_new(float *dem_p1_d,float *dem_p2_d,float *dem_p_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,float *tmp_perturb_vp_d,float *tmp_perturb_vs_d,float *tmp_perturb_density_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,float *tao_d,float *strain_p_d,float *strain_s_d,float dt,int inversion_para)
//cuda_cal_dem_parameter_viscoelastic_media_new<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p_all_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d,dt,inversion_para);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		float dt_real=dt/1000;
		float lame1,lame2;
		float mp=0.0;
		float ms=0.0;
		float np=0.0;
		float ns=0.0;

		float p1,p2,p3;
		float p4,p5,p6;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;
			
				mp=1.0*strain_p_d[in_idx]/tao_d[in_idx];
				ms=1.0*strain_s_d[in_idx]/tao_d[in_idx];

				np=1.0/tao_d[in_idx]*(1-mp);
				ns=1.0/tao_d[in_idx]*(1-ms);
				

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				
				if(inversion_para==1)
				{
					p1=tmp_perturb_lame1_d[in_idx]*lame1*1.0;

					p2=tmp_perturb_lame2_d[in_idx]*lame2*1.0;

					p3=tmp_perturb_den_d[in_idx]*s_density_d[in_idx]*1.0;				
				}
				
				if(inversion_para==2)
				{
					p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*1.0;

					p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*1.0;

					p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;
				}

				if(inversion_para==3)
				{
					p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*s_density_d[in_idx]*1.0;

					p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*s_density_d[in_idx]*1.0;

					p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;
				}

				if(inversion_para==1)
				{
					p1=1.0*p1;

					p2=1.0*p2;

					p3=1.0*p3;
				}

				if(inversion_para==2)
				{
					p1=2.0*s_density_d[in_idx]*s_velocity_d[in_idx]*p4-4.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*(1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;

					//p2=-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p2=2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

				if(inversion_para==3)
				{
					//p1=2.0*s_density_d[in_idx]*s_velocity_d[in_idx]*p4-4.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*(1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;
					p1=2*s_velocity_d[in_idx]*p4-4*s_velocity1_d[in_idx]*p5+(-1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]+2*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;
					//p2=2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;
					p2=2.0*s_velocity1_d[in_idx]*p5-1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

					dem_p1_d[in_idx]=(-1.0)*p3*vx_t_d[in_idx];
					dem_p2_d[in_idx]=(-1.0)*p3*vz_t_d[in_idx];

					dem_p_d[2*dimx*dimz+in_idx]=1.0*dt_real*((p1+2*p2)*mp*vx_x_d[in_idx]+((p1+2*p2)*mp-2*p2*ms)*vz_z_d[in_idx]);
		
					dem_p_d[3*dimx*dimz+in_idx]=1.0*dt_real*((p1+2*p2)*mp*vz_z_d[in_idx]+((p1+2*p2)*mp-2*p2*ms)*vx_x_d[in_idx]);

					dem_p_d[4*dimx*dimz+in_idx]=1.0*dt_real*(p2*ms*(vx_z_d[in_idx]+vz_x_d[in_idx]));


					dem_p_d[5*dimx*dimz+in_idx]=1.0*dt_real*((p1+2*p2)*np*vx_x_d[in_idx]+((p1+2*p2)*np-2*p2*ns)*vz_z_d[in_idx]);

					dem_p_d[6*dimx*dimz+in_idx]=1.0*dt_real*((p1+2*p2)*np*vz_z_d[in_idx]+((p1+2*p2)*np-2*p2*ns)*vx_x_d[in_idx]);

					dem_p_d[7*dimx*dimz+in_idx]=1.0*dt_real*(p2*ns*(vx_z_d[in_idx]+vz_x_d[in_idx]));
				
		}
}

__global__ void cuda_cal_multiply(float *tmp_perturb_den_d,float *s_density_d,float *dem_p1_d,int dimx,int dimz)
////////////////cuda_cal_multiply<<<dimGrid,dimBlock>>>(tmp_perturb_den_d,s_density_d,dem_p1_d,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		//float m;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;
			
				tmp_perturb_den_d[in_idx]=tmp_perturb_den_d[in_idx]*s_density_d[in_idx];

				dem_p1_d[in_idx]=tmp_perturb_den_d[in_idx];
		}
}

__global__ void cuda_bell_smoothz_new(float *g, float *smg, int rbell, int nx, int nz)
/*< smoothing with gaussian function >*/
{
	int i;
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id=iz+ix*nz;

	if(ix<nx && iz<nz)
	{
		float s=0;
		float sum=0;
		for(i=-rbell; i<=rbell; i++) if(iz+i>=0 && iz+i<nz) sum+=expf(-(1.0*i*i)/2.0/rbell);

		for(i=-rbell; i<=rbell; i++) if(iz+i>=0 && iz+i<nz) s+=expf(-(1.0*i*i)/2.0/rbell)/sum*g[id+i];
		smg[id]=s;
	}
}

__global__ void cuda_bell_smoothx_new(float *g, float *smg, int rbell, int nx, int nz)
/*< smoothing with gaussian function >*/
{
	int i;
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id=iz+ix*nz;

	if(ix<nx && iz<nz)
	{
		float s=0;
		float sum=0.0;
		for(i=-rbell; i<=rbell; i++) if(ix+i>=0 && ix+i<nx) sum+=expf(-(1.0*i*i)/2.0/rbell);

		for(i=-rbell; i<=rbell; i++) if(ix+i>=0 && ix+i<nx) s+=expf(-(1.0*i*i)/2.0/rbell)/sum*g[id+i*nz];
		smg[id]=s;
	}
}

__global__ void cuda_bell_smooth_2d(float *g, float *smg, int rbell, int nx, int nz)
/*< smoothing with gaussian function >*/
{
	int im,in;
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id=iz+ix*nz;

	float distance;

	if(ix<nx && iz<nz)
	{
		float s=0;
		float sum=0.0;

		for(im=-rbell; im<=rbell; im++) 
			for(in=-rbell; in<=rbell; in++) 		
		if(ix+im>=0 && ix+im<nx && iz+in>=0 && iz+in<nz) 
		{
			distance=im*im+in*in;

			sum+=expf(-(1.0*distance)/2/rbell);
		}

		for(im=-rbell; im<=rbell; im++) 
			for(in=-rbell; in<=rbell; in++) 		
		if(ix+im>=0 && ix+im<nx && iz+in>=0 && iz+in<nz) 
		{
			distance=im*im+in*in;

			s+=expf(-(1.0*distance)/2/rbell)/sum*g[id+im*nz+in];;
		}		
		
		smg[id]=s;
	}
}

__global__ void cuda_get_partly_mode_boundary(float *velocity_all_d,float *wf_d,int nx,int nz,int receiver_x_cord,int receiver_interval,int receiver_num,int nx_append_new,int nz_append,int boundary_left,int boundary_up)
{
	
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;

	if(ix<nnx&&iz<nz)
	{
		id=ix*nz+iz;

		id1=(ix+boundary_left+receiver_x_cord)*nz_append+iz+boundary_up;

		wf_d[id]=velocity_all_d[id1];	
	}
}

__global__ void cuda_get_partly_mode_boundary_z1_z2(float *velocity_all_d,float *wf_d,int nx,int nz,int receiver_x_cord,int receiver_interval,int receiver_num,int nx_append_new,int nz_append,int boundary_left,int boundary_up,int z1,int z2)
//cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
{
	
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;

	if(ix<nnx&&iz<nz)
	{
		id=ix*nz+iz;

		id1=(ix+boundary_left+receiver_x_cord)*nz_append+iz+boundary_up;

		//if(iz>=z1&&iz<=z2)	wf_d[id]=velocity_all_d[id1];
		if(iz>=0&&iz<=z2)	wf_d[id]=velocity_all_d[id1];

		else			wf_d[id]=velocity_all_d[(ix+boundary_left+receiver_x_cord)*nz_append+z2+boundary_up];	
	}
}

__global__ void cuda_get_constant_mode(float *velocity_all_d,float *velocity_d,int nx,int nz)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;

	if(ix<nx&&iz<nz)
	{
		id=ix*nz+iz;

		velocity_d[id]=velocity_all_d[0];	
	}

}

__global__ void cuda_get_partly_mode(float *velocity_all_d,float *wf_d,int nx,int nz,int receiver_x_cord,int receiver_interval,int receiver_num)
{
	
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;

	if(ix<nnx&&iz<nz)
	{
		id=ix*nz+iz;

		id1=(ix+receiver_x_cord)*nz+iz;

		wf_d[id]=velocity_all_d[id1];	
	}
}

__global__ void cuda_get_partly_mode_z1_z2(float *velocity_all_d,float *wf_d,int nx,int nz,int receiver_x_cord,int receiver_interval,int receiver_num,int z1,int z2)
//cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_vp_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
{
	
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;

	if(ix<nnx&&iz<nz)
	{
		id=ix*nz+iz;

		id1=(ix+receiver_x_cord)*nz+iz;
		
		//if(iz>=z1&&iz<=z2)	wf_d[id]=velocity_all_d[id1];

		if(iz>=0&&iz<=z2)	wf_d[id]=velocity_all_d[id1];

		//else			wf_d[id]=velocity_all_d[(ix+receiver_x_cord)*nz+z2];
		else			wf_d[id]=0.0;	
	}
}

__global__ void cuda_sum_new_acqusition(float *all_vresultpp_d,float *vresultpp_d,int nx,int nz,int receiver_x_cord,int receiver_interval,int receiver_num)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;
	
	if(ix<nnx&&iz<nz)
	{
		id=ix*nz+iz;

		id1=(ix+receiver_x_cord)*nz+iz;

		all_vresultpp_d[id1]+=vresultpp_d[id];	
	}
}

__global__ void cuda_sum_new_acqusition_illum(float *d_illum_new,float *d_illum,int nx,int nz,int nx_append_new,int nz_append,int boundary_left,int boundary_up,int receiver_x_cord,int receiver_interval,int receiver_num)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id,id1;

	int nnx;
	nnx=receiver_interval*receiver_num;
	
	if(ix<nnx&&iz<nz)
	{
		id=(ix+boundary_left)*nz_append+iz+boundary_up;

		id1=(ix+receiver_x_cord+boundary_left)*nz_append+iz+boundary_up;

		d_illum_new[id1]+=d_illum[id];	
	}
}

__global__ void cauda_zero_acqusition_left(float *obs_shot_x_d,int acqusition_left,int receiver_x_cord,int receiver_interval,int receiver_num,int lt)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;

	if(ix<receiver_num&&iz<lt)
	{
		id=ix*lt+iz;

		if(ix>(receiver_num*receiver_interval-acqusition_left))
			obs_shot_x_d[id]=0;

	}
}

__global__ void cauda_zero_acqusition_right(float *obs_shot_x_d,int acqusition_right,int receiver_x_cord,int receiver_interval,int receiver_num,int lt)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;

	if(ix<receiver_num&&iz<lt)
	{
		id=ix*lt+iz;

		if(ix<acqusition_right)
			obs_shot_x_d[id]=0;
	}
}

__global__ void cauda_zero_acqusition_left_and_right(float *obs_shot_x_d,int offset_left,int offset_right,int source_x_cord,int receiver_offset,int receiver_num,int lt)
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;

	if(ix<receiver_num&&iz<lt)
	{
		id=ix*lt+iz;

		if(offset_left>receiver_offset)
		{
			if(ix>=0&&ix<(offset_left-receiver_offset))			
				obs_shot_x_d[id]=0;
		}

		if(offset_right>receiver_offset)
		{
			if(ix>=(source_x_cord+receiver_offset)&&ix<receiver_num)			
				obs_shot_x_d[id]=0;
		}		
	}
}

__global__ void cuda_expand_acqusition_left_and_right(float *velocity_d,int offset_left,int offset_right,int source_x_cord,int receiver_offset,int nx_size,int dimx,int dimz,int boundary_left,int boundary_up)
//cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(velocity_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;
	float change;

	if(ix<dimx&&iz<dimz)
	{
		//id=(ix+boundary_left)*dimz+iz+boundary_up;

		id=ix*dimz+iz;

		if(offset_left>receiver_offset)
		{
			change=velocity_d[(offset_left-receiver_offset+boundary_left)*dimz+iz];
		
			if(ix>=0&&ix<(offset_left-receiver_offset+boundary_left))			
				velocity_d[id]=change;
		}

		if(offset_right>receiver_offset)
		{
			change=velocity_d[(source_x_cord+receiver_offset+boundary_left)*dimz+iz];
			if(ix>=(source_x_cord+receiver_offset+boundary_left)&&ix<dimx)			
				velocity_d[id]=change;
		}	
	}
}

__global__ void smooth_acqusition(float *all_grad_density1_d,int nx,int nz,int *offset_left_d,int *offset_right_d,int *source_x_cord_d,int shot_num)
///smooth_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,nx,nz,offset_left_d,offset_right_d,source_x_cord_d,shot_num);
{

	int ix=threadIdx.x+blockIdx.x*blockDim.x;
	int iz=threadIdx.y+blockIdx.y*blockDim.y;

	int id;
	int ishot;

	if(ix<nx&&iz<nz)
	{
		//id=ix*nz+iz;

		for(ishot=0;ishot<shot_num;ishot++)
		{
			//if(source_x_cord_d[ishot]-offset_left_d[ishot]!=0)
			if((source_x_cord_d[ishot]-offset_left_d[ishot])!=0)
			{
				id=(source_x_cord_d[ishot]-offset_left_d[ishot])*nz+iz;
				all_grad_density1_d[id]=(all_grad_density1_d[id+nz]+all_grad_density1_d[id-nz])/2.0;
				//all_grad_density1_d[id]=0;
			}

			//if(source_x_cord_d[ishot]+offset_right_d[ishot]!=nx)
			if((source_x_cord_d[ishot]+offset_right_d[ishot])!=nx)
			{
				id=(source_x_cord_d[ishot]+offset_right_d[ishot])*nz+iz;
				all_grad_density1_d[id]=(all_grad_density1_d[id+nz]+all_grad_density1_d[id-nz])/2.0;
				//all_grad_density1_d[id]=0;
			}
		}
	}
}

__global__ void cuda_attenuation_truncation(float *grad_vp1_d,int nx,int nz,int offset_left,int offset_right,int receiver_offset)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	int distance_left;
	int distance_right;

	float m=0.0;

	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		
		distance_left=int(offset_left/3);
		distance_right=int(offset_right/3);

		if(receiver_offset!=0)
		{
			if(ix<=distance_left&&offset_left!=0)
			{
				m=1.0*(distance_left-ix);

				change=pow(cos(pai/2*m/distance_left),3);

				//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.1*m*m));

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}

			if(ix>=nx-distance_right&&offset_right!=0)
			{
				m=1.0*(ix-nx+distance_right);

				change=pow(cos(pai/2*m/distance_right),3);

				//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.1*m*m));

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}
	}
}

__global__ void cauda_zero_and_attenuation_truncation(float *grad_vp1_d,int nx,int nz,int offset_left,int offset_right,int receiver_offset,int offset_attenuation)
//cauda_zero_and_attenuation_truncation<<<dimGrid,dimBlock>>>(grad_den1_d,nx_size,nz,offset_left[ishot],offset_right[ishot],receiver_offset);
{
/////////////////////////attenuation for boundary value in part receiver <receiver_offset
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	int distance_left;
	int distance_right;
	
	//int beg,end;

	float m=0.0;

	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		
		if(offset_left>receiver_offset)
		{
			if(ix>=0&&ix<(offset_left-receiver_offset))
				grad_vp1_d[in_idx]=0;

			distance_left=int(receiver_offset/offset_attenuation);

			if(distance_left<=2)		distance_left=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

			if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0)
			{
				m=1.0*(offset_left-receiver_offset+distance_left-ix);

				change=pow(cos(pai/2*m/distance_left),2);				

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}

		if(offset_right>receiver_offset)
		{
			if(ix>=(offset_left+receiver_offset)&&ix<nx)
				grad_vp1_d[in_idx]=0;
			
			distance_right=int(receiver_offset/offset_attenuation);

			if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

			if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0)
			{
				m=1.0*(ix-(offset_left+receiver_offset-distance_right));

				change=pow(cos(pai/2*m/distance_right),2);

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}	
	
	}
}

__global__ void cauda_zero_and_attenuation_truncation_old(float *grad_vp1_d,int nx,int nz,int offset_left,int offset_right,int receiver_offset)
//cauda_zero_and_attenuation_truncation<<<dimGrid,dimBlock>>>(grad_den1_d,nx_size,nz,offset_left[ishot],offset_right[ishot],receiver_offset);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	int distance_left;
	int distance_right;
	
	//int beg,end;

	float m=0.0;

	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		
		if(offset_left>receiver_offset)
		{
			if(ix>=0&&ix<(offset_left-receiver_offset))
				grad_vp1_d[in_idx]=0;

			distance_left=int(receiver_offset/3);

			if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0)
			{
				m=1.0*(offset_left-receiver_offset+distance_left-ix);

				change=pow(cos(pai/2*m/distance_left),3);				

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}

		if(offset_right>receiver_offset)
		{
			if(ix>=(offset_left+receiver_offset)&&ix<nx)
				grad_vp1_d[in_idx]=0;
			
			distance_right=int(receiver_offset/3);

			if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0)
			{
				m=1.0*(ix-(offset_left+receiver_offset-distance_right));

				change=pow(cos(pai/2*m/distance_right),3);

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}	
	
	}
}
