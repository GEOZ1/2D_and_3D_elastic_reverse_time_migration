#include "hip/hip_runtime.h"
#include <time.h>
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <malloc.h>
#include <math.h>
#include "hipblas.h"
// includes, project
#include <hipfft/hipfft.h>
//#include <cutil_inline.h>
//#include <shrQATest.h>
#include "su.h"
#include "segy.h"
#include "Complex.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "zzzzz"
#include "elastic_2D_kernel_1.cu"
#include "elastic_2D_kernel_2.cu"
#include "elastic_2D_kernel_3.cu"
#include "elastic_adjoint_equation.cu"
#include "viscoelastic_equation.cu"

#define radius 6
#define pai 3.1415926
#define Block_Size  512	/* vector computation blocklength */
#define scale 9

/*********************** self documentation ******************************/
char *sdoc[] = {
"                                                                        ",
" this is a program to model elastic zhengyan by FD ",
" this was created by zhange in Daqing in 2015-09-06 ",
" Prestack visco/elastic LSRTM in 2017-8-31 ",
" Prestack           Prestack               Prestack"
"                                                                        ",
NULL};
/**************** end self doc *******************************************/
segy tr;

//static time_t t1,t2;
//----------------------------- main -------------------------------------------
int main(int argc, char **argv)
{
		//requestdoc(1);
		initargs(argc,argv);

		int iter_start,niter,iter,join_vs,join_den,join_shot,precon,precon_z1,precon_z2,laplace,laplace_compensate,rbell,inversion_para,inversion_den;
		int nx,nz,nx_append,nx_append_new,nz_append,nxnz,nxanza,nx_size_nz,nxa_new_nza,lt_rec;
		float dx,dz;
		int lt;
		float dt;
		float freq;
		int wavelet_length,wavelet_half;
		int shot_num,shot_start,shot_interval,shot_depth;
		int receiver_num,receiver_start,receiver_interval,receiver_depth,receiver_offset,receiver_mark;
		int vsp,shot_z_interval,receiver_z_interval,decomposition;/////////////////for vsp.............2017年03月14日 星期二 08时32分25秒
		int vsp_2,receiver_start_2,receiver_interval_2,receiver_depth_2,receiver_z_interval_2,receiver_num_2;//for vsp2...2017年03月17日 星期二 08时32分25秒
		int vsp_precon;//for vsp2...2017年03月17日 星期二 08时32分25秒  
		int boundary_up,boundary_down,boundary_left,boundary_right;
		float coe_attenuation,*cal_max,*obs_max,array_max;
		char filename[100],filename1[100];

		float smooth_time_vp,smooth_time_vs,smooth_time_density,smooth_time_qp,smooth_time_qs;////velocity smooth time
		int length_vp,length_vs,length_density,length_qp,length_qs;////velocity smooth time2017年08月15日 星期二 09时57分09秒 

		int cuda_code;
		float *rep;
		FILE *logfile=NULL,*rm_f_file=NULL;float mstimer=0.0,totaltime=0.0;
		logfile=fopen("log.txt","ab");//remember to free log file

		int ittt_beg=0,ittt_end=0;
		int join_wavefield,RTM_only;

		float gpumem=0.0,gpumem_residual=0.0,change=0.0;/////for check 2017年07月27日 星期四 10时11分40秒 
		int check_number,check_interval,check_residual,variable_number;/////for check 2017年07月27日 星期四 10时11分40秒 
		if(!getparint("variable_number",&variable_number))		err("variable_number is not set!");/////for check 2017年07月27日 星期四 10时11分40秒		
		
		int migration_type,modeling_type,accumulation,correlation_misfit,amplitude_error,amplitude_error_number,cut_direct_wave,shot_scale,offset_attenuation;///2017年07月27日 星期四 20时17分11秒 
		if(!getparint("migration_type",&migration_type))				err("migration_type is not set!");////QQQQQQQ2017年07月30日 星期日 09时23分52秒 
		if(!getparint("modeling_type",&modeling_type))				err("modeling_type is not set!");
		if(!getparint("accumulation",&accumulation))				err("accumulation is not set!");/////for add virtual source
		if(!getparint("correlation_misfit",&correlation_misfit))			err("correlation_misfit is not set!");/////for correlation_misfit 
		if(!getparint("amplitude_error",&amplitude_error))				err("amplitude_error is not set!");/////for correlation_misfit 
		if(!getparint("amplitude_error_number",&amplitude_error_number))		err("amplitude_error_number is not set!");/////for correlation_misfit 
		if(!getparint("shot_scale",&shot_scale))					err("shot_scale is not set!");/////for correlation_misfit 
		if(!getparint("cut_direct_wave",&cut_direct_wave))				err("cut_direct_wave is not set!");/////for correlation_misfit

		if(!getparint("offset_attenuation",&offset_attenuation))			err("offset_attenuation is not set!");/////for correlation_misfit 
		//migration_type=1  denote: viscoelastic migration 
		//migration_type=0  denote: elastic migration
		//modeling_type=1  denote:  viscoelastic modeling 
		//modeling_type=0  denote:  elastic modeling 
		if(!getparint("join_wavefield",&join_wavefield))		err("join_wavefield is not set!"); 
		if(!getparint("RTM_only",&RTM_only))			err("RTM_only is not set!"); 

		if(!getparint("cuda_code",&cuda_code))			err("cuda_code is not set!");
		hipSetDevice(cuda_code);

		if(!getparint("inversion_para",&inversion_para))		err("inversion_para is not set!");//////////   inversion      parameter
		if(inversion_para==0||inversion_para==1)			warn("inversion parameter is lame coefficient\n");
		if(inversion_para==2)					warn("inversion parameter is velocity\n");
		if(inversion_para==3)					warn("inversion parameter is impedance\n");

		if(!getparint("inversion_den",&inversion_den))		err("inversion_den is not set!");//////////   inversion      parameter
		if(inversion_den==0)						warn("inversion parameter have not density\n");
		if(inversion_den==1)						warn("inversion parameter have density\n");
		/////////
		if(!getparint("precon",&precon))				err("precon is not set!");
		if(!getparint("precon_z1",&precon_z1))			err("precon_z1 is not set!");
		if(!getparint("precon_z2",&precon_z2))			err("precon_z2 is not set!");

		if(!getparint("laplace",&laplace))				err("laplace is not set!");
		if(!getparint("laplace_compensate",&laplace_compensate))				err("laplace_compensate is not set!");

		if(!getparint("rbell",&rbell))				rbell=2;
		if(!getparint("iter_start",&iter_start))			err("iter_start is not set!");///////////for program died ,restart
		if(!getparint("niter",&niter))				err("niter is not set!");

		if(!getparint("join_vs",&join_vs))				err("join_vs is not set!");////join_vs=1 denote input s wave velocity(velocity1)
		if(!getparint("join_den",&join_den))			err("join_den is not set!");////join_den=1 denote input density
		if(!getparint("join_shot",&join_shot))			err("join_shot is not set!");////join_shot=1 denote obs_x_shot and obs_z_shot has gotten

		if(!getparint("nx",&nx))					err("nx is not set!");
		if(!getparint("nz",&nz))					err("nz is not set!");
		if(!getparfloat("dx",&dx))					err("dx is not set!");
		if(!getparfloat("dz",&dz))					err("dz is not set!");

		if(!getparint("lt",&lt))					err("lt is not set!");
		if(!getparfloat("dt",&dt))					err("dt is not set!");
		if(!getparfloat("freq",&freq))				err("freq is not set!");

		if(!getparint("shot_num",&shot_num))			err("shot_num is not set!");
		if(!getparint("shot_start",&shot_start))			err("shot_start is not set!");
		if(!getparint("shot_interval",&shot_interval))		err("shot_interval is not set!");
		if(!getparint("shot_depth",&shot_depth))			err("shot_depth is not set!");

		if(!getparint("receiver_num",&receiver_num))		err("receiver_num is not set!");
		if(!getparint("receiver_start",&receiver_start))		err("receiver_start is not set!");
		if(!getparint("receiver_interval",&receiver_interval))	err("receiver_interval is not set!");
		if(!getparint("receiver_depth",&receiver_depth))		err("receiver_depth is not set!");
		if(!getparint("receiver_offset",&receiver_offset))		err("receiver_offset is not set!");
		if(!getparint("receiver_mark",&receiver_mark))		err("receiver_mark is not set!");
/////////for vsp???/////////////////for vsp.............2017年03月14日 星期二 08时32分25秒
		if(!getparint("decomposition",&decomposition))			err("decomposition is not set!"); 
		if(!getparint("vsp",&vsp))						err("vsp is not set!");
		if(!getparint("receiver_z_interval",&receiver_z_interval))	err("receiver_z_interval is not set!");
		if(!getparint("shot_z_interval",&shot_z_interval))			err("shot_z_interval is not set!");

///////////////////////for vsp2222222222222222222
		if(!getparint("vsp_2",&vsp_2))						err("vsp_2 is not set!");
		if(vsp_2!=0)
		{
			if(!getparint("receiver_start_2",&receiver_start_2))		err("receiver_start_2 is not set!");
			if(!getparint("receiver_interval_2",&receiver_interval_2))	err("receiver_interval_2 is not set!");
			if(!getparint("receiver_depth_2",&receiver_depth_2))		err("receiver_depth_2 is not set!");
			if(!getparint("receiver_z_interval_2",&receiver_z_interval_2))	err("receiver_z_interval_2 is not set!");		
			if(!getparint("receiver_num_2",&receiver_num_2))			err("receiver_num_2 is not set!");
		}
		if(!getparint("vsp_precon",&vsp_precon))					err("vsp_precon is not set!");
/////////for vsp???/////////////////for vsp.............2017年03月14日 星期二 08时32分25秒 
		if(!getparint("boundary_up",&boundary_up))				err("boundary_up is not set!");
		if(!getparint("boundary_down",&boundary_down))			err("boundary_down is not set!");
		if(!getparint("boundary_left",&boundary_left))			err("boundary_left is not set!");
		if(!getparint("boundary_right",&boundary_right))			err("boundary_right is not set!");
		if(!getparfloat("coe_attenuation",&coe_attenuation))		err("coe_attenuation is not set!");

		//if(!getparint("smooth_time",&smooth_time))			err("smooth_time is not set!");

		if(!getparfloat("smooth_time_vp",&smooth_time_vp))			err("smooth_time_vp is not set!");///velocity smooth 2017年08月15日 星期二 09时57分09秒 

		if(!getparfloat("smooth_time_vs",&smooth_time_vs))			err("smooth_time_vs is not set!");///velocity smooth 2017年08月15日 星期二 09时57分09秒

		if(!getparfloat("smooth_time_density",&smooth_time_density))	err("smooth_time_density is not set!");///velocity smooth 2017年08月15日 星期二 09时57分09秒

		if(!getparfloat("smooth_time_qp",&smooth_time_qp))			err("smooth_time_qp is not set!");////QQQQQQQ2017年07月30日 星期日 09时23分52秒 

		if(!getparfloat("smooth_time_qs",&smooth_time_qs))			err("smooth_time_qs is not set!");
		
		char *velocity_name;
		if(!getparstring("velocity",&velocity_name))			err("can not read velocity model!");

		char *velocity1_name;
		if(!getparstring("velocity1",&velocity1_name))			err("can not read velocity1 model!");

		char *density_name;
		if(!getparstring("density",&density_name))				err("can not read density model!");

		char *qp_name;
		if(!getparstring("qp_model",&qp_name))				err("can not read qp_model!");////////QQQQ2017年07月27日 星期四 19时47分30秒

		char *qs_name;
		if(!getparstring("qs_model",&qs_name))				err("can not read qs_model!");////////QQQQQ2017年07月27日 星期四 19时47分30秒
		
		//////////2017年03月07日 星期二 21时46分03秒 
		char *s_velocity_name;
		if(smooth_time_vp==0)
		{		
			if(!getparstring("s_velocity",&s_velocity_name))		err("can not read s_velocity model!");

		}///velocity smooth 2017年08月15日 星期二 09时57分09秒

		char *s_velocity1_name;
		if(smooth_time_vs==0)
		{
			if(!getparstring("s_velocity1",&s_velocity1_name))		err("can not read s_velocity1 model!");
		}///velocity smooth 2017年08月15日 星期二 09时57分09秒

		char *s_density_name;
		if(smooth_time_density==0)
		{	
			if(!getparstring("s_density",&s_density_name))		err("can not read s_density model!");
		}///velocity smooth 2017年08月15日 星期二 09时57分09秒

		char *s_qp_name;
		if(smooth_time_qp==0)
		{		
			if(!getparstring("s_qp_model",&s_qp_name))			err("can not read s_qp_name model!");
		}///velocity smooth 2017年08月15日 星期二 09时57分09秒

		char *s_qs_name;
		if(smooth_time_qs==0)
		{
			if(!getparstring("s_qs_model",&s_qs_name))			err("can not read s_qs_name model!");
		}///velocity smooth 2017年08月15日 星期二 09时57分09秒

		if(join_vs==0)					fprintf(logfile,"vs has not been joined\n");
		if(join_vs==1)					fprintf(logfile,"vs has been joined\n");
		if(join_den==0)					fprintf(logfile,"density has not been joined\n");
		if(join_den==1)					fprintf(logfile,"density has been joined\n");
		if(inversion_para==0||inversion_para==1)		fprintf(logfile,"inversion parameter is lame coefficient\n");
		if(inversion_para==2)				fprintf(logfile,"inversion parameter is velocity\n");
		if(inversion_para==3)				fprintf(logfile,"inversion parameter is impedance\n");
		if(inversion_den==0)					fprintf(logfile,"inversion has not density\n");
		if(inversion_den==1)					fprintf(logfile,"inversion has density\n");

		/* creat timing variables on device */
		hipEvent_t start, stop;
  		hipEventCreate(&start);	
		hipEventCreate(&stop);			
//////////2017年03月07日 星期二 21时46分03秒 
		char *outfile_name;
		if(!getparstring("outfile_name",&outfile_name))    	err("can not read outfile_name!");
//////////////We compile this program, if there is no this file, a disadvantage that segmentation fault is not easy to find. But we can read this file and output "warn". Anthoer solution is opening this file in there and close this.

//////////////////////////////////////////////////////input 

		float *wavelet,*wavelet_integral;		
		wavelet=make_ricker_new(freq,dt/1000.0,&wavelet_length);
		wavelet_integral=make_ricker_new(freq,dt/1000.0,&wavelet_length);
		wavelet_half=wavelet_length/2;
		/*wavelet=alloc1float(200);
		wavelet_integral=alloc1float(200);
		set_zero_1d(wavelet,200);
		make_ricker_initial(wavelet,freq,dt,200);
		make_ricker_initial(wavelet_integral,freq,dt,200);
		wavelet_length=200;
		wavelet_half=wavelet_length/2;*/
		warn("Ricker wavelet is set   wavelet_length=%d,wavelet_half=%d\n",wavelet_length,wavelet_half);

		float *coe_opt;
		coe_opt=alloc1float(radius+1);
		make_coe_optimized_new(coe_opt);
		float *coe_opt1;
		coe_opt1=alloc1float(radius+1);
		make_coe_optimized1_new(coe_opt1);
		float  coe_x;
		coe_x=dt/(1000.0*dx);
		float  coe_z;
		coe_z=dt/(1000.0*dz);
		
		if(laplace_compensate!=0)/////////////////////////twice integral
		{
			intergrating_seismic(wavelet_integral,wavelet_length,1);
			intergrating_seismic(wavelet_integral,wavelet_length,1);
			//derivation(wavelet,wavelet_length,coe_opt);
		}

/////////////////////////////smoother
		float *tmp;
		tmp=alloc1float(nx*nz);
		fread_file_1d(tmp,nx,nz,velocity_name);
		//array_max=cpu_caculate_max(velocity,nx,nz);warn("velocity_max=%f\n",array_max); 	 fprintf(logfile,"velocity_max=%f\n",array_max);

		array_max=caculate_average_new(tmp,nx,nz);warn("velocity_average=%f\n",array_max); fprintf(logfile,"velocity_average=%f\n",array_max);
	
		int landa;///velocity smooth 2017年08月15日 星期二 09时57分09秒
		landa=int(array_max/2.0/(freq*1.0)/dx);warn("landa=%d\n",landa);     fprintf(logfile,"landa=%d\n",landa);

		//landa=int(2000.0/2.0*wavelet_length*0.001/dx);warn("landa=%d\n",landa);   fprintf(logfile,"landa=%d\n",landa);

		length_vp=int(landa*smooth_time_vp);
		length_vs=int(landa*smooth_time_vs);
		length_density=int(landa*smooth_time_density);
		length_qp=int(landa*smooth_time_qp);
		length_qs=int(landa*smooth_time_qs);

		landa=max(max(max(max(length_vp,length_vs),length_density),length_qp),length_qs);

		warn("max=%d,length_vp=%d,length_vs=%d,length_density=%d,length_qp=%d,length_qs=%d",
			landa,length_vp,length_vs,length_density,length_qp,length_qs);

		fprintf(logfile,"max=%d,length_vp=%d,length_vs=%d,length_density=%d,length_qp=%d,length_qs=%d\n",
			landa,length_vp,length_vs,length_density,length_qp,length_qs);
	
		if(landa>boundary_up)	boundary_up=landa;	
		if(landa>boundary_down)	boundary_down=landa;
		if(landa>boundary_left)	boundary_left=landa;
		if(landa>boundary_right)	boundary_right=landa;

		fclose(logfile);
		logfile=fopen("log.txt","ab");//remember to free log file	
/////////////////////////////smoother

/////////////////////new  acquisition way 2017年08月16日 星期三 20时31分29秒 
		int *source_x_cord;
		source_x_cord=alloc1int(shot_num);
		for(int is=0;is<shot_num;is++)
				source_x_cord[is]=shot_start+is*shot_interval;

		int *receiver_x_cord;
		receiver_x_cord=alloc1int(shot_num);
		if(0==receiver_offset)
		{
			for(int is=0;is<shot_num;is++)
				receiver_x_cord[is]=receiver_start;
		}
		else
		{
			for(int is=0;is<shot_num;is++)
			{
				receiver_x_cord[is]=source_x_cord[is]-receiver_offset;

				if(receiver_x_cord[is]<0)	receiver_x_cord[is]=0;/////////////////////new  acquisition way 2017年08月16日 星期三 20时31分29秒

				if(receiver_x_cord[is]+receiver_interval*receiver_num>=nx) receiver_x_cord[is]=nx-receiver_interval*receiver_num;//new  acquisition way
			} 
		}

		/*int *acqusition_left,*acqusition_right;
		acqusition_left=alloc1int(shot_num);memset(acqusition_left,0,shot_num*sizeof(int));
		acqusition_right=alloc1int(shot_num);memset(acqusition_right,0,shot_num*sizeof(int));
		if(0!=receiver_offset)
		{
			for(int is=0;is<shot_num;is++)
			{
				if(source_x_cord[is]-receiver_offset<0)	
					acqusition_left[is]=-1*(source_x_cord[is]-receiver_offset);

				//if(acqusition_left[is]!=0)	warn("acqusition_left[%d]=%d\n",is,acqusition_left[is]);

				if(source_x_cord[is]-receiver_offset+receiver_interval*receiver_num>=nx)	
					acqusition_right[is]=source_x_cord[is]-receiver_offset-receiver_x_cord[is];

				//if(acqusition_right[is]!=0)	warn("acqusition_right[%d]=%d\n",is,acqusition_right[is]);	
			}
		}*/

		int *offset_left,*offset_right;
		//int *offset_left_d,*offset_right_d,*source_x_cord_d;
		offset_left=alloc1int(shot_num);				memset(offset_left,0,shot_num*sizeof(int));
		offset_right=alloc1int(shot_num);				memset(offset_right,0,shot_num*sizeof(int));
		//hipMalloc(&offset_left_d,shot_num*sizeof(int));		hipMemcpy(offset_left_d,offset_left,shot_num*sizeof(int),hipMemcpyHostToDevice);
		//hipMalloc(&offset_right_d,shot_num*sizeof(int));	hipMemcpy(offset_right_d,offset_right,shot_num*sizeof(int),hipMemcpyHostToDevice);
		//hipMalloc(&source_x_cord_d,shot_num*sizeof(int));	hipMemcpy(source_x_cord_d,source_x_cord,shot_num*sizeof(int),hipMemcpyHostToDevice);
		if(0!=receiver_offset)
		{
			for(int is=0;is<shot_num;is++)
			{
				offset_left[is]=source_x_cord[is]-receiver_x_cord[is];

				offset_right[is]=receiver_x_cord[is]+receiver_num*receiver_interval-source_x_cord[is];	
			
				//if((is%20)==0)	warn("offset_left[%d]=%d,offset_right[%d]=%d\n",is,offset_left[is],is,offset_right[is]);
			}
		}
		//hipMalloc(&offset_left_d,shot_num*sizeof(int));		hipMemcpy(offset_left_d,offset_left,shot_num*sizeof(int),hipMemcpyHostToDevice);
		//hipMalloc(&offset_right_d,shot_num*sizeof(int));	hipMemcpy(offset_right_d,offset_right,shot_num*sizeof(int),hipMemcpyHostToDevice);
		//hipMalloc(&source_x_cord_d,shot_num*sizeof(int));	hipMemcpy(source_x_cord_d,source_x_cord,shot_num*sizeof(int),hipMemcpyHostToDevice);
/////////////////////new  acquisition way 2017年08月16日 星期三 20时31分29秒  
/////////////////////new  acquisition way 2017年08月16日 星期三 20时31分29秒  
		int *imaging_start,*imaging_size,*imaging_end,nx_size;
		imaging_start=alloc1int(shot_num);
		imaging_size=alloc1int(shot_num);
		imaging_end=alloc1int(shot_num);
		for(int is=0;is<shot_num;is++)
		{
			imaging_start[is]=receiver_x_cord[is];
			imaging_end[is]=receiver_x_cord[is]+receiver_interval*receiver_num;
			imaging_size[is]=imaging_end[is]-imaging_start[is];
			nx_size=imaging_size[0];
		}
			
		if(nx_size==nx)
		{
			nx_append=nx+boundary_left+boundary_right;
			nz_append=nz+boundary_up+boundary_down;
		}
		if(nx_size!=nx)
		{
			nx_append=nx_size+boundary_left+boundary_right;
			nz_append=nz+boundary_up+boundary_down;
		}			
/////////////////////new  acquisition way 2017年08月16日 星期三 20时31分29秒  

///////////////////////for vsp.............2017年03月14日 星期二 08时32分25秒 
		int *source_z_cord;
		source_z_cord=alloc1int(shot_num);
		for(int is=0;is<shot_num;is++)
				source_z_cord[is]=shot_depth+is*shot_z_interval;

		int *receiver_z_cord;
		receiver_z_cord=alloc1int(shot_num);
		for(int is=0;is<shot_num;is++)
				receiver_z_cord[is]=receiver_depth;
///////////////////////for vsp.............2017年03月14日 星期二 08时32分25秒

///////////////////////for vsp2222222222222222222
		//if(vsp_2!=0)
		//{
			int *receiver_x_cord_2;
			receiver_x_cord_2=alloc1int(shot_num);	
				for(int is=0;is<shot_num;is++)
					receiver_x_cord_2[is]=receiver_start_2;
			int *receiver_z_cord_2;
			receiver_z_cord_2=alloc1int(shot_num);
				for(int is=0;is<shot_num;is++)
					receiver_z_cord_2[is]=receiver_depth_2;
		//}
///////////////////////for vsp2222222222222222222
		
		nx_append_new=nx+boundary_left+boundary_right;
		nxnz=nx*nz;////////////////////////////////////////all mode
		nxa_new_nza=nx_append_new*nz_append;//////////all model +boundary			

		nx_size_nz=nx_size*nz;////////////////////////////////calculated mode
		nxanza=nx_append*nz_append;/////////////////////calculated mode+boundary

		lt_rec=lt*receiver_num;		

		warn("nxnz=%d,nx=%d,nz=%d,dx=%f,dz=%f",nxnz,nx,nz,dx,dz);
		warn("lt=%d,dt=%f,freq=%f",lt,dt,freq);
		warn("shot_num=%d,shot_start=%d",shot_num,shot_start);
		warn("shot_interval=%d,shot_depth=%d",shot_interval,shot_depth);
		warn("receiver_num=%d,receiver_start=%d",receiver_num,receiver_start);
		warn("receiver_interval=%d,receiver_depth=%d,receiver_offset=%d",receiver_interval,receiver_depth,receiver_offset);
		warn("coe_attenuation=%f",coe_attenuation);
		warn("nx_append=%d,nz_append=%d",nx_append,nz_append);
		
		
		float *shotgather,*shotgather1,*wf_shot;
		shotgather=alloc1float(lt_rec);		memset((void *) (shotgather), 0, lt_rec * sizeof (float));
		shotgather1=alloc1float(lt_rec);		memset((void *) (shotgather1), 0, lt_rec * sizeof (float));
		wf_shot=alloc1float(lt_rec);			memset((void *) (wf_shot), 0, lt_rec * sizeof (float));

		float *wf_append,*wf;
		wf_append=alloc1float(nxanza);		memset((void *) (wf_append), 0, nxanza * sizeof (float));

		wf=alloc1float(nx_size_nz);			memset((void *) (wf), 0, nx_size_nz * sizeof (float));

		float *wf_append_new,*wf_nxnz;
		wf_append_new=alloc1float(nxa_new_nza);	memset((void *) (wf_append_new), 0, nxa_new_nza * sizeof (float));
								//memset((void *) (wf_append), 0, nxa_new_nza * sizeof (float));
		wf_nxnz=alloc1float(nxnz);			memset((void *) (wf_nxnz), 0, nxnz * sizeof (float));

//////////////////parameters
		float *attenuation;		
		
		attenuation=alloc1float(nxanza);		memset((void *) (attenuation), 0, nxanza * sizeof (float));
		make_attenuation_new(attenuation,nx_size,nz,boundary_up,boundary_down,boundary_left,boundary_right,coe_attenuation);
///////////////parameters
//////////////fread_file_1d all  parameters
		float *velocity_all,*velocity1_all,*density_all,*qp_all,*qs_all;
		velocity_all=alloc1float(nxa_new_nza);	
		velocity1_all=alloc1float(nxa_new_nza);
		density_all=alloc1float(nxa_new_nza);
		qp_all=alloc1float(nxa_new_nza);
		qs_all=alloc1float(nxa_new_nza);
///////////////////for vp
					read_velocity_new(velocity_all,nx,nz,boundary_up,boundary_down,boundary_left,boundary_right,velocity_name);//read_real_vp
	
///////////////////for vs
		if(join_vs==1)	read_velocity_new(velocity1_all,nx,nz,boundary_up,boundary_down,boundary_left,boundary_right,velocity1_name);//read_real_vs

		else			vp_set_vs(velocity_all,velocity1_all,nx_append_new,nz_append);//set_real_vs by the  relation with vp

///////////////////for density
		if(join_den==1)	read_velocity_new(density_all,nx,nz,boundary_up,boundary_down,boundary_left,boundary_right,density_name);//read_real_density

		else			vp_set_density(velocity_all,density_all,nx_append_new,nz_append);//set_real_density by the  relation with vp

///////////////////for qp
		if(modeling_type!=0||migration_type!=0)
				{ 
					read_velocity_new(qp_all,nx,nz,boundary_up,boundary_down,boundary_left,boundary_right,qp_name);	
///////////////////for qs 
					read_velocity_new(qs_all,nx,nz,boundary_up,boundary_down,boundary_left,boundary_right,qs_name);
				}
////output nxanza size:
		write_file_1d(wavelet,wavelet_length,"./someoutput/wavelet2.bin");
		write_file_1d(wavelet_integral,wavelet_length,"./someoutput/wavelet3.bin");

		write_file_1d(attenuation,nxanza,"./someoutput/att.bin");

		write_file_1d(velocity_all,nxa_new_nza,"./someoutput/vp.bin");
	
		write_file_1d(velocity1_all,nxa_new_nza,"./someoutput/vs.bin");		

		write_file_1d(density_all,nxa_new_nza,"./someoutput/density.bin");		

		write_file_1d(qp_all,nxa_new_nza,"./someoutput/qp.bin");

		write_file_1d(qs_all,nxa_new_nza,"./someoutput/qs.bin");
////output nxanza size:
	
////output nxnz size:
		exchange(velocity_all,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
		write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-vp.bin");
		
		exchange(velocity1_all,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
		write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-vs.bin");
			
		exchange(density_all,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
		write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-density.bin");

		exchange(qp_all,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
		write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-qp.bin");
			
		exchange(qs_all,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
		write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-qs.bin");

		warn("Real_Velocity and Attenuation has been read!");
////output nxnz size:

		int nx_append_radius=nx_append-2*radius;//chu qu liangbian jie shu  
		int nz_append_radius=nz_append-2*radius;		

		dim3 dimBlock(32,16);

		dim3 dimGrid((nx_append+dimBlock.x-1)/dimBlock.x,(nz_append+dimBlock.y-1)/dimBlock.y);////cal

		dim3 dimGrid_new((nx_append_new+dimBlock.x-1)/dimBlock.x,(nz_append+dimBlock.y-1)/dimBlock.y);///all
		
		dim3 dimGrid_3nx_nz((3*nx+dimBlock.x-1)/dimBlock.x,(nz+dimBlock.y-1)/dimBlock.y);///for conjugated method

		dim3 dimGrid_3nx_size_nz((3*nx_size+dimBlock.x-1)/dimBlock.x,(nz+dimBlock.y-1)/dimBlock.y);///for conjugated method

		dim3 dimGrid_nx_nz((nx+dimBlock.x-1)/dimBlock.x,(nz+dimBlock.y-1)/dimBlock.y);

		//dim3 trans_dimGrid((nz_append+dimBlock.y-1)/dimBlock.y,(nx_append+dimBlock.x-1)/dimBlock.x);//smooth

		dim3 dimGrid_lt((receiver_num+dimBlock.x-1)/dimBlock.x,(lt+dimBlock.y-1)/dimBlock.y);///some operation on obs/cal seismic data

		int numofblock=((nx_append_radius+dimBlock.x-1)/dimBlock.x)*((nz_append_radius+dimBlock.y-1)/dimBlock.y);
		warn("num of block in x direction=%d",(nx_append_radius+dimBlock.x-1)/dimBlock.x);
		warn("num of block in z direction=%d",(nz_append_radius+dimBlock.y-1)/dimBlock.y);
		warn("num of block in total direction=%d",numofblock);
		
//////////////////////////////////////////correct velocity and smooth velocity2017年08月17日 星期四 08时40分12秒 
		float *velocity_all_d,*velocity1_all_d,*density_all_d,*qp_all_d,*qs_all_d;
		gpumem += (nxa_new_nza*5)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒
		hipMalloc(&velocity_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(velocity_all_d,velocity_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&velocity1_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(velocity1_all_d,velocity1_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&density_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(density_all_d,density_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&qp_all_d,nxa_new_nza*sizeof(float));		hipMemcpy(qp_all_d,qp_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&qs_all_d,nxa_new_nza*sizeof(float));		hipMemcpy(qs_all_d,qs_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);

		float *s_velocity_all_d,*s_velocity1_all_d,*s_density_all_d,*s_qp_all_d,*s_qs_all_d;
		gpumem += (nxa_new_nza*5)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒
		hipMalloc(&s_velocity_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(s_velocity_all_d,velocity_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_velocity1_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(s_velocity1_all_d,velocity1_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_density_all_d,nxa_new_nza*sizeof(float));	hipMemcpy(s_density_all_d,density_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_qp_all_d,nxa_new_nza*sizeof(float));		hipMemcpy(s_qp_all_d,qp_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_qs_all_d,nxa_new_nza*sizeof(float));		hipMemcpy(s_qs_all_d,qs_all,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);

		float *wf_d,*wf_append_d,*wf_nxnz_d,*wf_append_new_d;
		gpumem += (nxa_new_nza*2)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒
		hipMalloc(&wf_d,nx_size_nz*sizeof(float));		hipMemset(wf_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&wf_append_d,nxanza*sizeof(float));		hipMemset(wf_append_d,0,nxanza*sizeof(float));

		//hipMalloc(&wf_nxnz_d,nx_size_nz*sizeof(float));		hipMemset(wf_nxnz_d,0,nxnz*sizeof(float));
		hipMalloc(&wf_nxnz_d,nxnz*sizeof(float));			hipMemset(wf_nxnz_d,0,nxnz*sizeof(float));
		hipMalloc(&wf_append_new_d,nxa_new_nza*sizeof(float));	hipMemset(wf_append_new_d,0,nxa_new_nza*sizeof(float));

			////////////////////////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
			if(length_vp!=0)///////////vp
			{
				//cuda_bell_smoothx_new<<< dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_append_new_d,length_vp,nx_append_new,nz_append);
				//cuda_bell_smoothz_new<<< dimGrid_new,dimBlock>>>(wf_append_new_d,s_velocity_all_d,length_vp,nx_append_new,nz_append);

				cuda_bell_smooth_2d<<< dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_append_new_d,length_vp,nx_append_new,nz_append);	
				hipMemcpy(s_velocity_all_d,wf_append_new_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToDevice);
			}
				cuda_cal_window<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

			if(length_vs!=0)///////////vs
			{
				//cuda_bell_smoothx_new<<< dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_append_new_d,length_vs,nx_append_new,nz_append);
				//cuda_bell_smoothz_new<<< dimGrid_new,dimBlock>>>(wf_append_new_d,s_velocity1_all_d,length_vs,nx_append_new,nz_append);

				cuda_bell_smooth_2d<<< dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_append_new_d,length_vs,nx_append_new,nz_append);	
				hipMemcpy(s_velocity1_all_d,wf_append_new_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToDevice);	
			}
				cuda_cal_window<<<dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

			if(length_density!=0)///////////density
			{
				//cuda_bell_smoothx_new<<< dimGrid_new,dimBlock>>>(s_density_all_d,wf_append_new_d,length_density,nx_append_new,nz_append);
				//cuda_bell_smoothz_new<<< dimGrid_new,dimBlock>>>(wf_append_new_d,s_density_all_d,length_density,nx_append_new,nz_append);

				cuda_bell_smooth_2d<<< dimGrid_new,dimBlock>>>(s_density_all_d,wf_append_new_d,length_density,nx_append_new,nz_append);	
				hipMemcpy(s_density_all_d,wf_append_new_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToDevice);	
			}
				cuda_cal_window<<<dimGrid_new,dimBlock>>>(s_density_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid_new,dimBlock>>>(s_density_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

			if(length_qp!=0)///////////qp
			{
				//cuda_bell_smoothx_new<<< dimGrid_new,dimBlock>>>(s_qp_all_d,wf_append_new_d,length_qp,nx_append_new,nz_append);
				//cuda_bell_smoothz_new<<< dimGrid_new,dimBlock>>>(wf_append_new_d,s_qp_all_d,length_qp,nx_append_new,nz_append);

				cuda_bell_smooth_2d<<< dimGrid_new,dimBlock>>>(s_qp_all_d,wf_append_new_d,length_qp,nx_append_new,nz_append);	
				hipMemcpy(s_qp_all_d,wf_append_new_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToDevice);		
			}
				cuda_cal_window<<<dimGrid_new,dimBlock>>>(s_qp_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid_new,dimBlock>>>(s_qp_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

			if(length_qs!=0)///////////qs
			{
				//cuda_bell_smoothx_new<<< dimGrid_new,dimBlock>>>(s_qs_all_d,wf_append_new_d,length_qs,nx_append_new,nz_append);
				//cuda_bell_smoothz_new<<< dimGrid_new,dimBlock>>>(wf_append_new_d,s_qs_all_d,length_qs,nx_append_new,nz_append);

				cuda_bell_smooth_2d<<< dimGrid_new,dimBlock>>>(s_qs_all_d,wf_append_new_d,length_qs,nx_append_new,nz_append);	
				hipMemcpy(s_qs_all_d,wf_append_new_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToDevice);	
			}
				cuda_cal_window<<<dimGrid_new,dimBlock>>>(s_qs_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid_new,dimBlock>>>(s_qs_all_d,wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

				hipMemcpy(wf_append_new,s_velocity_all_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./someoutput/vp-s.bin");
				exchange(wf_append_new,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-vp-s.bin");

				hipMemcpy(wf_append_new,s_velocity1_all_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./someoutput/vs-s.bin");
				exchange(wf_append_new,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-vs-s.bin");

				hipMemcpy(wf_append_new,s_density_all_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./someoutput/density-s.bin");
				exchange(wf_append_new,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-density-s.bin");

				hipMemcpy(wf_append_new,s_qp_all_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./someoutput/qp-s.bin");
				exchange(wf_append_new,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-qp-s.bin");

				hipMemcpy(wf_append_new,s_qs_all_d,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./someoutput/qs-s.bin");
				exchange(wf_append_new,wf_nxnz,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
				write_file_1d(wf_nxnz,nxnz,"./someoutput/cut-qs-s.bin");

				warn("smooth velocity has been gotten!");
//////////////////////////////////////////correct velocity and smooth velocity2017年08月17日 星期四 08时40分12秒	
		
		float *qp_d,*qs_d,*s_qp_d,*s_qs_d,*tao_d,*strain_p_d,*strain_s_d,*modul_p_d,*modul_s_d;///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
		float *mem_p1_d,*mem_p2_d,*mem_xx1_d,*mem_xx2_d,*mem_zz1_d,*mem_zz2_d,*mem_xz1_d,*mem_xz2_d;
		float *rmem_p1_d,*rmem_p2_d,*rmem_xx1_d,*rmem_xx2_d,*rmem_zz1_d,*rmem_zz2_d,*rmem_xz1_d,*rmem_xz2_d;
		hipMalloc(&qp_d,nxanza*sizeof(float));	//hipMemcpy(qp_d,qp,nxanza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&qs_d,nxanza*sizeof(float));	//hipMemcpy(qs_d,qs,nxanza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_qp_d,nxanza*sizeof(float));	//hipMemcpy(s_qp_d,s_qp,nxanza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&s_qs_d,nxanza*sizeof(float));	//hipMemcpy(s_qs_d,s_qs,nxanza*sizeof(float),hipMemcpyHostToDevice);
		hipMalloc(&tao_d,nxanza*sizeof(float));
		hipMalloc(&strain_p_d,nxanza*sizeof(float));
		hipMalloc(&strain_s_d,nxanza*sizeof(float));
		hipMalloc(&modul_p_d,nxanza*sizeof(float));
		hipMalloc(&modul_s_d,nxanza*sizeof(float));///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 

		hipMalloc(&mem_p1_d,nxanza*sizeof(float));
		hipMalloc(&mem_p2_d,nxanza*sizeof(float));		
		hipMalloc(&mem_xx1_d,nxanza*sizeof(float));
		hipMalloc(&mem_xx2_d,nxanza*sizeof(float));
		hipMalloc(&mem_zz1_d,nxanza*sizeof(float));
		hipMalloc(&mem_zz2_d,nxanza*sizeof(float));
		hipMalloc(&mem_xz1_d,nxanza*sizeof(float));
		hipMalloc(&mem_xz2_d,nxanza*sizeof(float));

		hipMalloc(&rmem_p1_d,nxanza*sizeof(float));
		hipMalloc(&rmem_p2_d,nxanza*sizeof(float));		
		hipMalloc(&rmem_xx1_d,nxanza*sizeof(float));
		hipMalloc(&rmem_xx2_d,nxanza*sizeof(float));
		hipMalloc(&rmem_zz1_d,nxanza*sizeof(float));
		hipMalloc(&rmem_zz2_d,nxanza*sizeof(float));
		hipMalloc(&rmem_xz1_d,nxanza*sizeof(float));
		hipMalloc(&rmem_xz2_d,nxanza*sizeof(float));///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 

		gpumem += (nxanza*23)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒

////////////////////vsp 3.17
		//if(vsp_2!=0)
		//{
			float *obs_shot_x_d_2,*obs_shot_z_d_2,*cal_shot_x_d_2,*cal_shot_z_d_2,*res_shot_x_d_2,*res_shot_z_d_2;
			hipMalloc(&obs_shot_x_d_2,lt_rec*sizeof(float));		hipMemset(obs_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMalloc(&obs_shot_z_d_2,lt_rec*sizeof(float));		hipMemset(obs_shot_z_d_2,0,lt_rec*sizeof(float));
			hipMalloc(&cal_shot_x_d_2,lt_rec*sizeof(float));		hipMemset(cal_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMalloc(&cal_shot_z_d_2,lt_rec*sizeof(float));		hipMemset(cal_shot_z_d_2,0,lt_rec*sizeof(float));
			hipMalloc(&res_shot_x_d_2,lt_rec*sizeof(float));		hipMemset(res_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMalloc(&res_shot_z_d_2,lt_rec*sizeof(float));		hipMemset(res_shot_z_d_2,0,lt_rec*sizeof(float));
		//}
////////////////////vsp 3.17
		float *wavelet_d,*coe_opt_d,*coe_opt1_d,*s_velocity_d,*s_velocity1_d,*velocity_d,*velocity1_d,*attenuation_d,*density_d,*s_density_d;
		float *vx1_d,*vz1_d,*txx1_d,*tzz1_d,*txz1_d,*vx2_d,*vz2_d,*txx2_d,*tzz2_d,*txz2_d;
		float *rvx1_d,*rvz1_d,*rtxx1_d,*rtzz1_d,*rtxz1_d,*rvx2_d,*rvz2_d,*rtxx2_d,*rtzz2_d,*rtxz2_d;
		float *vx_t_d,*vz_t_d;///////vx of the direvation of time  vz of the direvation of time
		float *obs_shot_x_d,*obs_shot_z_d,*cal_shot_x_d,*cal_shot_z_d,*res_shot_x_d,*res_shot_z_d,*obs_shot_all_d,*cal_shot_all_d,*res_shot_all_d;
		float *cal_shot_x1_d,*cal_shot_z1_d,*obs_shot_x1_d,*obs_shot_z1_d,*res_shot_x1_d,*res_shot_z1_d,*res_shot_x2_d,*res_shot_z2_d;

		float *tmp_shot_x_d,*tmp_shot_z_d;
		float *adj_shot_x_d,*adj_shot_z_d;
		float *correlation_parameter_d;
		float *obj_parameter_d;
		//////////////tmp_shot_x_d:the sum of cal_shot in previous iteration for cross-correlation misfunction 2017年08月25日 星期五 09时28分54秒 

		hipMalloc(&cal_max,1*sizeof(float));
		hipMalloc(&obs_max,1*sizeof(float));
		hipMalloc(&wavelet_d,wavelet_length*sizeof(float));
		hipMalloc(&coe_opt_d,(radius+1)*sizeof(float));
		hipMalloc(&coe_opt1_d,(radius+1)*sizeof(float));
		hipMalloc(&velocity_d,nxanza*sizeof(float));
		hipMalloc(&velocity1_d,nxanza*sizeof(float));
		hipMalloc(&s_velocity_d,nxanza*sizeof(float));
		hipMalloc(&s_velocity1_d,nxanza*sizeof(float));
		hipMalloc(&attenuation_d,nxanza*sizeof(float));
		hipMalloc(&density_d,nxanza*sizeof(float));
		hipMalloc(&s_density_d,nxanza*sizeof(float));
		hipMalloc(&vx1_d,nxanza*sizeof(float));
		hipMalloc(&vz1_d,nxanza*sizeof(float));
		hipMalloc(&txx1_d,nxanza*sizeof(float));
		hipMalloc(&tzz1_d,nxanza*sizeof(float));
		hipMalloc(&txz1_d,nxanza*sizeof(float));
		hipMalloc(&vx2_d,nxanza*sizeof(float));
		hipMalloc(&vz2_d,nxanza*sizeof(float));
		hipMalloc(&txx2_d,nxanza*sizeof(float));
		hipMalloc(&tzz2_d,nxanza*sizeof(float));
		hipMalloc(&txz2_d,nxanza*sizeof(float));
		hipMalloc(&rvx1_d,nxanza*sizeof(float));
		hipMalloc(&rvz1_d,nxanza*sizeof(float));
		hipMalloc(&rtxx1_d,nxanza*sizeof(float));
		hipMalloc(&rtzz1_d,nxanza*sizeof(float));
		hipMalloc(&rtxz1_d,nxanza*sizeof(float));
		hipMalloc(&rvx2_d,nxanza*sizeof(float));
		hipMalloc(&rvz2_d,nxanza*sizeof(float));
		hipMalloc(&rtxx2_d,nxanza*sizeof(float));
		hipMalloc(&rtzz2_d,nxanza*sizeof(float));
		hipMalloc(&rtxz2_d,nxanza*sizeof(float));
		hipMalloc(&vx_t_d,nxanza*sizeof(float));
		hipMalloc(&vz_t_d,nxanza*sizeof(float));
		/*this is a falut smilar with free space happen error */
		hipMalloc(&obs_shot_x_d,lt_rec*sizeof(float));			hipMemset(obs_shot_x_d,0,lt_rec*sizeof(float));
		hipMalloc(&obs_shot_z_d,lt_rec*sizeof(float));			hipMemset(obs_shot_z_d,0,lt_rec*sizeof(float));
		hipMalloc(&cal_shot_x_d,lt_rec*sizeof(float));			hipMemset(cal_shot_x_d,0,lt_rec*sizeof(float));
		hipMalloc(&cal_shot_z_d,lt_rec*sizeof(float));			hipMemset(cal_shot_z_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_x_d,lt_rec*sizeof(float));			hipMemset(res_shot_x_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_z_d,lt_rec*sizeof(float));			hipMemset(res_shot_z_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_x1_d,lt_rec*sizeof(float));			hipMemset(res_shot_x1_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_z1_d,lt_rec*sizeof(float));			hipMemset(res_shot_z1_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_x2_d,lt_rec*sizeof(float));			hipMemset(res_shot_x2_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_z2_d,lt_rec*sizeof(float));			hipMemset(res_shot_z2_d,0,lt_rec*sizeof(float));
		hipMalloc(&obs_shot_all_d,lt_rec*sizeof(float));			hipMemset(obs_shot_all_d,0,lt_rec*sizeof(float));
		hipMalloc(&cal_shot_all_d,lt_rec*sizeof(float));			hipMemset(cal_shot_all_d,0,lt_rec*sizeof(float));
		hipMalloc(&res_shot_all_d,lt_rec*sizeof(float));			hipMemset(res_shot_all_d,0,lt_rec*sizeof(float));
		hipMalloc(&cal_shot_x1_d,lt_rec*sizeof(float));			hipMemset(cal_shot_x1_d,0,lt_rec*sizeof(float));
		hipMalloc(&cal_shot_z1_d,lt_rec*sizeof(float));			hipMemset(cal_shot_z1_d,0,lt_rec*sizeof(float));
		hipMalloc(&obs_shot_x1_d,lt_rec*sizeof(float));			hipMemset(obs_shot_x1_d,0,lt_rec*sizeof(float));
		hipMalloc(&obs_shot_z1_d,lt_rec*sizeof(float));			hipMemset(obs_shot_z1_d,0,lt_rec*sizeof(float));

		hipMalloc(&tmp_shot_x_d,lt_rec*sizeof(float));			hipMemset(tmp_shot_x_d,0,lt_rec*sizeof(float));
		hipMalloc(&tmp_shot_z_d,lt_rec*sizeof(float));			hipMemset(tmp_shot_z_d,0,lt_rec*sizeof(float));
		hipMalloc(&adj_shot_x_d,lt_rec*sizeof(float));			hipMemset(adj_shot_x_d,0,lt_rec*sizeof(float));
		hipMalloc(&adj_shot_z_d,lt_rec*sizeof(float));			hipMemset(adj_shot_z_d,0,lt_rec*sizeof(float));
		hipMalloc(&correlation_parameter_d,10*sizeof(float));		hipMemset(correlation_parameter_d,0,10*sizeof(float));
		hipMalloc(&obj_parameter_d,3*sizeof(float));			hipMemset(obj_parameter_d,0,3*sizeof(float));//for cross-correlation misfunction
		
		hipMemcpy(wavelet_d,wavelet,wavelet_length*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(coe_opt_d,coe_opt,(radius+1)*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(coe_opt1_d,coe_opt1,(radius+1)*sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(velocity_d,velocity,nxanza*sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(s_velocity_d,s_velocity,nxanza*sizeof(float),hipMemcpyHostToDevice);	
		//hipMemcpy(density_d,density,nxanza*sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(s_density_d,s_density,nxanza*sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(velocity1_d,velocity1,nxanza*sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(s_velocity1_d,s_velocity1,nxanza*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(attenuation_d,attenuation,nxanza*sizeof(float),hipMemcpyHostToDevice);


		gpumem += (nxanza*48)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒
		gpumem += (lt_rec*4)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒

		/////////////////when we used viscoelastic modeling or migration, we don't perform wavefield construction. why???
		/*float *vxu_d,*vxd_d,*vxl_d,*vxr_d,*vzu_d,*vzd_d,*vzl_d,*vzr_d;
		hipMalloc(&vxu_d,radius*nx_append*lt*sizeof(float));hipMemset(vxu_d,0,radius*nx_append*lt*sizeof(float));
		hipMalloc(&vxd_d,radius*nx_append*lt*sizeof(float));hipMemset(vxd_d,0,radius*nx_append*lt*sizeof(float));
		hipMalloc(&vzu_d,radius*nx_append*lt*sizeof(float));hipMemset(vzu_d,0,radius*nx_append*lt*sizeof(float));
		hipMalloc(&vzd_d,radius*nx_append*lt*sizeof(float));hipMemset(vzd_d,0,radius*nx_append*lt*sizeof(float));
		hipMalloc(&vxl_d,radius*nz_append*lt*sizeof(float));hipMemset(vxl_d,0,radius*nz_append*lt*sizeof(float));
		hipMalloc(&vxr_d,radius*nz_append*lt*sizeof(float));hipMemset(vxr_d,0,radius*nz_append*lt*sizeof(float));
		hipMalloc(&vzl_d,radius*nz_append*lt*sizeof(float));hipMemset(vzl_d,0,radius*nz_append*lt*sizeof(float));
		hipMalloc(&vzr_d,radius*nz_append*lt*sizeof(float));hipMemset(vzr_d,0,radius*nz_append*lt*sizeof(float));*/ 

////derivation for vx or vz
		float *vx_x_d,*vz_z_d,*vx_z_d,*vz_x_d;
		hipMalloc(&vx_x_d,nxanza*sizeof(float));hipMemset(vx_x_d,0,nxanza*sizeof(float));
		hipMalloc(&vz_z_d,nxanza*sizeof(float));hipMemset(vz_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vx_z_d,nxanza*sizeof(float));hipMemset(vx_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vz_x_d,nxanza*sizeof(float));hipMemset(vz_x_d,0,nxanza*sizeof(float));

////gradient or conjugate direction
///////////////2017年03月14日 星期二 20时21分13秒 波场分离的LSRTM
		float *tp2_d,*tp1_d,*vxp2_d,*vxp1_d,*vzp2_d,*vzp1_d,*vxs2_d,*vxs1_d,*vzs2_d,*vzs1_d;
		hipMalloc(&tp2_d,nxanza*sizeof(float));
		hipMalloc(&tp1_d,nxanza*sizeof(float));
		hipMalloc(&vxp2_d,nxanza*sizeof(float));
		hipMalloc(&vxp1_d,nxanza*sizeof(float));
		hipMalloc(&vzp2_d,nxanza*sizeof(float));
		hipMalloc(&vzp1_d,nxanza*sizeof(float));
		hipMalloc(&vxs2_d,nxanza*sizeof(float));
		hipMalloc(&vxs1_d,nxanza*sizeof(float));
		hipMalloc(&vzs2_d,nxanza*sizeof(float));
		hipMalloc(&vzs1_d,nxanza*sizeof(float));
			
		float *rtp2_d,*rtp1_d,*rvxp2_d,*rvxp1_d,*rvzp2_d,*rvzp1_d,*rvxs2_d,*rvxs1_d,*rvzs2_d,*rvzs1_d;
		hipMalloc(&rtp2_d,nxanza*sizeof(float));
		hipMalloc(&rtp1_d,nxanza*sizeof(float));
		hipMalloc(&rvxp2_d,nxanza*sizeof(float));
		hipMalloc(&rvxp1_d,nxanza*sizeof(float));
		hipMalloc(&rvzp2_d,nxanza*sizeof(float));
		hipMalloc(&rvzp1_d,nxanza*sizeof(float));
		hipMalloc(&rvxs2_d,nxanza*sizeof(float));
		hipMalloc(&rvxs1_d,nxanza*sizeof(float));
		hipMalloc(&rvzs2_d,nxanza*sizeof(float));
		hipMalloc(&rvzs1_d,nxanza*sizeof(float));

///////////////2017年03月14日 星期二 20时21分13秒 波场分离的LSRTM
		float *rvxp_integral_d,*rvzp_integral_d,*rvxs_integral_d,*rvzs_integral_d;
		hipMalloc(&rvxp_integral_d,nxanza*sizeof(float));hipMemset(rvxp_integral_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzp_integral_d,nxanza*sizeof(float));hipMemset(rvzp_integral_d,0,nxanza*sizeof(float));
		hipMalloc(&rvxs_integral_d,nxanza*sizeof(float));hipMemset(rvxs_integral_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzs_integral_d,nxanza*sizeof(float));hipMemset(rvzs_integral_d,0,nxanza*sizeof(float));

		float *rvxp_x_d,*rvzp_z_d,*rvxp_z_d,*rvzp_x_d;
		hipMalloc(&rvxp_x_d,nxanza*sizeof(float));hipMemset(rvxp_x_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzp_z_d,nxanza*sizeof(float));hipMemset(rvzp_z_d,0,nxanza*sizeof(float));
		hipMalloc(&rvxp_z_d,nxanza*sizeof(float));hipMemset(rvxp_z_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzp_x_d,nxanza*sizeof(float));hipMemset(rvzp_x_d,0,nxanza*sizeof(float));

		float *rvxs_x_d,*rvzs_z_d,*rvxs_z_d,*rvzs_x_d;
		hipMalloc(&rvxs_x_d,nxanza*sizeof(float));hipMemset(rvxs_x_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzs_z_d,nxanza*sizeof(float));hipMemset(rvzs_z_d,0,nxanza*sizeof(float));
		hipMalloc(&rvxs_z_d,nxanza*sizeof(float));hipMemset(rvxs_z_d,0,nxanza*sizeof(float));
		hipMalloc(&rvzs_x_d,nxanza*sizeof(float));hipMemset(rvzs_x_d,0,nxanza*sizeof(float));

		float *vxp_x_d,*vzp_z_d,*vxp_z_d,*vzp_x_d;
		hipMalloc(&vxp_x_d,nxanza*sizeof(float));hipMemset(vxp_x_d,0,nxanza*sizeof(float));
		hipMalloc(&vzp_z_d,nxanza*sizeof(float));hipMemset(vzp_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vxp_z_d,nxanza*sizeof(float));hipMemset(vxp_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vzp_x_d,nxanza*sizeof(float));hipMemset(vzp_x_d,0,nxanza*sizeof(float));

		float *vxs_x_d,*vzs_z_d,*vxs_z_d,*vzs_x_d;
		hipMalloc(&vxs_x_d,nxanza*sizeof(float));hipMemset(vxs_x_d,0,nxanza*sizeof(float));
		hipMalloc(&vzs_z_d,nxanza*sizeof(float));hipMemset(vzs_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vxs_z_d,nxanza*sizeof(float));hipMemset(vxs_z_d,0,nxanza*sizeof(float));
		hipMalloc(&vzs_x_d,nxanza*sizeof(float));hipMemset(vzs_x_d,0,nxanza*sizeof(float));

		float *vxp_t_d,*vzp_t_d,*vxs_t_d,*vzs_t_d;
		hipMalloc(&vxp_t_d,nxanza*sizeof(float));hipMemset(vxp_t_d,0,nxanza*sizeof(float));
		hipMalloc(&vzp_t_d,nxanza*sizeof(float));hipMemset(vzp_t_d,0,nxanza*sizeof(float));
		hipMalloc(&vxs_t_d,nxanza*sizeof(float));hipMemset(vxs_t_d,0,nxanza*sizeof(float));
		hipMalloc(&vzs_t_d,nxanza*sizeof(float));hipMemset(vzs_t_d,0,nxanza*sizeof(float));

		float *p_d,*s_d,*rp_d,*rs_d;
		hipMalloc(&p_d,nxanza*sizeof(float));		hipMemset(p_d,0,nxanza*sizeof(float));
		hipMalloc(&s_d,nxanza*sizeof(float));		hipMemset(s_d,0,nxanza*sizeof(float));
		hipMalloc(&rp_d,nxanza*sizeof(float));		hipMemset(rp_d,0,nxanza*sizeof(float));
		hipMalloc(&rs_d,nxanza*sizeof(float));		hipMemset(rs_d,0,nxanza*sizeof(float));
///////////////2017年03月14日 星期二 20时21分13秒 波场分离的LSRTM

////objective value
		float *obj_d;
		hipMalloc(&obj_d,3*sizeof(float));			hipMemset(obj_d,0,3*sizeof(float));///obj_d[0]:a=0  obj_d[1]:a=0.7  obj_d[2]:a=1.3 otherwise a=2/3

		float *obj_h,*obj_niter_h,*obj_niter_h1;
		obj_h=alloc1float(3);memset(obj_h,0,3*sizeof(float));///obj_d[0]:a=0  obj_d[1]:a=0.7  obj_d[2]:a=1.3 otherwise a=2/3
		obj_niter_h=alloc1float(niter);			memset(obj_niter_h,0,niter*sizeof(float));
		obj_niter_h1=alloc1float(niter);			memset(obj_niter_h1,0,niter*sizeof(float));
		float obj_exchange=0.0;//,obj_first=0;
////objective value

		
/////conjugate parameter
		float *beta_d,*alpha_d,*beta_step_d;	
		hipMalloc(&beta_d,3*sizeof(float));		hipMemset(beta_d,0,3*sizeof(float));/////make gradient direction transform to conjuagte direction
		hipMalloc(&alpha_d,1*sizeof(float));		hipMemset(alpha_d,0,1*sizeof(float));///////a step length of Hybrid conjugate direction 
		hipMalloc(&beta_step_d,3*sizeof(float));		hipMemset(beta_step_d,0,3*sizeof(float));/////assign different step for vp/vs/density

		float *d_alpha1,*d_alpha2;/////for beta_setp /* compute the numerator and the denominator of alpha: equations 5 and 12 */
		hipMalloc(&d_alpha1,lt_rec*sizeof(float));		hipMemset(d_alpha1,0,lt_rec*sizeof(float));
		hipMalloc(&d_alpha2,lt_rec*sizeof(float));		hipMemset(d_alpha2,0,lt_rec*sizeof(float));
		
		float *beta_h,*alpha_h,*beta_step_h;
		beta_h=alloc1float(3);					memset(beta_h,0,3*sizeof(float));
		alpha_h=alloc1float(1);					memset(alpha_h,0,1*sizeof(float));
		beta_step_h=alloc1float(3);					memset(beta_step_h,0,3*sizeof(float));
		
		float *epsil_d,*epsil_h;
		hipMalloc(&epsil_d,4*sizeof(float));			hipMemset(epsil_d,0,4*sizeof(float));	
		epsil_h=alloc1float(4);					memset(epsil_h,0,4*sizeof(float));
//////four small perturbation for vp or vx or density///epsil_d[0]:vp epsil_d[1]:vs  epsil_d[2]:density  epsil_d[3]:all


///////////////////////new acqusition way  2017年08月17日 星期四 10时06分44秒 
///////////final output lame parameterization
		float *perturb_lame1_d,*perturb_lame2_d,*perturb_den_d;
		hipMalloc(&perturb_lame1_d,nxnz*sizeof(float));		hipMemset(perturb_lame1_d,0,nxnz*sizeof(float));/////final output lame parameterization
		hipMalloc(&perturb_lame2_d,nxnz*sizeof(float));		hipMemset(perturb_lame2_d,0,nxnz*sizeof(float));/////final output lame parameterization
		hipMalloc(&perturb_den_d,nxnz*sizeof(float));		hipMemset(perturb_den_d,0,nxnz*sizeof(float));/////final output lame parameterization

///////////final output velocity or impedance parameterization
		float *perturb_vp_d,*perturb_vs_d,*perturb_density_d;
		hipMalloc(&perturb_vp_d,nxnz*sizeof(float));		hipMemset(perturb_vp_d,0,nxnz*sizeof(float));////final output velocity or impedance parameterization
		hipMalloc(&perturb_vs_d,nxnz*sizeof(float));		hipMemset(perturb_vs_d,0,nxnz*sizeof(float));////final output velocity or impedance parameterization
		hipMalloc(&perturb_density_d,nxnz*sizeof(float));		hipMemset(perturb_density_d,0,nxnz*sizeof(float));////final output velocity or impedance paramete
///////////final output


////////all gradient or conjugate direction
		float *all_grad_den_pp_d,*all_grad_lame1_pp_d,*all_grad_lame2_pp_d;		
		float *all_grad_den_ps_d,*all_grad_lame1_ps_d,*all_grad_lame2_ps_d;
		float *all_grad_den_sp_d,*all_grad_lame1_sp_d,*all_grad_lame2_sp_d;
		float *all_grad_den_ss_d,*all_grad_lame1_ss_d,*all_grad_lame2_ss_d;

		hipMalloc(&all_grad_den_pp_d,nxnz*sizeof(float));		hipMemset(all_grad_den_pp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame1_pp_d,nxnz*sizeof(float));	hipMemset(all_grad_lame1_pp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame2_pp_d,nxnz*sizeof(float));	hipMemset(all_grad_lame2_pp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_den_ps_d,nxnz*sizeof(float));		hipMemset(all_grad_den_ps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame1_ps_d,nxnz*sizeof(float));	hipMemset(all_grad_lame1_ps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame2_ps_d,nxnz*sizeof(float));	hipMemset(all_grad_lame2_ps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_den_sp_d,nxnz*sizeof(float));		hipMemset(all_grad_den_sp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame1_sp_d,nxnz*sizeof(float));	hipMemset(all_grad_lame1_sp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame2_sp_d,nxnz*sizeof(float));	hipMemset(all_grad_lame2_sp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_den_ss_d,nxnz*sizeof(float));		hipMemset(all_grad_den_ss_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame1_ss_d,nxnz*sizeof(float));	hipMemset(all_grad_lame1_ss_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame2_ss_d,nxnz*sizeof(float));	hipMemset(all_grad_lame2_ss_d,0,nxnz*sizeof(float));
////////////////when 波场分离的LSRTM 使用的时候
		//float *all_grad_den_d,*all_grad_lame1_d,*all_grad_lame2_d;
		float *all_grad_den1_d,*all_grad_lame11_d,*all_grad_lame22_d;
		float *all_conj_den_d,*all_conj_lame1_d,*all_conj_lame2_d;

		//hipMalloc(&all_grad_den_d,nxnz*sizeof(float));		hipMemset(all_grad_den_d,0,nxnz*sizeof(float));
		//hipMalloc(&all_grad_lame1_d,nxnz*sizeof(float));	hipMemset(all_grad_lame1_d,0,nxnz*sizeof(float));
		//hipMalloc(&all_grad_lame2_d,nxnz*sizeof(float));	hipMemset(all_grad_lame2_d,0,nxnz*sizeof(float));/////////the previous step

		hipMalloc(&all_grad_den1_d,nxnz*sizeof(float));		hipMemset(all_grad_den1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame11_d,nxnz*sizeof(float));		hipMemset(all_grad_lame11_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_lame22_d,nxnz*sizeof(float));		hipMemset(all_grad_lame22_d,0,nxnz*sizeof(float));/////////the current step

		hipMalloc(&all_conj_den_d,nxnz*sizeof(float));		hipMemset(all_conj_den_d,0,nxnz*sizeof(float));
		hipMalloc(&all_conj_lame1_d,nxnz*sizeof(float));		hipMemset(all_conj_lame1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_conj_lame2_d,nxnz*sizeof(float));		hipMemset(all_conj_lame2_d,0,nxnz*sizeof(float));/////////the current conjugate step

		//float *all_grad_density_d,*all_grad_vp_d ,*all_grad_vs_d;
		float *all_grad_density1_d,*all_grad_vp1_d,*all_grad_vs1_d;
		float *all_conj_density_d,*all_conj_vp_d,*all_conj_vs_d;

		float *all_hydrid_conj_d,*all_hydrid_grad1_d,*all_hydrid_grad2_d;

		//hipMalloc(&all_grad_density_d,nxnz*sizeof(float));	hipMemset(all_grad_density_d,0,nxnz*sizeof(float));
		//hipMalloc(&all_grad_vp_d,nxnz*sizeof(float));		hipMemset(all_grad_vp_d,0,nxnz*sizeof(float));
		//hipMalloc(&all_grad_vs_d,nxnz*sizeof(float));		hipMemset(all_grad_vs_d,0,nxnz*sizeof(float));/////////the previous step

		hipMalloc(&all_grad_density1_d,nxnz*sizeof(float));	hipMemset(all_grad_density1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_vp1_d,nxnz*sizeof(float));		hipMemset(all_grad_vp1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_grad_vs1_d,nxnz*sizeof(float));		hipMemset(all_grad_vs1_d,0,nxnz*sizeof(float));/////////the current step

		hipMalloc(&all_conj_vp_d,nxnz*sizeof(float));		hipMemset(all_conj_vp_d,0,nxnz*sizeof(float));	
		hipMalloc(&all_conj_vs_d,nxnz*sizeof(float));		hipMemset(all_conj_vs_d,0,nxnz*sizeof(float));	
		hipMalloc(&all_conj_density_d,nxnz*sizeof(float));	hipMemset(all_conj_density_d,0,nxnz*sizeof(float));/////////the current conjugate step

		hipMalloc(&all_hydrid_conj_d,3*nxnz*sizeof(float));	hipMemset(all_hydrid_conj_d,0,3*nxnz*sizeof(float));
		hipMalloc(&all_hydrid_grad1_d,3*nxnz*sizeof(float));	hipMemset(all_hydrid_grad1_d,0,3*nxnz*sizeof(float));
		hipMalloc(&all_hydrid_grad2_d,3*nxnz*sizeof(float));	hipMemset(all_hydrid_grad2_d,0,3*nxnz*sizeof(float));
////////all gradient or conjugate direction


////////migration result
		float *all_vresultpp_d,*all_vresultps_d,*all_vresultsp_d,*all_vresultss_d; 
		float *all_vresultppx_d,*all_vresultpsx_d,*all_vresultspx_d,*all_vresultssx_d,*all_vresultppz_d,*all_vresultpsz_d,*all_vresultspz_d,*all_vresultssz_d;
		hipMalloc(&all_vresultpp_d,nxnz*sizeof(float));	hipMemset(all_vresultpp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultps_d,nxnz*sizeof(float));	hipMemset(all_vresultps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultsp_d,nxnz*sizeof(float));	hipMemset(all_vresultsp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultss_d,nxnz*sizeof(float));	hipMemset(all_vresultss_d,0,nxnz*sizeof(float));

		hipMalloc(&all_vresultppx_d,nxnz*sizeof(float));	hipMemset(all_vresultppx_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultpsx_d,nxnz*sizeof(float));	hipMemset(all_vresultpsx_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultspx_d,nxnz*sizeof(float));	hipMemset(all_vresultspx_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultssx_d,nxnz*sizeof(float));	hipMemset(all_vresultssx_d,0,nxnz*sizeof(float));

		hipMalloc(&all_vresultppz_d,nxnz*sizeof(float));	hipMemset(all_vresultppz_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultpsz_d,nxnz*sizeof(float));	hipMemset(all_vresultpsz_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultspz_d,nxnz*sizeof(float));	hipMemset(all_vresultspz_d,0,nxnz*sizeof(float));
		hipMalloc(&all_vresultssz_d,nxnz*sizeof(float));	hipMemset(all_vresultssz_d,0,nxnz*sizeof(float));

		float *all_resultpp_d,*all_resultps_d,*all_resultps1_d,*all_resultps2_d,*all_resultsp_d,*all_resultsp1_d,*all_resultsp2_d,*all_resultss_d;		
		hipMalloc(&all_resultpp_d,nxnz*sizeof(float));	hipMemset(all_resultpp_d,0,nxnz*sizeof(float));

		hipMalloc(&all_resultps_d,nxnz*sizeof(float));	hipMemset(all_resultps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_resultps1_d,nxnz*sizeof(float));	hipMemset(all_resultps1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_resultps2_d,nxnz*sizeof(float));	hipMemset(all_resultps2_d,0,nxnz*sizeof(float));

		hipMalloc(&all_resultsp_d,nxnz*sizeof(float));	hipMemset(all_resultsp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_resultsp1_d,nxnz*sizeof(float));	hipMemset(all_resultsp1_d,0,nxnz*sizeof(float));
		hipMalloc(&all_resultsp2_d,nxnz*sizeof(float));	hipMemset(all_resultsp2_d,0,nxnz*sizeof(float));

		hipMalloc(&all_resultss_d,nxnz*sizeof(float));	hipMemset(all_resultss_d,0,nxnz*sizeof(float));
	
		float *all_result_tp_d;
		hipMalloc(&all_result_tp_d,nxnz*sizeof(float));	hipMemset(all_result_tp_d,0,nxnz*sizeof(float));

		gpumem += (nxnz*51)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒

////////migration result
///////////2017年03月18日 星期六 21时07分00秒Traditional   ERTM 
		float *resultpp_d,*resultps_d,*resultps1_d,*resultps2_d,*resultsp_d,*resultsp1_d,*resultsp2_d,*resultss_d;
		hipMalloc(&resultpp_d,nx_size_nz*sizeof(float));		hipMemset(resultpp_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&resultps_d,nx_size_nz*sizeof(float));		hipMemset(resultps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultps1_d,nx_size_nz*sizeof(float));		hipMemset(resultps1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultps2_d,nx_size_nz*sizeof(float));		hipMemset(resultps2_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&resultsp_d,nx_size_nz*sizeof(float));		hipMemset(resultsp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultsp1_d,nx_size_nz*sizeof(float));		hipMemset(resultsp1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultsp2_d,nx_size_nz*sizeof(float));		hipMemset(resultsp2_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&resultss_d,nx_size_nz*sizeof(float));		hipMemset(resultss_d,0,nx_size_nz*sizeof(float));
	
		float *result_tp_d;
		hipMalloc(&result_tp_d,nx_size_nz*sizeof(float));		hipMemset(result_tp_d,0,nx_size_nz*sizeof(float));

///////////2017年03月18日 星期六 21时07分00秒Traditional   ERTM
		float *vresultpp_d,*vresultps_d,*vresultsp_d,*vresultss_d; 
		float *vresultppx_d,*vresultpsx_d,*vresultspx_d,*vresultssx_d,*vresultppz_d,*vresultpsz_d,*vresultspz_d,*vresultssz_d;
		hipMalloc(&vresultpp_d,nx_size_nz*sizeof(float));		hipMemset(vresultpp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultps_d,nx_size_nz*sizeof(float));		hipMemset(vresultps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultsp_d,nx_size_nz*sizeof(float));		hipMemset(vresultsp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultss_d,nx_size_nz*sizeof(float));		hipMemset(vresultss_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&vresultppx_d,nx_size_nz*sizeof(float));	hipMemset(vresultppx_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultpsx_d,nx_size_nz*sizeof(float));	hipMemset(vresultpsx_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultspx_d,nx_size_nz*sizeof(float));	hipMemset(vresultspx_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultssx_d,nx_size_nz*sizeof(float));	hipMemset(vresultssx_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&vresultppz_d,nx_size_nz*sizeof(float));	hipMemset(vresultppz_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultpsz_d,nx_size_nz*sizeof(float));	hipMemset(vresultpsz_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultspz_d,nx_size_nz*sizeof(float));	hipMemset(vresultspz_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&vresultssz_d,nx_size_nz*sizeof(float));	hipMemset(vresultssz_d,0,nx_size_nz*sizeof(float));
///////////2017年03月18日 星期六 21时07分00秒Traditional   ERTM 

//////////////////////////////source_illumination or excitation amplitude imaging condition  2018年01月24日 星期三 20时33分21秒 
		float *down_vpp_x_d,*down_vpp_z_d,*down_vss_x_d,*down_vss_z_d;
		float *down_tp_d,*down_vpp_d,*down_vss_d,*down_pp_d,*down_ss_d,*down_xx_d,*down_zz_d; 
		hipMalloc(&down_vpp_x_d,nx_size_nz*sizeof(float));		hipMemset(down_vpp_x_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_vpp_z_d,nx_size_nz*sizeof(float));		hipMemset(down_vpp_z_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_vss_x_d,nx_size_nz*sizeof(float));		hipMemset(down_vss_x_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_vss_z_d,nx_size_nz*sizeof(float));		hipMemset(down_vss_z_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&down_tp_d,nx_size_nz*sizeof(float));			hipMemset(down_tp_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&down_vpp_d,nx_size_nz*sizeof(float));			hipMemset(down_vpp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_vss_d,nx_size_nz*sizeof(float));			hipMemset(down_vss_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&down_pp_d,nx_size_nz*sizeof(float));			hipMemset(down_pp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_ss_d,nx_size_nz*sizeof(float));			hipMemset(down_ss_d,0,nx_size_nz*sizeof(float));

		hipMalloc(&down_xx_d,nx_size_nz*sizeof(float));			hipMemset(down_xx_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&down_zz_d,nx_size_nz*sizeof(float));			hipMemset(down_zz_d,0,nx_size_nz*sizeof(float));
//////excitation:   	related function in kernel_3  excitation amplitude imaging condition
		float *ex_vresultpp_d,*ex_vresultps_d,*ex_result_tp_d,*ex_result_tp_old_d,*resultxx_d,*resultzz_d;
		float *com_ex_vresultpp_d,*com_ex_vresultps_d;
		hipMalloc(&com_ex_vresultpp_d,nx_size_nz*sizeof(float));		hipMemset(com_ex_vresultpp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&com_ex_vresultps_d,nx_size_nz*sizeof(float));		hipMemset(com_ex_vresultps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&ex_vresultpp_d,nx_size_nz*sizeof(float));		hipMemset(ex_vresultpp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&ex_vresultps_d,nx_size_nz*sizeof(float));		hipMemset(ex_vresultps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&ex_result_tp_d,nx_size_nz*sizeof(float));		hipMemset(ex_result_tp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&ex_result_tp_old_d,nx_size_nz*sizeof(float));		hipMemset(ex_result_tp_old_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultxx_d,nx_size_nz*sizeof(float));			hipMemset(resultxx_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&resultzz_d,nx_size_nz*sizeof(float));			hipMemset(resultzz_d,0,nx_size_nz*sizeof(float));

		float *all_ex_vresultpp_d,*all_ex_vresultps_d,*all_ex_result_tp_d,*all_ex_result_tp_old_d,*all_resultxx_d,*all_resultzz_d;
		float *all_com_ex_vresultpp_d,*all_com_ex_vresultps_d;
		hipMalloc(&all_com_ex_vresultpp_d,nxnz*sizeof(float));		hipMemset(all_com_ex_vresultpp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_com_ex_vresultps_d,nxnz*sizeof(float));		hipMemset(all_com_ex_vresultps_d,0,nxnz*sizeof(float));
		hipMalloc(&all_ex_vresultpp_d,nxnz*sizeof(float));		hipMemset(all_ex_vresultpp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_ex_vresultps_d,nxnz*sizeof(float));		hipMemset(all_ex_vresultps_d,0,nxnz*sizeof(float)); 
		hipMalloc(&all_ex_result_tp_d,nxnz*sizeof(float));		hipMemset(all_ex_result_tp_d,0,nxnz*sizeof(float));
		hipMalloc(&all_ex_result_tp_old_d,nxnz*sizeof(float));		hipMemset(all_ex_result_tp_old_d,0,nxnz*sizeof(float));
		hipMalloc(&all_resultxx_d,nxnz*sizeof(float));			hipMemset(all_resultxx_d,0,nxnz*sizeof(float)); 
		hipMalloc(&all_resultzz_d,nxnz*sizeof(float));			hipMemset(all_resultzz_d,0,nxnz*sizeof(float));  

		float *ex_amp_d,*ex_amp_x_d,*ex_amp_z_d,*ex_amp_tp_old_d,*ex_time_d;
		hipMalloc(&ex_amp_d,nxanza*sizeof(float));			hipMemset(ex_amp_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_amp_x_d,nxanza*sizeof(float));			hipMemset(ex_amp_x_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_amp_z_d,nxanza*sizeof(float));			hipMemset(ex_amp_z_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_amp_tp_old_d,nxanza*sizeof(float));			hipMemset(ex_amp_tp_old_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_time_d,nxanza*sizeof(float));			hipMemset(ex_time_d,0,nxanza*sizeof(float));
////////////////////////////2018年05月21日 星期一 10时22分36秒 
		float *ex_tp_time_d,*ex_amp_tp_d;
		hipMalloc(&ex_amp_tp_d,nxanza*sizeof(float));			hipMemset(ex_amp_tp_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_tp_time_d,nxanza*sizeof(float));			hipMemset(ex_tp_time_d,0,nxanza*sizeof(float));

		float *ex_angle_pp_d,*ex_angle_rpp_d,*ex_angle_rps_d;
		hipMalloc(&ex_angle_pp_d,nxanza*sizeof(float));			hipMemset(ex_angle_pp_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_angle_rpp_d,nxanza*sizeof(float));			hipMemset(ex_angle_rpp_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_angle_rps_d,nxanza*sizeof(float));			hipMemset(ex_angle_rps_d,0,nxanza*sizeof(float));
		float *ex_angle_pp1_d,*ex_angle_rpp1_d,*ex_angle_rps1_d;
		hipMalloc(&ex_angle_pp1_d,nxanza*sizeof(float));			hipMemset(ex_angle_pp1_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_angle_rpp1_d,nxanza*sizeof(float));			hipMemset(ex_angle_rpp1_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_angle_rps1_d,nxanza*sizeof(float));			hipMemset(ex_angle_rps1_d,0,nxanza*sizeof(float));
		float *para_max_d,p_printf;
		hipMalloc(&para_max_d,20*sizeof(float));				hipMemset(para_max_d,0,20*sizeof(float));

		float *ex_open_pp_d,*ex_open_ps_d,*ex_open_pp1_d,*ex_open_ps1_d;
		hipMalloc(&ex_open_pp_d,nxanza*sizeof(float));			hipMemset(ex_open_pp_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_open_ps_d,nxanza*sizeof(float));			hipMemset(ex_open_ps_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_open_pp1_d,nxanza*sizeof(float));			hipMemset(ex_open_pp1_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_open_ps1_d,nxanza*sizeof(float));			hipMemset(ex_open_ps1_d,0,nxanza*sizeof(float));
		float *ex_com_pp_sign_d,*ex_com_ps_sign_d;
		hipMalloc(&ex_com_pp_sign_d,nxanza*sizeof(float));			hipMemset(ex_com_pp_sign_d,0,nxanza*sizeof(float));
		hipMalloc(&ex_com_ps_sign_d,nxanza*sizeof(float));			hipMemset(ex_com_ps_sign_d,0,nxanza*sizeof(float));
//////////////////////////////

///////////////////////new acqusition way  2017年08月17日 星期四 10时06分44秒 
////lame1:langda,lame2:u
/////////////////Ren and Liu 2016 in geophysics  Xu and Mcmechan 2014 in geophysics
		float *grad_den_pp_d,*grad_lame1_pp_d,*grad_lame2_pp_d;		
		float *grad_den_ps_d,*grad_lame1_ps_d,*grad_lame2_ps_d;
		float *grad_den_sp_d,*grad_lame1_sp_d,*grad_lame2_sp_d;
		float *grad_den_ss_d,*grad_lame1_ss_d,*grad_lame2_ss_d;

		hipMalloc(&grad_den_pp_d,nx_size_nz*sizeof(float));	hipMemset(grad_den_pp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame1_pp_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame1_pp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame2_pp_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame2_pp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_den_ps_d,nx_size_nz*sizeof(float));	hipMemset(grad_den_ps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame1_ps_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame1_ps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame2_ps_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame2_ps_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_den_sp_d,nx_size_nz*sizeof(float));	hipMemset(grad_den_sp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame1_sp_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame1_sp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame2_sp_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame2_sp_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_den_ss_d,nx_size_nz*sizeof(float));	hipMemset(grad_den_ss_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame1_ss_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame1_ss_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame2_ss_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame2_ss_d,0,nx_size_nz*sizeof(float));

		float *grad_den_d,*grad_lame1_d,*grad_lame2_d;
		float *grad_den1_d,*grad_lame11_d,*grad_lame22_d;
	
		hipMalloc(&grad_den_d,nx_size_nz*sizeof(float));		hipMemset(grad_den_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame1_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame2_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame2_d,0,nx_size_nz*sizeof(float));/////////the previous step

		hipMalloc(&grad_den1_d,nx_size_nz*sizeof(float));		hipMemset(grad_den1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame11_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame11_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_lame22_d,nx_size_nz*sizeof(float));	hipMemset(grad_lame22_d,0,nx_size_nz*sizeof(float));/////////the current step	

		
		float *grad_vp1_d,*grad_vs1_d,*grad_density1_d;
		hipMalloc(&grad_vp1_d,nx_size_nz*sizeof(float));		hipMemset(grad_vp1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_vs1_d,nx_size_nz*sizeof(float));		hipMemset(grad_vs1_d,0,nx_size_nz*sizeof(float));
		hipMalloc(&grad_density1_d,nx_size_nz*sizeof(float));	hipMemset(grad_density1_d,0,nx_size_nz*sizeof(float));/////////the current step		
////lame1:langda,lame2:u		

////gradient or conjugate direction for lame1 and lame2 den	
		float *tmp_perturb_lame1_d,*tmp_perturb_lame2_d,*tmp_perturb_den_d;
		hipMalloc(&tmp_perturb_lame1_d,nxanza*sizeof(float));hipMemset(tmp_perturb_lame1_d,0,nxanza*sizeof(float));/////the tmp perturb result
		hipMalloc(&tmp_perturb_lame2_d,nxanza*sizeof(float));hipMemset(tmp_perturb_lame2_d,0,nxanza*sizeof(float));/////the tmp perturb result
		hipMalloc(&tmp_perturb_den_d,nxanza*sizeof(float));hipMemset(tmp_perturb_den_d,0,nxanza*sizeof(float));/////the tmp perturb result
	
		float *tmp_perturb_vp_d,*tmp_perturb_vs_d,*tmp_perturb_density_d;
		hipMalloc(&tmp_perturb_vp_d,nxanza*sizeof(float));hipMemset(tmp_perturb_vp_d,0,nxanza*sizeof(float));/////the tmp perturb result
		hipMalloc(&tmp_perturb_vs_d,nxanza*sizeof(float));hipMemset(tmp_perturb_vs_d,0,nxanza*sizeof(float));/////the tmp  perturb result
		hipMalloc(&tmp_perturb_density_d,nxanza*sizeof(float));hipMemset(tmp_perturb_density_d,0,nxanza*sizeof(float));/////the tmp perturb result
////gradient or conjugate direction for vp and vs density	

/////////for nomarlized
		float *d_illum,*d_illum_new,*r_d_illum;
		hipMalloc(&d_illum,nxanza*sizeof(float));			hipMemset(d_illum,0,nxanza*sizeof(float));

		hipMalloc(&r_d_illum,nxanza*sizeof(float));		hipMemset(r_d_illum,0,nxanza*sizeof(float));

		hipMalloc(&d_illum_new,nxa_new_nza*sizeof(float));	hipMemset(d_illum_new,0,nxa_new_nza*sizeof(float));		

//////////2016年11月20日 星期日 05时55分17秒 and optimize at 2017年01月03日 星期二 10时08分13秒 
		float *dem_p1_d,*dem_p2_d,*dem_p3_d,*dem_p4_d,*dem_p5_d,*dem_p6_d,*dem_p7_d,*dem_p8_d;
		hipMalloc(&dem_p1_d,nxanza*sizeof(float));hipMemset(dem_p1_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p2_d,nxanza*sizeof(float));hipMemset(dem_p2_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p3_d,nxanza*sizeof(float));hipMemset(dem_p3_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p4_d,nxanza*sizeof(float));hipMemset(dem_p4_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p5_d,nxanza*sizeof(float));hipMemset(dem_p5_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p6_d,nxanza*sizeof(float));hipMemset(dem_p6_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p7_d,nxanza*sizeof(float));hipMemset(dem_p7_d,0,nxanza*sizeof(float));
		hipMalloc(&dem_p8_d,nxanza*sizeof(float));hipMemset(dem_p8_d,0,nxanza*sizeof(float));

		float *dem_p_all_d;
		hipMalloc(&dem_p_all_d,8*nxanza*sizeof(float));hipMemset(dem_p_all_d,0,8*nxanza*sizeof(float));

//filter_signy_d  this is for Du or Li method for polarity correction
		/*float *sign_d,*sign1_d;float *signx_d,*signy_d,*signz_d,*filter_signx_d,*filter_signy_d,*filter_signz_d;
		hipMalloc(&sign_d,nxanza*sizeof(float));hipMemset(sign_d,0,nxanza*sizeof(float));
		hipMalloc(&sign1_d,nxanza*sizeof(float));hipMemset(sign1_d,0,nxanza*sizeof(float));
		hipMalloc(&signx_d,nxanza*sizeof(float));hipMemset(signx_d,0,nxanza*sizeof(float));
		hipMalloc(&signy_d,nxanza*sizeof(float));hipMemset(signy_d,0,nxanza*sizeof(float));
		hipMalloc(&signz_d,nxanza*sizeof(float));hipMemset(signz_d,0,nxanza*sizeof(float));
		hipMalloc(&filter_signx_d,nxanza*sizeof(float));hipMemset(filter_signx_d,0,nxanza*sizeof(float));
		hipMalloc(&filter_signy_d,nxanza*sizeof(float));hipMemset(filter_signy_d,0,nxanza*sizeof(float));
		hipMalloc(&filter_signz_d,nxanza*sizeof(float));hipMemset(filter_signz_d,0,nxanza*sizeof(float));*/
//define poynting vector for x or z component
		/*float *poyn_x_d,*poyn_z_d,*poyn_rx_d,*poyn_rz_d;
		hipMalloc(&poyn_x_d, nxanza*sizeof(float));
		hipMalloc(&poyn_z_d, nxanza*sizeof(float));
		hipMalloc(&poyn_rx_d,nxanza*sizeof(float));
		hipMalloc(&poyn_rz_d,nxanza*sizeof(float));*/
		float *poyn_px_d,*poyn_pz_d,*poyn_sx_d,*poyn_sz_d;
		hipMalloc(&poyn_px_d, nxanza*sizeof(float));		hipMemset(poyn_px_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_pz_d, nxanza*sizeof(float));		hipMemset(poyn_pz_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_sx_d,nxanza*sizeof(float));		hipMemset(poyn_sx_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_sz_d,nxanza*sizeof(float));		hipMemset(poyn_sz_d,0,nxanza*sizeof(float));

		float *poyn_rpx_d,*poyn_rpz_d,*poyn_rsx_d,*poyn_rsz_d;
		hipMalloc(&poyn_rpx_d, nxanza*sizeof(float));		hipMemset(poyn_rpx_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_rpz_d, nxanza*sizeof(float));		hipMemset(poyn_rpz_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_rsx_d,nxanza*sizeof(float));		hipMemset(poyn_rsx_d,0,nxanza*sizeof(float));
		hipMalloc(&poyn_rsz_d,nxanza*sizeof(float));		hipMemset(poyn_rsz_d,0,nxanza*sizeof(float));

		float *direction_px_d,*direction_pz_d,*direction_sx_d,*direction_sz_d;
		hipMalloc(&direction_px_d, nxanza*sizeof(float));		hipMemset(direction_px_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_pz_d, nxanza*sizeof(float));		hipMemset(direction_pz_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_sx_d,nxanza*sizeof(float));		hipMemset(direction_sx_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_sz_d,nxanza*sizeof(float));		hipMemset(direction_sz_d,0,nxanza*sizeof(float));

		float *direction_rpx_d,*direction_rpz_d,*direction_rsx_d,*direction_rsz_d;
		hipMalloc(&direction_rpx_d, nxanza*sizeof(float));	hipMemset(direction_rpx_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_rpz_d, nxanza*sizeof(float));	hipMemset(direction_rpz_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_rsx_d,nxanza*sizeof(float));		hipMemset(direction_rsx_d,0,nxanza*sizeof(float));
		hipMalloc(&direction_rsz_d,nxanza*sizeof(float));		hipMemset(direction_rsz_d,0,nxanza*sizeof(float));
///////////2017年03月12日 星期日 10时51分29秒Traditional   ERTM 

		gpumem += (nxanza*90)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒

/////for check 2017年07月27日 星期四 10时11分40秒
		gpumem += (nxanza*65)*sizeof(float)/1024.0/1024.0;/////for check 2017年07月27日 星期四 10时11分40秒
		
		//check_number,check_interval,check_residual;

		gpumem_residual=2700-gpumem;

		change=1.0*variable_number*lt*nx_size*nz*4.0/1024.0/1024.0;
		warn("GPU memory cost: %f (MB).",gpumem);
		warn("gpumem_residual memory cost: %f (MB).",gpumem_residual);
		warn("all memory cost: %f (MB).",change);

		if(change<=gpumem_residual)
		{
			check_number=1;
			check_interval=lt;
			check_residual=0;
		}

		if(change>gpumem_residual)
		{
			check_number=int(1.0*change/gpumem_residual+1);

			check_interval=int(1.0*lt/check_number);

			check_residual=int(lt-1.0*check_number*check_interval);
		}

		float *save_vx_x_d,*save_vx_z_d,*save_vx_t_d,*save_vz_x_d,*save_vz_z_d,*save_vz_t_d;
		float *save_vxp_d,*save_vxs_d,*save_vzp_d,*save_vzs_d;
		float *save_tp_d,*save_p_d,*save_s_d;		

		hipMalloc(&save_vx_x_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vx_x_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vx_z_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vx_z_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vx_t_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vx_t_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vz_x_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vz_x_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vz_z_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vz_z_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vz_t_d,check_interval*nx_size_nz*sizeof(float));	hipMemset(save_vz_t_d,0,check_interval*nx_size_nz*sizeof(float));

		hipMalloc(&save_vxp_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_vxp_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vxs_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_vxs_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vzp_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_vzp_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_vzs_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_vzs_d,0,check_interval*nx_size_nz*sizeof(float));
	
		hipMalloc(&save_tp_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_tp_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_p_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_p_d,0,check_interval*nx_size_nz*sizeof(float));
		hipMalloc(&save_s_d,check_interval*nx_size_nz*sizeof(float));		hipMemset(save_s_d,0,check_interval*nx_size_nz*sizeof(float));

		float *save_h;
		save_h=alloc1float(check_interval*nx_size_nz);				memset(save_h,0,check_interval*nx_size_nz*sizeof(float));				

		warn("check_number=%d,check_interval=%d,check_residual=%d\n.",check_number,check_interval,check_residual);

		float *packaging_d;
		hipMalloc(&packaging_d,20*sizeof(float));		hipMemset(packaging_d,0,20*sizeof(float));
/////for check 2017年07月27日 星期四 10时11分40秒

//////////////////////////To bring in large amplitude errors/////for correlation_misfit
		float *error_random,*error_random_d;/////for correlation_misfit
		error_random=alloc1float(receiver_num);				memset(error_random,0,receiver_num*sizeof(float));
		hipMalloc(&error_random_d,receiver_num*sizeof(float));		hipMemset(error_random_d,0,receiver_num*sizeof(float));
		
		cal_cpu_error_random(error_random,amplitude_error_number,receiver_num,receiver_interval);

		hipMemcpy(error_random_d,error_random,receiver_num*sizeof(float),hipMemcpyHostToDevice);

		write_file_1d(error_random,receiver_num,"./someoutput/error_random");
//////////////////To bring in source strength errors/////for correlation_misfit
		float *shot_scale_h,*shot_scale_d;
		shot_scale_h=alloc1float(shot_num);				memset(shot_scale_h,0,shot_num*sizeof(float));
		hipMalloc(&shot_scale_d,shot_num*sizeof(float));		hipMemset(shot_scale_d,0,shot_num*sizeof(float));

		if(shot_scale!=0)	cal_cpu_shot_scale(shot_scale_h,shot_num,shot_scale);

		hipMemcpy(shot_scale_d,shot_scale_h,shot_num*sizeof(float),hipMemcpyHostToDevice);

		write_file_1d(shot_scale_h,shot_num,"./someoutput/shot_scale");		
//////////////////////////To bring in large amplitude errors/////for correlation_misfit
			
		warn("******Start to Calculate******");
		warn("nx=%d,nz=%d",nx,nz);
		warn("boundary_up=%d,boundary_down=%d,boundary_left=%d,boundary_right=%d",boundary_up,boundary_down,boundary_left,boundary_right);
		warn("nx_append=%d,nz_append=%d",nx_append,nz_append);
		//clock_t start,finish;
		//double duration;
		//time(&t1);
		//start = clock();

		if(join_wavefield==1)	
		{

			system("mkdir wavefield1");
			system("mkdir wavefield1/0");	
			system("mkdir wavefield1/1");
			system("mkdir wavefield1/2");
			system("mkdir wavefield1/3");
			system("mkdir wavefield1/4");
			system("mkdir wavefield1/5");
		}

		cuda_packaging<<<10,1>>>(packaging_d,dx,dz,dt,coe_x,coe_z,nx_append_radius,nz_append_radius,coe_opt_d);
		/////////////////////////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 

		hipEventRecord(start);/* record starting time */
		int ishot=0,mark;
		wavelet_half=0;
		while(ishot<shot_num&&join_shot==0)//////join_shot=0 denote that there is no obs shots, we need simulate obs shots
		{					
				if(fmod((ishot*1.0),40.0)==0)		
				{
					//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 1,it=%d",ishot+1,(it+1)-wavelet_half);
					warn("ishot=%d",ishot+1);
					warn("shot cord (x:%d z:%d)",source_x_cord[ishot],shot_depth);
					warn("receiver cord [start (x:%d z:%d ) interval:%d number:%d]",receiver_x_cord[ishot],receiver_depth,receiver_interval,receiver_num);
				}

				if(cut_direct_wave==0||cut_direct_wave==1)
				{
					/////////////////////////////////////get constant mode;				
					cuda_get_constant_mode<<<dimGrid,dimBlock>>>(velocity_all_d,velocity_d,nx_append,nz_append);
					cuda_get_constant_mode<<<dimGrid,dimBlock>>>(velocity1_all_d,velocity1_d,nx_append,nz_append);
					cuda_get_constant_mode<<<dimGrid,dimBlock>>>(density_all_d,density_d,nx_append,nz_append);
					cuda_get_constant_mode<<<dimGrid,dimBlock>>>(qp_all_d,qp_d,nx_append,nz_append);
					cuda_get_constant_mode<<<dimGrid,dimBlock>>>(qs_all_d,qs_d,nx_append,nz_append);
///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
					cuda_cal_viscoelastic<<< dimGrid,dimBlock>>>(modul_p_d,modul_s_d,qp_d,qs_d,tao_d,strain_p_d,strain_s_d,freq,velocity_d,velocity1_d,density_d,nx_append,nz_append);
					
					memset((void *)(wf_append),0,nxanza*sizeof(float));
					hipMemset(vx1_d,0,nxanza*sizeof(float));
					hipMemset(vz1_d,0,nxanza*sizeof(float));
					hipMemset(txx1_d,0,nxanza*sizeof(float));
					hipMemset(tzz1_d,0,nxanza*sizeof(float));
					hipMemset(txz1_d,0,nxanza*sizeof(float));
					hipMemset(vx2_d,0,nxanza*sizeof(float));
					hipMemset(vz2_d,0,nxanza*sizeof(float));
					hipMemset(txx2_d,0,nxanza*sizeof(float));
					hipMemset(tzz2_d,0,nxanza*sizeof(float));
					hipMemset(txz2_d,0,nxanza*sizeof(float));

					hipMemset(tp2_d,0,nxanza*sizeof(float));
					hipMemset(tp1_d,0,nxanza*sizeof(float));
					hipMemset(vxp2_d,0,nxanza*sizeof(float));
					hipMemset(vxp1_d,0,nxanza*sizeof(float));
					hipMemset(vzp2_d,0,nxanza*sizeof(float));
					hipMemset(vzp1_d,0,nxanza*sizeof(float));
					hipMemset(vxs2_d,0,nxanza*sizeof(float));
					hipMemset(vxs1_d,0,nxanza*sizeof(float));
					hipMemset(vzs2_d,0,nxanza*sizeof(float));
					hipMemset(vzs1_d,0,nxanza*sizeof(float));

					hipMemset(mem_p1_d,0,nxanza*sizeof(float));
					hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
					hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
					hipMemset(mem_xz1_d,0,nxanza*sizeof(float));
					hipMemset(mem_p2_d,0,nxanza*sizeof(float));
					hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
					hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
					hipMemset(mem_xz2_d,0,nxanza*sizeof(float));///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒		
				
	    			for(int it=0;it<lt+wavelet_half;it++)
					{
							//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 1,it=%d",ishot+1,(it+1)-wavelet_half);
							
							if(it<wavelet_length)
							{
								//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],shot_depth,it,boundary_up,boundary_left,nz_append);
								//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for vsp or surface 2017年03月14日 星期二 08时41分20秒 

								add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for new acqusition way 2017年08月17日 星期四 09时10分03秒 
							}

							fwd_vx<<<dimGrid,dimBlock>>>(vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

							fwd_vz<<<dimGrid,dimBlock>>>(vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

							//fwd_vxp_vzp<<<dimGrid,dimBlock>>>(vxp2_d,vxp1_d,vzp2_d,vzp1_d,tp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,density_d);

							//vp_vs<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,nx_append_radius,nz_append_radius);

							if(modeling_type==0)	fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

							//else	fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

							//fwd_memory<<<dimGrid,dimBlock>>>(mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,vx2_d,vz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

							//fwd_txxzzxzpp_viscoelastic<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p1_d,mem_xx1_d,mem_zz1_d,mem_xz1_d);

							else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

							//decom<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,coe_opt_d,nx_append,nz_append,dx,dz);

							//decom_new<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,velocity_d,velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);

							if(it>=wavelet_half&&it<(lt+wavelet_half))
							{
								if(receiver_offset==0)
								{
									write_shot_x_z<<<receiver_num,1>>>(vx2_d,cal_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
									write_shot_x_z<<<receiver_num,1>>>(vz2_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								}

								else
								{
									write_shot_x_z_acqusition<<<receiver_num,1>>>(vx2_d,cal_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
									write_shot_x_z_acqusition<<<receiver_num,1>>>(vz2_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								}

							}
						
							rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
							rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
							rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
							rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
							rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

							rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
							rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
							rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
							rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
							rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

							rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
							rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
							rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
							rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;
					}

						if(ishot%40==0)
						{
							hipMemcpy(shotgather,cal_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct1_obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,cal_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct1_obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}
				}				
					
/////////////////////////////////////get correct vp;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(velocity_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(velocity_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

				hipMemcpy(wf,wf_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
				sprintf(filename,"./someoutput/cut-vp-%d.bin",ishot+1);
				write_file_1d(wf,nx_size_nz,filename);

				hipMemcpy(wf_append,velocity_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
				sprintf(filename,"./someoutput/vp-%d.bin",ishot+1);
				write_file_1d(wf_append,nxanza,filename);
/////////////////////////////////////get correct vs;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(velocity1_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(velocity1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get correct density;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(density_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get correct qp;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(qp_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(qp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get correct qs;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(qs_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(qs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

				/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
				{								
					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(velocity_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
					hipMemcpy(wf_append,velocity_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/vp-new-%d.bin",ishot+1);
					write_file_1d(wf_append,nxanza,filename);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(velocity1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(qp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(qs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);						
				}*/
///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
				cuda_cal_viscoelastic<<< dimGrid,dimBlock>>>(modul_p_d,modul_s_d,qp_d,qs_d,tao_d,strain_p_d,strain_s_d,freq,velocity_d,velocity1_d,density_d,nx_append,nz_append);	
				
				memset((void *)(wf_append),0,nxanza*sizeof(float));
				hipMemset(vx1_d,0,nxanza*sizeof(float));
				hipMemset(vz1_d,0,nxanza*sizeof(float));
				hipMemset(txx1_d,0,nxanza*sizeof(float));
				hipMemset(tzz1_d,0,nxanza*sizeof(float));
				hipMemset(txz1_d,0,nxanza*sizeof(float));
				hipMemset(vx2_d,0,nxanza*sizeof(float));
				hipMemset(vz2_d,0,nxanza*sizeof(float));
				hipMemset(txx2_d,0,nxanza*sizeof(float));
				hipMemset(tzz2_d,0,nxanza*sizeof(float));
				hipMemset(txz2_d,0,nxanza*sizeof(float));

				hipMemset(tp2_d,0,nxanza*sizeof(float));
				hipMemset(tp1_d,0,nxanza*sizeof(float));
				hipMemset(vxp2_d,0,nxanza*sizeof(float));
				hipMemset(vxp1_d,0,nxanza*sizeof(float));
				hipMemset(vzp2_d,0,nxanza*sizeof(float));
				hipMemset(vzp1_d,0,nxanza*sizeof(float));
				hipMemset(vxs2_d,0,nxanza*sizeof(float));
				hipMemset(vxs1_d,0,nxanza*sizeof(float));
				hipMemset(vzs2_d,0,nxanza*sizeof(float));
				hipMemset(vzs1_d,0,nxanza*sizeof(float));

				hipMemset(mem_p1_d,0,nxanza*sizeof(float));
				hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
				hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
				hipMemset(mem_xz1_d,0,nxanza*sizeof(float));
				hipMemset(mem_p2_d,0,nxanza*sizeof(float));
				hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
				hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
				hipMemset(mem_xz2_d,0,nxanza*sizeof(float));///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒		
			
    			for(int it=0;it<lt+wavelet_half;it++)
				{
						//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 1,it=%d",ishot+1,(it+1)-wavelet_half);
						
						if(it<wavelet_length)
						{
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],shot_depth,it,boundary_up,boundary_left,nz_append);
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for vsp or surface 2017年03月14日 星期二 08时41分20秒 

							add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for new acqusition way 2017年08月17日 星期四 09时10分03秒 
						}

						fwd_vx<<<dimGrid,dimBlock>>>(vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

						fwd_vz<<<dimGrid,dimBlock>>>(vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

						//fwd_vxp_vzp<<<dimGrid,dimBlock>>>(vxp2_d,vxp1_d,vzp2_d,vzp1_d,tp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,density_d);

						//vp_vs<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,nx_append_radius,nz_append_radius);

						if(modeling_type==0)	fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);

						//else	fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

						//fwd_memory<<<dimGrid,dimBlock>>>(mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,vx2_d,vz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

						//fwd_txxzzxzpp_viscoelastic<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p1_d,mem_xx1_d,mem_zz1_d,mem_xz1_d);

						else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,velocity_d,velocity1_d,tao_d,strain_p_d,strain_s_d);

						//sum_poynting<<<dimGrid,dimBlock>>>(poyn_px_d,poyn_pz_d,poyn_sx_d,poyn_sz_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,txx2_d,tzz2_d,txz2_d,tp2_d,nx_append_radius,nz_append_radius);
							
						//decom<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,coe_opt_d,nx_append,nz_append,dx,dz);

						//decom_new<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,velocity_d,velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);
						if(0==(it-wavelet_half)%100&&join_wavefield==1&&iter==0)
						{
							hipMemcpy(wf_append,vx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./wavefield1/0/vx-%d-shot_%d",ishot+1,it-wavelet_half);
							write_file_1d(wf_append,nxanza,filename);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size_nz,filename);
										
							hipMemcpy(wf_append,vxp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./wavefield1/0/vxp-%d-shot_%d",ishot+1,it-wavelet_half);
							write_file_1d(wf_append,nxanza,filename);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size_nz,filename);
						}

						if(it>=wavelet_half&&it<(lt+wavelet_half))
						{
							//write_shot<<<receiver_num,1>>>(vx2_d,vz2_d,obs_shot_x_d,obs_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dt,source_x_cord[ishot],velocity_d,wavelet_half);
							if(receiver_offset==0)
							{
								write_shot_x_z<<<receiver_num,1>>>(vx2_d,obs_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								write_shot_x_z<<<receiver_num,1>>>(vz2_d,obs_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
							}
							else
							{
								write_shot_x_z_acqusition<<<receiver_num,1>>>(vx2_d,obs_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								write_shot_x_z_acqusition<<<receiver_num,1>>>(vz2_d,obs_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
							}
//////////////vsp_2
							if(vsp_2!=0)
							{
								write_shot_x_z<<<receiver_num,1>>>(vx2_d,obs_shot_x_d_2,it-wavelet_half,lt,receiver_num_2,receiver_x_cord_2[ishot],receiver_interval_2,receiver_z_cord_2[ishot],receiver_z_interval_2,boundary_left,boundary_up,nz_append);///for vsp 2017年03月17日 星期二 08时46分12秒 
								write_shot_x_z<<<receiver_num,1>>>(vz2_d,obs_shot_z_d_2,it-wavelet_half,lt,receiver_num_2,receiver_x_cord_2[ishot],receiver_interval_2,receiver_z_cord_2[ishot],receiver_z_interval_2,boundary_left,boundary_up,nz_append);///for vsp 2017年03月17日 星期二 08时46分12秒 
							}
						}
					
						rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
						rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
						rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
						rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
						rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

						rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
						rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
						rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
						rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
						rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;
										
						rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
						rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
						rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
						rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;
				}
					
						if(ishot%40==0)
						{
							hipMemcpy(shotgather,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct_obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct_obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}

//cut direct wave
						if(cut_direct_wave==0||cut_direct_wave==1)
						{
							cal_sub_a_b_to_c<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,cal_shot_x_d,obs_shot_x_d,receiver_num,lt);

							cal_sub_a_b_to_c<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,cal_shot_z_d,obs_shot_z_d,receiver_num,lt);

							if(receiver_offset!=0)
							{							
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);							
							}

							hipMemcpy(shotgather,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}

						else///for vsp 2017年03月14日 星期二 08时55分03秒 
						{
							cut_direct_new1<<<dimGrid,dimBlock>>>(obs_shot_x_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half,cut_direct_wave);
							cut_direct_new1<<<dimGrid,dimBlock>>>(obs_shot_z_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half,cut_direct_wave);

							if(receiver_offset!=0)
							{							
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);							
							}
							hipMemcpy(shotgather,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}		

						if(amplitude_error!=0)//////////////////////////To bring in large amplitude errors/////for correlation_misfit
						{
							cuda_mul_error_random<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,error_random_d,receiver_interval,receiver_num,lt);
							cuda_mul_error_random<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,error_random_d,receiver_interval,receiver_num,lt);

							hipMemcpy(shotgather,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}

						if(shot_scale!=0)//////////////////////////To bring in large amplitude errors/////for correlation_misfit
						{
							//cuda_mul_shot_scale<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,ishot,shot_num,shot_scale,receiver_num,lt);
							//cuda_mul_shot_scale<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,ishot,shot_num,shot_scale,receiver_num,lt);

							cuda_mul_shot_scale_new<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,ishot,shot_num,shot_scale_d,receiver_num,lt);
							cuda_mul_shot_scale_new<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,ishot,shot_num,shot_scale_d,receiver_num,lt);

							hipMemcpy(shotgather,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}
//////////////////////////////////////////////////output:vsp2:::::::::
						if(vsp_2!=0)
						{
							hipMemcpy(shotgather,obs_shot_x_d_2,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d_2",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,obs_shot_z_d_2,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/obs_shot_z_%d_2",ishot+1);
							write_file_1d(shotgather,lt_rec,filename);
						}				
	
				ishot++;	
		}
		warn("modeling seismic is over");
		hipEventRecord(stop);/* record ending time */
  		hipEventSynchronize(stop);
  		hipEventElapsedTime(&mstimer, start, stop);
		totaltime+=mstimer*1e-3;

		warn("modeling finished: %f (s)\n", mstimer*1e-3);////////to current step has cost times
		fprintf(logfile,"modeling finished: %f (s)\n\n", mstimer*1e-3);////////the current step  cost times
		fclose(logfile);////important


		
		warn("iterative inversion begin");	
		//for(iter=0;iter<niter;iter++)
		for(iter=iter_start;iter<niter;iter++)
		{
			if(iter==1)
			{
				hipMemcpy(wf_append_new,d_illum_new,nxa_new_nza*sizeof(float),hipMemcpyDeviceToHost);
				write_file_1d(wf_append_new,nxa_new_nza,"./check_file/d_illum");
			}
			
			if(iter_start!=0)
			{
				fread_file_1d(wf_append_new,nx_append_new,nz_append,"./check_file/d_illum");
				hipMemcpy(d_illum_new,wf_append_new,nxa_new_nza*sizeof(float),hipMemcpyHostToDevice);
			}
///////////////////2017年03月31日 星期五 08时20分42秒 
			if(iter>=5)
			{
				if(iter==5)	system("mkdir ./someoutput/save");

				rm_f_file=fopen("rm-f-shot","wb");//////cal_shot_*_iter_1  res_shot_*_iter_1
				fprintf(rm_f_file,"#!/bin/sh\n");

				fprintf(rm_f_file,"cp -r 	./someoutput/bin/res_shot_*_%d_iter_%d   ./someoutput/save\n",int((shot_num+1)/2.0),iter-2);
				fprintf(rm_f_file,"cp -r 	./someoutput/bin/cal_shot_*_%d_iter_%d   ./someoutput/save\n",int((shot_num+1)/2.0),iter-2);
				fprintf(rm_f_file,"cp -r 	./someoutput/bin/tmp_shot_*_%d_iter_%d   ./someoutput/save\n",int((shot_num+1)/2.0),iter-2);
				fprintf(rm_f_file,"cp -r 	./someoutput/bin/adj_shot_*_%d_iter_%d   ./someoutput/save\n",int((shot_num+1)/2.0),iter-2);
				//fprintf(rm_f_file,"cp -r 	./someoutput/bin/adj1_shot_*_%d_iter_%d   ./someoutput/save\n",int((shot_num+1)/2.0),iter-2);
				

				fprintf(rm_f_file,"rm -f 	./someoutput/bin/res_shot_*_iter_%d\n",iter-2);////////rm -f res
				fprintf(rm_f_file,"rm -f     ./someoutput/bin/cal_shot_*_iter_%d\n",iter-2);////////rm -f cal
				fprintf(rm_f_file,"rm -f     ./someoutput/bin/tmp_shot_*_iter_%d\n",iter-2);////////rm -f tmp
				fprintf(rm_f_file,"rm -f     ./someoutput/bin/adj_shot_*_iter_%d\n",iter-2);////////rm -f tmp
				//fprintf(rm_f_file,"rm -f     ./someoutput/bin/adj1_shot_*_iter_%d\n",iter-2);////////rm -f tmp

				fclose(rm_f_file);////important

				system("sh rm-f-shot");
			}
///////////////////2017年03月31日 星期五 08时20分42秒
			
			//warn("1\n");

			logfile=fopen("log.txt","ab");//remember to free log file			

			hipEventRecord(start);/* record starting time */

			if(laplace_compensate!=0)
			{
				hipMemcpy(wavelet_d,wavelet_integral,wavelet_length*sizeof(float),hipMemcpyHostToDevice);
			}/////////////////////////tiwce integral

//////
			hipMemset(obs_shot_x_d,0,lt_rec*sizeof(float));
			hipMemset(obs_shot_z_d,0,lt_rec*sizeof(float));
			hipMemset(cal_shot_x_d,0,lt_rec*sizeof(float));
			hipMemset(cal_shot_z_d,0,lt_rec*sizeof(float));
			hipMemset(res_shot_x_d,0,lt_rec*sizeof(float));
			hipMemset(res_shot_z_d,0,lt_rec*sizeof(float));
			hipMemset(tmp_shot_x_d,0,lt_rec*sizeof(float));
			hipMemset(tmp_shot_z_d,0,lt_rec*sizeof(float));				
			hipMemset(adj_shot_x_d,0,lt_rec*sizeof(float));
			hipMemset(adj_shot_z_d,0,lt_rec*sizeof(float));

			hipMemset(obs_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMemset(obs_shot_z_d_2,0,lt_rec*sizeof(float));
			hipMemset(cal_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMemset(cal_shot_z_d_2,0,lt_rec*sizeof(float));
			hipMemset(res_shot_x_d_2,0,lt_rec*sizeof(float));
			hipMemset(res_shot_z_d_2,0,lt_rec*sizeof(float));
//////
////setz zero 
			if(decomposition!=0)
			{
				hipMemset(all_grad_den_pp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame1_pp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame2_pp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_den_ps_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame1_ps_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame2_ps_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_den_sp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame1_sp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame2_sp_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_den_ss_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame1_ss_d,0,nxnz*sizeof(float));
				hipMemset(all_grad_lame2_ss_d,0,nxnz*sizeof(float));
			}

			hipMemset(all_grad_lame11_d,0,nxnz*sizeof(float));
			hipMemset(all_grad_lame22_d,0,nxnz*sizeof(float));
			hipMemset(all_grad_den1_d,0,nxnz*sizeof(float));

			hipMemset(all_grad_vp1_d,0,nxnz*sizeof(float));
			hipMemset(all_grad_vs1_d,0,nxnz*sizeof(float));
			hipMemset(all_grad_density1_d,0,nxnz*sizeof(float));
///exchange gradient
			hipMemcpy(all_hydrid_grad1_d,all_hydrid_grad2_d,3*nxnz*sizeof(float), hipMemcpyDeviceToDevice);
			hipMemset(all_hydrid_grad2_d,0,3*nxnz*sizeof(float));//////hybrid conjugated gradient method for exchange
///exchange gradient

///set_zero for objetive vaule
			hipMemset(obj_parameter_d,0,3*sizeof(float));
			hipMemset(obj_d,0,3*sizeof(float));
			memset(obj_h,0,3*sizeof(float));
///set_zero for objetive vaule

			ishot=0;
			while(ishot<shot_num)
			{		
				if(fmod((ishot*1.0),40.0)==0)		
				{
					//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 1,it=%d",ishot+1,(it+1)-wavelet_half);
					warn("ishot=%d",ishot+1);
					warn("shot cord (x:%d z:%d)",source_x_cord[ishot],shot_depth);
					warn("receiver cord [start (x:%d z:%d ) interval:%d number:%d]",receiver_x_cord[ishot],receiver_depth,receiver_interval,receiver_num);
					warn("imaging scope(start:%d size:%d end:%d)",imaging_start[ishot],imaging_size[ishot],imaging_end[ishot]);
				}

/////////////////////////////////////get smooth vp;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

				hipMemcpy(wf,wf_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
				sprintf(filename,"./someoutput/cut-vp-s-%d.bin",ishot+1);
				write_file_1d(wf,nx_size_nz,filename);

				hipMemcpy(wf_append,s_velocity_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
				sprintf(filename,"./someoutput/vp-s-%d.bin",ishot+1);
				write_file_1d(wf_append,nxanza,filename);

/////////////////////////////////////get smooth vs;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth density;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_density_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(s_density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qp;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_qp_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qs;				
				cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_qs_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
				cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

				/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
				{								
					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

					cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);						
				}*/

/////////////////////////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
				cuda_cal_viscoelastic<<< dimGrid,dimBlock>>>(modul_p_d,modul_s_d,s_qp_d,s_qs_d,tao_d,strain_p_d,strain_s_d,freq,s_velocity_d,s_velocity1_d,s_density_d,nx_append,nz_append);


				hipMemset(correlation_parameter_d,0,10*sizeof(float));/////////correlation misfit function  it is important for calculating adjiont source
				//////???????????????????????????????????
				hipMemset(obs_shot_x_d,0,lt_rec*sizeof(float));
				hipMemset(obs_shot_z_d,0,lt_rec*sizeof(float));
				hipMemset(cal_shot_x_d,0,lt_rec*sizeof(float));
				hipMemset(cal_shot_z_d,0,lt_rec*sizeof(float));
				hipMemset(res_shot_x_d,0,lt_rec*sizeof(float));
				hipMemset(res_shot_z_d,0,lt_rec*sizeof(float));
				hipMemset(tmp_shot_x_d,0,lt_rec*sizeof(float));
				hipMemset(tmp_shot_z_d,0,lt_rec*sizeof(float));
				hipMemset(adj_shot_x_d,0,lt_rec*sizeof(float));
				hipMemset(adj_shot_z_d,0,lt_rec*sizeof(float));

				hipMemset(obs_shot_x_d_2,0,lt_rec*sizeof(float));
				hipMemset(obs_shot_z_d_2,0,lt_rec*sizeof(float));
				hipMemset(cal_shot_x_d_2,0,lt_rec*sizeof(float));
				hipMemset(cal_shot_z_d_2,0,lt_rec*sizeof(float));
				hipMemset(res_shot_x_d_2,0,lt_rec*sizeof(float));
				hipMemset(res_shot_z_d_2,0,lt_rec*sizeof(float));
				//////???????????????????????????????????
			
				if(iter==0)
				{
					//////fread obs shot	
					sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
					fread_file_1d(shotgather,receiver_num,lt,filename);
					hipMemcpy(obs_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

					sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
					fread_file_1d(shotgather,receiver_num,lt,filename);
					hipMemcpy(obs_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
					//////fread obs shot

					////////for sn!=0 data
					if(receiver_offset!=0)
					{					
						cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
						cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);	
					}

					//hipMemcpy(res_shot_x_d,obs_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);
					//hipMemcpy(res_shot_z_d,obs_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);///2016年10月27日 星期四 05时07分14秒 
					cuda_cal_residuals_new<<<dimGrid_lt,dimBlock>>>(res_shot_x_d,obs_shot_x_d,cal_shot_x_d,receiver_num,lt);
					cuda_cal_residuals_new<<<dimGrid_lt,dimBlock>>>(res_shot_z_d,obs_shot_z_d,cal_shot_z_d,receiver_num,lt);

					/////cuda_adj_shot
					hipMemcpy(adj_shot_x_d,res_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);
					hipMemcpy(adj_shot_z_d,res_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);

					/////////output first residuals	
					hipMemcpy(shotgather,res_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);

					hipMemcpy(shotgather,res_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);
					/////////output first residuals


					/////////output tmp cal	
					hipMemcpy(shotgather,tmp_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/tmp_shot_x_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);

					hipMemcpy(shotgather,tmp_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/tmp_shot_z_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);
					/////////output tmp cal
				}

				if(correlation_misfit==0)
				{
					if(iter>0)
					{	
						//////fread iter+1 res_shot			
						sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

						sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
					
						/////cuda_adj_shot
						hipMemcpy(adj_shot_x_d,res_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);
						hipMemcpy(adj_shot_z_d,res_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToDevice);
					}
////////calculate objective value
					cuda_cal_objective<<<1, Block_Size>>>(&obj_d[0], res_shot_x_d, lt_rec);
					hipMemcpy(&obj_exchange,&obj_d[0],1*sizeof(float),hipMemcpyDeviceToHost);
					obj_h[0]+=obj_exchange;

					cuda_cal_objective<<<1, Block_Size>>>(&obj_d[0], res_shot_z_d, lt_rec);
					hipMemcpy(&obj_exchange,&obj_d[0],1*sizeof(float),hipMemcpyDeviceToHost);
					obj_h[0]+=obj_exchange;
////////calculate objective value
				}

				else
				{
					if(iter>0)//correlation misfit function
					{	
						//////fread obs shot	
						sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(obs_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

						sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(obs_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
						//////fread obs shot

						////////for sn!=0 data
						if(receiver_offset!=0)
						{
							cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
							cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
						}

						//////fread iter+1 tmp_shot			
						sprintf(filename,"./someoutput/bin/tmp_shot_x_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(tmp_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

						sprintf(filename,"./someoutput/bin/tmp_shot_z_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(tmp_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
					
						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,tmp_shot_x_d,lt_rec,&correlation_parameter_d[0]);
						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,tmp_shot_z_d,lt_rec,&correlation_parameter_d[0]);
						

						cuda_dot_sum<<<1,Block_Size>>>(obs_shot_x_d,obs_shot_x_d,lt_rec,&correlation_parameter_d[1]);
						cuda_dot_sum<<<1,Block_Size>>>(obs_shot_z_d,obs_shot_z_d,lt_rec,&correlation_parameter_d[1]);


						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,obs_shot_x_d,lt_rec,&correlation_parameter_d[2]);
						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,obs_shot_z_d,lt_rec,&correlation_parameter_d[2]);


						cuda_adj_shot<<<dimGrid_lt,dimBlock>>>(adj_shot_x_d,tmp_shot_x_d,obs_shot_x_d,receiver_num,lt,correlation_parameter_d);
						cuda_adj_shot<<<dimGrid_lt,dimBlock>>>(adj_shot_z_d,tmp_shot_z_d,obs_shot_z_d,receiver_num,lt,correlation_parameter_d);

						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,tmp_shot_x_d,lt_rec,&obj_parameter_d[0]);
						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,tmp_shot_z_d,lt_rec,&obj_parameter_d[0]);
						

						cuda_dot_sum<<<1,Block_Size>>>(obs_shot_x_d,obs_shot_x_d,lt_rec,&obj_parameter_d[1]);
						cuda_dot_sum<<<1,Block_Size>>>(obs_shot_z_d,obs_shot_z_d,lt_rec,&obj_parameter_d[1]);


						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,obs_shot_x_d,lt_rec,&obj_parameter_d[2]);
						cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,obs_shot_z_d,lt_rec,&obj_parameter_d[2]);

						if(ishot==shot_num-1)//correlation misfit function
						{	
							cuda_cal_correlation_objective<<<1,1>>>(&obj_d[0],obj_parameter_d);
							hipMemcpy(&obj_exchange,&obj_d[0],1*sizeof(float),hipMemcpyDeviceToHost);
							obj_h[0]=obj_exchange;
						}
					}
				}
					/////////output adj_shot	
					hipMemcpy(shotgather,adj_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/adj_shot_x_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);

					hipMemcpy(shotgather,adj_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
					sprintf(filename,"./someoutput/bin/adj_shot_z_%d_iter_%d",ishot+1,iter+1);
					write_file_1d(shotgather,lt_rec,filename);

					/*if(receiver_offset!=0)////////////////try to weigthed adjoint source
					{
						cut_direct<<<dimGrid,dimBlock>>>(adj_shot_x_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half);
						cut_direct<<<dimGrid,dimBlock>>>(adj_shot_z_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half);

						cuda_attenuation_adj<<<dimGrid_lt,dimBlock>>>(adj_shot_x_d,receiver_num,lt,offset_left[ishot],offset_right[ishot],receiver_offset);
						cuda_attenuation_adj<<<dimGrid_lt,dimBlock>>>(adj_shot_z_d,receiver_num,lt,offset_left[ishot],offset_right[ishot],receiver_offset);
						/////////output adj_shot	
						hipMemcpy(shotgather,adj_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./someoutput/bin/adj1_shot_x_%d_iter_%d",ishot+1,iter+1);
						write_file_1d(shotgather,lt_rec,filename);

						hipMemcpy(shotgather,adj_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./someoutput/bin/adj1_shot_z_%d_iter_%d",ishot+1,iter+1);
						write_file_1d(shotgather,lt_rec,filename);
					}*/

				//set zero
				hipMemset(vx1_d,0,nxanza*sizeof(float));
				hipMemset(vz1_d,0,nxanza*sizeof(float));
				hipMemset(txx1_d,0,nxanza*sizeof(float));
				hipMemset(tzz1_d,0,nxanza*sizeof(float));
				hipMemset(txz1_d,0,nxanza*sizeof(float));

				hipMemset(vx2_d,0,nxanza*sizeof(float));
				hipMemset(vz2_d,0,nxanza*sizeof(float));
				hipMemset(txx2_d,0,nxanza*sizeof(float));
				hipMemset(tzz2_d,0,nxanza*sizeof(float));
				hipMemset(txz2_d,0,nxanza*sizeof(float));

				hipMemset(mem_p1_d,0,nxanza*sizeof(float));
				hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
				hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
				hipMemset(mem_xz1_d,0,nxanza*sizeof(float));

				hipMemset(mem_p2_d,0,nxanza*sizeof(float));
				hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
				hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
				hipMemset(mem_xz2_d,0,nxanza*sizeof(float));

				hipMemset(rvx1_d,0,nxanza*sizeof(float));
				hipMemset(rvz1_d,0,nxanza*sizeof(float));
				hipMemset(rtxx1_d,0,nxanza*sizeof(float));
				hipMemset(rtzz1_d,0,nxanza*sizeof(float));
				hipMemset(rtxz1_d,0,nxanza*sizeof(float));

				hipMemset(rvx2_d,0,nxanza*sizeof(float));
				hipMemset(rvz2_d,0,nxanza*sizeof(float));
				hipMemset(rtxx2_d,0,nxanza*sizeof(float));
				hipMemset(rtzz2_d,0,nxanza*sizeof(float));
				hipMemset(rtxz2_d,0,nxanza*sizeof(float));

				hipMemset(rmem_p1_d,0,nxanza*sizeof(float));
				hipMemset(rmem_xx1_d,0,nxanza*sizeof(float));
				hipMemset(rmem_zz1_d,0,nxanza*sizeof(float));
				hipMemset(rmem_xz1_d,0,nxanza*sizeof(float));

				hipMemset(rmem_p2_d,0,nxanza*sizeof(float));
				hipMemset(rmem_xx2_d,0,nxanza*sizeof(float));
				hipMemset(rmem_zz2_d,0,nxanza*sizeof(float));
				hipMemset(rmem_xz2_d,0,nxanza*sizeof(float));

				if(iter==0||decomposition!=0)
				{
					hipMemset(tp2_d,0,nxanza*sizeof(float));
					hipMemset(tp1_d,0,nxanza*sizeof(float));
					hipMemset(vxp2_d,0,nxanza*sizeof(float));
					hipMemset(vxp1_d,0,nxanza*sizeof(float));
					hipMemset(vzp2_d,0,nxanza*sizeof(float));
					hipMemset(vzp1_d,0,nxanza*sizeof(float));
					hipMemset(vxs2_d,0,nxanza*sizeof(float));
					hipMemset(vxs1_d,0,nxanza*sizeof(float));
					hipMemset(vzs2_d,0,nxanza*sizeof(float));
					hipMemset(vzs1_d,0,nxanza*sizeof(float));

					hipMemset(rtp2_d,0,nxanza*sizeof(float));
					hipMemset(rtp1_d,0,nxanza*sizeof(float));
					hipMemset(rvxp2_d,0,nxanza*sizeof(float));
					hipMemset(rvxp1_d,0,nxanza*sizeof(float));
					hipMemset(rvzp2_d,0,nxanza*sizeof(float));
					hipMemset(rvzp1_d,0,nxanza*sizeof(float));
					hipMemset(rvxs2_d,0,nxanza*sizeof(float));
					hipMemset(rvxs1_d,0,nxanza*sizeof(float));
					hipMemset(rvzs2_d,0,nxanza*sizeof(float));
					hipMemset(rvzs1_d,0,nxanza*sizeof(float));

					hipMemset(vx_x_d,0,nxanza*sizeof(float));
					hipMemset(vx_z_d,0,nxanza*sizeof(float));
					hipMemset(vz_x_d,0,nxanza*sizeof(float));
					hipMemset(vz_z_d,0,nxanza*sizeof(float));
					hipMemset(vx_t_d,0,nxanza*sizeof(float));
					hipMemset(vz_t_d,0,nxanza*sizeof(float));
				
					hipMemset(rvxp_integral_d,0,nxanza*sizeof(float));
					hipMemset(rvzp_integral_d,0,nxanza*sizeof(float));
					hipMemset(rvxs_integral_d,0,nxanza*sizeof(float));
					hipMemset(rvzs_integral_d,0,nxanza*sizeof(float));

					hipMemset(grad_den_pp_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame1_pp_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame2_pp_d,0,nx_size_nz*sizeof(float));

					hipMemset(grad_den_ps_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame1_ps_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame2_ps_d,0,nx_size_nz*sizeof(float));

					hipMemset(grad_den_sp_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame1_sp_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame2_sp_d,0,nx_size_nz*sizeof(float));

					hipMemset(grad_den_ss_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame1_ss_d,0,nx_size_nz*sizeof(float));
					hipMemset(grad_lame2_ss_d,0,nx_size_nz*sizeof(float));
				}

				if(iter==0)/////////////for migration
				{	
					hipMemset(resultpp_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultps_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultps1_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultps2_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultsp_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultsp1_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultsp2_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultss_d,0,nx_size_nz*sizeof(float));
		
					hipMemset(result_tp_d,0,nx_size_nz*sizeof(float));		
					hipMemset(vresultpp_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultps_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultsp_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultss_d,0,nx_size_nz*sizeof(float));

					hipMemset(vresultppx_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultpsx_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultspx_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultssx_d,0,nx_size_nz*sizeof(float));

					hipMemset(vresultppz_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultpsz_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultspz_d,0,nx_size_nz*sizeof(float));
					hipMemset(vresultssz_d,0,nx_size_nz*sizeof(float));


					hipMemset(down_vpp_x_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_vpp_z_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_vss_x_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_vss_z_d,0,nx_size_nz*sizeof(float));

					hipMemset(down_tp_d,0,nx_size_nz*sizeof(float));

					hipMemset(down_vpp_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_vss_d,0,nx_size_nz*sizeof(float));

					hipMemset(down_pp_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_ss_d,0,nx_size_nz*sizeof(float));
				
					hipMemset(down_xx_d,0,nx_size_nz*sizeof(float));
					hipMemset(down_zz_d,0,nx_size_nz*sizeof(float));
//////excitation:   	related function in kernel_3	
					hipMemset(com_ex_vresultpp_d,0,nx_size_nz*sizeof(float));
					hipMemset(com_ex_vresultps_d,0,nx_size_nz*sizeof(float));
					hipMemset(ex_vresultpp_d,0,nx_size_nz*sizeof(float));
					hipMemset(ex_vresultps_d,0,nx_size_nz*sizeof(float));
					hipMemset(ex_result_tp_d,0,nx_size_nz*sizeof(float));
					hipMemset(ex_result_tp_old_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultxx_d,0,nx_size_nz*sizeof(float));
					hipMemset(resultzz_d,0,nx_size_nz*sizeof(float));

					hipMemset(ex_amp_d,0,nxanza*sizeof(float));
					hipMemset(ex_time_d,0,nxanza*sizeof(float));
					hipMemset(ex_amp_x_d,0,nxanza*sizeof(float));
					hipMemset(ex_amp_z_d,0,nxanza*sizeof(float));
					hipMemset(ex_amp_tp_old_d,0,nxanza*sizeof(float));

					hipMemset(ex_tp_time_d,0,nxanza*sizeof(float));
					hipMemset(ex_amp_tp_d,0,nxanza*sizeof(float));
			
					hipMemset(para_max_d,0,20*sizeof(float));

					hipMemset(ex_angle_pp_d,0,nxanza*sizeof(float));
					hipMemset(ex_angle_rpp_d,0,nxanza*sizeof(float));
					hipMemset(ex_angle_rps_d,0,nxanza*sizeof(float));

					hipMemset(ex_angle_pp1_d,0,nxanza*sizeof(float));
					hipMemset(ex_angle_rpp1_d,0,nxanza*sizeof(float));
					hipMemset(ex_angle_rps1_d,0,nxanza*sizeof(float));

					hipMemset(ex_open_pp_d,0,nxanza*sizeof(float));
					hipMemset(ex_open_ps_d,0,nxanza*sizeof(float));
					hipMemset(ex_open_pp1_d,0,nxanza*sizeof(float));
					hipMemset(ex_open_ps1_d,0,nxanza*sizeof(float));

					hipMemset(ex_com_pp_sign_d,0,nxanza*sizeof(float));
					hipMemset(ex_com_ps_sign_d,0,nxanza*sizeof(float));
				}

////setz zero for gradient
				hipMemset(grad_lame11_d,0,nx_size_nz*sizeof(float));
				hipMemset(grad_lame22_d,0,nx_size_nz*sizeof(float));
				hipMemset(grad_den1_d,0,nx_size_nz*sizeof(float));
//////setz zero for gradient

///////setz zero for gradient
				hipMemset(grad_vp1_d,0,nx_size_nz*sizeof(float));
				hipMemset(grad_vs1_d,0,nx_size_nz*sizeof(float));
				hipMemset(grad_density1_d,0,nx_size_nz*sizeof(float));/////////it is noted that set zero for calculating gradient
///////setz zero for gradient
				hipMemset(d_illum,0,nxanza*sizeof(float));///////////////for  every shot  and variation////so we must set zero

			if(iter==0)
			{
				for(int it=0;it<lt+wavelet_half;it++)
					{
						//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 2,it=%d",ishot+1,(it+1)-wavelet_half);

						if(it<wavelet_length)
						{
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],shot_depth,it,boundary_up,boundary_left,nz_append);
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for vsp or surface 2017年03月14日 星期二 08时41分20秒
							add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);////for new acqusition way 2017年08月17日 星期四 09时10分03秒
						}

						fwd_vx_new_new<<<dimGrid,dimBlock>>>(d_illum,vx_t_d,vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

						fwd_vz_new_new<<<dimGrid,dimBlock>>>(d_illum,vz_t_d,vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

						if(decomposition!=0||iter==0)
						{
							fwd_vxp_vzp<<<dimGrid,dimBlock>>>(vxp2_d,vxp1_d,vzp2_d,vzp1_d,tp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);

							vp_vs<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,nx_append_radius,nz_append_radius);

							//decom<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,coe_opt_d,nx_append,nz_append,dx,dz);
							//decom_new<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);
							sum_poynting<<<dimGrid,dimBlock>>>(poyn_px_d,poyn_pz_d,poyn_sx_d,poyn_sz_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,txx2_d,tzz2_d,txz2_d,tp2_d,nx_append_radius,nz_append_radius);

							cal_direction_2D_elastic<<<dimGrid,dimBlock>>>(direction_px_d,direction_pz_d,direction_sx_d,direction_sz_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,txx2_d,tzz2_d,txz2_d,tp2_d,nx_append_radius,nz_append_radius);
						}

						if(migration_type==0)	fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);	
						
						//if(migration_type==0)	fwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

						//else	fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);

						else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);

/////////////////////////////////////////////////////////////////////////////excitation imaging condition
						if(it>=wavelet_half)
						{
///1111
							//caculate_ex_amp_time<<<dimGrid,dimBlock>>>(p_down_d,ex_amp_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							//caculate_ex_amp_time_new<<<dimGrid,dimBlock>>>(vxp_down_d,vzp_down_d,ex_amp_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							//caculate_ex_x_z<<<dimGrid,dimBlock>>>(ex_amp_x_d,ex_amp_z_d,vxp_down_d,vzp_down_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);				
							//caculate_ex_angle<<<dimGrid,dimBlock>>>(ex_angle_d,angle_pp1_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);	
///2222			
							//caculate_ex_amp_time<<<dimGrid,dimBlock>>>(p_d,ex_amp_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							caculate_ex_amp_time_new<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,ex_amp_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							caculate_ex_tp_time_new<<<dimGrid,dimBlock>>>(tp1_d,ex_amp_tp_d,ex_tp_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							//caculate_ex_x_z_new<<<dimGrid,dimBlock>>>(ex_amp_x_d,ex_amp_z_d,vxp1_d,vzp1_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							caculate_ex_x_z<<<dimGrid,dimBlock>>>(ex_amp_x_d,ex_amp_z_d,vxp1_d,vzp1_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							caculate_ex_x_z<<<dimGrid,dimBlock>>>(ex_amp_tp_old_d,ex_amp_tp_old_d,tp1_d,tp1_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
								
							caculate_ex_angle_pp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_pp_d,poyn_px_d,poyn_pz_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);

							caculate_ex_angle_pp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_pp1_d,direction_px_d,direction_pz_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							//caculate_ex_angle<<<dimGrid,dimBlock>>>(ex_angle_d,angle_pp_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
							//caculate_ex_angle_new<<<dimGrid,dimBlock>>>(ex_angle1_d,angle_pp1_d,normal_angle_d,poyn_px_d,poyn_pz_d,ex_time_d,it-wavelet_half,nx_append_radius,nz_append_radius);
						}					
							
							/////for check 2017年07月27日 星期四 10时11分40秒
							if(0==(it-wavelet_half)%check_interval)
							{
								/////for check 2017年07月27日 星期四 10时11分40秒
								////////velocity
								hipMemcpy(wf_append,vx1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vx-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,vz1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,vxp1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vxp-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,vzp1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vzp-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								/*hipMemcpy(wf_append,vxs1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vxs-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,vzs1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/velocity/vzs-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);*/

								////////stress
								hipMemcpy(wf_append,txx1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/stress/txx-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,txz1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/stress/txz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,tzz1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/stress/tzz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,tp1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/stress/tp-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								////////memory
								hipMemcpy(wf_append,mem_p1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/memory/mem_p-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,mem_xx1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/memory/mem_xx-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,mem_zz1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/memory/mem_zz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);

								hipMemcpy(wf_append,mem_xz1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./check_file/memory/mem_xz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);								
							}
						
							if(0==(it-wavelet_half)%100&&join_wavefield==1&&iter==0)
							{	
								hipMemcpy(wf_append,vx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vx-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
									
								hipMemcpy(wf_append,vz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vz-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vxp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vxp-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vzp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vzp-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
						
								hipMemcpy(wf_append,vxs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vxs-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
				
								hipMemcpy(wf_append,vzs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/1/vzs-%d-shot_%d",ishot+1,it-wavelet_half);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
							}
						
							rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
							rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
							rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
							rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
							rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

							rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
							rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
							rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
							rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
							rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

							rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
							rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
							rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
							rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒
					}
			}
						cuda_cal_max<<<1,Block_Size>>>(&para_max_d[2],ex_amp_tp_old_d,nxanza);
						cuda_cal_max<<<1,Block_Size>>>(&para_max_d[0],ex_amp_tp_d,nxanza);
						cuda_cal_max<<<1,Block_Size>>>(&para_max_d[1],ex_amp_d,nxanza);

						hipMemcpy(&p_printf,&para_max_d[2],sizeof(float),hipMemcpyDeviceToHost);
						warn("tp_max_old=%f\n",p_printf);

						hipMemcpy(&p_printf,&para_max_d[0],sizeof(float),hipMemcpyDeviceToHost);
						warn("tp_max=%f\n",p_printf);

						hipMemcpy(&p_printf,&para_max_d[1],sizeof(float),hipMemcpyDeviceToHost);
						warn("amp_max=%f\n",p_printf);
						
						if(ishot%10==0)//outup excitation amp or time
						{	
////////amp
							hipMemcpy(wf_append,ex_amp_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							//sprintf(filename,"./someoutput/ex-amp");
							//write_file_1d(wf_append,nx_append*nz_append,filename);
							sprintf(filename,"./someoutput/cut-ex-amp_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
////////time
							hipMemcpy(wf_append,ex_tp_time_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-tp-time_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);

////////time
							hipMemcpy(wf_append,ex_time_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-time_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
////vxp
							hipMemcpy(wf_append,ex_amp_x_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-amp-vxp_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
///vzp
							hipMemcpy(wf_append,ex_amp_z_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-amp-vzp_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
///tp_new
							hipMemcpy(wf_append,ex_amp_tp_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-amp-tp_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
///tp_old
							hipMemcpy(wf_append,ex_amp_tp_old_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-amp-tp_old_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);

							hipMemcpy(wf_append,ex_angle_pp_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-angle-pp_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);

							hipMemcpy(wf_append,ex_angle_pp1_d,nx_append*nz_append*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-ex-angle-pp1_%d",ishot+1);
							exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							write_file_1d(wf,nx_size*nz,filename);
						}
						warn("forwrd_modeling_for_ERTM_is_over");
							
			for(int it=lt-1;it>=0;it--)
				{
					//if(fmod(it*1.0,1000)==0) warn("shot=%d,step=back,it=%d",ishot+1,it);

					if((0==fmod(it*1.0,check_interval)||(it==lt-1))&&it!=0)
					{
						hipMemset(vx1_d,0,nxanza*sizeof(float));
						hipMemset(vz1_d,0,nxanza*sizeof(float));
						hipMemset(txx1_d,0,nxanza*sizeof(float));
						hipMemset(tzz1_d,0,nxanza*sizeof(float));
						hipMemset(txz1_d,0,nxanza*sizeof(float));
						hipMemset(vx2_d,0,nxanza*sizeof(float));
						hipMemset(vz2_d,0,nxanza*sizeof(float));
						hipMemset(txx2_d,0,nxanza*sizeof(float));
						hipMemset(tzz2_d,0,nxanza*sizeof(float));
						hipMemset(txz2_d,0,nxanza*sizeof(float));

						hipMemset(tp2_d,0,nxanza*sizeof(float));
						hipMemset(tp1_d,0,nxanza*sizeof(float));
						hipMemset(vxp2_d,0,nxanza*sizeof(float));
						hipMemset(vxp1_d,0,nxanza*sizeof(float));
						hipMemset(vzp2_d,0,nxanza*sizeof(float));
						hipMemset(vzp1_d,0,nxanza*sizeof(float));
						hipMemset(vxs2_d,0,nxanza*sizeof(float));
						hipMemset(vxs1_d,0,nxanza*sizeof(float));
						hipMemset(vzs2_d,0,nxanza*sizeof(float));
						hipMemset(vzs1_d,0,nxanza*sizeof(float));

						hipMemset(mem_p1_d,0,nxanza*sizeof(float));
						hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
						hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
						hipMemset(mem_xz1_d,0,nxanza*sizeof(float));
						hipMemset(mem_p2_d,0,nxanza*sizeof(float));
						hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
						hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
						hipMemset(mem_xz2_d,0,nxanza*sizeof(float));

						if(0==fmod(it*1.0,check_interval))	
						{
							ittt_beg=it-check_interval;

							ittt_end=it;
						}

						else
						{
							if(check_residual==0)	
							{
								ittt_beg=it-check_interval+1;
								ittt_end=lt-1;
							}							
									
							else
							{				
								ittt_beg=it-check_residual+1;
								ittt_end=lt-1;
							}
						}
						
						//warn("ittt_beg=%d\n",ittt_beg);					
						//warn("ittt_end=%d\n",ittt_end);

						if(ittt_beg!=0)
						{
							///////velocity
							sprintf(filename,"./check_file/velocity/vx-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vx1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
						
							sprintf(filename,"./check_file/velocity/vz-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vz1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/velocity/vxp-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vxp1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/velocity/vzp-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vzp1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							/*sprintf(filename,"./check_file/velocity/vxs-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vxs1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/velocity/vzs-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(vzs1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);*/

							///////stress
							sprintf(filename,"./check_file/stress/txx-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(txx1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/stress/txz-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(txz1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/stress/tzz-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(tzz1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/stress/tp-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(tp1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							///////memory
							sprintf(filename,"./check_file/memory/mem_p-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(mem_p1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/memory/mem_xx-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(mem_xx1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/memory/mem_zz-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(mem_zz1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./check_file/memory/mem_xz-%d-shot_%d",ishot+1,ittt_beg);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(mem_xz1_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
						}

						//for(int ittt=ittt_beg;ittt<ittt_end;ittt++)
						for(int ittt=ittt_beg+1;ittt<ittt_end+1;ittt++)
						{
							//if(fmod((ittt+1.0),check_interval/2)==0) warn("shot=%d,step=forward,it=%d",ishot+1,(ittt+1));
/////////////////////////////////////////////recalculate and save wavefied						
							if(ittt<wavelet_length&&ittt!=ittt_beg+1)
							{
								add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],shot_depth,ittt,boundary_up,boundary_left,nz_append);
							}						

							fwd_vx_new<<<dimGrid,dimBlock>>>(vx_t_d,vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

							fwd_vz_new<<<dimGrid,dimBlock>>>(vz_t_d,vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
							if(decomposition!=0||iter==0)
							{
								fwd_vxp_vzp<<<dimGrid,dimBlock>>>(vxp2_d,vxp1_d,vzp2_d,vzp1_d,tp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);

								vp_vs<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,nx_append_radius,nz_append_radius);

								decom<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,coe_opt_d,nx_append,nz_append,dx,dz);

								//decom_new<<<dimGrid,dimBlock>>>(vx2_d,vz2_d,p_d,s_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);
							}

							if(migration_type==0)	fwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

							//else	fwd_txxzzxzpp_viscoelastic_and_memory<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);

							else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,packaging_d);
/////////////////////////////////////////////recalculate and save wavefied						
							mark=int(fmod(ittt*1.0,check_interval*1.0));

							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_x_d,&save_vx_x_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_z_d,&save_vx_z_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_t_d,&save_vx_t_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_x_d,&save_vz_x_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_z_d,&save_vz_z_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_t_d,&save_vz_t_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							if(iter==0||decomposition!=0)
							{
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vxp2_d,&save_vxp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vzp2_d,&save_vzp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vxs2_d,&save_vxs_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vzs2_d,&save_vzs_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(tp2_d,&save_tp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(p_d,&save_p_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(s_d,&save_s_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,0);
							}
/////////////////////////////////////////////recalculate and save wavefied
						
							if(0==ittt%100&&join_wavefield==1&&iter==0)
							{
								hipMemcpy(wf_append,vx_z_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vx-z-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vz_x_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vz-x-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vx-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								sprintf(filename,"./wavefield1/1/vx-%d-shot_%d",ishot+1,ittt);
								fread_file_1d(wf_append,nx_append,nz_append,filename);
								hipMemcpy(wf_append_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
								cal_sub_a_b_to_c<<<dimGrid,dimBlock>>>(vx2_d,wf_append_d,wf_append_d,nx_append,nz_append);
								hipMemcpy(wf_append,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vx-difference-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./wavefield1/1/vz-%d-shot_%d",ishot+1,ittt);
								fread_file_1d(wf_append,nx_append,nz_append,filename);
								hipMemcpy(wf_append_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
								cal_sub_a_b_to_c<<<dimGrid,dimBlock>>>(vz2_d,wf_append_d,wf_append_d,nx_append,nz_append);
								hipMemcpy(wf_append,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vz-difference-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
									
								hipMemcpy(wf_append,vxp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vxp-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vzp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vzp-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
									
								hipMemcpy(wf_append,vxs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vxs-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
	
								hipMemcpy(wf_append,vzs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vzs-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								sprintf(filename,"./wavefield1/1/vxp-%d-shot_%d",ishot+1,ittt);
								fread_file_1d(wf_append,nx_append,nz_append,filename);
								hipMemcpy(wf_append_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
								cal_sub_a_b_to_c<<<dimGrid,dimBlock>>>(vxp2_d,wf_append_d,wf_append_d,nx_append,nz_append);
								hipMemcpy(wf_append,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vxp-difference-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./wavefield1/1/vzs-%d-shot_%d",ishot+1,ittt);
								fread_file_1d(wf_append,nx_append,nz_append,filename);
								hipMemcpy(wf_append_d,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
								cal_sub_a_b_to_c<<<dimGrid,dimBlock>>>(vzs2_d,wf_append_d,wf_append_d,nx_append,nz_append);
								hipMemcpy(wf_append,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/2/vzs-difference-%d-shot_%d",ishot+1,ittt);
								write_file_1d(wf_append,nxanza,filename);
							}

								rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
								rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
								rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
								rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
								rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

								rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
								rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
								rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
								rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
								rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

								rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
								rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
								rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
								rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;
						}

								/*if(join_wavefield==1&&iter==0)
								{
									hipMemcpy(save_h,save_vx_x_d,check_interval*nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/save-vx-x-%d-shot_%d",ishot+1,it);
									write_file_1d(save_h,check_interval*nx_size_nz,filename);
								}*/
					}
					
/////////////////////////////////////////////recover/set wavefied						
							mark=int(fmod(it*1.0,check_interval*1.0));

							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_x_d,&save_vx_x_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_z_d,&save_vx_z_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vx_t_d,&save_vx_t_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_x_d,&save_vz_x_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_z_d,&save_vz_z_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vz_t_d,&save_vz_t_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							if(iter==0||decomposition!=0)
							{
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vxp2_d,&save_vxp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vzp2_d,&save_vzp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vxs2_d,&save_vxs_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(vzs2_d,&save_vzs_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(tp2_d,&save_tp_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(p_d,&save_p_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							save_and_set_wavefiled<<<dimGrid,dimBlock>>>(s_d,&save_s_d[mark*nx_size_nz],nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,1);
							}
////////////////////////////////////////////recover/set wavefied
						
						rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
						rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
						rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
						rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
						rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

						rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
						rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
						rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
						rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
						rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

						rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
						rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
						rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
						rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;///////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒
///2016年10月08日 星期六 10时03分28秒 伴随状态反传
												
							//wraddshot<<<receiver_num,1>>>(rvx2_d,rvz1_d,res_shot_x_d,res_shot_z_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_depth,receiver_num);
							//wraddshot_set<<<receiver_num,1>>>(rvx2_d,rvz1_d,res_shot_x_d,res_shot_z_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_depth,receiver_num);
						if(receiver_offset==0)
						{
							wraddshot_x_z<<<receiver_num,1>>>(rvx2_d,adj_shot_x_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,receiver_num,accumulation);//for vsp 2017年03月14日 星期二 09时02分11秒 
							wraddshot_x_z<<<receiver_num,1>>>(rvz2_d,adj_shot_z_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,receiver_num,accumulation);//for vsp 2017年03月14日 星期二 09时02分11秒 
						}
						else//correlation
						{
							wraddshot_x_z_acqusition<<<receiver_num,1>>>(rvx2_d,adj_shot_x_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,receiver_num,accumulation);//for vsp 2017年03月14日 星期二 09时02分11秒 
							wraddshot_x_z_acqusition<<<receiver_num,1>>>(rvz2_d,adj_shot_z_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,receiver_num,accumulation);//for vsp 2017年03月14日 星期二 09时02分11秒 
						}
					

						if(RTM_only==0)
						{
							if(migration_type==0)
							{	
								///2016年10月08日 星期六 10时03分28秒 伴随状态反传	
								//receiver wavefield reverse propagation   (vetor)						
								/*wraddshot<<<receiver_num,1>>>(rvx2_d,rvz2_d,res_shot_x_d,res_shot_z_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_depth,receiver_num);
										
								fwd_txxzzxz<<<dimGrid,dimBlock>>>(rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);		
								fwd_vx<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);						
								fwd_vz<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);*/
								adjoint_fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(rtp1_d,rtp2_d,rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

								adjoint_fwd_vx<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);

								adjoint_fwd_vz<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);

								if(decomposition!=0||iter==0)
								{
									fwd_vxp_vzp<<<dimGrid,dimBlock>>>(rvxp1_d,rvxp2_d,rvzp1_d,rvzp2_d,rtp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);////////////////////////why??????????????Wang's method works!!!!

									vp_vs<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,nx_append_radius,nz_append_radius);

									decom<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,coe_opt_d,nx_append,nz_append,dx,dz);

									//decom_new<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);///////////////divergence and curl operator  and amplitude correction in Li's method(2016)

									//poynting<<<dimGrid,dimBlock>>>(rtxx1_d,rtxz1_d,rtzz1_d,rvx1_d,rvz1_d,poyn_rz_d,poyn_rx_d,nx_append_radius,nz_append_radius);						
									//sum_poynting<<<dimGrid,dimBlock>>>(poyn_rpx_d,poyn_rpz_d,poyn_rsx_d,poyn_rsz_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,rtxx1_d,rtzz1_d,rtxz1_d,rtp1_d,nx_append_radius,nz_append_radius);
								}		
							}
					
							if(migration_type==1)
							{
								/////////////////////////////////adjoint equation in viscoelastic media  for viscoelastic LSRTM
								adjoint_fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(rtp1_d,rtp2_d,rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
							
								adjoint_fwd_memory<<<dimGrid,dimBlock>>>(rmem_p1_d,rmem_p2_d,rmem_xx1_d,rmem_xx2_d,rmem_zz1_d,rmem_zz2_d,rmem_xz1_d,rmem_xz2_d,rtp1_d,rtxx1_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
								adjoint_fwd_vx_viscoelastic<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,modul_p_d,modul_s_d,s_density_d,rmem_p1_d,rmem_xx1_d,rmem_zz1_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
								adjoint_fwd_vz_viscoelastic<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,modul_p_d,modul_s_d,s_density_d,rmem_p1_d,rmem_xx1_d,rmem_zz1_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
								if(decomposition!=0||iter==0)
								{
									fwd_vxp_vzp<<<dimGrid,dimBlock>>>(rvxp1_d,rvxp2_d,rvzp1_d,rvzp2_d,rtp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);	
										
									vp_vs<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,nx_append_radius,nz_append_radius);

									decom<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,coe_opt_d,nx_append,nz_append,dx,dz);

									//decom_new<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);///////////////divergence and curl operator  and amplitude correction in Li's method(2016)

									//poynting<<<dimGrid,dimBlock>>>(rtxx1_d,rtxz1_d,rtzz1_d,rvx1_d,rvz1_d,poyn_rz_d,poyn_rx_d,nx_append_radius,nz_append_radius);						
									//sum_poynting<<<dimGrid,dimBlock>>>(poyn_rpx_d,poyn_rpz_d,poyn_rsx_d,poyn_rsz_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,rtxx1_d,rtzz1_d,rtxz1_d,rtp1_d,nx_append_radius,nz_append_radius)
								}

								/////////////////////////////////receiver propgagation equation in viscoelastic media  for viscoelastic RTM
								/*receiver_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(rtp1_d,rtp2_d,rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,rmem_p1_d,rmem_p2_d,rmem_xx1_d,rmem_xx2_d,rmem_zz1_d,rmem_zz2_d,rmem_xz1_d,rmem_xz2_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);
			
								fwd_vx<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
							
								fwd_vz<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
											
								fwd_vxp_vzp<<<dimGrid,dimBlock>>>(rvxp1_d,rvxp2_d,rvzp1_d,rvzp2_d,rtp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);	
										
								vp_vs<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,nx_append_radius,nz_append_radius);

								decom<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,coe_opt_d,nx_append,nz_append,dx,dz);

								//decom_new<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);///////////////divergence and curl operator  and amplitude correction in Li's method(2016)

								//poynting<<<dimGrid,dimBlock>>>(rtxx1_d,rtxz1_d,rtzz1_d,rvx1_d,rvz1_d,poyn_rz_d,poyn_rx_d,nx_append_radius,nz_append_radius);*/
							}
						}
						
						else
						{
								fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(rtp1_d,rtp2_d,rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
		
								fwd_vx<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);						
								fwd_vz<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

								fwd_vxp_vzp<<<dimGrid,dimBlock>>>(rvxp1_d,rvxp2_d,rvzp1_d,rvzp2_d,rtp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);	
										
								vp_vs<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,nx_append_radius,nz_append_radius);

								decom<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,coe_opt_d,nx_append,nz_append,dx,dz);

								//decom_new<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rp_d,rs_d,s_velocity_d,s_velocity1_d,coe_opt_d,nx_append_radius,nz_append_radius,dx,dz);///////////////divergence and curl operator  and amplitude correction in Li's method(2016)

								//poynting<<<dimGrid,dimBlock>>>(rtxx1_d,rtxz1_d,rtzz1_d,rvx1_d,rvz1_d,poyn_rz_d,poyn_rx_d,nx_append_radius,nz_append_radius);						
								sum_poynting<<<dimGrid,dimBlock>>>(poyn_rpx_d,poyn_rpz_d,poyn_rsx_d,poyn_rsz_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,rtxx1_d,rtzz1_d,rtxz1_d,rtp1_d,nx_append_radius,nz_append_radius);

								cal_direction_2D_elastic<<<dimGrid,dimBlock>>>(direction_rpx_d,direction_rpz_d,direction_rsx_d,direction_rsz_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,rtxx1_d,rtzz1_d,rtxz1_d,rtp1_d,nx_append_radius,nz_append_radius);
						}					
		
							if(0==(it)%100&&join_wavefield==1&&iter==0)
							{
								hipMemcpy(wf_append,rvx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vx-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
																		
								hipMemcpy(wf_append,rvxp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vxp-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
										
								hipMemcpy(wf_append,rvxs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vxs-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,rvz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vz-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
																		
								hipMemcpy(wf_append,rvzp2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vzp-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
										
								hipMemcpy(wf_append,rvzs2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/vzs-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,poyn_rpz_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/pz-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,poyn_rsz_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/sz-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);


								hipMemcpy(wf_append,poyn_rpx_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/px-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,poyn_rsx_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/3/sx-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
							}

							rep=rvx1_d;rvx1_d=rvx2_d;rvx2_d=rep;
							rep=rvz1_d;rvz1_d=rvz2_d;rvz2_d=rep;
							rep=rtxx1_d;rtxx1_d=rtxx2_d;rtxx2_d=rep;
							rep=rtzz1_d;rtzz1_d=rtzz2_d;rtzz2_d=rep;
							rep=rtxz1_d;rtxz1_d=rtxz2_d;rtxz2_d=rep;

							rep=rtp1_d;rtp1_d=rtp2_d;rtp2_d=rep;
							rep=rvxp1_d;rvxp1_d=rvxp2_d;rvxp2_d=rep;
							rep=rvzp1_d;rvzp1_d=rvzp2_d;rvzp2_d=rep;
							rep=rvxs1_d;rvxs1_d=rvxs2_d;rvxs2_d=rep;
							rep=rvzs1_d;rvzs1_d=rvzs2_d;rvzs2_d=rep;/////fast...........................................

							rep=rmem_p1_d;rmem_p1_d=rmem_p2_d;rmem_p2_d=rep;
							rep=rmem_xx1_d;rmem_xx1_d=rmem_xx2_d;rmem_xx2_d=rep;
							rep=rmem_zz1_d;rmem_zz1_d=rmem_zz2_d;rmem_zz2_d=rep;
							rep=rmem_xz1_d;rmem_xz1_d=rmem_xz2_d;rmem_xz2_d=rep;


						if(iter==0)
						{
							imaging_correlation<<<dimGrid,dimBlock>>>(tp1_d,tp1_d,down_tp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
			
							imaging_correlation<<<dimGrid,dimBlock>>>(p_d,p_d,down_pp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(p_d,s_d,down_ss_d,nx_size,nz,nz_append,boundary_up,boundary_left);

							imaging_correlation<<<dimGrid,dimBlock>>>(vxp1_d,vxp1_d,down_vpp_x_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzp1_d,vzp1_d,down_vpp_z_d,nx_size,nz,nz_append,boundary_up,boundary_left);

							imaging_correlation<<<dimGrid,dimBlock>>>(vxs1_d,vxs1_d,down_vss_x_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzs1_d,vzs1_d,down_vss_z_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,vxp1_d,vzp1_d,down_vpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,vxs1_d,vzs1_d,down_vss_d,nx_size,nz,nz_append,boundary_up,boundary_left);
			
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,vxp1_d,vzp1_d,down_vpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,vxs1_d,vzs1_d,down_vss_d,nx_size,nz,nz_append,boundary_up,boundary_left);
///////////////////////////////////////////////////////////excitation imaing condition
///////////////////////////////////////////////////////one method to calculate reflected angle
							caculate_ex_angle_rp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_rpp_d,poyn_rpx_d,poyn_rpz_d,ex_time_d,it,nx_append_radius,nz_append_radius);
							caculate_ex_angle_rp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_rps_d,poyn_rsx_d,poyn_rsz_d,ex_time_d,it,nx_append_radius,nz_append_radius);
///////////////////////////////////////////////////////one method to calculate reflected angle
							caculate_ex_angle_rp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_rpp1_d,direction_rpx_d,direction_rpz_d,ex_time_d,it,nx_append_radius,nz_append_radius);
							caculate_ex_angle_rp_only_RTM<<<dimGrid,dimBlock>>>(ex_angle_rps1_d,direction_rsx_d,direction_rsz_d,ex_time_d,it,nx_append_radius,nz_append_radius);

							imaging_correlation_ex_2D<<<dimGrid,dimBlock>>>(ex_result_tp_d,ex_amp_tp_d,ex_tp_time_d,rtp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[0],it);

							imaging_correlation_ex_2D<<<dimGrid,dimBlock>>>(ex_result_tp_old_d,ex_amp_tp_old_d,ex_time_d,rtp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[0],it);


							imaging_inner_product_ex_2D<<<dimGrid,dimBlock>>>(ex_vresultpp_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxp2_d,rvzp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);
							imaging_inner_product_ex_2D<<<dimGrid,dimBlock>>>(ex_vresultps_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxs2_d,rvzs2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);

							//imaging_inner_product_ex_2D_new<<<dimGrid,dimBlock>>>(ex_vresultpp_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxp2_d,rvzp2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);
							//imaging_inner_product_ex_2D_new<<<dimGrid,dimBlock>>>(ex_vresultps_d,ex_amp_d,ex_amp_x_d,ex_amp_z_d,ex_time_d,rvxs2_d,rvzs2_d,nx_size,nz,nz_append,boundary_up,boundary_left,&para_max_d[1],it);

							//imaging_pp_compensate_dependent_angle_2D<<<dimGrid,dimBlock>>>(ex_angle_pp1_d,ex_angle_rpp1_d,com_ex_vresultpp_d,ex_vresultpp_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left,it);

							//imaging_ps_compensate_dependent_angle_2D<<<dimGrid,dimBlock>>>(ex_angle_pp1_d,ex_angle_rps1_d,com_ex_vresultps_d,ex_vresultps_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left,it);
							
							caculate_ex_open_pp_ps<<<dimGrid,dimBlock>>>(ex_open_pp_d,ex_angle_pp_d,ex_angle_rpp_d,nx_size,nz,nx_append,nz_append,boundary_up,boundary_left,it,ex_time_d);
							caculate_ex_open_pp_ps<<<dimGrid,dimBlock>>>(ex_open_ps_d,ex_angle_pp_d,ex_angle_rps_d,nx_size,nz,nx_append,nz_append,boundary_up,boundary_left,it,ex_time_d);

							caculate_ex_open_pp_ps<<<dimGrid,dimBlock>>>(ex_open_pp1_d,ex_angle_pp1_d,ex_angle_rpp1_d,nx_size,nz,nx_append,nz_append,boundary_up,boundary_left,it,ex_time_d);
							caculate_ex_open_pp_ps<<<dimGrid,dimBlock>>>(ex_open_ps1_d,ex_angle_pp1_d,ex_angle_rps1_d,nx_size,nz,nx_append,nz_append,boundary_up,boundary_left,it,ex_time_d);

//////////////////////////////////////////////////////////////xx or zz
							imaging_correlation_for_xxzz<<<dimGrid,dimBlock>>>(vxp1_d,vxs1_d,vxp1_d,vxs1_d,down_xx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation_for_xxzz<<<dimGrid,dimBlock>>>(vzp1_d,vzs1_d,vzp1_d,vzs1_d,down_zz_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation_for_xxzz<<<dimGrid,dimBlock>>>(vxp1_d,vxs1_d,rvxp1_d,rvxs1_d,resultxx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation_for_xxzz<<<dimGrid,dimBlock>>>(vzp1_d,vzs1_d,rvzp1_d,rvzs1_d,resultzz_d,nx_size,nz,nz_append,boundary_up,boundary_left);							
/////////////based on Li/Du' method 2016/2012  correction
							//ps8				
							/*set_sign_basedon_polarization_ps<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxs1_d,rvzs1_d,signx_d,signy_d,signz_d,nx_append_radius,nz_append_radius);					
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signx_d,filter_signx_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signy_d,filter_signy_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signz_d,filter_signz_d,nx_append_radius,nz_append_radius,scale);
							compare_sign<<<dimGrid,dimBlock>>>(filter_signx_d,filter_signy_d,filter_signz_d,sign_d,nx_append_radius,nz_append_radius);
							imaging_correlation_sign_ps<<<dimGrid,dimBlock>>>(p_d,rs_d,resultps2_d,sign_d,source_x_cord[ishot],nx_size,nz,nz_append,boundary_up,boundary_left);
//sp8
							set_sign_basedon_polarization_sp<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,rvxp1_d,rvzp1_d,signx_d,signy_d,signz_d,nx_append_radius,nz_append_radius);					
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signx_d,filter_signx_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signy_d,filter_signy_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signz_d,filter_signz_d,nx_append_radius,nz_append_radius,scale);
							compare_sign<<<dimGrid,dimBlock>>>(filter_signx_d,filter_signy_d,filter_signz_d,sign_d,nx_append_radius,nz_append_radius);
							imaging_correlation_sign<<<dimGrid,dimBlock>>>(s_d,rp_d,resultsp2_d,sign_d,nx_size,nz,nz_append,boundary_up,boundary_left);*/
//ADCIGS					
							/*set_sign_forps<<<dimGrid,dimBlock>>>(poyn_x_d,poyn_z_d,poyn_rx_d,poyn_rz_d,signx_d,signy_d,signz_d,nx_append_radius,nz_append_radius);				
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signx_d,filter_signx_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signy_d,filter_signy_d,nx_append_radius,nz_append_radius,scale);
							filter_sign_new_share<<<dimGrid,dimBlock>>>(signz_d,filter_signz_d,nx_append_radius,nz_append_radius,scale);
							compare_sign<<<dimGrid,dimBlock>>>(filter_signx_d,filter_signy_d,filter_signz_d,sign_d,nx_append_radius,nz_append_radius);
							imaging_correlation_sign<<<dimGrid,dimBlock>>>(p_d,rs_d,resultps2_d,sign_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_correlation_sign_ps<<<dimGrid,dimBlock>>>(p_d,rs_d,resultps2_d,sign_d,source_x_cord[ishot],nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation_sign<<<dimGrid,dimBlock>>>(s_d,rp_d,resultsp2_d,sign_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_correlation_sign_ps<<<dimGrid,dimBlock>>>(s_d,rp_d,resultsp2_d,sign_d,source_x_cord[ishot],nx_size,nz,nz_append,boundary_up,boundary_left);*/
/////////////based on Li/Du' method 2016/2012  correction
							//pp ps ps1
							imaging_correlation<<<dimGrid,dimBlock>>>(p_d,rp_d,resultpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(p_d,rs_d,resultps_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(s_d,rp_d,resultsp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(s_d,rs_d,resultss_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							
							imaging_correlation_source_x_cord<<<dimGrid,dimBlock>>>(p_d,rs_d,resultps1_d,nx_size,nz,nz_append,boundary_up,boundary_left,source_x_cord[ishot]-receiver_x_cord[ishot]);
							imaging_correlation_source_x_cord<<<dimGrid,dimBlock>>>(s_d,rp_d,resultsp1_d,nx_size,nz,nz_append,boundary_up,boundary_left,source_x_cord[ishot]-receiver_x_cord[ishot]);
							/////tp*tp
							imaging_correlation<<<dimGrid,dimBlock>>>(tp1_d,rtp1_d,result_tp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							/////vppx vppz
							imaging_correlation<<<dimGrid,dimBlock>>>(vxp1_d,rvxp1_d,vresultppx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzp1_d,rvzp1_d,vresultppz_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							/////vpsx vpsz
							imaging_correlation<<<dimGrid,dimBlock>>>(vxp1_d,rvxs1_d,vresultpsx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzp1_d,rvzs1_d,vresultpsz_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							/////vspx vspz
							imaging_correlation<<<dimGrid,dimBlock>>>(vxs1_d,rvxp1_d,vresultspx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzs1_d,rvzp1_d,vresultspz_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							/////vssx vssz
							imaging_correlation<<<dimGrid,dimBlock>>>(vxs1_d,rvxs1_d,vresultssx_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_correlation<<<dimGrid,dimBlock>>>(vzs1_d,rvzs1_d,vresultssz_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							/////vpp vps vsp vss
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxp1_d,rvzp1_d,vresultpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxs1_d,rvzs1_d,vresultps_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,rvxp1_d,rvzp1_d,vresultsp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							imaging_vector_correlation<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,rvxs1_d,rvzs1_d,vresultss_d,nx_size,nz,nz_append,boundary_up,boundary_left); 
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxp1_d,rvzp1_d,vresultpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxs1_d,rvzs1_d,vresultps_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,rvxp1_d,rvzp1_d,vresultsp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
							//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxs1_d,vzs1_d,rvxs1_d,rvzs1_d,vresultss_d,nx_size,nz,nz_append,boundary_up,boundary_left);
						}

						if(decomposition==0)
						{
							if(migration_type==0)
							{					
////cal_gradient_for_lame1
								//cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame11_d,rtxx2_d,rtzz2_d,vx_x_d,vz_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
	////cal_gradient_for_lame2
								//cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame22_d,rtxx2_d,rtxz2_d,rtzz2_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
////cal_gradient_for_density							
								//cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den1_d,vx_t_d,vz_t_d,rvx2_d,rvz2_d,dt,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

								cal_gradient_in_elastic_media<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

								//cal_gradient_in_elastic_media_new<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
							}

							if(migration_type==1)
							{
								cal_gradient_in_viscoelastic_media<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,rmem_xx2_d,rmem_xz2_d,rmem_zz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d);
								//cal_gradient_in_viscoelastic_media_new<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,rmem_xx2_d,rmem_xz2_d,rmem_zz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d);
							}
						}

						else
						{
////////////////////////////////////////////Ren 2016
							/*cal_derivation_x<<<dimGrid,dimBlock>>>(vx1_d,vx_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);
							cal_derivation_z<<<dimGrid,dimBlock>>>(vx1_d,vx_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);
							cal_derivation_z<<<dimGrid,dimBlock>>>(vz1_d,vz_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);
							cal_derivation_x<<<dimGrid,dimBlock>>>(vz1_d,vz_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);*/

///////////////////////////////forward vxp vzp for x or z direction derivation vxp vzp for x or z direction derivation
							cal_derivation_x<<<dimGrid,dimBlock>>>(vxp1_d,vxp_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_z<<<dimGrid,dimBlock>>>(vxp1_d,vxp_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);
	
							cal_derivation_z<<<dimGrid,dimBlock>>>(vzp1_d,vzp_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_x<<<dimGrid,dimBlock>>>(vzp1_d,vzp_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);

///////////////////////////////forward vxs vzs for x or z direction derivation			
							cal_derivation_x<<<dimGrid,dimBlock>>>(vxs1_d,vxs_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_z<<<dimGrid,dimBlock>>>(vxs1_d,vxs_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);
	
							cal_derivation_z<<<dimGrid,dimBlock>>>(vzs1_d,vzs_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_x<<<dimGrid,dimBlock>>>(vzs1_d,vzs_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);

///////////////////////////////back vxs vzs integral   for x or z direction derivation	
							sum_integral<<<dimGrid,dimBlock>>>(rvxp_integral_d,rvxp2_d,nx_append_radius,nz_append_radius);
	
							sum_integral<<<dimGrid,dimBlock>>>(rvzp_integral_d,rvzp2_d,nx_append_radius,nz_append_radius);

							sum_integral<<<dimGrid,dimBlock>>>(rvxs_integral_d,rvxs2_d,nx_append_radius,nz_append_radius);

							sum_integral<<<dimGrid,dimBlock>>>(rvzs_integral_d,rvzs2_d,nx_append_radius,nz_append_radius);

///////////////////////////////back vxs vzs integral   for x or z direction derivation	
							cal_derivation_x<<<dimGrid,dimBlock>>>(rvxp_integral_d,rvxp_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_z<<<dimGrid,dimBlock>>>(rvxp_integral_d,rvxp_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);
	
							cal_derivation_z<<<dimGrid,dimBlock>>>(rvzp_integral_d,rvzp_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_x<<<dimGrid,dimBlock>>>(rvzp_integral_d,rvzp_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);

///////////////////////////////back vxs vzs integral   for x or z direction derivation	
							cal_derivation_x<<<dimGrid,dimBlock>>>(rvxs_integral_d,rvxs_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_z<<<dimGrid,dimBlock>>>(rvxs_integral_d,rvxs_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);
	
							cal_derivation_z<<<dimGrid,dimBlock>>>(rvzs_integral_d,rvzs_z_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,0);

							cal_derivation_x<<<dimGrid,dimBlock>>>(rvzs_integral_d,rvzs_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,1);

							/*if(0==(it)%100&&join_wavefield==1&&iter==0)
							{
								hipMemcpy(wf_append,rvxp_integral_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield/3/vxp_integral-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								exchange(wf_append,wf,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size_nz,filename);
							}*/
////////////////////////////////////////////Ren 2016
////cal_gradient_for_density
								if(inversion_den!=0)
								{
									cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den_pp_d,vxp_t_d,vzp_t_d,rvxp2_d,rvzp2_d,dt,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den_ps_d,vxp_t_d,vzp_t_d,rvxs2_d,rvzs2_d,dt,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den_sp_d,vxs_t_d,vzs_t_d,rvxp2_d,rvzp2_d,dt,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den_ss_d,vxs_t_d,vzs_t_d,rvxs2_d,rvzs2_d,dt,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
								}
////cal_gradient_for_lame1
									//cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame11_d,rtxx2_d,rtzz2_d,vx_x_d,vz_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame1_pp_d,rvxp_x_d,rvzp_z_d,vxp_x_d,vzp_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

									cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame1_sp_d,rvxp_x_d,rvzp_z_d,vxs_x_d,vzs_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

									cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame1_ps_d,rvxs_x_d,rvzs_z_d,vxp_x_d,vzp_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

									cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame1_ss_d,rvxs_x_d,rvzs_z_d,vxs_x_d,vzs_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

////cal_gradient_for_lame2
									cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(rvxp_z_d,rvzp_x_d,wf_append_d,nx_append,nz_append);
									//cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame22_d,rtxx2_d,rtxz2_d,rtzz2_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame2_pp_d,rvxp_x_d,wf_append_d,rvzp_z_d,vxp_x_d,vzp_z_d,vxp_z_d,vzp_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

									cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame2_sp_d,rvxp_x_d,wf_append_d,rvzp_z_d,vxs_x_d,vzs_z_d,vxs_z_d,vzs_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);


									cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(rvxs_z_d,rvzs_x_d,wf_append_d,nx_append,nz_append);
									//cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame22_d,rtxx2_d,rtxz2_d,rtzz2_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
									cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame2_ps_d,rvxs_x_d,wf_append_d,rvzs_z_d,vxp_x_d,vzp_z_d,vxp_z_d,vzp_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);

									cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame2_ss_d,rvxs_x_d,wf_append_d,rvzs_z_d,vxs_x_d,vzs_z_d,vxs_z_d,vzs_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
						}
				}
//////////////////////////////////illumination
							if(iter==0)
							{	
//////////////////////////////////////////////////////////////2018年05月24日 星期四 20时18分34秒  compensate PP or  PS  new
								cuda_ex_com_pp_ps_sign<<<dimGrid,dimBlock>>>(ex_com_pp_sign_d,ex_open_pp1_d,nx_size,nz,nx_append,nz_append,0);
								cuda_ex_com_pp_ps_sign<<<dimGrid,dimBlock>>>(ex_com_ps_sign_d,ex_open_ps1_d,nx_size,nz,nx_append,nz_append,1);
								if(ishot==0)
								{
									sprintf(filename,"./someoutput/cut-ex-pp-sign_%d",ishot+1);
									hipMemcpy(wf_append,ex_com_pp_sign_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									write_file_1d(wf,nx_size*nz,filename);

									sprintf(filename,"./someoutput/cut-ex-ps-sign_%d",ishot+1);
									hipMemcpy(wf_append,ex_com_ps_sign_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									write_file_1d(wf,nx_size*nz,filename);
								}

/////////////////////////////////////////////////////////////////smooth pp sign								
								cuda_bell_smooth_2d<<<dimGrid_new,dimBlock>>>(ex_com_pp_sign_d,wf_append_d,20,nx_append,nz_append);	
								hipMemcpy(ex_com_pp_sign_d,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToDevice);
/////////////////////////////////////////////////////////////////smooth ps sign	
								cuda_bell_smooth_2d<<<dimGrid_new,dimBlock>>>(ex_com_ps_sign_d,wf_append_d,20,nx_append,nz_append);	
								hipMemcpy(ex_com_ps_sign_d,wf_append_d,nxanza*sizeof(float),hipMemcpyDeviceToDevice);

								if(ishot==0)
								{
									sprintf(filename,"./someoutput/cut-ex-pp-sign-smooth_%d",ishot+1);
									hipMemcpy(wf_append,ex_com_pp_sign_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									write_file_1d(wf,nx_size*nz,filename);

									sprintf(filename,"./someoutput/cut-ex-ps-sign-smooth_%d",ishot+1);
									hipMemcpy(wf_append,ex_com_ps_sign_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									write_file_1d(wf,nx_size*nz,filename);
								}

								imaging_pp_compensate_dependent_angle_2D_new<<<dimGrid,dimBlock>>>(ex_open_pp1_d,ex_com_pp_sign_d,com_ex_vresultpp_d,ex_vresultpp_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left);

								imaging_ps_compensate_dependent_angle_2D_new<<<dimGrid,dimBlock>>>(ex_open_ps1_d,ex_com_ps_sign_d,com_ex_vresultps_d,ex_vresultps_d,ex_time_d,nx_size,nz,nz_append,boundary_up,boundary_left);

//////////////////////////////////////////////////////////////2018年05月24日 星期四 20时18分34秒  compensate PP or  PS  new
					
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(result_tp_d,down_tp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultpp_d,down_pp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultps_d,down_pp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultsp_d,down_ss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultss_d,down_ss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultps1_d,down_pp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultps2_d,down_pp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultsp1_d,down_ss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultsp2_d,down_ss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultpp_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultps_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultsp_d,down_vss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultss_d,down_vss_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultppx_d,down_vpp_x_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultpsx_d,down_vpp_x_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultspx_d,down_vss_x_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultssx_d,down_vss_x_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultppz_d,down_vpp_z_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultpsz_d,down_vpp_z_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultspz_d,down_vss_z_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultssz_d,down_vss_z_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
//////////////////////////////////////////////////////////////////////excitation amplitude imaging condition  attenuation only
								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(ex_vresultpp_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(ex_vresultps_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(ex_result_tp_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(ex_result_tp_old_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(com_ex_vresultpp_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM_ex_amp<<<dimGrid,dimBlock>>>(com_ex_vresultps_d,down_vpp_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
////////////////////////////////////////////////////////////////////////////////xx or zz
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultxx_d,down_xx_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
								cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(resultzz_d,down_zz_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,1.0,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
//////////////////RTM
//////////////////////////////////////////////////////////////////////excitation amplitude imaging condition
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_d,ex_result_tp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_old_d,ex_result_tp_old_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_ex_vresultpp_d,ex_vresultpp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_ex_vresultps_d,ex_vresultps_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);

								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultpp_d,com_ex_vresultpp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultps_d,com_ex_vresultps_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
////////////////////////////////////////////////////////////////////////////////xx or zz
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultxx_d,resultxx_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultzz_d,resultzz_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);	
///////////////////////////////////inner prodcut
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_result_tp_d,result_tp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultpp_d,vresultpp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultps_d,vresultps_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultsp_d,vresultsp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultss_d,vresultss_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
		///////////////////////////////////inner prodcut xxxxxxxxxxxxxx
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultppx_d,vresultppx_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultpsx_d,vresultpsx_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultspx_d,vresultspx_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultssx_d,vresultssx_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
		///////////////////////////////////inner prodcut zzzzzzzzzzzzzzz
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultppz_d,vresultppz_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultpsz_d,vresultpsz_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultspz_d,vresultspz_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_vresultssz_d,vresultssz_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
		///////////////////////////////////conventional migration 
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultpp_d,resultpp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultps_d,resultps_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultsp_d,resultsp_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultss_d,resultss_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
		///////////////////////////////////corrected ps or sp imaging conditions
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultps1_d,resultps1_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultsp1_d,resultsp1_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultps2_d,resultps2_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
								cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_resultsp2_d,resultsp2_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
//////////////////RTM
							}
							
							if(ishot%10==0)
							{
								hipMemcpy(wf,grad_lame11_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename1,"./result/RTM/grad_lame11_%d_iter_%d",ishot+1,iter+1);
								write_file_1d(wf,nx_size_nz,filename1);

								hipMemcpy(wf,grad_lame22_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename1,"./result/RTM/grad_lame22_%d_iter_%d",ishot+1,iter+1);
								write_file_1d(wf,nx_size_nz,filename1);

								
								sprintf(filename,"./check_file/down_pp_%d",ishot+1);
								hipMemcpy(wf_append,down_pp_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./check_file/down_ss_%d",ishot+1);
								hipMemcpy(wf_append,down_ss_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./check_file/down_vpp_%d",ishot+1);
								hipMemcpy(wf_append,down_vpp_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./check_file/down_vss_%d",ishot+1);
								hipMemcpy(wf_append,down_vss_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./check_file/down_xx_%d",ishot+1);
								hipMemcpy(wf_append,down_xx_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./check_file/down_zz_%d",ishot+1);
								hipMemcpy(wf_append,down_zz_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_append,nxanza,filename);

								sprintf(filename,"./someoutput/cut-ex-open-pp_%d",ishot+1);
								hipMemcpy(wf_append,ex_open_pp_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);

								sprintf(filename,"./someoutput/cut-ex-open-pp1_%d",ishot+1);
								hipMemcpy(wf_append,ex_open_pp1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);

								sprintf(filename,"./someoutput/cut-ex-open-ps_%d",ishot+1);
								hipMemcpy(wf_append,ex_open_ps_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);

								sprintf(filename,"./someoutput/cut-ex-open-ps1_%d",ishot+1);
								hipMemcpy(wf_append,ex_open_ps1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);


								sprintf(filename,"./someoutput/cut-ex-angle-rpp_%d",ishot+1);
								hipMemcpy(wf_append,ex_angle_rpp_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);

								sprintf(filename,"./someoutput/cut-ex-angle-rps_%d",ishot+1);
								hipMemcpy(wf_append,ex_angle_rps_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);


								sprintf(filename,"./someoutput/cut-ex-angle-rpp1_%d",ishot+1);
								hipMemcpy(wf_append,ex_angle_rpp1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);

								sprintf(filename,"./someoutput/cut-ex-angle-rps1_%d",ishot+1);
								hipMemcpy(wf_append,ex_angle_rps1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								write_file_1d(wf,nx_size*nz,filename);
							}

					if(precon!=0)
					{	
						if(iter==0)
						{
							sprintf(filename,"./check_file/d_illum_%d",ishot+1);
							hipMemcpy(wf_append,d_illum,nxanza*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_append,nxanza,filename);
						}
			
						if(iter>0)
						{
							sprintf(filename,"./check_file/d_illum_%d",ishot+1);
							fread_file_1d(wf_append,nx_append,nz_append,filename);
							hipMemcpy(d_illum,wf_append,nxanza*sizeof(float),hipMemcpyHostToDevice);
						}
			
							cuda_scale_gradient_acqusition_new<<<dimGrid,dimBlock>>>(grad_lame11_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
							cuda_scale_gradient_acqusition_new<<<dimGrid,dimBlock>>>(grad_lame22_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
							cuda_scale_gradient_acqusition_new<<<dimGrid,dimBlock>>>(grad_den1_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);

						/*if(iter==0)
						{
							cuda_sum_new_acqusition_illum<<<dimGrid_new,dimBlock>>>(d_illum_new,d_illum,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,receiver_x_cord[ishot],receiver_interval,receiver_num);
						}*/
					}
						/*if(receiver_offset!=0&&offset_attenuation!=0)
						{
							cauda_zero_and_attenuation_truncation<<<dimGrid,dimBlock>>>(grad_den1_d,nx_size,nz,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
							cauda_zero_and_attenuation_truncation<<<dimGrid,dimBlock>>>(grad_lame11_d,nx_size,nz,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
							cauda_zero_and_attenuation_truncation<<<dimGrid,dimBlock>>>(grad_lame22_d,nx_size,nz,offset_left[ishot],offset_right[ishot],receiver_offset,offset_attenuation);
						}*/
////////////////////////////new acqusition//////////////////////gradient
						cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_den1_d,grad_den1_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_lame11_d,grad_lame11_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_sum_new_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_lame22_d,grad_lame22_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
//////////////////////gradient////////////////////////////new acqusition
						////////////////////The output is used to check artifacts in surface 
						ishot++;
			}

						if(inversion_den==0)
						{
							hipMemset(all_grad_den1_d,0,nxnz*sizeof(float));
						}

						if(iter==0)/////////////2017年03月12日 星期日 11时04分38秒    elastic RTM for PP and PS reflection
						{
								///////////////////////////////////////////////////cuda_attenuation_after_lap_new2lace
//////////////////////////////////////////////////excitation imaging condition	
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_old_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_vresultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_vresultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);

								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
////////////////////////////////////////////////////////////////////////////////xx or zz
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultxx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultzz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						
///////////////////////////////////////////////////vresult
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_result_tp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultsp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultss_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultppx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpsx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultspx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultssx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultppz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpsz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultspz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultssz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
///////////////////////////////////////////////////result
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultss_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
///////////////////////////////////////////////////correction
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps2_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp2_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);

								sprintf(filename1,"./result/RTM/initial-com-ex-vresultpp");
								hipMemcpy(wf_nxnz,all_com_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-com-ex-vresultps");
								hipMemcpy(wf_nxnz,all_com_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-ex-vresultpp");
								hipMemcpy(wf_nxnz,all_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-ex-vresultps");
								hipMemcpy(wf_nxnz,all_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-ex-result-tp");
								hipMemcpy(wf_nxnz,all_ex_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-ex-result-tp-old");
								hipMemcpy(wf_nxnz,all_ex_result_tp_old_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultxx");
								hipMemcpy(wf_nxnz,all_resultxx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultzz");
								hipMemcpy(wf_nxnz,all_resultzz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-result-tp");
								hipMemcpy(wf_nxnz,all_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);								

								sprintf(filename1,"./result/RTM/initial-resultpp");
								hipMemcpy(wf_nxnz,all_resultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultps",iter+1);
								hipMemcpy(wf_nxnz,all_resultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
						
								sprintf(filename1,"./result/RTM/initial-resultps1",iter+1);
								hipMemcpy(wf_nxnz,all_resultps1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultps2",iter+1);
								hipMemcpy(wf_nxnz,all_resultps2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultsp1",iter+1);
								hipMemcpy(wf_nxnz,all_resultsp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultsp2",iter+1);
								hipMemcpy(wf_nxnz,all_resultsp2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-resultss",iter+1);
								hipMemcpy(wf_nxnz,all_resultss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultpp");
								hipMemcpy(wf_nxnz,all_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultps");
								hipMemcpy(wf_nxnz,all_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultsp");
								hipMemcpy(wf_nxnz,all_vresultsp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultss");
								hipMemcpy(wf_nxnz,all_vresultss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultppx");
								hipMemcpy(wf_nxnz,all_vresultppx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultpsx");
								hipMemcpy(wf_nxnz,all_vresultpsx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultppz");
								hipMemcpy(wf_nxnz,all_vresultppz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/initial-vresultpsz");
								hipMemcpy(wf_nxnz,all_vresultpsz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

							//if(laplace==1)////RTM for laplace  is different LSRTM
							{
//////////////////////////////////////////////////////////excitation imaging condition
//////////////////////////////////////////////////////////ex_tp
								hipMemcpy(wf_nxnz_d,all_ex_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_ex_result_tp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////ex_tp_old
								hipMemcpy(wf_nxnz_d,all_ex_result_tp_old_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_old_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_ex_result_tp_old_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ex_pp
								hipMemcpy(wf_nxnz_d,all_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_ex_vresultpp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_ex_vresultpp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ex_ps						
								hipMemcpy(wf_nxnz_d,all_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_ex_vresultps_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_ex_vresultps_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ex_pp
								hipMemcpy(wf_nxnz_d,all_com_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultpp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_com_ex_vresultpp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ex_ps						
								hipMemcpy(wf_nxnz_d,all_com_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_com_ex_vresultps_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_com_ex_vresultps_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////xx
								hipMemcpy(wf_nxnz_d,all_resultxx_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultxx_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultxx_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////zz
								hipMemcpy(wf_nxnz_d,all_resultzz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultzz_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultzz_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////tp
								hipMemcpy(wf_nxnz_d,all_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_result_tp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_result_tp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////pp
								hipMemcpy(wf_nxnz_d,all_resultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultpp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultpp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ps						
								hipMemcpy(wf_nxnz_d,all_resultps_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultps_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultps_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////sp						
								hipMemcpy(wf_nxnz_d,all_resultsp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultsp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultsp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ss						
								hipMemcpy(wf_nxnz_d,all_resultss_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultss_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultss_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////ps1								
								hipMemcpy(wf_nxnz_d,all_resultps1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultps1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultps1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////ps2
								hipMemcpy(wf_nxnz_d,all_resultps2_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultps2_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultps2_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////sp1								
								hipMemcpy(wf_nxnz_d,all_resultsp1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultsp1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultsp1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////sp2								
								hipMemcpy(wf_nxnz_d,all_resultsp2_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_resultsp2_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_resultsp2_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////vpp
								hipMemcpy(wf_nxnz_d,all_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultpp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultpp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vps						
								hipMemcpy(wf_nxnz_d,all_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultps_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultps_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vsp								
								hipMemcpy(wf_nxnz_d,all_vresultsp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultsp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultsp_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vss
								hipMemcpy(wf_nxnz_d,all_vresultss_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultss_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultss_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////vppx
								hipMemcpy(wf_nxnz_d,all_vresultppx_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultppx_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultppx_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vpsx						
								hipMemcpy(wf_nxnz_d,all_vresultpsx_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultpsx_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultpsx_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vspx								
								hipMemcpy(wf_nxnz_d,all_vresultspx_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultspx_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultspx_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vssx
								hipMemcpy(wf_nxnz_d,all_vresultssx_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultssx_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultssx_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);

//////////////////////////////////////////////////////////vppz
								hipMemcpy(wf_nxnz_d,all_vresultppz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultppz_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultppz_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vpsz						
								hipMemcpy(wf_nxnz_d,all_vresultpsz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultpsz_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultpsz_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vspz								
								hipMemcpy(wf_nxnz_d,all_vresultspz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultspz_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultspz_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////vssz
								hipMemcpy(wf_nxnz_d,all_vresultssz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_vresultssz_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_vresultssz_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							}

///////////////////////////////////////////////////cuda_attenuation_after_lap_new2lace
//////////////////////////////////////////////////excitation imaging condition	
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_result_tp_old_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_vresultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_ex_vresultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultxx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultzz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						
///////////////////////////////////////////////////vresult
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_result_tp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultsp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultss_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultppx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpsx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultspx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultssx_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultppz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultpsz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultspz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_vresultssz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
///////////////////////////////////////////////////result
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultpp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultss_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
///////////////////////////////////////////////////correction
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultps2_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_resultsp2_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);					

/////////////////////excitation amplitude imaging condition
/////////////////////////////////////////ex_tp
								sprintf(filename1,"./result/RTM/ex-result-tp-lap");
								hipMemcpy(wf_nxnz,all_ex_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
							
								sprintf(filename1,"./result/RTM/ex-result-tp-old-lap");
								hipMemcpy(wf_nxnz,all_ex_result_tp_old_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/ex-vresultpp-lap");
								hipMemcpy(wf_nxnz,all_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/ex-vresultps-lap");
								hipMemcpy(wf_nxnz,all_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/com-ex-vresultpp-lap");
								hipMemcpy(wf_nxnz,all_com_ex_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/com-ex-vresultps-lap");
								hipMemcpy(wf_nxnz,all_com_ex_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
///////////////////////////////////////////xx
								sprintf(filename1,"./result/RTM/resultxx-lap");
								hipMemcpy(wf_nxnz,all_resultxx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
///////////////////////////////////////////zz
								sprintf(filename1,"./result/RTM/resultzz-lap");
								hipMemcpy(wf_nxnz,all_resultzz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
/////////////////////////////////////////tp					
								sprintf(filename1,"./result/RTM/result-tp-lap");
								hipMemcpy(wf_nxnz,all_result_tp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);										
////////////////tradtional method
								sprintf(filename1,"./result/RTM/resultpp-lap");
								hipMemcpy(wf_nxnz,all_resultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultps-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultsp-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultsp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultss-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultps1-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultps1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultps2-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultps2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultsp1-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultsp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/resultsp2-lap",iter+1);
								hipMemcpy(wf_nxnz,all_resultsp2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
////////////////inner method
								sprintf(filename1,"./result/RTM/vresultpp-lap");
								hipMemcpy(wf_nxnz,all_vresultpp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultps-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultsp-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultsp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultss-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
////////////////correlation method
								sprintf(filename1,"./result/RTM/vresultppx-lap");
								hipMemcpy(wf_nxnz,all_vresultppx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultpsx-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultpsx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultspx-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultspx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultssx-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultssx_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
////////////////correlation method
								sprintf(filename1,"./result/RTM/vresultppz-lap");
								hipMemcpy(wf_nxnz,all_vresultppz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultpsz-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultpsz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultspz-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultspz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/RTM/vresultssz-lap",iter+1);
								hipMemcpy(wf_nxnz,all_vresultssz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);		
						}

///////////////////////////all  mode:PP PS SP and SS
						if(decomposition!=0)
						{
///////////////////////////one  mode:PP 
								sprintf(filename1,"./result/gradient/grad_lame1_pp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame1_pp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_lame2_pp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame2_pp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_den_pp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_den_pp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
///////////////////////////one  mode:SS 
								sprintf(filename1,"./result/gradient/grad_lame1_ss_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame1_ss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_lame2_ss_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame2_ss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_den_ss_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_den_ss_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
///////////////////////////one  mode:PS
								sprintf(filename1,"./result/gradient/grad_lame1_ps_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame1_ps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_lame2_ps_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame2_ps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_den_ps_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_den_ps_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
//////////////////////////one  mode:SP
								sprintf(filename1,"./result/gradient/grad_lame1_sp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame1_sp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_lame2_sp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame2_sp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_den_sp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_den_sp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
						}

////////////////////////////////////output gradient				
							sprintf(filename1,"./result/gradient/grad_lame1_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_grad_lame11_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/grad_lame2_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_grad_lame22_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/grad_den_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_grad_den1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

/////////////////////////precondition scale
						/* compute the gradient of FWI by scaling, precondition incorporated here: equations 9 and 10 */
						if(inversion_para==0||inversion_para==1)
						{

							if(laplace==1)
							{
								hipMemcpy(wf_nxnz_d,all_grad_lame11_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_lame11_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_lame11_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////////////////////////////////////////						
								hipMemcpy(wf_nxnz_d,all_grad_lame22_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_lame22_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_lame22_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////////////////////////////////////////							
								hipMemcpy(wf_nxnz_d,all_grad_den1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_den1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_den1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							}
///////////////////////////////////////////////////grad_lame
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_lame11_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_lame22_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_den1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								sprintf(filename1,"./result/gradient/grad_lame111_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame11_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_lame222_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_lame22_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_den11_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_den1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);							
						}

////////////////////////////////////////////invert lame gradient to velocity  gradient and density gradient
						if(inversion_para==2)
						{			
							if(decomposition==0)
							{
								//invert_lame_to_velocity_para_new<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_velocity_para<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
							}

							if(decomposition!=0)
							{
								invert_lame_to_velocity_vp<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_lame1_pp_d,all_grad_lame2_pp_d,all_grad_den_pp_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_velocity_vs<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,all_grad_lame1_ps_d,all_grad_lame2_ps_d,all_grad_den_ps_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_velocity_density<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
							}

								sprintf(filename1,"./result/gradient/grad_vp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_vs_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vs1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_density_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_density1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);
						}

						if(inversion_para==3)
						{									
							if(decomposition==0)
							{
								//invert_lame_to_impedance_para_new<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_impedance_para<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
							}

							if(decomposition!=0)
							{
								invert_lame_to_impedance_vp<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_lame1_pp_d,all_grad_lame2_pp_d,all_grad_den_pp_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_impedance_vs<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,all_grad_lame1_ps_d,all_grad_lame2_ps_d,all_grad_den_ps_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);

								invert_lame_to_impedance_density<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
							}

								sprintf(filename1,"./result/gradient/grad_vp_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_vs_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vs1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_density_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_density1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);			
						}			
////////////////////////////////////////////invert lame gradient to velocity gradient and density gradient
	
						if(inversion_para==2||inversion_para==3)
						{
							//////////////////////////////////////////////vsp_precondition
							if(vsp_precon==1)
							{
								cuda_scale_gradient_new<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,r_d_illum,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,vsp_precon);
								cuda_scale_gradient_new<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,r_d_illum,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,vsp_precon);
								cuda_scale_gradient_new<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,r_d_illum,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,vsp_precon);			
							}
	
							if(laplace==1)
							{
								hipMemcpy(wf_nxnz_d,all_grad_vp1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_vp1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////////////////////////////////////////							
								hipMemcpy(wf_nxnz_d,all_grad_vs1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_vs1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
//////////////////////////////////////////////////////////////////////////////////////////////							
								hipMemcpy(wf_nxnz_d,all_grad_density1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
								cuda_laplace<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
								cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
								hipMemcpy(all_grad_density1_d,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							}
///////////////////////////////////////////////////grad_vp vs density
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
								cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);

								sprintf(filename1,"./result/gradient/grad_vp2_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_vs2_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vs1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_density2_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_density1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								/*smooth_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,nx,nz,offset_left_d,offset_right_d,source_x_cord_d,shot_num);
								smooth_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,nx,nz,offset_left_d,offset_right_d,source_x_cord_d,shot_num);
								smooth_acqusition<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,nx,nz,offset_left_d,offset_right_d,source_x_cord_d,shot_num);
								sprintf(filename1,"./result/gradient/grad_vp3_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vp1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_vs3_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_vs1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);

								sprintf(filename1,"./result/gradient/grad_density3_iter_%d",iter+1);
								hipMemcpy(wf_nxnz,all_grad_density1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
								write_file_1d(wf_nxnz,nxnz,filename1);*/								
						}


////////////////////YPL's conjugate method
						if(inversion_para==0||inversion_para==1)//////////iter_start!=0:restart
						{
							////new conjugate method
							calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_lame11_d,nx,nz,0);

							calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_lame22_d,nx,nz,1);

							if(inversion_den!=0)
							{
								calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_den1_d,nx,nz,2);
							}

							if(iter>0&&iter_start!=iter)//////////iter_start!=0:restart
							{								
								cuda_cal_beta_new<<<1, Block_Size>>>(beta_d,all_hydrid_grad1_d,all_hydrid_grad2_d,all_hydrid_conj_d,3*nxnz,0);
							}

							cuda_cal_conjgrad_new<<<dimGrid_3nx_nz,dimBlock>>>(all_hydrid_grad2_d,all_hydrid_conj_d,beta_d,3*nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_lame11_d,all_conj_lame1_d,beta_d,nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_lame22_d,all_conj_lame2_d,beta_d,nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_den1_d,all_conj_den_d,beta_d,nx,nz,0);

							sprintf(filename1,"./result/gradient/conj_lame1_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_lame1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/conj_lame2_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_lame2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/conj_den_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_den_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);
						}
					
						if(inversion_para==2||inversion_para==3)//////////iter_start!=0:restart 
						{	
							////new conjugate method
							calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_vp1_d,nx,nz,0);

							calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_vs1_d,nx,nz,1);

							if(inversion_den!=0)
							{
								calcualte_hydrid_grad<<<dimGrid_new,dimBlock>>>(all_hydrid_grad2_d,all_grad_density1_d,nx,nz,2);
							}
	
							if (iter>0&&iter_start!=iter)  
							{
								cuda_cal_beta_new<<<1, Block_Size>>>(beta_d,all_hydrid_grad1_d,all_hydrid_grad2_d,all_hydrid_conj_d,3*nxnz,0);
							}
						
							cuda_cal_conjgrad_new<<<dimGrid_3nx_nz,dimBlock>>>(all_hydrid_grad2_d,all_hydrid_conj_d,beta_d,3*nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_conj_vp_d,beta_d,nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_vs1_d,all_conj_vs_d,beta_d,nx,nz,0);
							cuda_cal_conjgrad_new<<<dimGrid_new,dimBlock>>>(all_grad_density1_d,all_conj_density_d,beta_d,nx,nz,0);

							sprintf(filename1,"./result/gradient/conj_vp_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_vp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/conj_vs_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_vs_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);

							sprintf(filename1,"./result/gradient/conj_density_iter_%d",iter+1);
							hipMemcpy(wf_nxnz,all_conj_density_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							write_file_1d(wf_nxnz,nxnz,filename1);	
						}
					
////////////////////////////////////////////2017年09月11日 星期一 12时05分48秒   it is important for set zero
						if(inversion_den==0)
						{
							hipMemset(all_conj_den_d,0,nxnz*sizeof(float));
							hipMemset(all_conj_density_d,0,nxnz*sizeof(float));
							warn("density parameter is no inversion\n");
						}
////////////////////YPL's conjugate method

//直接拿扰动结果正演得到地震记录，对于正演为线性（如：Born线性正演），可以这样计算！！！，但是非线性必须给定扰动（如正常的正演模拟）（给定微小扰动，计算lame1的最优步长：），意义是一样！！！/////////this process refer to Claerbout	YPL in 2015
				if(laplace_compensate!=0)/////////////////////////real wavelet
				{
					hipMemcpy(wavelet_d,wavelet,wavelet_length*sizeof(float),hipMemcpyHostToDevice);
				}
				ishot=0;
				hipMemset(d_alpha1, 0, lt_rec*sizeof(float));
				hipMemset(d_alpha2, 0, lt_rec*sizeof(float));
				hipMemset(correlation_parameter_d,0,10*sizeof(float));///////important
				while(ishot<shot_num)
				{
					if(cut_direct_wave==1)
					{
/////////////////////////////////////one_born_modeling
/////////////////////////////////////get smooth vp;				
						cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

						/*hipMemcpy(wf,wf_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./someoutput/cut1-vp-%d.bin",ishot+1);
						write_file_1d(wf,nx_size_nz,filename);

						hipMemcpy(wf_append,s_velocity_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./someoutput/vp1-%d.bin",ishot+1);
						write_file_1d(wf_append,nxanza,filename);*/

/////////////////////////////////////get smooth vs;				
						cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth density;				
						cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_density_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(s_density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qp;				
						cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_qp_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qs;				
						cuda_get_partly_mode_boundary_z1_z2<<<dimGrid_new,dimBlock>>>(s_qs_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

						/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
						{								
							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);						
						}*/

/////////////////////////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
						cuda_cal_viscoelastic<<< dimGrid,dimBlock>>>(modul_p_d,modul_s_d,s_qp_d,s_qs_d,tao_d,strain_p_d,strain_s_d,freq,s_velocity_d,s_velocity1_d,s_density_d,nx_append,nz_append);
						
						if(inversion_para==0||inversion_para==1)
						{
/////////////////////////////////////get perturbed lame1;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_lame1_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_lame1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_lame2_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_lame2_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_den_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_den_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
							{								
								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_lame1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_lame2_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_den_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
							}*/
						}

						if(inversion_para==2||inversion_para==3)
						{
/////////////////////////////////////get perturbed lame1;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_vp_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_vp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
							/*hipMemcpy(wf,wf_d,nx_size_nz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/cut-perturb-vp-%d.bin",ishot+1);
							write_file_1d(wf,nx_size_nz,filename);

							hipMemcpy(wf_append,tmp_perturb_vp_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/perturb-vp-%d.bin",ishot+1);
							write_file_1d(wf_append,nxanza,filename);*/
/////////////////////////////////////get perturbed lame2;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_vs_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_vs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
							cuda_get_partly_mode_z1_z2<<<dimGrid_new,dimBlock>>>(all_conj_density_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,precon_z1,precon_z2);
							cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

							/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
							{								
								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_vp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_vs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

								cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
							}*/
						}

						memset((void *)(wf_append),0,nxanza*sizeof(float));

						hipMemset(vx1_d,0,nxanza*sizeof(float));
						hipMemset(vz1_d,0,nxanza*sizeof(float));
						hipMemset(txx1_d,0,nxanza*sizeof(float));
						hipMemset(tzz1_d,0,nxanza*sizeof(float));
						hipMemset(txz1_d,0,nxanza*sizeof(float));

						hipMemset(vx2_d,0,nxanza*sizeof(float));
						hipMemset(vz2_d,0,nxanza*sizeof(float));
						hipMemset(txx2_d,0,nxanza*sizeof(float));
						hipMemset(tzz2_d,0,nxanza*sizeof(float));
						hipMemset(txz2_d,0,nxanza*sizeof(float));
		
						hipMemset(tp2_d,0,nxanza*sizeof(float));
						hipMemset(tp1_d,0,nxanza*sizeof(float));
						hipMemset(vxp2_d,0,nxanza*sizeof(float));
						hipMemset(vxp1_d,0,nxanza*sizeof(float));
						hipMemset(vzp2_d,0,nxanza*sizeof(float));
						hipMemset(vzp1_d,0,nxanza*sizeof(float));
						hipMemset(vxs2_d,0,nxanza*sizeof(float));
						hipMemset(vxs1_d,0,nxanza*sizeof(float));
						hipMemset(vzs2_d,0,nxanza*sizeof(float));
						hipMemset(vzs1_d,0,nxanza*sizeof(float));

						hipMemset(mem_p1_d,0,nxanza*sizeof(float));
						hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
						hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
						hipMemset(mem_xz1_d,0,nxanza*sizeof(float));
						hipMemset(mem_p2_d,0,nxanza*sizeof(float));
						hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
						hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
						hipMemset(mem_xz2_d,0,nxanza*sizeof(float));

						hipMemset(rvx1_d,0,nxanza*sizeof(float));
						hipMemset(rvz1_d,0,nxanza*sizeof(float));
						hipMemset(rtxx1_d,0,nxanza*sizeof(float));
						hipMemset(rtzz1_d,0,nxanza*sizeof(float));
						hipMemset(rtxz1_d,0,nxanza*sizeof(float));

						hipMemset(rvx2_d,0,nxanza*sizeof(float));
						hipMemset(rvz2_d,0,nxanza*sizeof(float));
						hipMemset(rtxx2_d,0,nxanza*sizeof(float));
						hipMemset(rtzz2_d,0,nxanza*sizeof(float));
						hipMemset(rtxz2_d,0,nxanza*sizeof(float));

						hipMemset(rtp2_d,0,nxanza*sizeof(float));
						hipMemset(rtp1_d,0,nxanza*sizeof(float));
						hipMemset(rvxp2_d,0,nxanza*sizeof(float));
						hipMemset(rvxp1_d,0,nxanza*sizeof(float));
						hipMemset(rvzp2_d,0,nxanza*sizeof(float));
						hipMemset(rvzp1_d,0,nxanza*sizeof(float));
						hipMemset(rvxs2_d,0,nxanza*sizeof(float));
						hipMemset(rvxs1_d,0,nxanza*sizeof(float));
						hipMemset(rvzs2_d,0,nxanza*sizeof(float));
						hipMemset(rvzs1_d,0,nxanza*sizeof(float));

						hipMemset(rmem_p1_d,0,nxanza*sizeof(float));
						hipMemset(rmem_xx1_d,0,nxanza*sizeof(float));
						hipMemset(rmem_zz1_d,0,nxanza*sizeof(float));
						hipMemset(rmem_xz1_d,0,nxanza*sizeof(float));
						hipMemset(rmem_p2_d,0,nxanza*sizeof(float));
						hipMemset(rmem_xx2_d,0,nxanza*sizeof(float));
						hipMemset(rmem_zz2_d,0,nxanza*sizeof(float));
						hipMemset(rmem_xz2_d,0,nxanza*sizeof(float));

						hipMemset(vx_x_d,0,nxanza*sizeof(float));
						hipMemset(vx_z_d,0,nxanza*sizeof(float));
						hipMemset(vz_x_d,0,nxanza*sizeof(float));
						hipMemset(vz_z_d,0,nxanza*sizeof(float));

						hipMemset(vx_t_d,0,nxanza*sizeof(float));
						hipMemset(vz_t_d,0,nxanza*sizeof(float));

				for(int it=0;it<lt+wavelet_half;it++)
						{
							//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 2,it=%d",ishot+1,(it+1)-wavelet_half);
							if(it<wavelet_length)
							{
								//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],shot_depth,it,boundary_up,boundary_left,nz_append);
								//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);///for vsp 2017年03月14日 星期二 08时55分59秒 
								add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);///for vsp 2017年03月14日 星期二 08时55分59秒 
							}	
								fwd_vx_new<<<dimGrid,dimBlock>>>(vx_t_d,vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

								fwd_vz_new<<<dimGrid,dimBlock>>>(vz_t_d,vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);	

								if(migration_type==0)	fwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

								else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,packaging_d);

								if(0==(it)%100&&join_wavefield==1&&iter==0)
								{
									hipMemcpy(wf_append,vx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/4/vx-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);
											
									hipMemcpy(wf_append,vz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/4/vz-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);

									hipMemcpy(wf_append,vz_z_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/4/vz-z-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);

									hipMemcpy(wf_append,vz_x_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/4/vz-x-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);

									hipMemcpy(wf_append,vz_t_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/4/vz-t-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);
								}

							rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
							rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
							rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
							rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
							rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

							rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
							rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
							rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
							rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
							rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

							rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
							rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
							rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
							rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;
	///////////////////////demigration to calculate cal_shots!!!!!!!!!!
							if(migration_type==0)
							{	
								cuda_cal_dem_parameter_elastic_media<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,inversion_para);

								demig_fwd_txxzzxz_mul<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

								demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

								demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);			
							}

							if(migration_type==1)
							{
								cuda_cal_dem_parameter_viscoelastic_media_new<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p_all_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d,dt,inversion_para);

								demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(rtp2_d,rtp1_d,rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,dem_p_all_d);

								demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

								demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);
							}

								if(0==(it)%100&&join_wavefield==1&&iter==0)
								{
									hipMemcpy(wf_append,rvx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/5/vx-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);
											
									hipMemcpy(wf_append,rvz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/5/vz-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);

									hipMemcpy(wf_append,dem_p3_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
									sprintf(filename,"./wavefield1/5/dem-p3-%d-shot_%d",ishot+1,it);
									write_file_1d(wf_append,nxanza,filename);
									//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
									//write_file_1d(wf,nx_size_nz,filename);
								}

							if(it>=wavelet_half&&it<(lt+wavelet_half))
							{
									//write_shot<<<receiver_num,1>>>(rvx2_d,rvz2_d,cal_shot_x_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dt,source_x_cord[ishot],s_velocity_d,wavelet_half);
								if(receiver_offset==0)
								{
									write_shot_x_z<<<receiver_num,1>>>(rvx2_d,cal_shot_x1_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
									write_shot_x_z<<<receiver_num,1>>>(rvz2_d,cal_shot_z1_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒
								}
								else
								{
									write_shot_x_z_acqusition<<<receiver_num,1>>>(rvx2_d,cal_shot_x1_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
									write_shot_x_z_acqusition<<<receiver_num,1>>>(rvz2_d,cal_shot_z1_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒
								}
							}

							rep=rvx1_d;rvx1_d=rvx2_d;rvx2_d=rep;
							rep=rvz1_d;rvz1_d=rvz2_d;rvz2_d=rep;
							rep=rtxx1_d;rtxx1_d=rtxx2_d;rtxx2_d=rep;
							rep=rtzz1_d;rtzz1_d=rtzz2_d;rtzz2_d=rep;
							rep=rtxz1_d;rtxz1_d=rtxz2_d;rtxz2_d=rep;

							rep=rtp1_d;rtp1_d=rtp2_d;rtp2_d=rep;
							rep=rvxp1_d;rvxp1_d=rvxp2_d;rvxp2_d=rep;
							rep=rvzp1_d;rvzp1_d=rvzp2_d;rvzp2_d=rep;
							rep=rvxs1_d;rvxs1_d=rvxs2_d;rvxs2_d=rep;
							rep=rvzs1_d;rvzs1_d=rvzs2_d;rvzs2_d=rep;/////fast...........................................

							rep=rmem_p1_d;rmem_p1_d=rmem_p2_d;rmem_p2_d=rep;
							rep=rmem_xx1_d;rmem_xx1_d=rmem_xx2_d;rmem_xx2_d=rep;
							rep=rmem_zz1_d;rmem_zz1_d=rmem_zz2_d;rmem_zz2_d=rep;
							rep=rmem_xz1_d;rmem_xz1_d=rmem_xz2_d;rmem_xz2_d=rep;
						}

							if(ishot%20==0)
							{		
								/////////output cal shots
								hipMemcpy(shotgather,cal_shot_x1_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./someoutput/bin/direct1_cal_shot_x_%d_iter_%d",ishot+1,iter+1);
								write_file_1d(shotgather,lt_rec,filename);
								hipMemcpy(shotgather,cal_shot_z1_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./someoutput/bin/direct1_cal_shot_z_%d_iter_%d",ishot+1,iter+1);
								write_file_1d(shotgather,lt_rec,filename);
								/////////output cal shots
							}
					}

/////////////////////////////////////another_born_modeling
/////////////////////////////////////get smooth vp;				
					cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_velocity_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
					cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth vs;				
					cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_velocity1_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
					cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth density;				
					cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_density_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
					cuda_cal_expand<<<dimGrid,dimBlock>>>(s_density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qp;				
					cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_qp_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
					cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

/////////////////////////////////////get smooth qs;				
					cuda_get_partly_mode_boundary<<<dimGrid_new,dimBlock>>>(s_qs_all_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num,nx_append_new,nz_append,boundary_left,boundary_up);
					cuda_cal_expand<<<dimGrid,dimBlock>>>(s_qs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

					/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
					{								
						cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

						cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_velocity1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

						cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

						cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

						cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(s_qs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);						
					}*/

/////////////////////////////////QQQQQQQQQQQQQQQ2017年07月27日 星期四 10时11分40秒 
					cuda_cal_viscoelastic<<< dimGrid,dimBlock>>>(modul_p_d,modul_s_d,s_qp_d,s_qs_d,tao_d,strain_p_d,strain_s_d,freq,s_velocity_d,s_velocity1_d,s_density_d,nx_append,nz_append);
					
					if(inversion_para==0||inversion_para==1)
					{
/////////////////////////////////////get perturbed lame1;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_lame1_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_lame1_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_lame2_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_lame2_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_den_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_den_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

						/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
						{								
							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_lame1_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_lame2_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_den_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
						}*/
					}

					if(inversion_para==2||inversion_para==3)
					{
/////////////////////////////////////get perturbed lame1;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_vp_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_vp_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_vs_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_vs_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
/////////////////////////////////////get perturbed lame2;
						cuda_get_partly_mode<<<dimGrid_new,dimBlock>>>(all_conj_density_d,wf_d,nx,nz,receiver_x_cord[ishot],receiver_interval,receiver_num);
						cuda_cal_expand<<<dimGrid,dimBlock>>>(tmp_perturb_density_d,wf_d,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);

						/*if((receiver_offset!=0)||(offset_left[ishot]>receiver_offset)||(offset_right[ishot]>receiver_offset))
						{								
							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_vp_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_vs_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);

							cuda_expand_acqusition_left_and_right<<<dimGrid_new,dimBlock>>>(tmp_perturb_density_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,nx_size,nx_append,nz_append,boundary_left,boundary_up);
						}*/
					}

					if(correlation_misfit==0)
					{
						////read real residual
						sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

						sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
						////read real residual
					}
					
					else
					{	
						///////////////////it is noted that  the first iteration is conventional LSRTM
						if(iter==0)
						{
							////read real residual
							sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(res_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(res_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
							////read real residual
						}
						
						if(iter>0)
						{
							//////fread obs shot	
							sprintf(filename,"./someoutput/bin/obs_shot_x_%d",ishot+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(obs_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/obs_shot_z_%d",ishot+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(obs_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
							//////fread obs shot
								
							////////for sn!=0 data
							if(receiver_offset!=0)
							{
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(obs_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
							}

							//////fread iter+1 tmp_shot			
							sprintf(filename,"./someoutput/bin/tmp_shot_x_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(tmp_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/tmp_shot_z_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(tmp_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
						}				
					}

					if(vsp_2!=0)
					{
						////read real residual
						sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d_2",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_x_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

						sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d_2",ishot+1,iter+1);
						fread_file_1d(shotgather,receiver_num,lt,filename);
						hipMemcpy(res_shot_z_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
						////read real residual
					}

					memset((void *)(wf_append),0,nxanza*sizeof(float));

					hipMemset(vx1_d,0,nxanza*sizeof(float));
					hipMemset(vz1_d,0,nxanza*sizeof(float));
					hipMemset(txx1_d,0,nxanza*sizeof(float));
					hipMemset(tzz1_d,0,nxanza*sizeof(float));
					hipMemset(txz1_d,0,nxanza*sizeof(float));

					hipMemset(vx2_d,0,nxanza*sizeof(float));
					hipMemset(vz2_d,0,nxanza*sizeof(float));
					hipMemset(txx2_d,0,nxanza*sizeof(float));
					hipMemset(tzz2_d,0,nxanza*sizeof(float));
					hipMemset(txz2_d,0,nxanza*sizeof(float));
	
					hipMemset(tp2_d,0,nxanza*sizeof(float));
					hipMemset(tp1_d,0,nxanza*sizeof(float));
					hipMemset(vxp2_d,0,nxanza*sizeof(float));
					hipMemset(vxp1_d,0,nxanza*sizeof(float));
					hipMemset(vzp2_d,0,nxanza*sizeof(float));
					hipMemset(vzp1_d,0,nxanza*sizeof(float));
					hipMemset(vxs2_d,0,nxanza*sizeof(float));
					hipMemset(vxs1_d,0,nxanza*sizeof(float));
					hipMemset(vzs2_d,0,nxanza*sizeof(float));
					hipMemset(vzs1_d,0,nxanza*sizeof(float));

					hipMemset(mem_p1_d,0,nxanza*sizeof(float));
					hipMemset(mem_xx1_d,0,nxanza*sizeof(float));
					hipMemset(mem_zz1_d,0,nxanza*sizeof(float));
					hipMemset(mem_xz1_d,0,nxanza*sizeof(float));
					hipMemset(mem_p2_d,0,nxanza*sizeof(float));
					hipMemset(mem_xx2_d,0,nxanza*sizeof(float));
					hipMemset(mem_zz2_d,0,nxanza*sizeof(float));
					hipMemset(mem_xz2_d,0,nxanza*sizeof(float));

					hipMemset(rvx1_d,0,nxanza*sizeof(float));
					hipMemset(rvz1_d,0,nxanza*sizeof(float));
					hipMemset(rtxx1_d,0,nxanza*sizeof(float));
					hipMemset(rtzz1_d,0,nxanza*sizeof(float));
					hipMemset(rtxz1_d,0,nxanza*sizeof(float));

					hipMemset(rvx2_d,0,nxanza*sizeof(float));
					hipMemset(rvz2_d,0,nxanza*sizeof(float));
					hipMemset(rtxx2_d,0,nxanza*sizeof(float));
					hipMemset(rtzz2_d,0,nxanza*sizeof(float));
					hipMemset(rtxz2_d,0,nxanza*sizeof(float));

					hipMemset(rtp2_d,0,nxanza*sizeof(float));
					hipMemset(rtp1_d,0,nxanza*sizeof(float));
					hipMemset(rvxp2_d,0,nxanza*sizeof(float));
					hipMemset(rvxp1_d,0,nxanza*sizeof(float));
					hipMemset(rvzp2_d,0,nxanza*sizeof(float));
					hipMemset(rvzp1_d,0,nxanza*sizeof(float));
					hipMemset(rvxs2_d,0,nxanza*sizeof(float));
					hipMemset(rvxs1_d,0,nxanza*sizeof(float));
					hipMemset(rvzs2_d,0,nxanza*sizeof(float));
					hipMemset(rvzs1_d,0,nxanza*sizeof(float));

					hipMemset(rmem_p1_d,0,nxanza*sizeof(float));
					hipMemset(rmem_xx1_d,0,nxanza*sizeof(float));
					hipMemset(rmem_zz1_d,0,nxanza*sizeof(float));
					hipMemset(rmem_xz1_d,0,nxanza*sizeof(float));
					hipMemset(rmem_p2_d,0,nxanza*sizeof(float));
					hipMemset(rmem_xx2_d,0,nxanza*sizeof(float));
					hipMemset(rmem_zz2_d,0,nxanza*sizeof(float));
					hipMemset(rmem_xz2_d,0,nxanza*sizeof(float));

					hipMemset(vx_x_d,0,nxanza*sizeof(float));
					hipMemset(vx_z_d,0,nxanza*sizeof(float));
					hipMemset(vz_x_d,0,nxanza*sizeof(float));
					hipMemset(vz_z_d,0,nxanza*sizeof(float));

					hipMemset(vx_t_d,0,nxanza*sizeof(float));
					hipMemset(vz_t_d,0,nxanza*sizeof(float));

			for(int it=0;it<lt+wavelet_half;it++)
					{
						//if(fmod((it+1.0)-wavelet_half,1000.0)==0) warn("shot=%d,step=forward 2,it=%d",ishot+1,(it+1)-wavelet_half);
						if(it<wavelet_length)
						{
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],shot_depth,it,boundary_up,boundary_left,nz_append);
							//add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);///for vsp 2017年03月14日 星期二 08时55分59秒 
							add_source<<<1,1>>>(txx1_d,tzz1_d,wavelet_d,source_x_cord[ishot]-receiver_x_cord[ishot],source_z_cord[ishot],it,boundary_up,boundary_left,nz_append);///for vsp 2017年03月14日 星期二 08时55分59秒 
						}	
							fwd_vx_new<<<dimGrid,dimBlock>>>(vx_t_d,vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

							fwd_vz_new<<<dimGrid,dimBlock>>>(vz_t_d,vz2_d,vz1_d,tzz1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);	

							if(migration_type==0)	fwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

							else	fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,modul_p_d,modul_s_d,attenuation_d,s_density_d,mem_p2_d,mem_p1_d,mem_xx2_d,mem_xx1_d,mem_zz2_d,mem_zz1_d,mem_xz2_d,mem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,packaging_d);

							if(0==(it)%100&&join_wavefield==1&&iter==0)
							{
								hipMemcpy(wf_append,vx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/4/vx-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);
										
								hipMemcpy(wf_append,vz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/4/vz-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vz_z_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/4/vz-z-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vz_x_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/4/vz-x-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,vz_t_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/4/vz-t-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);
							}

						rep=vx1_d;vx1_d=vx2_d;vx2_d=rep;
						rep=vz1_d;vz1_d=vz2_d;vz2_d=rep;
						rep=txx1_d;txx1_d=txx2_d;txx2_d=rep;
						rep=tzz1_d;tzz1_d=tzz2_d;tzz2_d=rep;
						rep=txz1_d;txz1_d=txz2_d;txz2_d=rep;

						rep=tp1_d;tp1_d=tp2_d;tp2_d=rep;
						rep=vxp1_d;vxp1_d=vxp2_d;vxp2_d=rep;
						rep=vzp1_d;vzp1_d=vzp2_d;vzp2_d=rep;
						rep=vxs1_d;vxs1_d=vxs2_d;vxs2_d=rep;
						rep=vzs1_d;vzs1_d=vzs2_d;vzs2_d=rep;

						rep=mem_p1_d;mem_p1_d=mem_p2_d;mem_p2_d=rep;
						rep=mem_xx1_d;mem_xx1_d=mem_xx2_d;mem_xx2_d=rep;
						rep=mem_zz1_d;mem_zz1_d=mem_zz2_d;mem_zz2_d=rep;
						rep=mem_xz1_d;mem_xz1_d=mem_xz2_d;mem_xz2_d=rep;
///////////////////////demigration to calculate cal_shots!!!!!!!!!!
						if(migration_type==0)
						{
							/*if(inversion_para==0)
							{
							cuda_cal_dem_parameter<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius);
							}
					
							if(inversion_para==1)
							{
							cuda_cal_dem_parameter_lame<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
							}

							if(inversion_para==2)
							{
							cuda_cal_dem_parameter_velocity<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
							}

							if(inversion_para==3)
							{
							cuda_cal_dem_parameter_impedance<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
							}*/

							cuda_cal_dem_parameter_elastic_media<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,inversion_para);

							demig_fwd_txxzzxz_mul<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							/*demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_txxzzxz_mul<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);*/

							/*cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p1_d,rvx1_d,rvx1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p2_d,rvz1_d,rvz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p3_d,rtxx1_d,rtxx1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p4_d,rtzz1_d,rtzz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p5_d,rtxz1_d,rtxz1_d,nx_append,nz_append);

							fwd_txxzzxz<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);

							fwd_vx<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
							fwd_vz<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);*/
						}

						if(migration_type==1)
						{
							/*cuda_cal_dem_parameter_viscoelastic_media<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,dem_p6_d,dem_p7_d,dem_p8_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d,dt,inversion_para);
							//cuda_cal_multiply<<<dimGrid,dimBlock>>>(tmp_perturb_den_d,s_density_d,dem_p1_d,nx_append_radius,nz_append_radius);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p1_d,rvx1_d,rvx1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p2_d,rvz1_d,rvz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p3_d,rtxx1_d,rtxx1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p4_d,rtzz1_d,rtzz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p5_d,rtxz1_d,rtxz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p6_d,rmem_xx1_d,rmem_xx1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p7_d,rmem_zz1_d,rmem_zz1_d,nx_append,nz_append);

							cal_sum_a_b_to_c<<<dimGrid,dimBlock>>>(dem_p8_d,rmem_xz1_d,rmem_xz1_d,nx_append,nz_append);
							fwd_vx<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);
							fwd_vz<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d);
							fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(rtp2_d,rtp1_d,rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx2_d,rvz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d);*/

							cuda_cal_dem_parameter_viscoelastic_media_new<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p_all_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,tao_d,strain_p_d,strain_s_d,dt,inversion_para);

							demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(rtp2_d,rtp1_d,rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,dem_p_all_d);

							demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							/*demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz1_d,rtxz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);

							demig_fwd_txxzzxzpp_viscoelastic_and_memory_3parameterization<<<dimGrid,dimBlock>>>(rtp2_d,rtp1_d,rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx2_d,rvz2_d,modul_p_d,modul_s_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,rmem_p2_d,rmem_p1_d,rmem_xx2_d,rmem_xx1_d,rmem_zz2_d,rmem_zz1_d,rmem_xz2_d,rmem_xz1_d,s_velocity_d,s_velocity1_d,tao_d,strain_p_d,strain_s_d,dem_p_all_d);*/
						}

							if(0==(it)%100&&join_wavefield==1&&iter==0)
							{
								hipMemcpy(wf_append,rvx2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/5/vx-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);
										
								hipMemcpy(wf_append,rvz2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/5/vz-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,dem_p1_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/5/dem-p1-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,dem_p2_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/5/dem-p2-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);

								hipMemcpy(wf_append,dem_p3_d,nxanza*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./wavefield1/5/dem-p3-%d-shot_%d",ishot+1,it);
								write_file_1d(wf_append,nxanza,filename);
								//exchange(wf_append,wf,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up);
								//write_file_1d(wf,nx_size_nz,filename);
							}

						if(it>=wavelet_half&&it<(lt+wavelet_half))
						{
								//write_shot<<<receiver_num,1>>>(rvx2_d,rvz2_d,cal_shot_x_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dt,source_x_cord[ishot],s_velocity_d,wavelet_half);
							if(receiver_offset==0)
							{
								write_shot_x_z<<<receiver_num,1>>>(rvx2_d,cal_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								write_shot_x_z<<<receiver_num,1>>>(rvz2_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒
							}
							else
							{
								write_shot_x_z_acqusition<<<receiver_num,1>>>(rvx2_d,cal_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
								write_shot_x_z_acqusition<<<receiver_num,1>>>(rvz2_d,cal_shot_z_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒
							}
						}

						rep=rvx1_d;rvx1_d=rvx2_d;rvx2_d=rep;
						rep=rvz1_d;rvz1_d=rvz2_d;rvz2_d=rep;
						rep=rtxx1_d;rtxx1_d=rtxx2_d;rtxx2_d=rep;
						rep=rtzz1_d;rtzz1_d=rtzz2_d;rtzz2_d=rep;
						rep=rtxz1_d;rtxz1_d=rtxz2_d;rtxz2_d=rep;

						rep=rtp1_d;rtp1_d=rtp2_d;rtp2_d=rep;
						rep=rvxp1_d;rvxp1_d=rvxp2_d;rvxp2_d=rep;
						rep=rvzp1_d;rvzp1_d=rvzp2_d;rvzp2_d=rep;
						rep=rvxs1_d;rvxs1_d=rvxs2_d;rvxs2_d=rep;
						rep=rvzs1_d;rvzs1_d=rvzs2_d;rvzs2_d=rep;/////fast...........................................

						rep=rmem_p1_d;rmem_p1_d=rmem_p2_d;rmem_p2_d=rep;
						rep=rmem_xx1_d;rmem_xx1_d=rmem_xx2_d;rmem_xx2_d=rep;
						rep=rmem_zz1_d;rmem_zz1_d=rmem_zz2_d;rmem_zz2_d=rep;
						rep=rmem_xz1_d;rmem_xz1_d=rmem_xz2_d;rmem_xz2_d=rep;
					}

						if(ishot%20==0)
						{
							hipMemcpy(shotgather,cal_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct_cal_shot_x_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,cal_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/direct_cal_shot_z_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
						}

						if(cut_direct_wave==0)
						{
							if(receiver_offset!=0)
							{
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
							}

							/////////output cal shots
							hipMemcpy(shotgather,cal_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_x_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,cal_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_z_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							/////////output cal shots
						}

						if(cut_direct_wave==1)
						{
							cal_sub_a_b_to_c<<<dimGrid_lt,dimBlock>>>(cal_shot_x_d,cal_shot_x1_d,cal_shot_x_d,receiver_num,lt);

							cal_sub_a_b_to_c<<<dimGrid_lt,dimBlock>>>(cal_shot_z_d,cal_shot_z1_d,cal_shot_z_d,receiver_num,lt);

							if(receiver_offset!=0)
							{
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
							}

							/////////output cal shots
							hipMemcpy(shotgather,cal_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_x_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,cal_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_z_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							/////////output cal shots
						}
		
						if(cut_direct_wave!=0&&cut_direct_wave!=1)///for vsp 2017年03月14日 星期二 08时55分03秒 
						{
							cut_direct_new1<<<dimGrid,dimBlock>>>(cal_shot_x_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half,cut_direct_wave);
							cut_direct_new1<<<dimGrid,dimBlock>>>(cal_shot_z_d,lt,source_x_cord[ishot],shot_depth,receiver_num,receiver_depth,receiver_x_cord[ishot],receiver_interval,boundary_left,boundary_up,nz_append,dx,dz,dt,velocity_d,wavelet_half,cut_direct_wave);

							if(receiver_offset!=0)
							{
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_x_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
								cauda_zero_acqusition_left_and_right<<<dimGrid_lt,dimBlock>>>(cal_shot_z_d,offset_left[ishot],offset_right[ishot],source_x_cord[ishot],receiver_offset,receiver_num,lt);
							}

							/////////output cal shots
							hipMemcpy(shotgather,cal_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_x_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							hipMemcpy(shotgather,cal_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/cal_shot_z_%d_iter_%d",ishot+1,iter+1);
							write_file_1d(shotgather,lt_rec,filename);
							/////////output cal shots
						}

						if(correlation_misfit==0)//correlation
						{
							cuda_sum_alpha12_new_for_lsrtm<<<dimGrid_lt,dimBlock>>>(d_alpha1,d_alpha2,cal_shot_x_d,obs_shot_x_d,res_shot_x_d,receiver_num,lt);
							cuda_sum_alpha12_new_for_lsrtm<<<dimGrid_lt,dimBlock>>>(d_alpha1,d_alpha2,cal_shot_z_d,obs_shot_z_d,res_shot_z_d,receiver_num,lt);
						}

						else///correlation
						{
							if(iter==0)///////////////////it is noted that  the first iteration is conventional LSRTM
							{
								cuda_sum_alpha12_new_for_lsrtm<<<dimGrid_lt,dimBlock>>>(d_alpha1,d_alpha2,cal_shot_x_d,obs_shot_x_d,res_shot_x_d,receiver_num,lt);
								cuda_sum_alpha12_new_for_lsrtm<<<dimGrid_lt,dimBlock>>>(d_alpha1,d_alpha2,cal_shot_z_d,obs_shot_z_d,res_shot_z_d,receiver_num,lt);
							}				
							
							if(iter>0)
							{
								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,tmp_shot_x_d,lt_rec,&correlation_parameter_d[0]);
								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,tmp_shot_z_d,lt_rec,&correlation_parameter_d[0]);//tmp*tmp

								cuda_dot_sum<<<1,Block_Size>>>(obs_shot_x_d,obs_shot_x_d,lt_rec,&correlation_parameter_d[1]);
								cuda_dot_sum<<<1,Block_Size>>>(obs_shot_z_d,obs_shot_z_d,lt_rec,&correlation_parameter_d[1]);//obs*obs

								cuda_dot_sum<<<1,Block_Size>>>(cal_shot_x_d,cal_shot_x_d,lt_rec,&correlation_parameter_d[2]);
								cuda_dot_sum<<<1,Block_Size>>>(cal_shot_z_d,cal_shot_z_d,lt_rec,&correlation_parameter_d[2]);//cal*cal

								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,obs_shot_x_d,lt_rec,&correlation_parameter_d[3]);
								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,obs_shot_z_d,lt_rec,&correlation_parameter_d[3]);//tmp*obs	

								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_x_d,cal_shot_x_d,lt_rec,&correlation_parameter_d[4]);
								cuda_dot_sum<<<1,Block_Size>>>(tmp_shot_z_d,cal_shot_z_d,lt_rec,&correlation_parameter_d[4]);//tmp*cal

								cuda_dot_sum<<<1,Block_Size>>>(cal_shot_x_d,obs_shot_x_d,lt_rec,&correlation_parameter_d[5]);
								cuda_dot_sum<<<1,Block_Size>>>(cal_shot_z_d,obs_shot_z_d,lt_rec,&correlation_parameter_d[5]);//cal*obs
							}
						}

						ishot++;
				}

						if(correlation_misfit==0)
						{
							cuda_cal_alpha_new_for_lsrtm<<<1, Block_Size>>>(beta_step_d,d_alpha1,d_alpha2,epsil_d,lt_rec,0);
						}

						else
						{
							///////////////////it is noted that  the first iteration is conventional LSRTM
							if(iter==0)
							{
								cuda_cal_alpha_new_for_lsrtm<<<1, Block_Size>>>(beta_step_d,d_alpha1,d_alpha2,epsil_d,lt_rec,0);
							}
							
							if(iter>0)////correlation
							{
								cuda_cal_alpha_new_for_correlation_lsrtm<<<1,1>>>(beta_step_d,correlation_parameter_d,0);
							}
						}
					
						/////for ELSRTM 
//////update res_shots:res_shot_x res_shot_x1
///////////////////////////////* update the res_shots according to previous res_shots, gradient/conjugate gradient  and estimated stepsize */
						ishot=0;
						while(ishot<shot_num)
						{	
							/////////read residuals	
							sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(res_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(res_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);
			
						
							/////////read tmp shot
							sprintf(filename,"./someoutput/bin/tmp_shot_x_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(tmp_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/tmp_shot_z_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(tmp_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);


							/////////read cal shot
							sprintf(filename,"./someoutput/bin/cal_shot_x_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(cal_shot_x_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

							sprintf(filename,"./someoutput/bin/cal_shot_z_%d_iter_%d",ishot+1,iter+1);
							fread_file_1d(shotgather,receiver_num,lt,filename);
							hipMemcpy(cal_shot_z_d,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);



							cuda_update_shots<<<dimGrid_lt,dimBlock>>>(res_shot_x_d,cal_shot_x_d,beta_step_d,receiver_num,lt,0);
							cuda_update_shots<<<dimGrid_lt,dimBlock>>>(res_shot_z_d,cal_shot_z_d,beta_step_d,receiver_num,lt,0);

							/////////output residuals
							hipMemcpy(shotgather,res_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d",ishot+1,iter+2);
							write_file_1d(shotgather,lt_rec,filename);

							hipMemcpy(shotgather,res_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d",ishot+1,iter+2);
							write_file_1d(shotgather,lt_rec,filename);



							cuda_update_tmp_shots<<<dimGrid_lt,dimBlock>>>(tmp_shot_x_d,cal_shot_x_d,beta_step_d,receiver_num,lt,0);
							cuda_update_tmp_shots<<<dimGrid_lt,dimBlock>>>(tmp_shot_z_d,cal_shot_z_d,beta_step_d,receiver_num,lt,0);
							/////////for cross-correlation misfunction 2017年08月25日 星期五 09时28分54秒 
							/////////output tmp_cal
							hipMemcpy(shotgather,tmp_shot_x_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/tmp_shot_x_%d_iter_%d",ishot+1,iter+2);
							write_file_1d(shotgather,lt_rec,filename);

							hipMemcpy(shotgather,tmp_shot_z_d,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./someoutput/bin/tmp_shot_z_%d_iter_%d",ishot+1,iter+2);
							write_file_1d(shotgather,lt_rec,filename);////

							if(vsp_2!=0)
							{
								/////////read residuals	
								sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d_2",ishot+1,iter+1);
								fread_file_1d(shotgather,receiver_num,lt,filename);
								hipMemcpy(res_shot_x_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d_2",ishot+1,iter+1);
								fread_file_1d(shotgather,receiver_num,lt,filename);
								hipMemcpy(res_shot_z_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

								/////////read cal shot
								sprintf(filename,"./someoutput/bin/cal_shot_x_%d_iter_%d_2",ishot+1,iter+1);
								fread_file_1d(shotgather,receiver_num,lt,filename);
								hipMemcpy(cal_shot_x_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./someoutput/bin/cal_shot_z_%d_iter_%d_2",ishot+1,iter+1);
								fread_file_1d(shotgather,receiver_num,lt,filename);
								hipMemcpy(cal_shot_z_d_2,shotgather,lt_rec*sizeof(float),hipMemcpyHostToDevice);

								cuda_update_shots<<<dimGrid_lt,dimBlock>>>(res_shot_x_d_2,cal_shot_x_d_2,beta_step_d,receiver_num,lt,0);
								cuda_update_shots<<<dimGrid_lt,dimBlock>>>(res_shot_z_d_2,cal_shot_z_d_2,beta_step_d,receiver_num,lt,0);

								/////////output residuals
								hipMemcpy(shotgather,res_shot_x_d_2,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./someoutput/bin/res_shot_x_%d_iter_%d_2",ishot+1,iter+2);
								write_file_1d(shotgather,lt_rec,filename);

								hipMemcpy(shotgather,res_shot_z_d_2,lt_rec*sizeof(float),hipMemcpyDeviceToHost);
								sprintf(filename,"./someoutput/bin/res_shot_z_%d_iter_%d_2",ishot+1,iter+2);
								write_file_1d(shotgather,lt_rec,filename);
							}			
							ishot++;
						}
//////////////////////////iter_start!=0  restart 因为程序中断，需要重新开始！！！！！！！！！！！！！！！！！
						if(iter_start!=0)///////////read current result to restart
						{
							sprintf(filename,"./result/obj_niter-%d",iter);
							fread_file_1d(obj_niter_h,1,niter,filename);

							sprintf(filename,"./result/obj_niter1-%d",iter);
							fread_file_1d(obj_niter_h1,1,niter,filename);

							if(inversion_para==1||inversion_para==0)
							{
								sprintf(filename,"./result/result-lame1-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_lame1_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./result/result-lame2-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_lame2_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./result/result-den-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_den_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);
							}
						
							if(inversion_para==2||inversion_para==3)
							{
								sprintf(filename,"./result/result-vp-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_vp_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./result/result-vs-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_vs_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);

								sprintf(filename,"./result/result-density-%d",iter);
								fread_file_1d(wf_nxnz,nx,nz,filename);	
								hipMemcpy(perturb_density_d,wf_nxnz,nxnz*sizeof(float),hipMemcpyHostToDevice);
							}
						}
//////////////////////////iter_start!=0  restart 因为程序中断，需要重新开始！！！！！！！！！！！！！！！！！

///////////////////////////////* update the image model according to previous image model, gradient/conjugate gradient  and estimated stepsize */
					if(inversion_para==1||inversion_para==0)
					{
						/////for ELSRTM
						/*cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_lame1_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_lame2_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_den_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);


						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_lame1_d,all_conj_lame1_d,beta_step_d,nx,nz,0);
//////update lame2
						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_lame2_d,all_conj_lame2_d,beta_step_d,nx,nz,0);		
//////update den
						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_den_d,all_conj_den_d,beta_step_d,nx,nz,0);*/
//////update lame1						
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_lame1_d,all_conj_lame1_d,beta_step_d,nx,nz,0,precon_z2);
//////update lame2
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_lame2_d,all_conj_lame2_d,beta_step_d,nx,nz,0,precon_z2);		
//////update den
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_den_d,all_conj_den_d,beta_step_d,nx,nz,0,precon_z2);					
						/////for ELSRTM 
					}
		
					if(inversion_para==2||inversion_para==3)
					{
						/////for ELSRTM
						/*cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_vp_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_vs_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
						cuda_attenuation_after_lap_new<<<dimGrid_new,dimBlock>>>(all_conj_density_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);


						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_vp_d,all_conj_vp_d,beta_step_d,nx,nz,0);
//////update velocity1
						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_vs_d,all_conj_vs_d,beta_step_d,nx,nz,0);
//////update density
						cuda_update_shots<<<dimGrid_new,dimBlock>>>(perturb_density_d,all_conj_density_d,beta_step_d,nx,nz,0);*/
//////update velocity						
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_vp_d,all_conj_vp_d,beta_step_d,nx,nz,0,precon_z2);
//////update velocity1
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_vs_d,all_conj_vs_d,beta_step_d,nx,nz,0,precon_z2);
//////update density
						cuda_update_shots_new<<<dimGrid_new,dimBlock>>>(perturb_density_d,all_conj_density_d,beta_step_d,nx,nz,0,precon_z2);
						/////for ELSRTM 
					}
			
///////////////////////////////////////one iteration is over
					hipMemcpy(beta_h,beta_d,3*sizeof(float),hipMemcpyDeviceToHost);
					hipMemcpy(beta_step_h,beta_step_d,3*sizeof(float),hipMemcpyDeviceToHost);
					hipMemcpy(epsil_h,epsil_d,4*sizeof(float),hipMemcpyDeviceToHost);
					//hipMemcpy(beta_h,beta_d,3*sizeof(float),hipMemcpyDeviceToHost);

						////time consume
						warn("iterative times is =%d",iter+1);
						fprintf(logfile,"iterative times=%d\n",iter+1);

					if(correlation_misfit==0)
					{
						if(iter==0) 	
						{
							obj_niter_h1[iter]=obj_h[0];

							obj_niter_h[iter]=1.0;					
						}

						else		
						{
							obj_niter_h1[iter]=obj_h[0];

							obj_niter_h[iter]=obj_niter_h1[iter]/obj_niter_h1[0];
						}

						/* output important information at each FWI iteration */
						//warn("obj=%f  beta=%f  epsil=%f  alpha=%f",obj, beta, epsil, alpha);

						////normolized objection vaule
						warn("normlaized_obj=%f",obj_niter_h[iter]*100);
						fprintf(logfile,"normlaized_obj=%f\n",obj_niter_h[iter]);

						////objection vaule
						warn("obj=%f",obj_niter_h1[iter]);
						fprintf(logfile,"obj=%f\n",obj_niter_h1[iter]);
					}

					else
					{
						obj_niter_h[iter]=obj_h[0];

						////normolized objection vaule
						warn("normlaized_obj=%f",obj_niter_h[iter]*100);
						fprintf(logfile,"normlaized_obj=%f\n",obj_niter_h[iter]);

						////objection vaule
						warn("obj=%f",obj_niter_h[iter]);
						fprintf(logfile,"obj=%f\n",obj_niter_h[iter]);
					}


					////conjugated method and overall'step
					warn("beta_vp=%f  beta_vs=%f  beta_density=%f ", beta_h[0],beta_h[1],beta_h[2]);
					warn("overall'step=%f",beta_step_h[0]);
					
					fprintf(logfile,"overall'step=%f beta_vp=%f  beta_vs=%f  beta_density=%f\n",beta_step_h[0],beta_h[0],beta_h[1],beta_h[2]);

					hipEventRecord(stop);/* record ending time */
  					hipEventSynchronize(stop);
  					hipEventElapsedTime(&mstimer, start, stop);
					totaltime+=mstimer*1e-3;

					warn("Programe is done, total time cost: %f (s)", totaltime);////////to current step has cost times
					fprintf(logfile,"iteration %d finished: %f (s)\n\n",iter+1, mstimer*1e-3);////////the current step  cost times

					//warn("epsil_vp=%f  epsil_step_vs=%f  epsil_step_density=%f ",epsil_h[0],epsil_h[1],epsil_h[2]);
					//warn("iteration %d finished: %f (s)",iter+1, mstimer*1e-3);
					//fprintf(logfile,"obj=%f  beta=%f  epsil=%f  alpha=%f\n",obj, beta, epsil, alpha);
					//fprintf(logfile,"iteration %d finished: %f (s)\n\n",iter+1, mstimer*1e-3);

////////////////output update lame coefficient
					if(inversion_para==1||inversion_para==0)
					{
						hipMemcpy(wf_nxnz,perturb_lame1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-lame1-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);
	
						hipMemcpy(wf_nxnz,perturb_lame2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-lame2-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);

						hipMemcpy(wf_nxnz,perturb_den_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-den-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);
					}
////////////////output update velocity
					if(inversion_para==2||inversion_para==3)
					{
						hipMemcpy(wf_nxnz,perturb_vp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-vp-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);
	
						hipMemcpy(wf_nxnz,perturb_vs_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-vs-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);

						hipMemcpy(wf_nxnz,perturb_density_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-density-%d",iter+1);
						write_file_1d(wf_nxnz,nxnz,filename);
					}

					sprintf(filename,"./result/obj_niter-%d",iter+1);
					write_file_1d(obj_niter_h,niter,filename);

					sprintf(filename,"./result/obj_niter1-%d",iter+1);
					write_file_1d(obj_niter_h1,niter,filename);

					fclose(logfile);////important

////////////////output update velocity and lame coefficient after laplace operator
					if(laplace==0)
					{
						if(inversion_para==2||inversion_para==3)
						{
							//////////1111111
							hipMemcpy(wf_nxnz_d,perturb_vp_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_vp_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-vp1-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);

							/////////2222222222
							hipMemcpy(wf_nxnz_d,perturb_vs_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_vs_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-vs1-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);

							//////////333333333
							hipMemcpy(wf_nxnz_d,perturb_density_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_density_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-density1-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);
						}

						if(inversion_para==0||inversion_para==1)
						{
							//////////1111111
							hipMemcpy(wf_nxnz_d,perturb_lame1_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_lame1_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-lame11-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);

							/////////2222222222
							hipMemcpy(wf_nxnz_d,perturb_lame2_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_lame2_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-lame22-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);

							//////////333333333
							hipMemcpy(wf_nxnz_d,perturb_den_d,nxnz*sizeof(float),hipMemcpyDeviceToDevice);
							cuda_laplace<<<dimGrid_new,dimBlock>>>(perturb_den_d,wf_nxnz_d,s_velocity_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,dx,dz,1,laplace);
							cuda_lap<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,laplace);
							cuda_attenuation_after_lap_new2<<<dimGrid_new,dimBlock>>>(wf_nxnz_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up,precon_z1,precon_z2);
							hipMemcpy(wf_nxnz,wf_nxnz_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
							sprintf(filename,"./result/result-den1-%d",iter+1);
							write_file_1d(wf_nxnz,nxnz,filename);
						}
					}		
	}	
					write_file_1d(obj_niter_h,niter,"./result/obj_niter");

					write_file_1d(obj_niter_h1,niter,"./result/obj_niter1");

////////////////output update lame coefficient
					if(inversion_para==1||inversion_para==0)
					{
						hipMemcpy(wf_nxnz,perturb_lame1_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-lame1");
						write_file_1d(wf_nxnz,nxnz,filename);
	
						hipMemcpy(wf_nxnz,perturb_lame2_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-lame2");
						write_file_1d(wf_nxnz,nxnz,filename);

						hipMemcpy(wf_nxnz,perturb_den_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-den");
						write_file_1d(wf_nxnz,nxnz,filename);
					}					
////////////////output update velocity
					if(inversion_para==2||inversion_para==3)
					{
						hipMemcpy(wf_nxnz,perturb_vp_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-vp");
						write_file_1d(wf_nxnz,nxnz,filename);
	
						hipMemcpy(wf_nxnz,perturb_vs_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-vs");
						write_file_1d(wf_nxnz,nxnz,filename);

						hipMemcpy(wf_nxnz,perturb_density_d,nxnz*sizeof(float),hipMemcpyDeviceToHost);
						sprintf(filename,"./result/result-density");
						write_file_1d(wf,nxnz,filename);
					}
			
		//finish = clock();
		//time(&t2);
		//warn("time is = %f\n",difftime(t2,t1));
		//duration = (double)(finish - start)/CLOCKS_PER_SEC;
		//warn( "CUDA duration time is =%f seconds\n", duration );
		//warn("get the shot gather");
		logfile=fopen("log.txt","ab");//remember to free log file
		fprintf(logfile,"Programe is done, total time cost: %f (s)\n", totaltime);
		fclose(logfile);////important

		/* destroy timing varibles */
		hipEventDestroy(start);
		hipEventDestroy(stop);

		

		hipFree(s_density_d);
		hipFree(s_velocity_d);
		hipFree(s_velocity1_d);

		hipFree(obs_shot_x_d);
		hipFree(obs_shot_z_d);
		hipFree(cal_shot_x_d);
		hipFree(cal_shot_z_d);
		hipFree(res_shot_x_d);
		hipFree(res_shot_z_d);

		hipFree(attenuation_d);

		hipFree(density_d);
		hipFree(velocity_d);
		hipFree(velocity1_d);
		
		hipFree(coe_opt_d);
		hipFree(coe_opt1_d);
		hipFree(wavelet_d);

		hipFree(vx1_d);
		hipFree(vz1_d);
		hipFree(txx1_d);
		hipFree(tzz1_d);
		hipFree(txz1_d);
		hipFree(vx2_d);
		hipFree(vz2_d);
		hipFree(txx2_d);
		hipFree(tzz2_d);
		hipFree(txz2_d);

		/*hipFree(vxu_d);
		hipFree(vxd_d);
		hipFree(vxr_d);
		hipFree(vxl_d);
		hipFree(vzu_d);
		hipFree(vzd_d);
		hipFree(vzr_d);
		hipFree(vzl_d);*/

		hipFree(rvx1_d);
		hipFree(rvz1_d);
		hipFree(rtxx1_d);
		hipFree(rtzz1_d);
		hipFree(rtxz1_d);
		hipFree(rvx2_d);
		hipFree(rvz2_d);
		hipFree(rtxx2_d);
		hipFree(rtzz2_d);
		hipFree(rtxz2_d);
	
		//warn("free1\n");
	
		free1int(receiver_x_cord);
		
		//warn("free2\n");		

		free1int(source_x_cord);

		//warn("free3\n");

		free1float(shotgather);
		free1float(shotgather1);
		//warn("free4\n");

		free1float(wf_append);
		//warn("free5\n");

		free1float(wf);
		//warn("free6\n");

		free1float(attenuation);	
		//warn("free7\n");

		/*free1float(velocity);
		free1float(velocity1);
		free1float(density);

		free1float(s_density);
		free1float(s_velocity);
		free1float(s_velocity1);*/

		free1float(coe_opt);
		free1float(wavelet);

		warn("free_over\n");
		warn("***end***\n");
} 
