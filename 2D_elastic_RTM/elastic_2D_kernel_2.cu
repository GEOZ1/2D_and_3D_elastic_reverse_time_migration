#include "hip/hip_runtime.h"
__constant__ const int BDIMX1=32;
__constant__ const int BDIMY1=16;
__constant__ const int radius1=6;
__constant__ const float pai=3.1415926;
__constant__ const int filter_scale=9;


__global__ void sum_poynting(float *poyn_px_d,float *poyn_pz_d,float *poyn_sx_d,float *poyn_sz_d,float *vxp1_d,float *vzp1_d,float *vxs1_d,float *vzs1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *tp1_d,int dimx,int dimz)
//sum_poynting<<<dimGrid,dimBlock>>>(poyn_rpx_d,poyn_rpz_d,poyn_rsx_d,poyn_rsz_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,rtxx1_d,rtzz1_d,rtxz1_d,rtp1_d,nx_append_radius,nz_append_radius);
//poyn_px_d,poyn_pz_d,poyn_sx_d,poyn_sz_d,vxp1_d,vzp1_d,vxs1_d,vzs1_d,txx1_d,tzz1_d,txz1_d,tp1_d,nx_append_radius1,nz_append_radius1
//poyn_rpx_d,poyn_rpz_d,poyn_rsx_d,poyn_rsz_d,rvxp2_d,rvzp2_d,rvxs2_d,rvzs2_d,rtxx2_d,rtzz2_d,rtxz2_d,rtp2_d,nx_append_radius1,nz_append_radius1
//(poyn_px_d,poyn_pz_d,poyn_sx_d,poyn_sz_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,txx2_d,tzz2_d,txz2_d,tp2_d,nx_append_radius1,nz_append_radius1);
{

		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
			
			poyn_px_d[in_idx]=-1*tp1_d[in_idx]*vxp1_d[in_idx];
			
			poyn_pz_d[in_idx]=-1*tp1_d[in_idx]*vzp1_d[in_idx];

			//poyn_px_d[in_idx]=-1*tp1_d[in_idx]*(vxp1_d[in_idx]+vxp1_d[in_idx+dimz])/2.0;
			
			//poyn_pz_d[in_idx]=-1*tp1_d[in_idx]*(vzp1_d[in_idx]+vzp1_d[in_idx+1])/2.0;
			
			//poyn_px_d[in_idx]=-1*tp1_d[in_idx]*(vxp1_d[in_idx]+vxp1_d[in_idx-dimz])/2.0;
			
			//poyn_pz_d[in_idx]=-1*tp1_d[in_idx]*(vzp1_d[in_idx]+vzp1_d[in_idx-1])/2.0;



			poyn_sx_d[in_idx]=-1*((txx1_d[in_idx]-tp1_d[in_idx])*vxs1_d[in_idx]+txz1_d[in_idx]*vzs1_d[in_idx]);
			
			poyn_sz_d[in_idx]=-1*((tzz1_d[in_idx]-tp1_d[in_idx])*vzs1_d[in_idx]+txz1_d[in_idx]*vxs1_d[in_idx]);
			
			//poyn_sx_d[in_idx]=-1*((txx1_d[in_idx]-tp1_d[in_idx])*(vxs1_d[in_idx]+vxs1_d[in_idx+dimz])/2.0+(txz1_d[in_idx]+txz1_d[in_idx+1]+txz1_d[in_idx+dimz]+txz1_d[in_idx+1+dimz])*(vzs1_d[in_idx]+vzs1_d[in_idx+1])/2.0/4.0);
			
			//poyn_sz_d[in_idx]=-1*((tzz1_d[in_idx]-tp1_d[in_idx])*(vzs1_d[in_idx]+vzs1_d[in_idx+1])/2.0+(txz1_d[in_idx]+txz1_d[in_idx+1]+txz1_d[in_idx+dimz]+txz1_d[in_idx+1+dimz])*(vxs1_d[in_idx]+vxs1_d[in_idx+dimz])/2.0/4.0);
			
			//poyn_sx_d[in_idx]=-1*((txx1_d[in_idx]-tp1_d[in_idx])*(vxs1_d[in_idx]+vxs1_d[in_idx-dimz])/2.0+(txz1_d[in_idx]+txz1_d[in_idx-1]+txz1_d[in_idx-dimz]+txz1_d[in_idx-1-dimz])*(vzs1_d[in_idx]+vzs1_d[in_idx-1])/2.0/4.0);
			
			//poyn_sz_d[in_idx]=-1*((tzz1_d[in_idx]-tp1_d[in_idx])*(vzs1_d[in_idx]+vzs1_d[in_idx-1])/2.0+(txz1_d[in_idx]+txz1_d[in_idx-1]+txz1_d[in_idx-dimz]+txz1_d[in_idx-1-dimz])*(vxs1_d[in_idx]+vxs1_d[in_idx-dimz])/2.0/4.0);
		}
}
__global__ void poynting(float *txx2_d,float *txz2_d,float *tzz2_d,float *vx2_d,float *vz2_d,float *pz_d,float *px_d,int dimx,int dimz)
//(txx2_d,txz2_d,tzz2_d,vx2_d,vz2_d,poyn_z_d,poyn_x_d,nx_append_radius1,nz_append_radius1);
{

		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
			
			//pz_d[in_idx]=-(txz2_d[in_idx]*vx2_d[in_idx]+tzz2_d[in_idx]*vz2_d[in_idx]);

			//px_d[in_idx]=-(txx2_d[in_idx]*vx2_d[in_idx]+txz2_d[in_idx]*vz2_d[in_idx]);
			
			pz_d[in_idx]=-((txz2_d[in_idx]+txz2_d[in_idx+1]+txz2_d[in_idx+dimz]+txz2_d[in_idx+1+dimz])*(vx2_d[in_idx]+vx2_d[in_idx+1])/2.0/4.0+tzz2_d[in_idx]*(vz2_d[in_idx]+vz2_d[in_idx+1])/2.0);

			px_d[in_idx]=-(txx2_d[in_idx]*(vx2_d[in_idx]+vx2_d[in_idx+dimz])/2.0+(txz2_d[in_idx]+txz2_d[in_idx+1]+txz2_d[in_idx+dimz]+txz2_d[in_idx+1+dimz])*(vz2_d[in_idx]+vz2_d[in_idx+1])/2.0/4.0);
			
			//pz_d[in_idx]=-((txz2_d[in_idx]+txz2_d[in_idx-1]+txz2_d[in_idx-dimz]+txz2_d[in_idx-1-dimz])*(vx2_d[in_idx]+vx2_d[in_idx-1])/2.0/4.0+tzz2_d[in_idx]*(vz2_d[in_idx]+vz2_d[in_idx-1])/2.0);

			//px_d[in_idx]=-(txx2_d[in_idx]*(vx2_d[in_idx]+vx2_d[in_idx-dimz])/2.0+(txz2_d[in_idx]+txz2_d[in_idx-1]+txz2_d[in_idx-dimz]+txz2_d[in_idx-1-dimz])*(vz2_d[in_idx]+vz2_d[in_idx-1])/2.0/4.0);
		}
}

__global__ void scalar_poynting(float *rpx_d,float *rpz_d,float *rvx2,float *rvz2,float *wf2_d,float *coe_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;
				dimz=dimz+2*radius1;
				ix=ix+radius1;
				iz=iz+radius1;
				in_idx=ix*dimz+iz;		

				rpx_d[in_idx]=-rvx2[in_idx]*wf2_d[in_idx];
				rpz_d[in_idx]=-rvz2[in_idx]*wf2_d[in_idx];
		}
}

__global__ void cal_direction_2D_elastic(float *poyn_px_d,float *poyn_pz_d,float *poyn_sx_d,float *poyn_sz_d,float *vxp1_d,float *vzp1_d,float *vxs1_d,float *vzs1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *tp1_d,int dimx,int dimz)
///cal_direction_2D_elastic<<<dimGrid,dimBlock>>>(direction_px_d,direction_pz_d,direction_sx_d,direction_sz_d,vxp2_d,vzp2_d,vxs2_d,vzs2_d,txx2_d,tzz2_d,txz2_d,tp2_d,nx_append_radius,nz_append_radius);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
////cross product
////one is x component , another is z component
		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;

			if(vzp1_d[in_idx]>=0)
			{
				poyn_px_d[in_idx]=vxp1_d[in_idx];
				
				poyn_pz_d[in_idx]=vzp1_d[in_idx];
			}

			else
			{
				poyn_px_d[in_idx]=-1.0*vxp1_d[in_idx];
				
				poyn_pz_d[in_idx]=-1.0*vzp1_d[in_idx];
			}

			if(vxs1_d[in_idx]>=0)
			{
				poyn_sx_d[in_idx]=vzs1_d[in_idx];
				
				poyn_sz_d[in_idx]=-1.0*vxs1_d[in_idx];
			}

			else
			{				
				poyn_sx_d[in_idx]=-1.0*vzs1_d[in_idx];
				
				poyn_sz_d[in_idx]=vxs1_d[in_idx];	
			}
		}
}

__global__ void divergence_old(float *p_d,float *p1_d,float *coe_d,float dx,float dz,int dimx,int dimz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;


		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=p_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=p_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=p_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=p_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=p_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}

				__syncthreads();
			
				/*float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);*/

				/*float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);*/
////two choice
				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);			

				p1_d[in_idx]=-1*sumx/dx;

		}

}

__global__ void divergence_new(float *p_d,float *p1_d,float *normalx1_d,float *normalz1_d,float *coe_d,float dx,float dz,int dimx,int dimz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=p_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=p_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=p_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=p_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=p_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}

				__syncthreads();
			
				/*float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);*/
////two choice
				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);	

				p1_d[in_idx]=(sumx*normalz1_d[in_idx]-sumz*normalx1_d[in_idx]);

		}

}

__global__ void real_divergence(float *p_d,float *p_x_d,float *p_z_d,float dx,float dz,int dimx,int dimz,float *coe_d)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;


		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=p_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=p_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=p_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=p_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=p_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}

				__syncthreads();
////one choice
				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);
					
					p_x_d[in_idx]=sumx/dx;
					p_z_d[in_idx]=sumz/dz;					
		}
}
__global__ void curl_old(float *s_d,float *s1_d,float *coe_d,float dx,float dz,int dimx,int dimz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=s_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=s_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=s_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=s_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=s_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}

				__syncthreads();
			
				/* float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);*/

				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);

				s1_d[in_idx]=sumx/dx;

		}

}

__global__ void curl_new(float *s_d,float *s1_d,float *normalx1_d,float *normalz1_d,float *coe_d,float dx,float dz,int dimx,int dimz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=s_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=s_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=s_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=s_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=s_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				
				__syncthreads();
			
				/* float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);*/
					
				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);

				s1_d[in_idx]=sumx*normalz1_d[in_idx]-sumz*normalx1_d[in_idx];

		}

}

__global__ void real_curl(float *s_d,float *s_x_d,float *s_z_d,float dx,float dz,int dimx,int dimz,float *coe_d)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;


		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=s_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=s_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=s_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=s_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=s_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}

				__syncthreads();

				float    sumx=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data[tz+1][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-6][tx]);
					
					s_x_d[in_idx]=sumz/dx;
					s_z_d[in_idx]=sumx/dz;
					
		}
}

__global__ void decom(float *wf1_d,float *f1_d,float *wfp_d,float *wfs_d,float *coe_d,int nx_append,int nz_append,float dx,float dz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];
		__shared__ float s_data1[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;

		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<nx_append-2*radius1&&iz<nz_append-2*radius1)
		{
			ix=ix+radius1;
			iz=iz+radius1;			
			in_idx=ix*nz_append+iz;	

			__syncthreads();

			s_data[tz][tx]=wf1_d[in_idx];
			s_data1[tz][tx]=f1_d[in_idx];
				
			if(threadIdx.y<radius1)
			{
				s_data[threadIdx.y][threadIdx.x]=wf1_d[in_idx-radius1-radius1*nz_append];//up
				s_data[threadIdx.y][threadIdx.x+2*radius1]=wf1_d[in_idx-radius1+radius1*nz_append];//up
				s_data[threadIdx.y+BDIMY1+radius1][threadIdx.x]=wf1_d[in_idx+BDIMY1-radius1*nz_append];//down
				s_data[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=wf1_d[in_idx+BDIMY1+radius1*nz_append];//down

				s_data1[threadIdx.y][threadIdx.x]=f1_d[in_idx-radius1-radius1*nz_append];//up
				s_data1[threadIdx.y][threadIdx.x+2*radius1]=f1_d[in_idx-radius1+radius1*nz_append];//up
				s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x]=f1_d[in_idx+BDIMY1-radius1*nz_append];//down
				s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=f1_d[in_idx+BDIMY1+radius1*nz_append];//down

			}
			if(threadIdx.x<radius1)
			{
				s_data[tz][threadIdx.x]=wf1_d[in_idx-radius1*nz_append];//g_input[in_idx-radius1];//left
				s_data[tz][threadIdx.x+BDIMX1+radius1]=wf1_d[in_idx+BDIMX1*nz_append];//g_input[in_idx+BDIMX1];//right
				s_data1[tz][threadIdx.x]=f1_d[in_idx-radius1*nz_append];//g_input[in_idx-radius1];//left
				s_data1[tz][threadIdx.x+BDIMX1+radius1]=f1_d[in_idx+BDIMX1*nz_append];//g_input[in_idx+BDIMX1];//right
			}
			
			__syncthreads();

//p wave
		float   sum=coe_d[1]*(s_data[tz][tx+1]-s_data[tz][tx]);		
		       sum+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
		       sum+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
		       sum+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
		       sum+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
		       sum+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

		float  sum1=coe_d[1]*(s_data1[tz][tx]-  s_data1[tz-1][tx]);
		      sum1+=coe_d[2]*(s_data1[tz+1][tx]-s_data1[tz-2][tx]);
		      sum1+=coe_d[3]*(s_data1[tz+2][tx]-s_data1[tz-3][tx]);
		      sum1+=coe_d[4]*(s_data1[tz+3][tx]-s_data1[tz-4][tx]);
		      sum1+=coe_d[5]*(s_data1[tz+4][tx]-s_data1[tz-5][tx]);
		      sum1+=coe_d[6]*(s_data1[tz+5][tx]-s_data1[tz-6][tx]);
		wfp_d[in_idx]=(1.0/dx)*sum+(1.0/dz)*sum1;

//s wave
		float  sum3=coe_d[1]*(s_data[tz+1][tx]-s_data[tz][tx]);
		      sum3+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
		      sum3+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
		      sum3+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
		      sum3+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
		      sum3+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);

		float   sum2=coe_d[1]*(s_data1[tz][tx]-  s_data1[tz][tx-1]);		
		       sum2+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
		       sum2+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
		       sum2+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
		       sum2+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
		       sum2+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);
		       
		/*float  sum3=coe_d[1]*(s_data[tz][tx]-s_data[tz-1][tx]);
		      sum3+=coe_d[2]*(s_data[tz+1][tx]-s_data[tz-2][tx]);
		      sum3+=coe_d[3]*(s_data[tz+2][tx]-s_data[tz-3][tx]);
		      sum3+=coe_d[4]*(s_data[tz+3][tx]-s_data[tz-4][tx]);
		      sum3+=coe_d[5]*(s_data[tz+4][tx]-s_data[tz-5][tx]);
		      sum3+=coe_d[6]*(s_data[tz+5][tx]-s_data[tz-6][tx]);

		float   sum2=coe_d[1]*(s_data1[tz][tx+1]-  s_data1[tz][tx]);		
		       sum2+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
		       sum2+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
		       sum2+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
		       sum2+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
		       sum2+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);*/

		wfs_d[in_idx]=(1.0/dz)*sum3-(1.0/dx)*sum2;

		}
}

__global__ void decom_new(float *vx1_d,float *vz1_d,float *p_d,float *s_d,float *velocity_d,float *velocity1_d,float *coe_d,int dimx,int dimz,float dx,float dz)
{
		__shared__ float s_data1[BDIMY1+2*radius1][BDIMX1+2*radius1];
		__shared__ float s_data2[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		float s_velocity,s_velocity1;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;

		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx1_d[in_idx];
				s_data2[tz][tx]=vz1_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data1[threadIdx.y][tx]=vx1_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data1[threadIdx.y+BDIMY1+radius1][tx]=vx1_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down

						s_data2[threadIdx.y][tx]=vz1_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data2[threadIdx.y+BDIMY1+radius1][tx]=vz1_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data1[tz][threadIdx.x]=vx1_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data1[tz][threadIdx.x+BDIMX1+radius1]=vx1_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right

						s_data2[tz][threadIdx.x]=vz1_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data2[tz][threadIdx.x+BDIMX1+radius1]=vz1_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				
				s_velocity=velocity_d[in_idx];
				s_velocity1=(velocity1_d[in_idx]+velocity1_d[in_idx+1]+velocity1_d[in_idx+dimz]+velocity1_d[in_idx+1+dimz])/4.0;
				__syncthreads();
				
//p wave
		float   sum=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);		
		       sum+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
		       sum+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
		       sum+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
		       sum+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
		       sum+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

		float  sum1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
		      sum1+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
		      sum1+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
		      sum1+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
		      sum1+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
		      sum1+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
		     
		/*  float   sum=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);		
		       sum+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
		       sum+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
		       sum+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
		       sum+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
		       sum+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

		float  sum1=coe_d[1]*(s_data2[tz+1][tx]-s_data2[tz][tx]);
		      sum1+=coe_d[2]*(s_data2[tz+2][tx]-s_data2[tz-1][tx]);
		      sum1+=coe_d[3]*(s_data2[tz+3][tx]-s_data2[tz-2][tx]);
		      sum1+=coe_d[4]*(s_data2[tz+4][tx]-s_data2[tz-3][tx]);
		      sum1+=coe_d[5]*(s_data2[tz+5][tx]-s_data2[tz-4][tx]);
		      sum1+=coe_d[6]*(s_data2[tz+6][tx]-s_data2[tz-5][tx]);*/
		      
			p_d[in_idx]=s_velocity*(1.0/dx)*sum+s_velocity*(1.0/dz)*sum1;

//s wave
		float  sum3=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
		      sum3+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
		      sum3+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
		      sum3+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
		      sum3+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
		      sum3+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

		float   sum2=coe_d[1]*(s_data2[tz][tx]-s_data2[tz][tx-1]);		
		       sum2+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
		       sum2+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
		       sum2+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
		       sum2+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
		       sum2+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);
		 
		/* float  sum3=coe_d[1]*(s_data1[tz][tx]-s_data1[tz-1][tx]);
		      sum3+=coe_d[2]*(s_data1[tz+1][tx]-s_data1[tz-2][tx]);
		      sum3+=coe_d[3]*(s_data1[tz+2][tx]-s_data1[tz-3][tx]);
		      sum3+=coe_d[4]*(s_data1[tz+3][tx]-s_data1[tz-4][tx]);
		      sum3+=coe_d[5]*(s_data1[tz+4][tx]-s_data1[tz-5][tx]);
		      sum3+=coe_d[6]*(s_data1[tz+5][tx]-s_data1[tz-6][tx]);*/

		/*float   sum2=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);		
		       sum2+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
		       sum2+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
		       sum2+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
		       sum2+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
		       sum2+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);*/
		       
			s_d[in_idx]=s_velocity1*(1.0/dz)*sum3-s_velocity1*(1.0/dx)*sum2;	
		}		
}

__global__ void save_all_wavefield(float *fws_d,float *vx2_d,int it,int dimx,int dimz)
{

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;iz=iz+radius1;
			dimx=dimx+2*radius1;dimz=dimz+2*radius1;		
			in_idx=ix*dimz+iz;
						
			fws_d[it*dimx*dimz+in_idx]=vx2_d[in_idx];
		}
}

__global__ void set_all_wavefield(float *fws_d,float *vx2_d,int it,int dimx,int dimz)
{

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;iz=iz+radius1;
			dimx=dimx+2*radius1;dimz=dimz+2*radius1;		
			in_idx=ix*dimz+iz;
						
			vx2_d[in_idx]=fws_d[it*dimx*dimz+in_idx];
		}
}

__global__ void filter_sign_new(float *signx_d,float *filter_signx_d,int dimx,int dimz,int scale)
{
		__shared__ float s_data[filter_scale+2*radius1][filter_scale+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[threadIdx.y+2*radius1-1][threadIdx.x+2*radius1-1]=0.0;
		s_data[threadIdx.y+2*radius1-1][threadIdx.x]=0.0;
		s_data[threadIdx.y][threadIdx.x+2*radius1-1]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				filter_signx_d[in_idx]=0.0;

				__syncthreads();
				
				s_data[tz][tx]=signx_d[in_idx];
		
				if(threadIdx.y<radius1)
				{
					s_data[threadIdx.y][tx]=signx_d[in_idx-radius1];
					s_data[threadIdx.y+filter_scale+radius1][tx]=signx_d[in_idx+filter_scale];
				}
				if(threadIdx.x<radius1)
				{
					s_data[tz][threadIdx.x]=signx_d[in_idx-radius1*dimz];
					s_data[tz][threadIdx.x+filter_scale+radius1]=signx_d[in_idx+filter_scale*dimz];
				}
			
				__syncthreads();

				for(int m=-filter_scale/2;m<=filter_scale/2;m++)
					for(int n=-filter_scale/2;n<=filter_scale/2;n++)
						filter_signx_d[in_idx]+=s_data[tz-m][tx-n];////m is replaced by n
		}		
}

__global__ void filter_sign_new_share(float *signx_d,float *filter_signx_d,int dimx,int dimz,int scale)
{
		__shared__ float s_data1[BDIMY1+2*radius1][BDIMX1+2*radius1];
		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				filter_signx_d[in_idx]=0.0;
				
				__syncthreads();

				s_data1[tz][tx]=signx_d[in_idx];

				if(threadIdx.y<radius1)
				{
					s_data1[threadIdx.y][threadIdx.x]=signx_d[in_idx-radius1-radius1*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius1]=signx_d[in_idx-radius1+radius1*dimz];//up
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x]=signx_d[in_idx+BDIMY1-radius1*dimz];//down
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=signx_d[in_idx+BDIMY1+radius1*dimz];//down
				}
				if(threadIdx.x<radius1)
				{
					s_data1[tz][threadIdx.x]=signx_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
					s_data1[tz][threadIdx.x+BDIMX1+radius1]=signx_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				__syncthreads();

				for(int m=-4;m<=4;m++)
					for(int n=-4;n<=4;n++)
						filter_signx_d[in_idx]+=s_data1[tz+m][tx+n];////m is replaced by n
		}		
}

__global__ void compare_sign(float *filter_signx_d,float *filter_signy_d,float *filter_signz_d,float *sign_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
			if(filter_signx_d[in_idx]>=filter_signy_d[in_idx]&&filter_signx_d[in_idx]>=filter_signz_d[in_idx])	sign_d[in_idx]= 1;
			if(filter_signy_d[in_idx]>filter_signx_d[in_idx]&&filter_signy_d[in_idx]>filter_signz_d[in_idx])	sign_d[in_idx]=-1;
			if(filter_signz_d[in_idx]>filter_signx_d[in_idx]&&filter_signz_d[in_idx]>filter_signy_d[in_idx])	sign_d[in_idx]= 0;
		}
}

__global__ void set_sign_basedon_polarization_ps(float *vxp1_d,float *vzp1_d,float *rvxs1_d,float *rvzs1_d,float *signx_d,float *signy_d,float *signz_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float set;
////cross product
////one is x component , another is z component
		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
			if(vzp1_d[in_idx]>=0&&rvxs1_d[in_idx]>=0)	set=-1*(vxp1_d[in_idx]*rvxs1_d[in_idx]+vzp1_d[in_idx]*rvzs1_d[in_idx]);
			if(vzp1_d[in_idx]>=0&&rvxs1_d[in_idx]<0)	set=1*(vxp1_d[in_idx]*rvxs1_d[in_idx]+vzp1_d[in_idx]*rvzs1_d[in_idx]);
			if(vzp1_d[in_idx]<0&&rvxs1_d[in_idx]>0)	set=1*(vxp1_d[in_idx]*rvxs1_d[in_idx]+vzp1_d[in_idx]*rvzs1_d[in_idx]);
			if(vzp1_d[in_idx]<0&&rvxs1_d[in_idx]<0)	set=-1*(vxp1_d[in_idx]*rvxs1_d[in_idx]+vzp1_d[in_idx]*rvzs1_d[in_idx]);
			
			if(set>0)	
				{
					signx_d[in_idx]=1;
					signy_d[in_idx]=0;
					signz_d[in_idx]=0;
				}
			if(set<0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=1;
					signz_d[in_idx]=0;
				}
			if(set==0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=0;
					signz_d[in_idx]=1;
				}
		}
}

__global__ void set_sign_basedon_polarization_sp(float *vxs1_d,float *vzs1_d,float *rvxp1_d,float *rvzp1_d,float *signx_d,float *signy_d,float *signz_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float set;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
////cross product
////one is x component , another is z component
			if(vxs1_d[in_idx]<=0&&rvzp1_d[in_idx]<=0)	set=(vzs1_d[in_idx]*rvzp1_d[in_idx]+vxs1_d[in_idx]*rvxp1_d[in_idx]);
			if(vxs1_d[in_idx]<0&&rvzp1_d[in_idx]>0)	set=-1*(vzs1_d[in_idx]*rvzp1_d[in_idx]+vxs1_d[in_idx]*rvxp1_d[in_idx]);
			if(vxs1_d[in_idx]>0&&rvzp1_d[in_idx]<0)	set=-1*(vzs1_d[in_idx]*rvzp1_d[in_idx]+vxs1_d[in_idx]*rvxp1_d[in_idx]);
			if(vxs1_d[in_idx]>0&&rvzp1_d[in_idx]>0)	set=(vzs1_d[in_idx]*rvzp1_d[in_idx]+vxs1_d[in_idx]*rvxp1_d[in_idx]);
			if(set>0)	
				{
					signx_d[in_idx]=1;
					signy_d[in_idx]=0;
					signz_d[in_idx]=0;
				}
			if(set<0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=1;
					signz_d[in_idx]=0;
				}
			if(set==0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=0;
					signz_d[in_idx]=1;
				}
		}
}

//cross-product for sign
__global__ void set_sign_forps(float *poyn_x_d,float *poyn_z_d,float *poyn_rx_d,float *poyn_rz_d,float *signx_d,float *signy_d,float *signz_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float set;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
			set=10000000*poyn_x_d[in_idx]*poyn_rz_d[in_idx]-10000000*poyn_z_d[in_idx]*poyn_rx_d[in_idx];
//cross-product for sign			
			if(set>0)	
				{
					signx_d[in_idx]=1;
					signy_d[in_idx]=0;
					signz_d[in_idx]=0;
				}
			if(set<0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=1;
					signz_d[in_idx]=0;
				}
			if(set==0)	
				{
					signx_d[in_idx]=0;
					signy_d[in_idx]=0;
					signz_d[in_idx]=1;
				}
		}
}

__global__ void normalized(float *normalx_d,float *normalz_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float sum;
		if(ix<dimx&&iz<dimz)
		{
			dimx=dimx+2*radius1;dimz=dimz+2*radius1;
			ix=ix+radius1;iz=iz+radius1;
			in_idx=ix*dimz+iz;
			sum=sqrt(normalx_d[in_idx]*normalx_d[in_idx]+normalz_d[in_idx]*normalz_d[in_idx]);
			if(sum!=0)
			{
				normalx_d[in_idx]=normalx_d[in_idx]/sum;
				normalz_d[in_idx]=normalz_d[in_idx]/sum;
			}
		}
}
__global__ void caculate_normal_basedon_resultpp(float *result_h_d,float *result_old_d,float *normalx1_d,float *normalz1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,float *coe)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int in_idx1;
		
		if(ix<nx-2*radius1&&iz<nz-2*radius1)
		{
			ix=ix+radius1;iz=iz+radius1;
			in_idx1=ix*nz+iz;
			in_idx=(ix+boundary_left)*dimz+boundary_up+iz;

			float  sumx_h=coe[1]*(result_h_d[in_idx1+1*dimz]-result_h_d[in_idx1]);
				sumx_h+=coe[2]*(result_h_d[in_idx1+2*dimz]-result_h_d[in_idx1-1*dimz]);
				sumx_h+=coe[3]*(result_h_d[in_idx1+3*dimz]-result_h_d[in_idx1-2*dimz]);
				sumx_h+=coe[4]*(result_h_d[in_idx1+4*dimz]-result_h_d[in_idx1-3*dimz]);
				sumx_h+=coe[5]*(result_h_d[in_idx1+5*dimz]-result_h_d[in_idx1-4*dimz]);
				sumx_h+=coe[6]*(result_h_d[in_idx1+6*dimz]-result_h_d[in_idx1-5*dimz]);
			float  sumz_h=coe[1]*(result_h_d[in_idx1+1]-result_h_d[in_idx1]);
				sumz_h+=coe[2]*(result_h_d[in_idx1+2]-result_h_d[in_idx1-1]);
				sumz_h+=coe[3]*(result_h_d[in_idx1+3]-result_h_d[in_idx1-2]);
				sumz_h+=coe[4]*(result_h_d[in_idx1+4]-result_h_d[in_idx1-3]);
				sumz_h+=coe[5]*(result_h_d[in_idx1+5]-result_h_d[in_idx1-4]);
				sumz_h+=coe[6]*(result_h_d[in_idx1+6]-result_h_d[in_idx1-5]);
			
			float  sumx_o=coe[1]*(result_old_d[in_idx1+1*dimz]-result_old_d[in_idx1]);
				sumx_o+=coe[2]*(result_old_d[in_idx1+2*dimz]-result_old_d[in_idx1-1*dimz]);
				sumx_o+=coe[3]*(result_old_d[in_idx1+3*dimz]-result_old_d[in_idx1-2*dimz]);
				sumx_o+=coe[4]*(result_old_d[in_idx1+4*dimz]-result_old_d[in_idx1-3*dimz]);
				sumx_o+=coe[5]*(result_old_d[in_idx1+5*dimz]-result_old_d[in_idx1-4*dimz]);
				sumx_o+=coe[6]*(result_old_d[in_idx1+6*dimz]-result_old_d[in_idx1-5*dimz]);
			float  sumz_o=coe[1]*(result_old_d[in_idx1+1]-result_old_d[in_idx1]);
				sumz_o+=coe[2]*(result_old_d[in_idx1+2]-result_old_d[in_idx1-1]);
				sumz_o+=coe[3]*(result_old_d[in_idx1+3]-result_old_d[in_idx1-2]);
				sumz_o+=coe[4]*(result_old_d[in_idx1+4]-result_old_d[in_idx1-3]);
				sumz_o+=coe[5]*(result_old_d[in_idx1+5]-result_old_d[in_idx1-4]);
				sumz_o+=coe[6]*(result_old_d[in_idx1+6]-result_old_d[in_idx1-5]);
				
			/*float  sumx_h=(result_h_d[in_idx1+1*dimz]-result_h_d[in_idx1]);
			float  sumz_h=(result_h_d[in_idx1+1]-result_h_d[in_idx1]);
			float  sumx_o=(result_old_d[in_idx1+1*dimz]-result_old_d[in_idx1]);
			float  sumz_o=(result_old_d[in_idx1+1]-result_old_d[in_idx1]);*/
			
				normalx1_d[in_idx]=(sumx_o*result_h_d[in_idx1]-sumx_h*result_old_d[in_idx1]);//(result_h_d[in_idx1]*result_h_d[in_idx1]+result_old_d[in_idx1]*result_old_d[in_idx1]);
				normalz1_d[in_idx]=(sumz_o*result_h_d[in_idx1]-sumz_h*result_old_d[in_idx1]);//(result_h_d[in_idx1]*result_h_d[in_idx1]+result_old_d[in_idx1]*result_old_d[in_idx1]);
		}
}

__global__ void caculate_normal_basedon_resultpp_new(float *result_h_d,float *result_old_d,float *normalx1_d,float *normalz1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,float *coe)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		
		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius1;
			iz=iz+radius1;
			dimx=dimx+2*radius1;
			dimz=dimz+2*radius1;
			in_idx=ix*dimz+iz;
///////note that  :   1-0,2- -1,3- -2,4- -3;
			/*float  sumx_h=coe[1]*(result_h_d[in_idx+1*dimz]-result_h_d[in_idx]);
				sumx_h+=coe[2]*(result_h_d[in_idx+2*dimz]-result_h_d[in_idx-1*dimz]);
				sumx_h+=coe[3]*(result_h_d[in_idx+3*dimz]-result_h_d[in_idx-2*dimz]);
				sumx_h+=coe[4]*(result_h_d[in_idx+4*dimz]-result_h_d[in_idx-3*dimz]);
				sumx_h+=coe[5]*(result_h_d[in_idx+5*dimz]-result_h_d[in_idx-4*dimz]);
				sumx_h+=coe[6]*(result_h_d[in_idx+6*dimz]-result_h_d[in_idx-5*dimz]);
			float  sumz_h=coe[1]*(result_h_d[in_idx+1]-result_h_d[in_idx]);
				sumz_h+=coe[2]*(result_h_d[in_idx+2]-result_h_d[in_idx-1]);
				sumz_h+=coe[3]*(result_h_d[in_idx+3]-result_h_d[in_idx-2]);
				sumz_h+=coe[4]*(result_h_d[in_idx+4]-result_h_d[in_idx-3]);
				sumz_h+=coe[5]*(result_h_d[in_idx+5]-result_h_d[in_idx-4]);
				sumz_h+=coe[6]*(result_h_d[in_idx+6]-result_h_d[in_idx-5]);
			
			float  sumx_o=coe[1]*(result_old_d[in_idx+1*dimz]-result_old_d[in_idx]);
				sumx_o+=coe[2]*(result_old_d[in_idx+2*dimz]-result_old_d[in_idx-1*dimz]);
				sumx_o+=coe[3]*(result_old_d[in_idx+3*dimz]-result_old_d[in_idx-2*dimz]);
				sumx_o+=coe[4]*(result_old_d[in_idx+4*dimz]-result_old_d[in_idx-3*dimz]);
				sumx_o+=coe[5]*(result_old_d[in_idx+5*dimz]-result_old_d[in_idx-4*dimz]);
				sumx_o+=coe[6]*(result_old_d[in_idx+6*dimz]-result_old_d[in_idx-5*dimz]);
			float  sumz_o=coe[1]*(result_old_d[in_idx+1]-result_old_d[in_idx]);
				sumz_o+=coe[2]*(result_old_d[in_idx+2]-result_old_d[in_idx-1]);
				sumz_o+=coe[3]*(result_old_d[in_idx+3]-result_old_d[in_idx-2]);
				sumz_o+=coe[4]*(result_old_d[in_idx+4]-result_old_d[in_idx-3]);
				sumz_o+=coe[5]*(result_old_d[in_idx+5]-result_old_d[in_idx-4]);
				sumz_o+=coe[6]*(result_old_d[in_idx+6]-result_old_d[in_idx-5]);*/
			float sumx_h=coe[0]*result_h_d[in_idx];
				sumx_h+=coe[1]*(result_h_d[in_idx+1*dimz]-result_h_d[in_idx-1*dimz]);
				sumx_h+=coe[2]*(result_h_d[in_idx+2*dimz]-result_h_d[in_idx-2*dimz]);
				sumx_h+=coe[3]*(result_h_d[in_idx+3*dimz]-result_h_d[in_idx-3*dimz]);
				sumx_h+=coe[4]*(result_h_d[in_idx+4*dimz]-result_h_d[in_idx-4*dimz]);
				sumx_h+=coe[5]*(result_h_d[in_idx+5*dimz]-result_h_d[in_idx-5*dimz]);
				sumx_h+=coe[6]*(result_h_d[in_idx+6*dimz]-result_h_d[in_idx-6*dimz]);
				
			float sumz_h=coe[0]*result_h_d[in_idx];
				sumz_h+=coe[1]*(result_h_d[in_idx+1]-result_h_d[in_idx-1]);
				sumz_h+=coe[2]*(result_h_d[in_idx+2]-result_h_d[in_idx-2]);
				sumz_h+=coe[3]*(result_h_d[in_idx+3]-result_h_d[in_idx-3]);
				sumz_h+=coe[4]*(result_h_d[in_idx+4]-result_h_d[in_idx-4]);
				sumz_h+=coe[5]*(result_h_d[in_idx+5]-result_h_d[in_idx-5]);
				sumz_h+=coe[6]*(result_h_d[in_idx+6]-result_h_d[in_idx-6]);
			
			float sumx_o=coe[0]*result_old_d[in_idx];
				sumx_o+=coe[1]*(result_old_d[in_idx+1*dimz]-result_old_d[in_idx-1*dimz]);
				sumx_o+=coe[2]*(result_old_d[in_idx+2*dimz]-result_old_d[in_idx-2*dimz]);
				sumx_o+=coe[3]*(result_old_d[in_idx+3*dimz]-result_old_d[in_idx-3*dimz]);
				sumx_o+=coe[4]*(result_old_d[in_idx+4*dimz]-result_old_d[in_idx-4*dimz]);
				sumx_o+=coe[5]*(result_old_d[in_idx+5*dimz]-result_old_d[in_idx-5*dimz]);
				sumx_o+=coe[6]*(result_old_d[in_idx+6*dimz]-result_old_d[in_idx-6*dimz]);
				
			float sumz_o=coe[0]*result_old_d[in_idx];
				sumz_o+=coe[1]*(result_old_d[in_idx+1]-result_old_d[in_idx-1]);
				sumz_o+=coe[2]*(result_old_d[in_idx+2]-result_old_d[in_idx-2]);
				sumz_o+=coe[3]*(result_old_d[in_idx+3]-result_old_d[in_idx-3]);
				sumz_o+=coe[4]*(result_old_d[in_idx+4]-result_old_d[in_idx-4]);
				sumz_o+=coe[5]*(result_old_d[in_idx+5]-result_old_d[in_idx-5]);
				sumz_o+=coe[6]*(result_old_d[in_idx+6]-result_old_d[in_idx-6]);
				
			/*float  sumx_h=(result_h_d[in_idx+1*dimz]-result_h_d[in_idx-1*dimz]);
			float  sumz_h=(result_h_d[in_idx+1]-result_h_d[in_idx-1]);
			float  sumx_o=(result_old_d[in_idx+1*dimz]-result_old_d[in_idx-1*dimz]);
			float  sumz_o=(result_old_d[in_idx+1]-result_old_d[in_idx-1]);*/
			
				normalx1_d[in_idx]=(sumx_o*result_h_d[in_idx]-sumx_h*result_old_d[in_idx])/(result_h_d[in_idx]*result_h_d[in_idx]+result_old_d[in_idx]*result_old_d[in_idx]);
				normalz1_d[in_idx]=(sumz_o*result_h_d[in_idx]-sumz_h*result_old_d[in_idx])/(result_h_d[in_idx]*result_h_d[in_idx]+result_old_d[in_idx]*result_old_d[in_idx]);
		}
}

__global__ void caculate_normal(float *normalx_d,float *normalz_d,float *poyn_px_d,float *poyn_pz_d,float *poyn_rpx_d,float *poyn_rpz_d,int dimx,int dimz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius;
			iz=iz+radius;
			dimx=dimx+2*radius;
			dimz=dimz+2*radius;
			in_idx=ix*dimz+iz;

			normalx_d[in_idx]=poyn_rpx_d[in_idx]-poyn_px_d[in_idx];
			normalz_d[in_idx]=poyn_rpz_d[in_idx]-poyn_pz_d[in_idx];
		}
}

__global__ void caculate_angle_open(float *angle_open_d,float *poyn_px_d,float *poyn_pz_d,float *poyn_rpx_d,float *poyn_rpz_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sum;

		if(ix<dimx&&iz<dimz)
		{
			dimx=dimx+2*radius1;dimz=dimz+2*radius1;
			ix=ix+radius1;iz=iz+radius1;
			in_idx=ix*dimz+iz;
			angle_open_d[in_idx]=0.0;
			sum=0.0;
					
			sum=(poyn_px_d[in_idx]*poyn_rpx_d[in_idx]+poyn_pz_d[in_idx]*poyn_rpz_d[in_idx]);
			
			angle_open_d[in_idx]=90*float(acos(sum))/pai;
		}
}

__global__ void caculate_angle_pp_real(float *angle_pp_d,float *normal_angle_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				
				angle_pp_d[in_idx]=angle_pp_d[in_idx]-normal_angle_d[in_idx];
		}
}


__global__ void fwd_smooth(float *input_d,int dimx,int dimz)
{
		__shared__ float s_data[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data[tz][tx]=input_d[in_idx];

				if(threadIdx.y<radius1)
				{
						s_data[threadIdx.y][tx]=input_d[in_idx-radius1];//g_input[in_idx-radius1*dimx];//up
						s_data[threadIdx.y+BDIMY1+radius1][tx]=input_d[in_idx+BDIMY1];//g_input[in_idx+BDIMY1*dimx];//down
				}
				if(threadIdx.x<radius1)
				{
						s_data[tz][threadIdx.x]=input_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
						s_data[tz][threadIdx.x+BDIMX1+radius1]=input_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				
				__syncthreads();

				input_d[in_idx]=(s_data[tz][tx]+s_data[tz+1][tx+1]+s_data[tz+1][tx-1]+s_data[tz-1][tx+1]+s_data[tz-1][tx-1])/5.0;
		}
}

__global__ void set_adcigs_imagingpp_angle(float *p_d,float *rp_d,float *p_adcigs_pp_d,float *n_adcigs_pp_d,float *imageup_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;

		if(ix<nx&&iz<nz&&iz>10)
		//if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=0.0;
				
			//if(angle_open_d[in_idx]<=60)	imageup_d[ix*nz+iz]=p_d[in_idx]*rp_d[in_idx];
			//if(poyn_z_d[in_idx]>=0&&poyn_rz_d[in_idx]<=0)	imageup_d[ix*nz+iz]=p_d[in_idx]*rp_d[in_idx];
			//if(poyn_z_d[in_idx]<=0&&poyn_rz_d[in_idx]>=0)	imageup_d[ix*nz+iz]=p_d[in_idx]*rp_d[in_idx];	
			if(angle_open_d[in_idx]<=90)	imageup_d[ix*nz+iz]=p_d[in_idx]*rp_d[in_idx]*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
///every time  only one angle need to caculate		
			set=r_angle*nx*nz+ix*nz+iz;
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) p_adcigs_pp_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) n_adcigs_pp_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		}
		
}

__global__ void set_adcigs_imagingps_angle(float *p_d,float *rs_d,float *p_adcigs_ps_d,float *n_adcigs_ps_d,float *imageup_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//angle_pp_d,p_d,rs_d,adcigs_ps_d,rimageupps_d,rimagedownps_d,angle_pp_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;

		if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=0.0;
			
			if(angle_pp_d[in_idx]>=0)
			{
				imageup_d[ix*nz+iz]=p_d[in_idx]*rs_d[in_idx];
			}
			if(angle_pp_d[in_idx]<0)
			{
				imageup_d[ix*nz+iz]=-1*p_d[in_idx]*rs_d[in_idx];
			}	
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
	
			set=r_angle*nx*nz+ix*nz+iz;
			
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) p_adcigs_ps_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) n_adcigs_ps_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		
		}
}

__global__ void set_adcigs_imagingpp_angle_new(float *vxp_d,float *vzp_d,float *rvxp_d,float *rvzp_d,float *p_adcigs_pp_d,float *n_adcigs_pp_d,float *imageup_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//(vxp1_d,vzp1_d,rvxp1_d,rvzp1_d,p_adcigs_pp_d,n_adcigs_pp_d,rimageuppp_d,rimagedownpp_d,angle_pp_d,angle_open_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		
		//float sign;

		if(ix<nx&&iz<nz&&iz>10)
		//if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=0.0;
			
			//if(vzp_d[in_idx]*rvzp_d[in_idx]<=0)	sign=-1;
			//if(vzp_d[in_idx]*rvzp_d[in_idx]>0)	sign=+1;
				
			//if(angle_open_d[in_idx]<=90)	imageup_d[ix*nz+iz]=(fabs(vxp_d[in_idx]*rvxp_d[in_idx])*sign+vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			if(angle_open_d[in_idx]<=90)	imageup_d[ix*nz+iz]=(vxp_d[in_idx]*rvxp_d[in_idx]+vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			//if(angle_open_d[in_idx]<=90)	imageup_d[ix*nz+iz]=(vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
///every time  only one angle need to caculate		
			set=r_angle*nx*nz+ix*nz+iz;
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) p_adcigs_pp_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) n_adcigs_pp_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		}
}

__global__ void set_adcigs_imagingps_angle_new(float *vxp_d,float *vzp_d,float *rvxs_d,float *rvzs_d,float *p_adcigs_ps_d,float *n_adcigs_ps_d,float *imageup_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//(vxp1_d,vzp1_d,rvxs1_d,rvzs1_d,p_adcigs_pp_d,n_adcigs_pp_d,rimageuppp_d,rimagedownpp_d,angle_pp_d,angle_open_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		//float sign;

		if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=0.0;
			
			//if(vxp_d[in_idx]*rvxs_d[in_idx]<=0)	sign=-1;
			//if(vxp_d[in_idx]*rvxs_d[in_idx]>0)	sign=+1;
			
			//imageup_d[ix*nz+iz]=vxp_d[in_idx]*rvxs_d[in_idx]+sign*fabs(vzp_d[in_idx]*rvzs_d[in_idx]);
			
			imageup_d[ix*nz+iz]=vxp_d[in_idx]*rvxs_d[in_idx]+vzp_d[in_idx]*rvzs_d[in_idx];
			
			//imageup_d[ix*nz+iz]=vxp_d[in_idx]*rvxs_d[in_idx];
				
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
	
			set=r_angle*nx*nz+ix*nz+iz;
			
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) p_adcigs_ps_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) n_adcigs_ps_d[set]+=imageup_d[ix*nz+iz]*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		
		}
}

__global__ void imagingadd_angle(float *adcigs_pp_d,float *imagedown_d,int nx,int nz,int angle_num,float average)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;
		int set;		

		if(ix<nx&&iz<nz)
		{
			for(int iangle=0;iangle<angle_num;iangle++)
				{
					set=iangle*nx*nz+ix*nz+iz;		
					adcigs_pp_d[set]=adcigs_pp_d[set]/(imagedown_d[ix*nz+iz]+average);
				}
		}
}

__global__ void imagingadd_angle_new(float *adcigs_pp_d,float *imagedown_d,int nx,int nz,int angle_num,float average)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;
		int set;		

		if(ix<nx&&iz<nz)
		{
			for(int iangle=0;iangle<2*angle_num;iangle++)
				{
					set=iangle*nx*nz+ix*nz+iz;		
					adcigs_pp_d[set]=adcigs_pp_d[set]/(imagedown_d[ix*nz+iz]+average);
				}
		}
}

__global__ void output_someangle(float *adcigs_d,float *wf_d,int angle_start,int angle_end,int nx,int nz)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		if(ix<nx&&iz<nz)
		{
			for(int iangle=angle_start;iangle<angle_end;iangle++)
			{				
				in_idx=iangle*nx*nz+ix*nz+iz;
				wf_d[ix*nz+iz]+=adcigs_d[in_idx];
			}
		}
}

__global__ void output_cdpangle(float *adcigs_d,float *adcigs_cdp_d,int cdp_location,int angle_num,int dangle,int nx,int nz)
{
		int iangle=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		if(iangle<angle_num&&iz<nz)
		{
			//in_idx=cdp_location*nz*angle_num+iangle*nz+iz;
			//adcigs_cdp_d[iangle*nz+iz]=adcigs_d[in_idx];

			in_idx=iangle*nx*nz+cdp_location*nz+iz;
			adcigs_cdp_d[iangle*nz+iz]=adcigs_d[in_idx];
		}
}

__global__ void caculate_angle_base_on_direction_least_square(float *angle_pp_d,float *poyn_px_d,float *poyn_pz_d,int dimx,int dimz,int scale)
//angle_pp_d,normal_x_d,normal_z_d,poyn_px_d,poyn_pz_d,nx_append_radius1,nz_append_radius1
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx,sumz;
		int m,n;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				angle_pp_d[in_idx]=0;
///note that  x/z
//least_square size of scale	  
				for(m=-4;m<=4;m++)
					for(n=-4;n<=4;n++)
						{
							sumx=sumx+poyn_px_d[in_idx+m+n*dimz]*poyn_pz_d[in_idx+m+n*dimz];
							sumz=sumz+poyn_pz_d[in_idx+m+n*dimz]*poyn_pz_d[in_idx+m+n*dimz];
						}
				if(sumz!=0)	angle_pp_d[in_idx]=float(atan(double(sumx*1.0/sumz)))*180/pai;			
		}
}

__global__ void caculate_angle_base_on_direction_least_square_share(float *angle_pp_d,float *poyn_px_d,float *poyn_pz_d,int dimx,int dimz,int scale)
//angle_pp_d,normal_x_d,normal_z_d,poyn_px_d,poyn_pz_d,nx_append_radius1,nz_append_radius1
{
		__shared__ float s_data1[BDIMY1+2*radius1][BDIMX1+2*radius1];
		__shared__ float s_data2[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx,sumz;
		int m,n;
		
		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				sumx=0.0;
				sumz=0.0;
				angle_pp_d[in_idx]=0;
				
				__syncthreads();

				s_data1[tz][tx]=poyn_px_d[in_idx];
				s_data2[tz][tx]=poyn_pz_d[in_idx];

				if(threadIdx.y<radius1)
				{
					s_data1[threadIdx.y][threadIdx.x]=poyn_px_d[in_idx-radius1-radius1*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius1]=poyn_px_d[in_idx-radius1+radius1*dimz];//up
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x]=poyn_px_d[in_idx+BDIMY1-radius1*dimz];//down
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=poyn_px_d[in_idx+BDIMY1+radius1*dimz];//down

					s_data2[threadIdx.y][threadIdx.x]=poyn_pz_d[in_idx-radius1-radius1*dimz];//up
					s_data2[threadIdx.y][threadIdx.x+2*radius1]=poyn_pz_d[in_idx-radius1+radius1*dimz];//up
					s_data2[threadIdx.y+BDIMY1+radius1][threadIdx.x]=poyn_pz_d[in_idx+BDIMY1-radius1*dimz];//down
					s_data2[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=poyn_pz_d[in_idx+BDIMY1+radius1*dimz];//down

				}
				if(threadIdx.x<radius1)
				{
					s_data1[tz][threadIdx.x]=poyn_px_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
					s_data1[tz][threadIdx.x+BDIMX1+radius1]=poyn_px_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				
					s_data2[tz][threadIdx.x]=poyn_pz_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
					s_data2[tz][threadIdx.x+BDIMX1+radius1]=poyn_pz_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				__syncthreads();
///note that  x/z
//least_square size of scale	  
				for(m=-4;m<=4;m++)
					for(n=-4;n<=4;n++)
						{
							sumx=sumx+s_data1[tz+m][tx+n]*s_data2[tz+m][tx+n];
							sumz=sumz+s_data2[tz+m][tx+n]*s_data2[tz+m][tx+n];
						}
				if(sumz!=0)	angle_pp_d[in_idx]=float(atan(double(sumx*1.0/sumz)))*180/pai;			
		}
}

__global__ void caculate_normal_base_on_direction_least_square(float *poyn_px_d,float *poyn_pz_d,int dimx,int dimz,int scale)
//angle_pp_d,normal_x_d,normal_z_d,poyn_px_d,poyn_pz_d,nx_append_radius1,nz_append_radius1
{
		__shared__ float s_data1[BDIMY1+2*radius1][BDIMX1+2*radius1];
		__shared__ float s_data2[BDIMY1+2*radius1][BDIMX1+2*radius1];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		double sumx,sumz;
		int m,n;
		
		int tx = threadIdx.x+radius1;
		int tz = threadIdx.y+radius1;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data1[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX1+2*radius1-1-threadIdx.x]=0.0;
		s_data2[BDIMY1+2*radius1-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
				dimx=dimx+2*radius1;dimz=dimz+2*radius1;
				ix=ix+radius1;iz=iz+radius1;
				in_idx=ix*dimz+iz;
				
				__syncthreads();

				s_data1[tz][tx]=poyn_px_d[in_idx];
				s_data2[tz][tx]=poyn_pz_d[in_idx];

				if(threadIdx.y<radius1)
				{
					s_data1[threadIdx.y][threadIdx.x]=poyn_px_d[in_idx-radius1-radius1*dimz];//up
					s_data1[threadIdx.y][threadIdx.x+2*radius1]=poyn_px_d[in_idx-radius1+radius1*dimz];//up
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x]=poyn_px_d[in_idx+BDIMY1-radius1*dimz];//down
					s_data1[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=poyn_px_d[in_idx+BDIMY1+radius1*dimz];//down

					s_data2[threadIdx.y][threadIdx.x]=poyn_pz_d[in_idx-radius1-radius1*dimz];//up
					s_data2[threadIdx.y][threadIdx.x+2*radius1]=poyn_pz_d[in_idx-radius1+radius1*dimz];//up
					s_data2[threadIdx.y+BDIMY1+radius1][threadIdx.x]=poyn_pz_d[in_idx+BDIMY1-radius1*dimz];//down
					s_data2[threadIdx.y+BDIMY1+radius1][threadIdx.x+2*radius1]=poyn_pz_d[in_idx+BDIMY1+radius1*dimz];//down

				}
				if(threadIdx.x<radius1)
				{
					s_data1[tz][threadIdx.x]=poyn_px_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
					s_data1[tz][threadIdx.x+BDIMX1+radius1]=poyn_px_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				
					s_data2[tz][threadIdx.x]=poyn_pz_d[in_idx-radius1*dimz];//g_input[in_idx-radius1];//left
					s_data2[tz][threadIdx.x+BDIMX1+radius1]=poyn_pz_d[in_idx+BDIMX1*dimz];//g_input[in_idx+BDIMX1];//right
				}
				__syncthreads();
				
				for(m=-4;m<=4;m++)
					for(n=-4;n<=4;n++)
						{
							sumx=sumx+s_data1[tz+m][tx+n];
						
							sumz=sumz+s_data2[tz+m][tx+n];
						}
				
				poyn_px_d[in_idx]=sumx;
				poyn_pz_d[in_idx]=sumz;		
		}
}



__global__ void set_adcigs_pp(float *p_d,float *rp_d,float *r_adcigs_pp_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		float image;

		if(ix<nx&&iz<nz&&iz>10)
		//if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
				
			//if(angle_open_d[in_idx]<=60)	image=p_d[in_idx]*rp_d[in_idx];	
			if(angle_open_d[in_idx]<=90)	image=p_d[in_idx]*rp_d[in_idx]*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
///every time  only one angle need to caculate		
			set=r_angle*nx*nz+ix*nz+iz;
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) r_adcigs_pp_d[angle_num*nx*nz+set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) r_adcigs_pp_d[angle_num*nx*nz-set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		}
		
}

__global__ void set_adcigs_ps(float *p_d,float *rs_d,float *r_adcigs_ps_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//angle_pp_d,p_d,rs_d,adcigs_ps_d,rimageupps_d,rimagedownps_d,angle_pp_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		float image;

		if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			if(angle_pp_d[in_idx]>=0)
			{
				image=p_d[in_idx]*rs_d[in_idx];
			}
			if(angle_pp_d[in_idx]<0)
			{
				image=-1*p_d[in_idx]*rs_d[in_idx];
			}	
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
	
			set=r_angle*nx*nz+ix*nz+iz;
			
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) r_adcigs_ps_d[angle_num*nx*nz+set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) r_adcigs_ps_d[angle_num*nx*nz-set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		
		}
}

__global__ void set_adcigs_pp_new(float *vxp_d,float *vzp_d,float *rvxp_d,float *rvzp_d,float *r_adcigs_pp_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//(vxp1_d,vzp1_d,rvxp1_d,rvzp1_d,p_adcigs_pp_d,n_adcigs_pp_d,rimageuppp_d,rimagedownpp_d,angle_pp_d,angle_open_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		
		//float sign;
		float image;

		if(ix<nx&&iz<nz&&iz>10)
		//if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			//if(vzp_d[in_idx]*rvzp_d[in_idx]<=0)	sign=-1;
			//if(vzp_d[in_idx]*rvzp_d[in_idx]>0)	sign=+1;
				
			//if(angle_open_d[in_idx]<=90)	image=(fabs(vxp_d[in_idx]*rvxp_d[in_idx])*sign+vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			if(angle_open_d[in_idx]<=90)	image=(vxp_d[in_idx]*rvxp_d[in_idx]+vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			//if(angle_open_d[in_idx]<=90)	image=(vzp_d[in_idx]*rvzp_d[in_idx])*float(float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180)))*float(cos(double(pai*angle_open_d[in_idx]/180))));
			
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
///every time  only one angle need to caculate		
			set=r_angle*nx*nz+ix*nz+iz;
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) r_adcigs_pp_d[angle_num*nx*nz+set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) r_adcigs_pp_d[angle_num*nx*nz-set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		}
}

__global__ void set_adcigs_ps_new(float *vxp_d,float *vzp_d,float *rvxs_d,float *rvzs_d,float *r_adcigs_ps_d,float *angle_pp_d,float *angle_open_d,int source_x_cord,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left,int angle_num,int dangle)
//(vxp1_d,vzp1_d,rvxs1_d,rvzs1_d,p_adcigs_pp_d,n_adcigs_pp_d,rimageuppp_d,rimagedownpp_d,angle_pp_d,angle_open_d,nx,nz,nx_append,nz_append,boundary_up,boundary_left,angle_num,dangle);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		int set;
		int r_angle;
		//float sign;
		float image;

		if(ix<nx&&iz<nz&&iz>10&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			//if(vxp_d[in_idx]*rvxs_d[in_idx]<=0)	sign=-1;
			//if(vxp_d[in_idx]*rvxs_d[in_idx]>0)	sign=+1;
			
			//image=vxp_d[in_idx]*rvxs_d[in_idx]+sign*fabs(vzp_d[in_idx]*rvzs_d[in_idx]);
			
			image=vxp_d[in_idx]*rvxs_d[in_idx]+vzp_d[in_idx]*rvzs_d[in_idx];
			
			//image=vxp_d[in_idx]*rvxs_d[in_idx];
				
			r_angle=int((fabs(angle_pp_d[in_idx])+0.5)/dangle);
	
			set=r_angle*nx*nz+ix*nz+iz;
			
			if(r_angle<angle_num&&angle_pp_d[in_idx]>=0) r_adcigs_ps_d[angle_num*nx*nz+set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
			if(r_angle<angle_num&&angle_pp_d[in_idx]<0) r_adcigs_ps_d[angle_num*nx*nz-set]+=image*exp(-(fabs(angle_pp_d[in_idx])-r_angle*dangle)*(fabs(angle_pp_d[in_idx])-r_angle*dangle)/8);
		
		}
}
