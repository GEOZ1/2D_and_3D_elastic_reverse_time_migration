
#include <hip/hip_runtime.h>
__constant__ const int BDIMX=32;
__constant__ const int BDIMY=16;
__constant__ const int radius=6;

__global__ void add_source(float *txx_d,float *tzz_d,float *wavelet_d,int source_x_cord,int shot_depth,int it,int boundary_up,int boundary_left,int nz_append)
{
		txx_d[(boundary_left+source_x_cord)*nz_append+(boundary_up+shot_depth)]+=1000000000*wavelet_d[it];
		tzz_d[(boundary_left+source_x_cord)*nz_append+(boundary_up+shot_depth)]+=1000000000*wavelet_d[it];		
}

__global__ void wraddshot(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int nz_append,int boundary_up,int boundary_left,int receiver_start,int receiver_interval,int receiver_depth,int receiver_num)
//__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
			wfr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]+=shotgather_d[ix*lt+it];
			 fr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]+=shotgather1_d[ix*lt+it];
		}
}

__global__ void wraddshot_set(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int nz_append,int boundary_up,int boundary_left,int receiver_start,int receiver_interval,int receiver_depth,int receiver_num)
//__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
			wfr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]=shotgather_d[ix*lt+it];
			 fr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]=shotgather1_d[ix*lt+it];
		}
}

__global__ void wraddshot_x_z(float *wfr_d,float *shotgather_d,int it,int lt,int nz_append,int boundary_up,int boundary_left,int receiver_x_cord,int receiver_interval,int receiver_z_cord,int receiver_z_interval,int receiver_num,int mark)
//__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		int id_x,id_z,id;

		if(ix<receiver_num)		
		{
				id_x=boundary_left+receiver_x_cord+ix*receiver_interval;
				id_z=boundary_up+receiver_z_cord+ix*receiver_z_interval;

				id=id_x*nz_append+id_z;

				if(mark==0)	wfr_d[id]=shotgather_d[ix*lt+it];
				else		wfr_d[id]+=shotgather_d[ix*lt+it];
		}	
}

__global__ void wraddshot_x_z_acqusition(float *wfr_d,float *shotgather_d,int it,int lt,int nz_append,int boundary_up,int boundary_left,int receiver_x_cord,int receiver_interval,int receiver_z_cord,int receiver_z_interval,int receiver_num,int mark)
//__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		int id_x,id_z,id;

		if(ix<receiver_num)		
		{
				id_x=boundary_left+ix*receiver_interval;
				id_z=boundary_up+receiver_z_cord+ix*receiver_z_interval;

				id=id_x*nz_append+id_z;

				if(mark==0)	wfr_d[id]=shotgather_d[ix*lt+it];
				else		wfr_d[id]+=shotgather_d[ix*lt+it];
		}	
}

__global__ void wraddshot_ls(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int nz_append,int boundary_up,int boundary_left,int receiver_start,int receiver_interval,int receiver_depth,int receiver_num)
//__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
			wfr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]+=-1*shotgather_d[ix*lt+it];
			 fr_d[(boundary_left+receiver_start+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]+=-1*shotgather1_d[ix*lt+it];
		}
}

__global__ void write_shot(float *wf_d,float *f_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_x_cord,int receiver_interval,int boundary_left,int boundary_up,int nz_append,float dx,float dt,int source_x_cord,float *velocity1_d,float wavelet_half)
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
				shotgather_d[ix*lt+it]=wf_d[(boundary_left+receiver_x_cord+ix*receiver_interval)*nz_append+boundary_up+receiver_depth];
				shotgather1_d[ix*lt+it]=f_d[(boundary_left+receiver_x_cord+ix*receiver_interval)*nz_append+boundary_up+receiver_depth];
		}	
}

__global__ void write_shot_x_z(float *wf_d,float *shotgather_d,int it,int lt,int receiver_num,int receiver_x_cord,int receiver_interval,int receiver_z_cord,int receiver_z_interval,int boundary_left,int boundary_up,int nz_append)
//write_shot_x_z<<<receiver_num,1>>>(vx2_d,obs_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
{
		int ix=blockIdx.x;

		int id_x,id_z,id;

		if(ix<receiver_num)		
		{
				id_x=boundary_left+receiver_x_cord+ix*receiver_interval;
				id_z=boundary_up+receiver_z_cord+ix*receiver_z_interval;

				id=id_x*nz_append+id_z;

				shotgather_d[ix*lt+it]=wf_d[id];
		}	
}

__global__ void write_shot_x_z_acqusition(float *wf_d,float *shotgather_d,int it,int lt,int receiver_num,int receiver_x_cord,int receiver_interval,int receiver_z_cord,int receiver_z_interval,int boundary_left,int boundary_up,int nz_append)
//write_shot_x_z<<<receiver_num,1>>>(vx2_d,obs_shot_x_d,it-wavelet_half,lt,receiver_num,receiver_x_cord[ishot],receiver_interval,receiver_z_cord[ishot],receiver_z_interval,boundary_left,boundary_up,nz_append);///for vsp 2017年03月14日 星期二 08时46分12秒 
{
		int ix=blockIdx.x;

		int id_x,id_z,id;

		if(ix<receiver_num)		
		{
				id_x=boundary_left+ix*receiver_interval;
				id_z=boundary_up+receiver_z_cord+ix*receiver_z_interval;

				id=id_x*nz_append+id_z;

				shotgather_d[ix*lt+it]=wf_d[id];
		}	
}


__global__ void wraddshot_new(float *wfr_d,float *fr_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
//(rwf1_d,rwf1_d,res_shot_d,res_shot_d,it,lt,nz_append,boundary_up,boundary_left,receiver_x_cord[ishot],receiver_interval,receiver_depth,receiver_num);
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
			wfr_d[(boundary_left+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]=shotgather_d[ix*lt+it];
			 fr_d[(boundary_left+ix*receiver_interval)*nz_append+boundary_up+receiver_depth]=shotgather1_d[ix*lt+it];
		}
}

__global__ void write_shot_new(float *wf_d,float *f_d,float *shotgather_d,float *shotgather1_d,int it,int lt,int receiver_num,int receiver_depth,int receiver_interval,int boundary_left,int boundary_up,int nz_append)
{
		int ix=blockIdx.x;

		if(ix<receiver_num)		
		{
				shotgather_d[ix*lt+it]=wf_d[(boundary_left+ix*receiver_interval)*nz_append+boundary_up+receiver_depth];
				shotgather1_d[ix*lt+it]=f_d[(boundary_left+ix*receiver_interval)*nz_append+boundary_up+receiver_depth];
		}	
}

__global__ void cut_direct(float *shotgather_d,int lt,int source_x_cord,int shot_depth,int receiver_num,int receiver_depth,int receiver_x_cord,int receiver_interval,int boundary_left,int boundary_up,int nz_append,float dx,float dz,float dt,float *velocity_d,int wavelet_half)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;

		int mark;

		float dt_real;
		dt_real=dt/1000;
		float distance;
		int time;

		if(ix<receiver_num)
		{
			
			distance=sqrt((receiver_x_cord+ix*receiver_interval-source_x_cord)*(receiver_x_cord+ix*receiver_interval-source_x_cord)*dx*dx*1.0+(shot_depth-receiver_depth)*(shot_depth-receiver_depth)*dz*dz);	
	
			time=distance*1.0/velocity_d[(ix+boundary_left)*nz_append+boundary_up]/dt_real;

			//for(mark=time;mark<time+250;mark++)
			for(mark=0;mark<time+250;mark++)
			//for(mark=0;mark<time+2*wavelet_half;mark++)
			shotgather_d[ix*lt+mark]=0;

			//for(mark=time+200;mark<=time+220;mark++)
			//shotgather_d[ix*lt+mark]=shotgather_d[ix*lt+mark]*float(exp(1.0*(mark-time-220.0)/0.10));
		}
}

__global__ void cut_direct_new1(float *shotgather_d,int lt,int source_x_cord,int shot_depth,int receiver_num,int receiver_depth,int receiver_x_cord,int receiver_interval,int boundary_left,int boundary_up,int nz_append,float dx,float dz,float dt,float *velocity_d,int wavelet_half,int cut_direct_wave)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;

		int mark;

		float dt_real;
		dt_real=dt/1000;
		float distance;
		int time;

		if(ix<receiver_num)
		{
			
			distance=sqrt((receiver_x_cord+ix*receiver_interval-source_x_cord)*(receiver_x_cord+ix*receiver_interval-source_x_cord)*dx*dx*1.0+(shot_depth-receiver_depth)*(shot_depth-receiver_depth)*dz*dz);	
	
			time=distance*1.0/velocity_d[(ix+boundary_left)*nz_append+boundary_up]/dt_real;

			//for(mark=time;mark<time+250;mark++)
			//for(mark=0;mark<time+200;mark++)
			//for(mark=0;mark<time+2*wavelet_half;mark++)
			for(mark=0;mark<time+cut_direct_wave;mark++)
			shotgather_d[ix*lt+mark]=0;

			//for(mark=time+200;mark<=time+220;mark++)
			//shotgather_d[ix*lt+mark]=shotgather_d[ix*lt+mark]*float(exp(1.0*(mark-time-220.0)/0.10));
		}
}

__global__ void cut_direct_new(float *shotgather_d,int lt,int source_x_cord,int shot_depth,int receiver_num,int receiver_depth,int receiver_x_cord,int receiver_interval,int boundary_left,int boundary_up,int nz_append,float dx,float dz,float dt,float *velocity_d,int wavelet_half)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;

		int mark;

		float dt_real;
		dt_real=dt/1000;
		float distance;
		int time;

		if(ix<receiver_num)
		{
			
			distance=sqrt((receiver_x_cord+ix*receiver_interval-source_x_cord)*(receiver_x_cord+ix*receiver_interval-source_x_cord)*dx*dx*1.0+(shot_depth-receiver_depth)*(shot_depth-receiver_depth)*dz*dz);	
	
			time=distance*1.0/velocity_d[(ix+boundary_left)*nz_append+boundary_up]/dt_real;

			//for(mark=time;mark<time+250;mark++)
			for(mark=0;mark<time+250;mark++)
				shotgather_d[ix*lt+mark]=shotgather_d[ix*lt+mark]*float(exp(-1.0*(time+250-mark)/(time+250)));
		}
}

__global__ void replace(float *c1,float *c2,float *c3,float *c4,float *c5,float *c6,float *c7,float *c8,float *c9,float *c10,int nx_append,int nz_append)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float change1;
		float change2;
		float change3;
		float change4;
		float change5;

		if((ix<nx_append)&&(iz<nz_append))
		{
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*(nz_append+2*radius)+iz;
				change1   =c1[in_idx];
				c1[in_idx]=c2[in_idx];
				c2[in_idx]=    change1;

				change2   =c3[in_idx];
				c3[in_idx]=c4[in_idx];
				c4[in_idx]=    change2;

				change3   =c5[in_idx];
				c5[in_idx]=c6[in_idx];
				c6[in_idx]=    change3;

				change4   =c7[in_idx];
				c7[in_idx]=c8[in_idx];
				c8[in_idx]=    change4;

				change5   =c9[in_idx];
				c9[in_idx]=c10[in_idx];
				c10[in_idx]=    change5;			
		}
}

__global__ void replace_2wf(float *c1,float *c2,int nx_append,int nz_append)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;
		float change1;
		
		if((ix<nx_append)&&(iz<nz_append))
		{
				ix=ix+radius;iz=iz+radius;
				in_idx=ix*(nz_append+2*radius)+iz;
				change1   =c1[in_idx];
				c1[in_idx]=c2[in_idx];
				c2[in_idx]=    change1;			
		}
}


__global__ void fwd_vxp_vzp_vxs_vzs(float *vxp2_d,float *vxp1_d,float *vzp2_d,float *vzp1_d,float *vxs2_d,float *vxs1_d,float *vzs2_d,float *vzs1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//vxp2_d,vxp1_d,vzp2_d,vzp1_d,vxs2_d,vxs1_d,vzs2_d,vzs1_d,txx1_d,tzz1_d,txz1_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data3[BDIMY+2*radius][BDIMX+2*radius];
		
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;
		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data3[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1_d[in_idx];
				s_data2[tz][tx]=tzz1_d[in_idx];
				s_data3[tz][tx]=txz1_d[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=tzz1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=tzz1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data3[threadIdx.y][tx]=txz1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data3[threadIdx.y+BDIMY+radius][tx]=txz1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=tzz1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=tzz1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data3[tz][threadIdx.x]=txz1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data3[tz][threadIdx.x+BDIMX+radius]=txz1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
			
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

//sumx1:the derivation of x direction of txx
				float    sumx1=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);
//sumx2:the derivation of x direction of tzz
				float    sumx2=coe_d[1]*(s_data2[tz][tx]  -s_data2[tz][tx-1]);
					sumx2+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx2+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx2+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx2+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx2+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);
//sumx3:the derivation of x direction of txz
				float    sumx3=coe_d[1]*(s_data3[tz][tx+1]-s_data3[tz][tx]);
					sumx3+=coe_d[2]*(s_data3[tz][tx+2]-s_data3[tz][tx-1]);
					sumx3+=coe_d[3]*(s_data3[tz][tx+3]-s_data3[tz][tx-2]);
					sumx3+=coe_d[4]*(s_data3[tz][tx+4]-s_data3[tz][tx-3]);
					sumx3+=coe_d[5]*(s_data3[tz][tx+5]-s_data3[tz][tx-4]);
					sumx3+=coe_d[6]*(s_data3[tz][tx+6]-s_data3[tz][tx-5]);
//sumz1:the derivation of z direction of txx				
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);
//sumz2:the derivation of z direction of tzz			
				float    sumz2=coe_d[1]*(s_data2[tz+1][tx]-s_data2[tz][tx]);
					sumz2+=coe_d[2]*(s_data2[tz+2][tx]-s_data2[tz-1][tx]);
					sumz2+=coe_d[3]*(s_data2[tz+3][tx]-s_data2[tz-2][tx]);
					sumz2+=coe_d[4]*(s_data2[tz+4][tx]-s_data2[tz-3][tx]);
					sumz2+=coe_d[5]*(s_data2[tz+5][tx]-s_data2[tz-4][tx]);
					sumz2+=coe_d[6]*(s_data2[tz+6][tx]-s_data2[tz-5][tx]);
//sumz3:the derivation of z direction of txz
				float    sumz3=coe_d[1]*(s_data3[tz][tx]-s_data3[tz-1][tx]);  ////s_data2..... is  a   fault  
					sumz3+=coe_d[2]*(s_data3[tz+1][tx]-s_data3[tz-2][tx]);
					sumz3+=coe_d[3]*(s_data3[tz+2][tx]-s_data3[tz-3][tx]);
					sumz3+=coe_d[4]*(s_data3[tz+3][tx]-s_data3[tz-4][tx]);
					sumz3+=coe_d[5]*(s_data3[tz+4][tx]-s_data3[tz-5][tx]);
					sumz3+=coe_d[6]*(s_data3[tz+5][tx]-s_data3[tz-6][tx]);

					      
				vxp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxp1_d[in_idx]+
(1.0/density_d[in_idx])*(s_velocity/(2*s_velocity-2*s_velocity1))*(sumx1*coe_x+sumx2*coe_x));

				vzp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzp1_d[in_idx]+
(1.0/density_d[in_idx])*(s_velocity/(2*s_velocity-2*s_velocity1))*(sumz1*coe_z+sumz2*coe_z));
				
				vxs2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxs1_d[in_idx]+
(1.0/density_d[in_idx])*(sumz3*coe_z-(s_velocity)/(2*s_velocity-2*s_velocity1)*sumx2*coe_x+(s_velocity-2*s_velocity1)/(2*s_velocity-2*s_velocity1)*sumx1*coe_x));

				vzs2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzs1_d[in_idx]+
(1.0/density_d[in_idx])*(sumx3*coe_x-(s_velocity)/(2*s_velocity-2*s_velocity1)*sumz1*coe_z+(s_velocity-2*s_velocity1)/(2*s_velocity-2*s_velocity1)*sumz2*coe_z));
		}
}

__global__ void fwd_sum_vx_vz(float *vx2_d,float *vz2_d,float *vxp2_d,float *vzp2_d,float *vxs2_d,float *vzs2_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius;
			iz=iz+radius;
			dimx=dimx+2*radius;
			dimz=dimz+2*radius;			
			in_idx=ix*dimz+iz;

			vx2_d[in_idx]=vxs2_d[in_idx]+vxp2_d[in_idx];
			vz2_d[in_idx]=vzs2_d[in_idx]+vzp2_d[in_idx];			
		}

}

__global__ void fwd_txxzzxz(float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void fwd_txxzzxz_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float dx,float dz,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx1_d,vz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				/////vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				////vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;

				vz_x_d[in_idx]=sumx1*1.0/dx;

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}


__global__ void rfwd_vxp_vzp_vxs_vzs(float *vxp2_d,float *vxp1_d,float *vzp2_d,float *vzp1_d,float *vxs2_d,float *vxs1_d,float *vzs2_d,float *vzs1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//vxp2_d,vxp1_d,vzp2_d,vzp1_d,vxs2_d,vxs1_d,vzs2_d,vzs1_d,txx1_d,tzz1_d,txz1_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,density_d
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data3[BDIMY+2*radius][BDIMX+2*radius];
		
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;
		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data3[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1_d[in_idx];
				s_data2[tz][tx]=tzz1_d[in_idx];
				s_data3[tz][tx]=txz1_d[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=tzz1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=tzz1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data3[threadIdx.y][tx]=txz1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data3[threadIdx.y+BDIMY+radius][tx]=txz1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=tzz1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=tzz1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data3[tz][threadIdx.x]=txz1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data3[tz][threadIdx.x+BDIMX+radius]=txz1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
			
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

//sumx1:the derivation of x direction of txx
				float    sumx1=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);
//sumx2:the derivation of x direction of tzz
				float    sumx2=coe_d[1]*(s_data2[tz][tx]  -s_data2[tz][tx-1]);
					sumx2+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx2+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx2+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx2+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx2+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);
//sumx3:the derivation of x direction of txz
				float    sumx3=coe_d[1]*(s_data3[tz][tx+1]-s_data3[tz][tx]);
					sumx3+=coe_d[2]*(s_data3[tz][tx+2]-s_data3[tz][tx-1]);
					sumx3+=coe_d[3]*(s_data3[tz][tx+3]-s_data3[tz][tx-2]);
					sumx3+=coe_d[4]*(s_data3[tz][tx+4]-s_data3[tz][tx-3]);
					sumx3+=coe_d[5]*(s_data3[tz][tx+5]-s_data3[tz][tx-4]);
					sumx3+=coe_d[6]*(s_data3[tz][tx+6]-s_data3[tz][tx-5]);
//sumz1:the derivation of z direction of txx				
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);
//sumz2:the derivation of z direction of tzz			
				float    sumz2=coe_d[1]*(s_data2[tz+1][tx]-s_data2[tz][tx]);
					sumz2+=coe_d[2]*(s_data2[tz+2][tx]-s_data2[tz-1][tx]);
					sumz2+=coe_d[3]*(s_data2[tz+3][tx]-s_data2[tz-2][tx]);
					sumz2+=coe_d[4]*(s_data2[tz+4][tx]-s_data2[tz-3][tx]);
					sumz2+=coe_d[5]*(s_data2[tz+5][tx]-s_data2[tz-4][tx]);
					sumz2+=coe_d[6]*(s_data2[tz+6][tx]-s_data2[tz-5][tx]);
//sumz3:the derivation of z direction of txz
				float    sumz3=coe_d[1]*(s_data3[tz][tx]-s_data3[tz-1][tx]);  ////s_data2..... is  a   fault  
					sumz3+=coe_d[2]*(s_data3[tz+1][tx]-s_data3[tz-2][tx]);
					sumz3+=coe_d[3]*(s_data3[tz+2][tx]-s_data3[tz-3][tx]);
					sumz3+=coe_d[4]*(s_data3[tz+3][tx]-s_data3[tz-4][tx]);
					sumz3+=coe_d[5]*(s_data3[tz+4][tx]-s_data3[tz-5][tx]);
					sumz3+=coe_d[6]*(s_data3[tz+5][tx]-s_data3[tz-6][tx]);

					      
				vxp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxp1_d[in_idx]-
(1.0/density_d[in_idx])*(s_velocity/(2*s_velocity-2*s_velocity1))*(sumx1*coe_x+sumx2*coe_x));

				vzp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzp1_d[in_idx]-
(1.0/density_d[in_idx])*(s_velocity/(2*s_velocity-2*s_velocity1))*(sumz1*coe_z+sumz2*coe_z));
				
				vxs2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxs1_d[in_idx]-
(1.0/density_d[in_idx])*(sumz3*coe_z-(s_velocity)/(2*s_velocity-2*s_velocity1)*sumx2*coe_x+(s_velocity-2*s_velocity1)/(2*s_velocity-2*s_velocity1)*sumx1*coe_x));

				vzs2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzs1_d[in_idx]-
(1.0/density_d[in_idx])*(sumx3*coe_x-(s_velocity)/(2*s_velocity-2*s_velocity1)*sumz1*coe_z+(s_velocity-2*s_velocity1)/(2*s_velocity-2*s_velocity1)*sumz2*coe_z));
		}
}


__global__ void rfwd_txxzzxz(float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-
						s_velocity*density_d[in_idx]*sumx*coe_x-(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void rfwd_txxzzxz_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float dx,float dz,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//rfwd_txxzzxz_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,txx1_d,txx2_d,tzz1_d,tzz2_d,txz1_d,txz2_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-
						s_velocity*density_d[in_idx]*sumx*coe_x-(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);
					
				/////vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				////vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;

				vz_x_d[in_idx]=sumx1*1.0/dx;
			

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void fwd_vx(float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		float density;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
		
				density=density_d[in_idx];			
		
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void fwd_vx_new(float* vx_t_d,float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vx_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;
		}
}

__global__ void fwd_vx_new_new(float *d_illum,float* vx_t_d,float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vx_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;

				d_illum[in_idx]+=vx2[in_idx]*vx2[in_idx];
		}
}

__global__ void fwd_vz(float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
			
		float density;

		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=tzz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=tzz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				density=density_d[in_idx];

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void fwd_vz_new(float *vz_t_d,float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=tzz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=tzz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vz_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;
		}
}

__global__ void fwd_vz_new_new(float *d_illum,float *vz_t_d,float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=tzz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=tzz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vz_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;

				d_illum[in_idx]+=vz2[in_idx]*vz2[in_idx];
		}
}

__global__ void fwd_vxp_vzp(float *vxp2_d,float *vxp1_d,float *vzp2_d,float *vzp1_d,float *tp1_d,float coe_x,float coe_z,float dx,float dz,float dt,float *attenuation,float *coe_d,int dimx,int dimz,float *density_d)
//fwd_vxp_vzp<<<dimGrid,dimBlock>>>(rvxp1_d,rvxp2_d,rvzp1_d,rvzp2_d,rtp1_d,coe_x,coe_z,dx,dz,dt,attenuation_d,coe_opt_d,nx_append_radius,nz_append_radius,s_density_d);	
{
		__shared__ float s_data[BDIMY+2*radius][BDIMX+2*radius];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float dt_real=dt/1000;

		float s_attenuation;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;

		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius,iz=iz+radius;
			dimx=dimx+2*radius;dimz=dimz+2*radius;	
			in_idx=ix*dimz+iz;
			__syncthreads();

			s_data[tz][tx]=tp1_d[in_idx];
				
			if(threadIdx.y<radius)
			{
				s_data[threadIdx.y][tx]=tp1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
				s_data[threadIdx.y+BDIMY+radius][tx]=tp1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
			}
			if(threadIdx.x<radius)
			{
				s_data[tz][threadIdx.x]=tp1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
				s_data[tz][threadIdx.x+BDIMX+radius]=tp1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
			}
			
			s_attenuation=attenuation[in_idx];
			__syncthreads();


			float    	sumx= coe_d[1]*(s_data[tz][tx]-s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+1]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+2]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+3]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+4]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+5]-s_data[tz][tx-6]);

			float    	sumz=coe_d[1]* (s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);


			/*float    	sumx=coe_d[1]*(s_data[tz][tx+1]- s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

			float    	sumz=coe_d[1]*(s_data[tz][tx]-  s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+1][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+2][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+3][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+4][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+5][tx]-s_data[tz-6][tx]);*/


			//vxp2_d[in_idx]=vxp1_d[in_idx]+(1.0/density_d[in_idx])*sumx*coe_x;
			vxp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxp1_d[in_idx]+(1.0/density_d[in_idx])*sumx*coe_x);

			vzp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzp1_d[in_idx]+(1.0/density_d[in_idx])*sumz*coe_z);
			//vzp2_d[in_idx]=vzp1_d[in_idx]+(1.0/density_d[in_idx])*sumz*coe_z;
		}
}

__global__ void vp_vs(float *vx2_d,float *vz2_d,float *vxp2_d,float *vzp2_d,float *vxs2_d,float *vzs2_d,int dimx,int dimz)
//vp_vs<<<dimGrid,dimBlock>>>(rvx1_d,rvz1_d,rvxp1_d,rvzp1_d,rvxs1_d,rvzs1_d,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius;
			iz=iz+radius;
			dimx=dimx+2*radius;
			dimz=dimz+2*radius;			
			in_idx=ix*dimz+iz;

			vxs2_d[in_idx]=vx2_d[in_idx]-vxp2_d[in_idx];

			vzs2_d[in_idx]=vz2_d[in_idx]-vzp2_d[in_idx];			
		}
}

__global__ void fwd_txxzzxzpp(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;	

		float density;

		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				density=density_d[in_idx];

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density*sumx*coe_x+(s_velocity-2*s_velocity1)*density*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density*sumx*coe_x+s_velocity*density*sumz*coe_z);//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+s_velocity*density*sumx*coe_x+s_velocity*density*sumz*coe_z);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void fwd_txxzzxzpp_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float dx,float dz,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//fwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp2_d,tp1_d,txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}
				
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
				
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+s_velocity*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z);	
					
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				//vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				//vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;/////no 

				vz_x_d[in_idx]=sumx1*1.0/dx;/////no 

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void rfwd_vx(float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-  s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]-1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void rfwd_vx_new(float *d_illum,float *d_illum_t,float *vx_t_d,float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=txx1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=txx1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-  s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]-1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vx_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;

				d_illum_t[in_idx]=d_illum_t[in_idx]+vx_t_d[in_idx]*vx_t_d[in_idx];

				d_illum[in_idx]=d_illum[in_idx]+vx2[in_idx]*vx2[in_idx];
		}
}

__global__ void rfwd_vz(float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=tzz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=tzz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]-1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void rfwd_vz_new(float *d_illum,float *d_illum_t,float *vz_t_d,float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=tzz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=txz1[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=tzz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=txz1[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]-1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));

				vz_t_d[in_idx]=1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)/dt_real;

				d_illum_t[in_idx]=d_illum_t[in_idx]+vz_t_d[in_idx]*vz_t_d[in_idx];

				d_illum[in_idx]=d_illum[in_idx]+vz2[in_idx]*vz2[in_idx];
		}
}

__global__ void rfwd_vxp_vzp(float *vxp2_d,float *vxp1_d,float *vzp2_d,float *vzp1_d,float *tp1_d,float coe_x,float coe_z,float dx,float dz,float dt,float *attenuation,float *coe_d,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data[BDIMY+2*radius][BDIMX+2*radius];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float dt_real=dt/1000;

		float s_attenuation;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;

		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius,iz=iz+radius;
			dimx=dimx+2*radius;dimz=dimz+2*radius;	
			in_idx=ix*dimz+iz;
			__syncthreads();

			s_data[tz][tx]=tp1_d[in_idx];
				
			if(threadIdx.y<radius)
			{
				s_data[threadIdx.y][tx]=tp1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
				s_data[threadIdx.y+BDIMY+radius][tx]=tp1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
			}
			if(threadIdx.x<radius)
			{
				s_data[tz][threadIdx.x]=tp1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
				s_data[tz][threadIdx.x+BDIMX+radius]=tp1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
			}
			
			s_attenuation=attenuation[in_idx];
			__syncthreads();


			float    	sumx= coe_d[1]*(s_data[tz][tx]-	 s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+1]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+2]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+3]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+4]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+5]-s_data[tz][tx-6]);

			float    	sumz=coe_d[1]* (s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);


			/*float    	sumx=coe_d[1]*(s_data[tz][tx+1]- s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

			float    	sumz=coe_d[1]*(s_data[tz][tx]-  s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+1][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+2][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+3][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+4][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+5][tx]-s_data[tz-6][tx]);*/


			//vxp2_d[in_idx]=vxp1_d[in_idx]+(1.0/density_d[in_idx])*sumx*coe_x;
			vxp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxp1_d[in_idx]-(1.0/density_d[in_idx])*sumx*coe_x);

			vzp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzp1_d[in_idx]-(1.0/density_d[in_idx])*sumz*coe_z);
			//vzp2_d[in_idx]=vzp1_d[in_idx]+(1.0/density_d[in_idx])*sumz*coe_z;
		}
}

__global__ void rfwd_vxp_vzp_new(float *vxp_t_d,float *vzp_t_d,float *vxs_t_d,float *vzs_t_d,float *vx_t_d,float *vz_t_d,float *vxp2_d,float *vxp1_d,float *vzp2_d,float *vzp1_d,float *tp1_d,float coe_x,float coe_z,float dx,float dz,float dt,float *attenuation,float *coe_d,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data[BDIMY+2*radius][BDIMX+2*radius];

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float dt_real=dt/1000;

		float s_attenuation;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;

		s_data[tz][tx]=0.0;
		s_data[threadIdx.y][threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<dimx&&iz<dimz)
		{
			ix=ix+radius,iz=iz+radius;
			dimx=dimx+2*radius;dimz=dimz+2*radius;	
			in_idx=ix*dimz+iz;
			__syncthreads();

			s_data[tz][tx]=tp1_d[in_idx];
				
			if(threadIdx.y<radius)
			{
				s_data[threadIdx.y][tx]=tp1_d[in_idx-radius];//g_input[in_idx-radius*dimx];//up
				s_data[threadIdx.y+BDIMY+radius][tx]=tp1_d[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
			}
			if(threadIdx.x<radius)
			{
				s_data[tz][threadIdx.x]=tp1_d[in_idx-radius*dimz];//g_input[in_idx-radius];//left
				s_data[tz][threadIdx.x+BDIMX+radius]=tp1_d[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
			}
			
			s_attenuation=attenuation[in_idx];
			__syncthreads();


			float    	sumx= coe_d[1]*(s_data[tz][tx]-	 s_data[tz][tx-1]);
					sumx+=coe_d[2]*(s_data[tz][tx+1]-s_data[tz][tx-2]);
					sumx+=coe_d[3]*(s_data[tz][tx+2]-s_data[tz][tx-3]);
					sumx+=coe_d[4]*(s_data[tz][tx+3]-s_data[tz][tx-4]);
					sumx+=coe_d[5]*(s_data[tz][tx+4]-s_data[tz][tx-5]);
					sumx+=coe_d[6]*(s_data[tz][tx+5]-s_data[tz][tx-6]);

			float    	sumz=coe_d[1]* (s_data[tz+1][tx]-s_data[tz][tx]);
					sumz+=coe_d[2]*(s_data[tz+2][tx]-s_data[tz-1][tx]);
					sumz+=coe_d[3]*(s_data[tz+3][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[4]*(s_data[tz+4][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[5]*(s_data[tz+5][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[6]*(s_data[tz+6][tx]-s_data[tz-5][tx]);


			/*float    	sumx=coe_d[1]*(s_data[tz][tx+1]- s_data[tz][tx]);
					sumx+=coe_d[2]*(s_data[tz][tx+2]-s_data[tz][tx-1]);
					sumx+=coe_d[3]*(s_data[tz][tx+3]-s_data[tz][tx-2]);
					sumx+=coe_d[4]*(s_data[tz][tx+4]-s_data[tz][tx-3]);
					sumx+=coe_d[5]*(s_data[tz][tx+5]-s_data[tz][tx-4]);
					sumx+=coe_d[6]*(s_data[tz][tx+6]-s_data[tz][tx-5]);

			float    	sumz=coe_d[1]*(s_data[tz][tx]-  s_data[tz-1][tx]);
					sumz+=coe_d[2]*(s_data[tz+1][tx]-s_data[tz-2][tx]);
					sumz+=coe_d[3]*(s_data[tz+2][tx]-s_data[tz-3][tx]);
					sumz+=coe_d[4]*(s_data[tz+3][tx]-s_data[tz-4][tx]);
					sumz+=coe_d[5]*(s_data[tz+4][tx]-s_data[tz-5][tx]);
					sumz+=coe_d[6]*(s_data[tz+5][tx]-s_data[tz-6][tx]);*/


			//vxp2_d[in_idx]=vxp1_d[in_idx]+(1.0/density_d[in_idx])*sumx*coe_x;
			vxp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vxp1_d[in_idx]-(1.0/density_d[in_idx])*sumx*coe_x);

			vzp2_d[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*vzp1_d[in_idx]-(1.0/density_d[in_idx])*sumz*coe_z);
			//vzp2_d[in_idx]=vzp1_d[in_idx]+(1.0/density_d[in_idx])*sumz*coe_z;

			vxp_t_d[in_idx]=1.0/density_d[in_idx]*sumx*coe_x;

			vzp_t_d[in_idx]=1.0/density_d[in_idx]*sumz*coe_z;

			vxs_t_d[in_idx]=vx_t_d[in_idx]-vxp_t_d[in_idx];

			vzs_t_d[in_idx]=vz_t_d[in_idx]-vzp_t_d[in_idx];
		}
}

__global__ void rfwd_txxzzxzpp(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
//txx2_d,txx1_d,tzz2_d,tzz1_d,txz2_d,txz1_d,vx2_d,vz2_d,velocity_d,velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				
				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-
						s_velocity*density_d[in_idx]*sumx*coe_x-(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
					
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]-s_velocity*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);				
	
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void rfwd_txxzzxzpp_new(float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float dx,float dz,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,int nx,int nz,int boundary_left,int boundary_up,float *density_d)
//rfwd_txxzzxzpp_new<<<dimGrid,dimBlock>>>(vx_x_d,vx_z_d,vz_x_d,vz_z_d,tp1_d,tp2_d,txx1_d,txx2_d,tzz1_d,tzz2_d,txz1_d,txz2_d,vx2_d,vz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,dx,dz,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,nx,nz,boundary_left,boundary_up,s_density_d);
{
		__shared__ float s_data1[BDIMY+2*radius][BDIMX+2*radius];
		__shared__ float s_data2[BDIMY+2*radius][BDIMX+2*radius];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		
		//float s_velocity3;
		//float s_velocity4;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius;
		int tz = threadIdx.y+radius;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data1[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX+2*radius-1-threadIdx.x]=0.0;
		s_data2[BDIMY+2*radius-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius;dimz=dimz+2*radius;
				ix=ix+radius;iz=iz+radius;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data1[threadIdx.y+BDIMY+radius][tx]=vx2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius];//g_input[in_idx-radius*dimx];//up
						s_data2[threadIdx.y+BDIMY+radius][tx]=vz2[in_idx+BDIMY];//g_input[in_idx+BDIMY*dimx];//down
				}
				if(threadIdx.x<radius)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data1[tz][threadIdx.x+BDIMX+radius]=vx2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius*dimz];//g_input[in_idx-radius];//left
						s_data2[tz][threadIdx.x+BDIMX+radius]=vz2[in_idx+BDIMX*dimz];//g_input[in_idx+BDIMX];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				
				//s_velocity=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+dimz]*velocity_d[in_idx+dimz])/2.0;
				//s_velocity1=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+dimz]*velocity1_d[in_idx+dimz])/2.0;
				
				//s_velocity3=(velocity_d[in_idx]*velocity_d[in_idx]+velocity_d[in_idx+1]*velocity_d[in_idx+1])/2.0;
				//s_velocity4=(velocity1_d[in_idx]*velocity1_d[in_idx]+velocity1_d[in_idx+1]*velocity1_d[in_idx+1])/2.0;
				
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx_x_d[in_idx]=sumx*1.0/dx;

				vz_z_d[in_idx]=sumz*1.0/dz;


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]-
						s_velocity*density_d[in_idx]*sumx*coe_x-(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]-
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);//sumx  and  sumz 
					
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]-s_velocity*density_d[in_idx]*sumx*coe_x-s_velocity*density_d[in_idx]*sumz*coe_z);				
	
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				/////vx_z_d[in_idx]=sumx1*1.0/dz;//This is a fault, which leads to the distortion of the graident of the vs

				////vz_x_d[in_idx]=sumz1*1.0/dx;//This is a fault, which leads to the distortion of the graident of the vs

				vx_z_d[in_idx]=sumz1*1.0/dz;

				vz_x_d[in_idx]=sumx1*1.0/dx;

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]-s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z));
		}
}

__global__ void save_wfud(float *wf2_d,float *wfu_d,float *wfd_d,float *f2_d,float *fu_d,float *fd_d,int it,int lt,int nz,int nx_append,int nz_append,int boundary_up,int mark)
{
		int ix=blockIdx.x;

		wfu_d[mark*lt*nx_append+ix*lt+it]=wf2_d[ix*nz_append+boundary_up-1-mark];
		 fu_d[mark*lt*nx_append+ix*lt+it]= f2_d[ix*nz_append+boundary_up-1-mark];
		//wfu_d[mark*lt*nx_append+ix*lt+it]=0.0;
		// fu_d[mark*lt*nx_append+ix*lt+it]=0.0;
		
		wfd_d[mark*lt*nx_append+ix*lt+it]=wf2_d[ix*nz_append+boundary_up+nz+mark];
		 fd_d[mark*lt*nx_append+ix*lt+it]= f2_d[ix*nz_append+boundary_up+nz+mark];
}
//
__global__ void save_wflr(float *wf2_d,float *wfl_d,float *wfr_d,float *f2_d,float *fl_d,float *fr_d,int it,int lt,int nx_append,int nz_append,int boundary_left,int boundary_right,int mark)
{
		int iz=blockIdx.x;

		wfl_d[mark*lt*nz_append+iz*lt+it]=wf2_d[(boundary_left-mark-1)*nz_append+iz];
		 fl_d[mark*lt*nz_append+iz*lt+it]= f2_d[(boundary_left-mark-1)*nz_append+iz];

		wfr_d[mark*lt*nz_append+iz*lt+it]=wf2_d[(nx_append-boundary_right+mark)*nz_append+iz];
		 fr_d[mark*lt*nz_append+iz*lt+it]= f2_d[(nx_append-boundary_right+mark)*nz_append+iz];
}	

//
__global__ void set_wfud(float *wf2_d,float *wfu_d,float *wfd_d,float *f2_d,float *fu_d,float *fd_d,int it,int lt,int nz,int nx_append,int nz_append,int boundary_up,int mark)
{
		int ix=blockIdx.x;

		wf2_d[ix*nz_append+boundary_up-1-mark]=wfu_d[mark*lt*nx_append+ix*lt+it];
		 f2_d[ix*nz_append+boundary_up-1-mark]= fu_d[mark*lt*nx_append+ix*lt+it];

		wf2_d[ix*nz_append+boundary_up+nz+mark]=wfd_d[mark*lt*nx_append+ix*lt+it];
		 f2_d[ix*nz_append+boundary_up+nz+mark]= fd_d[mark*lt*nx_append+ix*lt+it];
}
//
__global__ void set_wflr(float *wf2_d,float *wfl_d,float *wfr_d,float *f2_d,float *fl_d,float *fr_d,int it,int lt,int nx_append,int nz_append,int boundary_left,int boundary_right,int mark)
{
		int iz=blockIdx.x;

		wf2_d[(boundary_left-mark-1)*nz_append+iz]=wfl_d[mark*nz_append*lt+iz*lt+it];
		 f2_d[(boundary_left-mark-1)*nz_append+iz]= fl_d[mark*nz_append*lt+iz*lt+it];

		wf2_d[(nx_append-boundary_right+mark)*nz_append+iz]=wfr_d[mark*nz_append*lt+iz*lt+it];
		 f2_d[(nx_append-boundary_right+mark)*nz_append+iz]= fr_d[mark*nz_append*lt+iz*lt+it];
}

__global__ void imaging_down_vector(float *vxp1_d,float *vzp1_d,float *image_down_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			image_down_d[ix*nz+iz]=image_down_d[ix*nz+iz]+vxp1_d[in_idx]*vxp1_d[in_idx]+vzp1_d[in_idx]*vzp1_d[in_idx];
		}
}

__global__ void imaging_down_correlation(float *p_d,float *image_down_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			image_down_d[ix*nz+iz]=image_down_d[ix*nz+iz]+p_d[in_idx]*p_d[in_idx];
		}
}

__global__ void imaging_vector_correlation(float *px1_d,float *pz1_d,float *rpx1_d,float *rpz1_d,float *imageup_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
//imaging_vector<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxp2_d,rvzp2_d,rimageup9_d,rimagedown9_d,nx,nz,nz_append,boundary_up,boundary_left);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]+px1_d[in_idx]*rpx1_d[in_idx]+pz1_d[in_idx]*rpz1_d[in_idx];
		}
}

__global__ void imaging_vector_correlation_new(float *px1_d,float *pz1_d,float *rpx1_d,float *rpz1_d,float *imageup_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
//imaging_vector_correlation_new<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxp1_d,rvzp1_d,vresultpp_d,nx_size,nz,nz_append,boundary_up,boundary_left);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		float change_pz,change_rpz;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;

			change_pz=1.0*(pz1_d[in_idx+1]+pz1_d[in_idx-1]+pz1_d[in_idx+nz_append]+pz1_d[in_idx-nz_append])/4.0;

			change_rpz=1.0*(rpz1_d[in_idx+1]+rpz1_d[in_idx-1]+rpz1_d[in_idx+nz_append]+rpz1_d[in_idx-nz_append])/4.0;
			
			imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]+px1_d[in_idx]*rpx1_d[in_idx]+change_pz*change_rpz;
		}
}

__global__ void imaging_correlation(float *p_d,float *rp_d,float *image_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			image_d[ix*nz+iz]=image_d[ix*nz+iz]+p_d[in_idx]*rp_d[in_idx];
		}
}

__global__ void imaging_correlation_source_x_cord(float *p_d,float *rp_d,float *image_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left,int source_x_cord)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			if(ix<source_x_cord)		image_d[ix*nz+iz]=image_d[ix*nz+iz]+p_d[in_idx]*rp_d[in_idx];

			else				image_d[ix*nz+iz]=image_d[ix*nz+iz]-p_d[in_idx]*rp_d[in_idx];
			
		}
}

__global__ void imaging_correlation_sign(float *p_d,float *rs_d,float *image_d,float *sign_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			if(sign_d[in_idx]>=0)	image_d[ix*nz+iz]=image_d[ix*nz+iz]+p_d[in_idx]*rs_d[in_idx];
			if(sign_d[in_idx]<0)	image_d[ix*nz+iz]=image_d[ix*nz+iz]-1*p_d[in_idx]*rs_d[in_idx];
		}
}

__global__ void imaging_correlation_sign_ps(float *p_d,float *rs_d,float *image_d,float *sign_d,int source_x_cord,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			if(sign_d[in_idx]>=0)	image_d[ix*nz+iz]=image_d[ix*nz+iz]+p_d[in_idx]*rs_d[in_idx];
			if(sign_d[in_idx]<0)	image_d[ix*nz+iz]=image_d[ix*nz+iz]-1*p_d[in_idx]*rs_d[in_idx];
		}
}

__global__ void imaging_vector_correlation_ps(float *px1_d,float *pz1_d,float *rpx1_d,float *rpz1_d,float *imageup_d,int source_x_cord,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
//imaging_vector<<<dimGrid,dimBlock>>>(vxp1_d,vzp1_d,rvxp2_d,rvzp2_d,rimageup9_d,rimagedown9_d,nx,nz,nz_append,boundary_up,boundary_left);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]+px1_d[in_idx]*rpx1_d[in_idx]+pz1_d[in_idx]*rpz1_d[in_idx];
		}
}

__global__ void imaging_correlation_ps(float *p_d,float *rp_d,float *image_d,int source_x_cord,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz&&iz>0.8*(ix-source_x_cord)&&iz>0.8*(source_x_cord-ix))
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			
			image_d[ix*nz+iz]=image_d[ix*nz+iz]+p_d[in_idx]*rp_d[in_idx];
		}
}

__global__ void imagingadd(float *imageup_d,float *imagedown_d,int nx,int nz,float *max_d,float average)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;
		
		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=ix*nz+iz;
			
			//imageup_d[in_idx]=imageup_d[in_idx]/(imagedown_d[in_idx]);
			
			imageup_d[in_idx]=imageup_d[in_idx]/(imagedown_d[in_idx]+average);

			//imageup_d[in_idx]=imageup_d[in_idx]/(0.5*max_d[iz]);

			//imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]/(imagedown_d[ix*nz+iz]+damping*max_d[iz]);
			//imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]/(imagedown_d[ix*nz+iz]+0.001);
						
			//imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]/(imagedown_d[ix*nz+iz]+0.5*max_d[iz]);
					
			//imageup_d[ix*nz+iz]=imageup_d[ix*nz+iz]/(imagedown_d[ix*nz+iz]+damping*global_max);
										
			
		}
}

__global__ void imaging_dot_product(float *p_x_d,float *p_z_d,float *rs_x_d,float *rs_z_d,float *imageup_d,float *imagedown_d,int nx,int nz,int nx_append,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		int in_idx;

		if(ix<nx&&iz<nz)
		{
			in_idx=(boundary_left+ix)*nz_append+iz+boundary_up;
			imageup_d[ix*nz+iz]+=p_x_d[in_idx]*rs_x_d[in_idx]+p_z_d[in_idx]*rs_z_d[in_idx];

			imagedown_d[ix*nz+iz]+=p_x_d[in_idx]*p_x_d[in_idx]+p_z_d[in_idx]*p_z_d[in_idx];
		}
}

__global__ void imaging_old(float *wf_d,float *wfr_d,float *imageup_d,float *imagedown_d,int nx,int nz,int nz_append,int boundary_up,int boundary_left)
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iz=blockIdx.y*blockDim.y+threadIdx.y;

		if(ix<nx&&iz<nz)
		{
			imageup_d[ix*nz+iz]+=wf_d[(boundary_left+ix)*nz_append+iz+boundary_up]*wfr_d[(boundary_left+ix)*nz_append+iz+boundary_up];

			imagedown_d[ix*nz+iz]+=wf_d[(boundary_left+ix)*nz_append+iz+boundary_up]*wf_d[(boundary_left+ix)*nz_append+iz+boundary_up];
			//imagedown_d[ix*nz+iz]+=wfr_d[(boundary_left+ix)*nz_append+iz+boundary_up]*wfr_d[(boundary_left+ix)*nz_append+iz+boundary_up];
		}
}

