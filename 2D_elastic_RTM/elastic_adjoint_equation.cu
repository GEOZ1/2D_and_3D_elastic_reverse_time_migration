#include "hip/hip_runtime.h"
__constant__ const int BDIMX2=32;
__constant__ const int BDIMY2=16;
__constant__ const int radius2=6;
__constant__ const int Block_Size=512;
#define EPS 1e-30

__global__ void cuda_mul_error_random(float *obs_shot_x_d,float *error_random_d,int receiver_interval,int receiver_num,int lt)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	if((ix<receiver_num)&&(iz<lt))
	{
		in_idx=ix*lt+iz;
			
		obs_shot_x_d[in_idx]=obs_shot_x_d[in_idx]*error_random_d[ix];
	}
}

__global__ void cuda_mul_shot_scale(float *obs_shot_x_d,int ishot,int shot_num,int shot_scale,int receiver_num,int lt)
//cuda_mul_shot_scale<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,ishot,shot_num,shot_scale,receiver_num,lt);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	if((ix<receiver_num)&&(iz<lt))
	{
		in_idx=ix*lt+iz;
			
		if(ishot<shot_num/2)	obs_shot_x_d[in_idx]=obs_shot_x_d[in_idx]*1.0/6;

		else			obs_shot_x_d[in_idx]=obs_shot_x_d[in_idx]*6.0;
	}
}

__global__ void cuda_mul_shot_scale_new(float *obs_shot_x_d,int ishot,int shot_num,float *shot_scale_d,int receiver_num,int lt)
//cuda_mul_shot_scale<<<dimGrid_lt,dimBlock>>>(obs_shot_x_d,ishot,shot_num,shot_scale,receiver_num,lt);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	if((ix<receiver_num)&&(iz<lt))
	{
		in_idx=ix*lt+iz;
			
			obs_shot_x_d[in_idx]=obs_shot_x_d[in_idx]*shot_scale_d[ishot];
	}
}

__global__ void cuda_laplace(float *input,float *output,float *velocity_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,float dx,float dz,int mark,int laplace)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int id;
		float s_velocity;	
		float up1,down1,left1,right1,self,result;	

		if(ix>0&&ix<nx-1&&iz>0&&iz<nz-1)
		{
			id=ix*nz+iz;

			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			__syncthreads();

			self=-1.0*input[id];
			up1=-1.0*input[id-1];
			down1=-1.0*input[id+1];

			left1=-1.0*input[id-nz];
			right1=-1.0*input[id+nz];
			s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
			
			result=(right1+left1-2*self)*1.0/dx/dx+(up1+down1-2*self)*1.0/dz/dz;			

			__syncthreads();

			if(mark==0)	output[id]=s_velocity*result/4.0;

			if(mark==1)	output[id]=2500*2500/4*result;
		}		
}

__global__ void cuda_lap(float *wf_d,int nx,int nz,int laplace)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int id;

		if(ix==0||ix==nx-1||iz==0||iz==nz-1)
		{
			
				id=ix*nz+iz;	

				/*if(ix<=1)	wf_d[id]=wf_d[id+1*nz];

				if(ix>=nx-1)	wf_d[id]=wf_d[id-1*nz];

				if(iz<=1)	wf_d[id]=wf_d[id+1];

				if(iz>=nz-1)	wf_d[id]=wf_d[id-1];*/
				////////////mabye  some artifact may be introduced into boundary.2017年03月23日 星期四 21时25分10秒 

				if(ix==0)	wf_d[id]=wf_d[id+1*nz];

				if(ix==nx-1)	wf_d[id]=wf_d[id-1*nz];

				if(iz==0)	wf_d[id]=wf_d[id+1];

				if(iz==nz-1)	wf_d[id]=wf_d[id-1];

				/*if(ix==0)	wf_d[id]=wf_d[id+1*nz]*0.6666;

				if(ix==nx-1)	wf_d[id]=wf_d[id-1*nz]*0.6666;

				if(iz==0)	wf_d[id]=wf_d[id+1]*0.6666;

				if(iz==nz-1)	wf_d[id]=wf_d[id-1]*0.6666;*/
			
		}

}

/*__global__ void cuda_laplace(float *input,float *velocity_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,float dx,float dz,int mark)
{
		__shared__ float s_data1[BDIMY2+2][BDIMX2+2];
		
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int id;

		float s_velocity;
		float result;

		int tx = threadIdx.x+1;
		int tz = threadIdx.y+1;

		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*1-threadIdx.y][BDIMX2+2*1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*1-threadIdx.y][threadIdx.x]=0.0;

		if(ix<nx-2&&iz<nz-2)
		{		
			ix=ix+1;iz=iz+1;
			in_idx=ix*nz+iz;//iz*dimx+ix;

			id=(ix+boundary_left)*dimz+iz+boundary_up;

			__syncthreads();

			s_data1[tz][tx]=input[in_idx];
			

			if(threadIdx.y<1)
			{
					s_data1[threadIdx.y][tx]=input[in_idx-1];//g_input[in_idx-radius2*dimx];//up
					s_data1[threadIdx.y+BDIMY2+1][tx]=input[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
			}
			if(threadIdx.x<1)
			{
					s_data1[tz][threadIdx.x]=input[in_idx-1*dimz];//g_input[in_idx-radius2];//left
					s_data1[tz][threadIdx.x+BDIMX2+1]=input[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
			}

			s_velocity=1;//velocity_d[id]*velocity_d[id];
				
			__syncthreads();

			result=(s_data1[tz+1][tx]+s_data1[tz-1][tx]-2.0*s_data1[tz][tx])/(dz*dz*1.0)+(s_data1[tz][tx+1]+s_data1[tz][tx-1]-2.0*s_data1[tz][tx])/(dx*dx*1.0);

			if(mark==0)	input[in_idx]=-1.0*result*s_velocity/4.0;

			if(mark==1)	input[in_idx]=-1.0*result*2500*2500/4.0;

		}
}*/

__global__ void cal_sum_a_b_to_c(float *vx_x_d,float *vz_z_d,float *wf_append_d,int dimx,int dimz)
///cal_sum_a_b_to_c<<<dimGrid_lt, dimBlock>>>(cal_shot_x_d,cal_shot_z_d,cal_shot_all_d,receiver_num,lt);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
			in_idx = ix*dimz+iz;//iz*dimx+ix;
			
			wf_append_d[in_idx]=vx_x_d[in_idx]+vz_z_d[in_idx];
		}
}

__global__ void cal_sub_a_b_to_c(float *vx_x_d,float *vz_z_d,float *wf_append_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
			in_idx = ix*dimz+iz;//iz*dimx+ix;
			
			wf_append_d[in_idx]=vx_x_d[in_idx]-vz_z_d[in_idx];
		}
}

__global__ void cal_mul_a_b_to_c(float *vx_x_d,float *vz_z_d,float *wf_append_d,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
			in_idx = ix*dimz+iz;//iz*dimx+ix;
			
			wf_append_d[in_idx]=1.0*vx_x_d[in_idx]*vz_z_d[in_idx];
		}
}

//////2016年10月20日 星期四 00时23分35秒 弹性波反偏移算子 based on Zhou 2012 and Ren 2016/// Zongcai Feng, Gerard T:five equation!!!!
/////similar with the first-order velocity-stress equation
__global__ void demig_fwd_vx(float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)//,float *vx,float *perturb_density,int mark_density)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius2,nz_append_radius2
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void demig_fwd_vx_new(float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *perturb_density_d,float *vx_t_d)//,float *vx,float *perturb_density,int mark_density)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius2,nz_append_radius2
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;
		float perturb_density;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				perturb_density=perturb_density_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)-perturb_density*vx_t_d[in_idx]);
		}
}

__global__ void demig_fwd_vz(float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z));
		}
}

__global__ void demig_fwd_vz_new(float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *perturb_density_d,float *vz_t_d)
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;
		float perturb_density;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				perturb_density=perturb_density_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)-perturb_density*vz_t_d[in_idx]);
		}
}

__global__ void demig_fwd_txxzzxz(float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *perturb_lame1_d,float *perturb_lame2_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d)
//demig_fwd_txxzzxz<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius2,nz_append_radius2,s_density_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float perturb_lame1;
		float perturb_lame2;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=vx2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=vz2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=vx2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=vz2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				perturb_lame1=perturb_lame1_d[in_idx];
				perturb_lame2=perturb_lame2_d[in_idx];

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z+
						(perturb_lame1+2*perturb_lame2)*vx_x_d[in_idx]*dt_real+perturb_lame1*vz_z_d[in_idx]*dt_real);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z+
						(perturb_lame1+2*perturb_lame2)*vz_z_d[in_idx]*dt_real+perturb_lame1*vx_x_d[in_idx]*dt_real);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+
						perturb_lame2*vx_z_d[in_idx]*dt_real+perturb_lame2*vz_x_d[in_idx]*dt_real);
		}
}
//////2017年01月04日 星期三 09时24分28秒 弹性波反偏移算子 based on Zhou 2012 and Ren 2016/// Zongcai Feng, Gerard T:five equation!!!!
/////similar with the first-order velocity-stress equation
__global__ void demig_fwd_vx_mul(float *vx2,float *vx1,float *txx1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *d1_d,float *d2_d,float *d3_d,float *d4_d,float *d5_d)
//demig_fwd_vx_mul<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)+1.0*d1_d[in_idx]*dt_real/density_d[in_idx]);
		}
}

__global__ void demig_fwd_vz_mul(float *vz2,float *vz1,float *tzz1,float *txz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *d1_d,float *d2_d,float *d3_d,float *d4_d,float *d5_d)
//demig_fwd_vz_mul<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtzz2_d,rtxz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
	//	float s_velocity;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=tzz1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();

				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+1.0/density_d[in_idx]*(sumx*coe_x+sumz*coe_z)+1.0*d2_d[in_idx]*dt_real/density_d[in_idx]);
		}
}

__global__ void demig_fwd_txxzzxz_mul(float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d,float *d1_d,float *d2_d,float *d3_d,float *d4_d,float *d5_d)
//demig_fwd_txxzzxz_mul<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d,dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;

		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=vx2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=vz2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=vx2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=vz2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		

				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);

				float    sumz=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);


				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+
						s_velocity*density_d[in_idx]*sumx*coe_x+(s_velocity-2*s_velocity1)*density_d[in_idx]*sumz*coe_z+
						d3_d[in_idx]*dt_real);//s_velocity  and  s_velocity1

				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+
						(s_velocity-2*s_velocity1)*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z+
						d4_d[in_idx]*dt_real);//sumx  and  sumz 
							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-  s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);

				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+s_velocity1*density_d[in_idx]*(sumx1*coe_x+sumz1*coe_z)+
						d5_d[in_idx]*dt_real);
		}
}


///////2016年11月20日 星期日 05时59分52秒 
__global__ void cuda_cal_dem_parameter(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,int dimx,int dimz)
//cuda_cal_dem_parameter<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				dem_p1_d[in_idx]=-1.0*tmp_perturb_den_d[in_idx]*vx_t_d[in_idx];
				dem_p2_d[in_idx]=-1.0*tmp_perturb_den_d[in_idx]*vz_t_d[in_idx];

				dem_p3_d[in_idx]=((tmp_perturb_lame1_d[in_idx]+2*tmp_perturb_lame2_d[in_idx])*vx_x_d[in_idx]+tmp_perturb_lame1_d[in_idx]*vz_z_d[in_idx]);

				dem_p4_d[in_idx]=((tmp_perturb_lame1_d[in_idx]+2*tmp_perturb_lame2_d[in_idx])*vz_z_d[in_idx]+tmp_perturb_lame1_d[in_idx]*vx_x_d[in_idx]);

				dem_p5_d[in_idx]=(tmp_perturb_lame2_d[in_idx]*vx_z_d[in_idx]+tmp_perturb_lame2_d[in_idx]*vz_x_d[in_idx]);
		}
}
/////2016年11月28日 星期一 05时14分07秒 
__global__ void cuda_cal_dem_parameter_new(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx1_d,float *vz1_d,float *txx1_d,float *txz1_d,float *tzz1_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,float *velocity_d,float *velocity1_d,float *s_density_d,int dimx,int dimz)
//<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx1_d,vz1_d,txx1_d,tzz1_d,txz1_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		float s_velocity,s_velocity1;
		float d_x=0,d_z=0;


		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;
			
				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];

				d_x=(s_velocity*s_density_d[in_idx]*txx1_d[in_idx]-(s_velocity-2*s_velocity1)*s_density_d[in_idx]*tzz1_d[in_idx])/(4*s_velocity1*s_density_d[in_idx]*(s_velocity-s_velocity1)*s_density_d[in_idx]);

				d_z=(s_velocity*s_density_d[in_idx]*tzz1_d[in_idx]-(s_velocity-2*s_velocity1)*s_density_d[in_idx]*txx1_d[in_idx])/(4*s_velocity1*s_density_d[in_idx]*(s_velocity-s_velocity1)*s_density_d[in_idx]);

				dem_p1_d[in_idx]=tmp_perturb_den_d[in_idx]*vx1_d[in_idx];
				dem_p2_d[in_idx]=tmp_perturb_den_d[in_idx]*vz1_d[in_idx];

				dem_p3_d[in_idx]=-1.0*((tmp_perturb_lame1_d[in_idx]+2*tmp_perturb_lame2_d[in_idx])*d_x+tmp_perturb_lame1_d[in_idx]*d_z);

				dem_p4_d[in_idx]=-1.0*((tmp_perturb_lame1_d[in_idx]+2*tmp_perturb_lame2_d[in_idx])*d_z+tmp_perturb_lame1_d[in_idx]*d_x);

				dem_p5_d[in_idx]=-1.0*tmp_perturb_lame2_d[in_idx]*txz1_d[in_idx]/(s_velocity1*s_density_d[in_idx]);
		}
}

__global__ void cuda_cal_dem_parameter_lame(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cuda_cal_dem_parameter_lame<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float lame1,lame2;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];

				dem_p1_d[in_idx]=-1.0*tmp_perturb_den_d[in_idx]*vx_t_d[in_idx]*s_density_d[in_idx];
				dem_p2_d[in_idx]=-1.0*tmp_perturb_den_d[in_idx]*vz_t_d[in_idx]*s_density_d[in_idx];

				//dem_p3_d[in_idx]=((tmp_perturb_lame1_d[in_idx]+2*tmp_perturb_lame2_d[in_idx])*vx_x_d[in_idx]+tmp_perturb_lame1_d[in_idx]*vz_z_d[in_idx]);
				dem_p3_d[in_idx]=1.0*lame1*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_lame1_d[in_idx]+2.0*lame2*vx_x_d[in_idx]*tmp_perturb_lame2_d[in_idx];

				dem_p4_d[in_idx]=1.0*lame1*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_lame1_d[in_idx]+2.0*lame2*vz_z_d[in_idx]*tmp_perturb_lame2_d[in_idx];

				dem_p5_d[in_idx]=1.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_lame2_d[in_idx];
		}
}

__global__ void cuda_cal_dem_parameter_velocity(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_vp_d,float *tmp_perturb_vs_d,float *tmp_perturb_density_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cuda_cal_dem_parameter_velocity<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float lame1,lame2;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];

				dem_p1_d[in_idx]=-1.0*tmp_perturb_density_d[in_idx]*vx_t_d[in_idx]*s_density_d[in_idx];
				dem_p2_d[in_idx]=-1.0*tmp_perturb_density_d[in_idx]*vz_t_d[in_idx]*s_density_d[in_idx];

				dem_p3_d[in_idx]=1.0*((lame1+2.0*lame2)*vx_x_d[in_idx]+lame1*vz_z_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vz_z_d[in_idx]*tmp_perturb_vs_d[in_idx];

				dem_p4_d[in_idx]=1.0*((lame1+2.0*lame2)*vz_z_d[in_idx]+lame1*vx_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vx_x_d[in_idx]*tmp_perturb_vs_d[in_idx];

				dem_p5_d[in_idx]=1.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						//+2.0*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////这个错误，找了一天，2017年01月10日 星期二 11时10分09秒 
						+2.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////this error spend one day!!!!!，
		}
}

__global__ void cuda_cal_dem_parameter_impedance(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_vp_d,float *tmp_perturb_vs_d,float *tmp_perturb_density_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cuda_cal_dem_parameter<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,nx_append_radius,nz_append_radius);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		float lame1,lame2;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];

				dem_p1_d[in_idx]=-1.0*tmp_perturb_density_d[in_idx]*vx_t_d[in_idx]*s_density_d[in_idx];
				dem_p2_d[in_idx]=-1.0*tmp_perturb_density_d[in_idx]*vz_t_d[in_idx]*s_density_d[in_idx];

				/*dem_p3_d[in_idx]=1.0*((lame1+2.0*lame2)*vx_x_d[in_idx]+lame1*vz_z_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vz_z_d[in_idx]*tmp_perturb_vs_d[in_idx];*/

				dem_p3_d[in_idx]=-1.0*((lame1+2.0*lame2)*vx_x_d[in_idx]+lame1*vz_z_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vz_z_d[in_idx]*tmp_perturb_vs_d[in_idx];

				/*dem_p4_d[in_idx]=1.0*((lame1+2.0*lame2)*vz_z_d[in_idx]+lame1*vx_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vx_x_d[in_idx]*tmp_perturb_vs_d[in_idx];*/

				dem_p4_d[in_idx]=-1.0*((lame1+2.0*lame2)*vz_z_d[in_idx]+lame1*vx_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						+2.0*(lame1+2.0*lame2)*(vx_x_d[in_idx]+vz_z_d[in_idx])*tmp_perturb_vp_d[in_idx]
						-4.0*lame2*vx_x_d[in_idx]*tmp_perturb_vs_d[in_idx];

				/*dem_p5_d[in_idx]=1.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						//+2.0*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////这个错误，找了一天，2017年01月10日 星期二 11时10分09秒 
						+2.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////this error spend one day!!!!!，*/

				dem_p5_d[in_idx]=-1.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_density_d[in_idx]
						//+2.0*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////这个错误，找了一天，2017年01月10日 星期二 11时10分09秒 
						+2.0*lame2*(vx_z_d[in_idx]+vz_x_d[in_idx])*tmp_perturb_vs_d[in_idx];/////this error spend one day!!!!!，
		}
}

//////2016年10月08日 星期六 09时53分37秒   一阶速度应力方程的伴随状态方程
__global__ void adjoint_fwd_vx(float *vx2,float *vx1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//adjoint_fwd_vx<<<dimGrid,dimBlock>>>(rvx1_d,rvx2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人
//////////////注意伴随状态方程左边存在密度，所以用来反传计算伴随波场跟密度没有关系？？？？？？？？？？

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumx:the derivation of x direction of txx
				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

////sumxz:the derivation of z direction of txz
				float    sumxz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumxz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumxz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumxz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumxz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumxz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
////sumx1:the derivation of x direction of tzz
				float    sumx1=coe_d[1]*(s_data3[tz][tx]-s_data3[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data3[tz][tx+1]-s_data3[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data3[tz][tx+2]-s_data3[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data3[tz][tx+3]-s_data3[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data3[tz][tx+4]-s_data3[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data3[tz][tx+5]-s_data3[tz][tx-6]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+(s_velocity*sumx*coe_x+(s_velocity-2*s_velocity1)*sumx1*coe_x+s_velocity1*sumxz*coe_z));
		}
}

__global__ void adjoint_fwd_vx_illum(float *r_d_illum,float *vx2,float *vx1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//adjoint_fwd_vx<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人
//////////////注意伴随状态方程左边存在密度，所以用来反传计算伴随波场跟密度没有关系？？？？？？？？？？

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumx:the derivation of x direction of txx
				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

////sumxz:the derivation of z direction of txz
				float    sumxz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumxz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumxz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumxz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumxz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumxz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
////sumx1:the derivation of x direction of tzz
				float    sumx1=coe_d[1]*(s_data3[tz][tx]-s_data3[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data3[tz][tx+1]-s_data3[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data3[tz][tx+2]-s_data3[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data3[tz][tx+3]-s_data3[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data3[tz][tx+4]-s_data3[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data3[tz][tx+5]-s_data3[tz][tx-6]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+(s_velocity*sumx*coe_x+(s_velocity-2*s_velocity1)*sumx1*coe_x+s_velocity1*sumxz*coe_z));

				r_d_illum[in_idx]+=vx2[in_idx]*vx2[in_idx];
		}
}

__global__ void adjoint_fwd_vz(float *vz2,float *vz1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//adjoint_fwd_vz<<<dimGrid,dimBlock>>>(rvz1_d,rvz2_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumz:the derivation of z direction of tzz   ///////data3:tzz1/////// 
				float    sumz=coe_d[1]*(s_data3[tz+1][tx]-s_data3[tz][tx]);
					sumz+=coe_d[2]*(s_data3[tz+2][tx]-s_data3[tz-1][tx]);
					sumz+=coe_d[3]*(s_data3[tz+3][tx]-s_data3[tz-2][tx]);
					sumz+=coe_d[4]*(s_data3[tz+4][tx]-s_data3[tz-3][tx]);
					sumz+=coe_d[5]*(s_data3[tz+5][tx]-s_data3[tz-4][tx]);
					sumz+=coe_d[6]*(s_data3[tz+6][tx]-s_data3[tz-5][tx]);

////sumz1:the derivation of z direction of txx/////data1:txx1////
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

////sumx:the derivation of x direction of txz/////data2:txz1////					
				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+(s_velocity*sumz*coe_z+(s_velocity-2*s_velocity1)*sumz1*coe_z+s_velocity1*sumx*coe_x));
		}
}

__global__ void adjoint_fwd_vz_illum(float *r_d_illum,float *vz2,float *vz1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//adjoint_fwd_vz<<<dimGrid,dimBlock>>>(rvz2_d,rvz1_d,rtxx2_d,rtxz2_d,rtzz2_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=txx1[in_idx];
				s_data2[tz][tx]=txz1[in_idx];
				s_data3[tz][tx]=tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=txx1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=txx1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=txz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=txz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=tzz1[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=tzz1[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=txx1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=txx1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=txz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=txz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=tzz1[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=tzz1[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumz:the derivation of z direction of tzz   ///////data3:tzz1/////// 
				float    sumz=coe_d[1]*(s_data3[tz+1][tx]-s_data3[tz][tx]);
					sumz+=coe_d[2]*(s_data3[tz+2][tx]-s_data3[tz-1][tx]);
					sumz+=coe_d[3]*(s_data3[tz+3][tx]-s_data3[tz-2][tx]);
					sumz+=coe_d[4]*(s_data3[tz+4][tx]-s_data3[tz-3][tx]);
					sumz+=coe_d[5]*(s_data3[tz+5][tx]-s_data3[tz-4][tx]);
					sumz+=coe_d[6]*(s_data3[tz+6][tx]-s_data3[tz-5][tx]);

////sumz1:the derivation of z direction of txx/////data1:txx1////
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

////sumx:the derivation of x direction of txz/////data2:txz1////					
				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+(s_velocity*sumz*coe_z+(s_velocity-2*s_velocity1)*sumz1*coe_z+s_velocity1*sumx*coe_x));

				r_d_illum[in_idx]+=vz2[in_idx]*vz2[in_idx];
		}
}


__global__ void adjoint_fwd_txxzzxz(float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//adjoint_fwd_txxzzxz<<<dimGrid,dimBlock>>>(rtxx2_d,rtxx1_d,rtzz2_d,rtzz1_d,rtxz2_d,rtxz1_d,rvx1_d,rvz1_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		//float s_velocity;
		//float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=vx2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=vz2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=vx2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=vz2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				//s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		
/////////sumx:the derivation of x direction of vx
				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);
/////////sumz:the derivation of z direction of vz
				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+sumx*coe_x);
				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+sumz*coe_z);

/////////sumx1:the derivation of x direction of vz							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);
/////////sumz1:the derivation of z direction of vx
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+sumx1*coe_x+sumz1*coe_z);
		}
}

__global__ void adjoint_fwd_txxzzxzpp(float *tp2,float *tp1,float *txx2,float *txx1,float *tzz2,float *tzz1,float *txz2,float *txz1,float *vx2,float *vz2,float *velocity_d,float *velocity1_d,float *attenuation_d,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *density_d)
//adjoint_fwd_txxzzxzpp<<<dimGrid,dimBlock>>>(rtp1_d,rtp2_d,rtxx1_d,rtxx2_d,rtzz1_d,rtzz2_d,rtxz1_d,rtxz2_d,rvx2_d,rvz2_d,s_velocity_d,s_velocity1_d,attenuation_d,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		float s_velocity;
		//float s_velocity1;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx2[in_idx];
				s_data2[tz][tx]=vz2[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=vx2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=vx2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=vz2[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=vz2[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=vx2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=vx2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=vz2[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=vz2[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}

				s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				//s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];
				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
		
/////////sumx:the derivation of x direction of vx
				float    sumx=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);
					sumx+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
					sumx+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
					sumx+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
					sumx+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
					sumx+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);
/////////sumz:the derivation of z direction of vz
				float    sumz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);

				txx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*txx1[in_idx]+sumx*coe_x);
				tzz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tzz1[in_idx]+sumz*coe_z);

				//tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+sumx*coe_x+sumz*coe_z);	
				tp2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*((1.0-s_attenuation*dt_real/2.0)*tp1[in_idx]+s_velocity*density_d[in_idx]*sumx*coe_x+s_velocity*density_d[in_idx]*sumz*coe_z);

/////////sumx1:the derivation of x direction of vz							
				float    sumx1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data2[tz][tx+1]-s_data2[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data2[tz][tx+2]-s_data2[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data2[tz][tx+3]-s_data2[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data2[tz][tx+4]-s_data2[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data2[tz][tx+5]-s_data2[tz][tx-6]);
/////////sumz1:the derivation of z direction of vx
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

				txz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
						((1.0-s_attenuation*dt_real/2.0)*txz1[in_idx]+sumx1*coe_x+sumz1*coe_z);
		}
}

//////2016年11月03日 星期四 19时12分09秒 add new gradient_x and_z for lame1 lame2    一阶速度应力方程的伴随状态方程
__global__ void adjoint_fwd_vx_new(float *vx2,float *vx1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//adjoint_fwd_vx<<<dimGrid,dimBlock>>>(rvx2_d,rvx1_d,rtxx1_d,rtxz1_d,rtzz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d);
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		//float s_velocity;
		//float s_velocity1;
		float s_density;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int up,down;
		int left,right;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();
//////////////////////////////////////////////////////////lame1:(velocity_d[in_idx]*velocity_d[in_idx]-2*velocity1_d[in_idx]*velocity1_d[in_idx])/density_d[in_idx]
//////////////////////////////////////////////////////////lame2:velocity1_d[in_idx]*velocity1_d[in_idx]/density_d[in_idx]
//////////////////////////////////////////////////////////lame1+2*lame2:velocity_d[in_idx]*velocity_d[in_idx]/density_d[in_idx]
				s_data1[tz][tx]=velocity_d[in_idx]*velocity_d[in_idx]/density_d[in_idx]*txx1[in_idx];
				s_data2[tz][tx]=velocity1_d[in_idx]*velocity1_d[in_idx]/density_d[in_idx]*txz1[in_idx];
				s_data3[tz][tx]=(velocity_d[in_idx]*velocity_d[in_idx]-2*velocity1_d[in_idx]*velocity1_d[in_idx])/density_d[in_idx]*tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						up=in_idx-radius2;
						down=in_idx+BDIMY2;
						s_data1[threadIdx.y][tx]=velocity_d[up]*velocity_d[up]/density_d[up]*txx1[up];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=velocity_d[down]*velocity_d[down]/density_d[down]*txx1[down];//g_input[in_idx+BDIMY2*dimx];//down
						s_data2[threadIdx.y][tx]=velocity1_d[up]*velocity1_d[up]/density_d[up]*txz1[up];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=velocity1_d[down]*velocity1_d[down]/density_d[down]*txz1[down];//g_input[in_idx+BDIMY2*dimx];//down
						s_data3[threadIdx.y][tx]=(velocity_d[up]*velocity_d[up]-2*velocity1_d[up]*velocity1_d[up])/density_d[up]*tzz1[up];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=(velocity_d[down]*velocity_d[down]-2*velocity1_d[down]*velocity1_d[down])/density_d[down]*tzz1[down];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						left=in_idx-radius2*dimz;
						right=in_idx+BDIMX2*dimz;
						s_data1[tz][threadIdx.x]=velocity_d[left]*velocity_d[left]/density_d[left]*txx1[left];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=velocity_d[right]*velocity_d[right]/density_d[right]*txx1[right];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=velocity1_d[left]*velocity1_d[left]/density_d[left]*txz1[left];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=velocity1_d[right]*velocity1_d[right]/density_d[right]*txz1[right];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=(velocity_d[left]*velocity_d[left]-2*velocity1_d[left]*velocity1_d[left])/density_d[left]*tzz1[left];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=(velocity_d[right]*velocity_d[right]-2*velocity1_d[right]*velocity1_d[right])/density_d[right]*tzz1[right];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				//s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人
				s_density=density_d[in_idx];
//////////////注意伴随状态方程左边存在密度，所以用来反传计算伴随波场跟密度没有关系？？？？？？？？？？

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumx:the derivation of x direction of txx
				float    sumx=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);
					sumx+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
					sumx+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
					sumx+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
					sumx+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
					sumx+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);

////sumxz:the derivation of z direction of txz
				float    sumxz=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
					sumxz+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
					sumxz+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
					sumxz+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
					sumxz+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
					sumxz+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
////sumx1:the derivation of x direction of tzz
				float    sumx1=coe_d[1]*(s_data3[tz][tx]-s_data3[tz][tx-1]);
					sumx1+=coe_d[2]*(s_data3[tz][tx+1]-s_data3[tz][tx-2]);
					sumx1+=coe_d[3]*(s_data3[tz][tx+2]-s_data3[tz][tx-3]);
					sumx1+=coe_d[4]*(s_data3[tz][tx+3]-s_data3[tz][tx-4]);
					sumx1+=coe_d[5]*(s_data3[tz][tx+4]-s_data3[tz][tx-5]);
					sumx1+=coe_d[6]*(s_data3[tz][tx+5]-s_data3[tz][tx-6]);

				vx2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vx1[in_idx]+(s_density*sumx*coe_x+s_density*sumx1*coe_x+s_density*sumxz*coe_z));
		}
}

__global__ void adjoint_fwd_vz_new(float *vz2,float *vz1,float *txx1,float *txz1,float *tzz1,float *attenuation_d,float dx,float dz,float dt,float *coe_d,float coe_x,float coe_z,int dimx,int dimz,float *velocity_d,float *velocity1_d,float *density_d)
//vx2_d,vx1_d,txx1_d,txz1_d,attenuation_d,dx,dz,dt,coe_opt_d,coe_x,coe_z,nx_append_radius2,nz_append_radius2
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];
		__shared__ float s_data3[BDIMY2+2*radius2][BDIMX2+2*radius2];
		float dt_real=dt/1000;
		//float s_velocity;
		//float s_velocity1;
		float s_density;
		float s_attenuation;

		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int up,down;
		int left,right;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;
		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		s_data3[tz][tx]=0.0;
		s_data3[threadIdx.y][threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data3[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();
//////////////////////////////////////////////////////////lame1:(velocity_d[in_idx]*velocity_d[in_idx]-2*velocity1_d[in_idx]*velocity1_d[in_idx])/density_d[in_idx]
//////////////////////////////////////////////////////////lame2:velocity1_d[in_idx]*velocity1_d[in_idx]/density_d[in_idx]
//////////////////////////////////////////////////////////lame1+2*lame2:velocity_d[in_idx]*velocity_d[in_idx]/density_d[in_idx]
				s_data1[tz][tx]=(velocity_d[in_idx]*velocity_d[in_idx]-2*velocity1_d[in_idx]*velocity1_d[in_idx])/density_d[in_idx]*txx1[in_idx];
				s_data2[tz][tx]=velocity1_d[in_idx]*velocity1_d[in_idx]/density_d[in_idx]*txz1[in_idx];
				s_data3[tz][tx]=velocity_d[in_idx]*velocity_d[in_idx]/density_d[in_idx]*tzz1[in_idx];

				if(threadIdx.y<radius2)
				{
						up=in_idx-radius2;
						down=in_idx+BDIMY2;
						s_data1[threadIdx.y][tx]=(velocity_d[up]*velocity_d[up]-2*velocity1_d[up]*velocity1_d[up])/density_d[up]*txx1[up];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=(velocity_d[down]*velocity_d[down]-2*velocity1_d[down]*velocity1_d[down])/density_d[down]*txx1[down];//g_input[in_idx+BDIMY2*dimx];//down

						s_data2[threadIdx.y][tx]=velocity1_d[up]*velocity1_d[up]/density_d[up]*txz1[up];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=velocity1_d[down]*velocity1_d[down]/density_d[down]*txz1[down];//g_input[in_idx+BDIMY2*dimx];//down

						s_data3[threadIdx.y][tx]=velocity_d[up]*velocity_d[up]/density_d[up]*tzz1[up];//g_input[in_idx-radius2*dimx];//up
						s_data3[threadIdx.y+BDIMY2+radius2][tx]=velocity_d[down]*velocity_d[down]/density_d[down]*tzz1[down];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						left=in_idx-radius2*dimz;
						right=in_idx+BDIMX2*dimz;
						s_data1[tz][threadIdx.x]=(velocity_d[left]*velocity_d[left]-2*velocity1_d[left]*velocity1_d[left])/density_d[left]*txx1[left];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=(velocity_d[right]*velocity_d[right]-2*velocity1_d[right]*velocity1_d[right])/density_d[right]*txx1[right];//g_input[in_idx+BDIMX2];//right

						s_data2[tz][threadIdx.x]=velocity1_d[left]*velocity1_d[left]/density_d[left]*txz1[left];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=velocity1_d[right]*velocity1_d[right]/density_d[right]*txz1[right];//g_input[in_idx+BDIMX2];//right

						s_data3[tz][threadIdx.x]=velocity_d[left]*velocity_d[left]/density_d[left]*tzz1[left];//g_input[in_idx-radius2];//left
						s_data3[tz][threadIdx.x+BDIMX2+radius2]=velocity_d[right]*velocity_d[right]/density_d[right]*tzz1[right];//g_input[in_idx+BDIMX2];//right
				}

				//s_velocity=velocity_d[in_idx]*velocity_d[in_idx];
				//s_velocity1=velocity1_d[in_idx]*velocity1_d[in_idx];//////s_velocity1=velocity1_d[in_idx]*velocity_d[in_idx];     error//气人
				s_density=density_d[in_idx];

				s_attenuation=attenuation_d[in_idx];
				__syncthreads();
/////data1:txx1///////data2:txz1///////data3:tzz1///////
////sumz:the derivation of z direction of tzz   ///////data3:tzz1/////// 
				float    sumz=coe_d[1]*(s_data3[tz+1][tx]-s_data3[tz][tx]);
					sumz+=coe_d[2]*(s_data3[tz+2][tx]-s_data3[tz-1][tx]);
					sumz+=coe_d[3]*(s_data3[tz+3][tx]-s_data3[tz-2][tx]);
					sumz+=coe_d[4]*(s_data3[tz+4][tx]-s_data3[tz-3][tx]);
					sumz+=coe_d[5]*(s_data3[tz+5][tx]-s_data3[tz-4][tx]);
					sumz+=coe_d[6]*(s_data3[tz+6][tx]-s_data3[tz-5][tx]);

////sumz1:the derivation of z direction of txx/////data1:txx1////
				float    sumz1=coe_d[1]*(s_data1[tz+1][tx]-s_data1[tz][tx]);
					sumz1+=coe_d[2]*(s_data1[tz+2][tx]-s_data1[tz-1][tx]);
					sumz1+=coe_d[3]*(s_data1[tz+3][tx]-s_data1[tz-2][tx]);
					sumz1+=coe_d[4]*(s_data1[tz+4][tx]-s_data1[tz-3][tx]);
					sumz1+=coe_d[5]*(s_data1[tz+5][tx]-s_data1[tz-4][tx]);
					sumz1+=coe_d[6]*(s_data1[tz+6][tx]-s_data1[tz-5][tx]);

////sumx:the derivation of x direction of txz/////data2:txz1////					
				float    sumx=coe_d[1]*(s_data2[tz][tx+1]-s_data2[tz][tx]);
					sumx+=coe_d[2]*(s_data2[tz][tx+2]-s_data2[tz][tx-1]);
					sumx+=coe_d[3]*(s_data2[tz][tx+3]-s_data2[tz][tx-2]);
					sumx+=coe_d[4]*(s_data2[tz][tx+4]-s_data2[tz][tx-3]);
					sumx+=coe_d[5]*(s_data2[tz][tx+5]-s_data2[tz][tx-4]);
					sumx+=coe_d[6]*(s_data2[tz][tx+6]-s_data2[tz][tx-5]);

				vz2[in_idx]=1.0/(1.0+s_attenuation*dt_real/2.0)*
				((1.0-s_attenuation*dt_real/2.0)*vz1[in_idx]+(s_density*sumz*coe_z+s_density*sumz1*coe_z+s_density*sumx*coe_x));
		}
}

//cal_derivation_x<<<dimGrid,dimBlock>>>(vx1_d,vx_x_d,coe_opt_d,dx,dz,nx_append_radius,nz_append_radius,mark);
__global__ void cal_derivation_x(float *vx1_d,float *vx_x_d,float *coe_d,float dx,float dz,int dimx,int dimz,int mark)
{
		__shared__ float s_data1[BDIMY2+2*radius2][BDIMX2+2*radius2];

		float sum=0;
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;

		s_data1[tz][tx]=0.0;
		s_data1[threadIdx.y][threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data1[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data1[tz][tx]=vx1_d[in_idx];


				if(threadIdx.y<radius2)
				{
						s_data1[threadIdx.y][tx]=vx1_d[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data1[threadIdx.y+BDIMY2+radius2][tx]=vx1_d[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data1[tz][threadIdx.x]=vx1_d[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data1[tz][threadIdx.x+BDIMX2+radius2]=vx1_d[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}
				__syncthreads();
				
	if(mark==0)
	{
		       sum=coe_d[1]*(s_data1[tz][tx+1]-s_data1[tz][tx]);		
		       sum+=coe_d[2]*(s_data1[tz][tx+2]-s_data1[tz][tx-1]);
		       sum+=coe_d[3]*(s_data1[tz][tx+3]-s_data1[tz][tx-2]);
		       sum+=coe_d[4]*(s_data1[tz][tx+4]-s_data1[tz][tx-3]);
		       sum+=coe_d[5]*(s_data1[tz][tx+5]-s_data1[tz][tx-4]);
		       sum+=coe_d[6]*(s_data1[tz][tx+6]-s_data1[tz][tx-5]);
	}

	
	if(mark==1)	
	{		     
		       sum=coe_d[1]*(s_data1[tz][tx]-s_data1[tz][tx-1]);		
		       sum+=coe_d[2]*(s_data1[tz][tx+1]-s_data1[tz][tx-2]);
		       sum+=coe_d[3]*(s_data1[tz][tx+2]-s_data1[tz][tx-3]);
		       sum+=coe_d[4]*(s_data1[tz][tx+3]-s_data1[tz][tx-4]);
		       sum+=coe_d[5]*(s_data1[tz][tx+4]-s_data1[tz][tx-5]);
		       sum+=coe_d[6]*(s_data1[tz][tx+5]-s_data1[tz][tx-6]);
	}
		      
			vx_x_d[in_idx]=(1.0/dx)*sum;
		}		
}

__global__ void cal_derivation_z(float *vz1_d,float *vz_z_d,float *coe_d,float dx,float dz,int dimx,int dimz,int mark)
{
		__shared__ float s_data2[BDIMY2+2*radius2][BDIMX2+2*radius2];

		float sum1=0;
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		int tx = threadIdx.x+radius2;
		int tz = threadIdx.y+radius2;

		s_data2[tz][tx]=0.0;
		s_data2[threadIdx.y][threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[threadIdx.y][BDIMX2+2*radius2-1-threadIdx.x]=0.0;
		s_data2[BDIMY2+2*radius2-1-threadIdx.y][threadIdx.x]=0.0;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;

				__syncthreads();

				s_data2[tz][tx]=vz1_d[in_idx];

				if(threadIdx.y<radius2)
				{
						s_data2[threadIdx.y][tx]=vz1_d[in_idx-radius2];//g_input[in_idx-radius2*dimx];//up
						s_data2[threadIdx.y+BDIMY2+radius2][tx]=vz1_d[in_idx+BDIMY2];//g_input[in_idx+BDIMY2*dimx];//down
				}
				if(threadIdx.x<radius2)
				{
						s_data2[tz][threadIdx.x]=vz1_d[in_idx-radius2*dimz];//g_input[in_idx-radius2];//left
						s_data2[tz][threadIdx.x+BDIMX2+radius2]=vz1_d[in_idx+BDIMX2*dimz];//g_input[in_idx+BDIMX2];//right
				}
		
				__syncthreads();
	
	if(mark==0)
	{
		      sum1=coe_d[1]*(s_data2[tz][tx]-s_data2[tz-1][tx]);
		      sum1+=coe_d[2]*(s_data2[tz+1][tx]-s_data2[tz-2][tx]);
		      sum1+=coe_d[3]*(s_data2[tz+2][tx]-s_data2[tz-3][tx]);
		      sum1+=coe_d[4]*(s_data2[tz+3][tx]-s_data2[tz-4][tx]);
		      sum1+=coe_d[5]*(s_data2[tz+4][tx]-s_data2[tz-5][tx]);
		      sum1+=coe_d[6]*(s_data2[tz+5][tx]-s_data2[tz-6][tx]);
		     
	}
	if(mark==1)
	{
		      sum1=coe_d[1]*(s_data2[tz+1][tx]-s_data2[tz][tx]);
		      sum1+=coe_d[2]*(s_data2[tz+2][tx]-s_data2[tz-1][tx]);
		      sum1+=coe_d[3]*(s_data2[tz+3][tx]-s_data2[tz-2][tx]);
		      sum1+=coe_d[4]*(s_data2[tz+4][tx]-s_data2[tz-3][tx]);
		      sum1+=coe_d[5]*(s_data2[tz+5][tx]-s_data2[tz-4][tx]);
		      sum1+=coe_d[6]*(s_data2[tz+6][tx]-s_data2[tz-5][tx]);
	}	      
			vz_z_d[in_idx]=(1.0/dz)*sum1;
		}		
}


__global__ void cuda_cal_objective(float *obj, float *err, int ng)
/*< calculate the value of objective function: obj >*/
{
  	__shared__ float  sdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?err[id]:0.0f;
		sdata[tid] += a*a;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) sdata[tid] += sdata[tid + s]; __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; }
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; }
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; }
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; }
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; }
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; }
    	}
     
    	if (tid == 0) { *obj=sdata[0]; }
}


__global__ void cuda_cal_correlation_objective(float *obj, float *obj_parameter_d)
/*< calculate the value of objective function: obj >*/
{
	*obj=float(-1.0*obj_parameter_d[2]/sqrt(obj_parameter_d[0])/sqrt(obj_parameter_d[1]));
}


__global__ void cuda_adj_shot(float *adj_shot_x_d,float *tmp_shot_x_d,float *obs_shot_x_d,int receiver_num,int lt,float *correlation_parameter_d)
{
	
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	float a,b,c;

	a=correlation_parameter_d[0];///tmp*tmp	
	b=correlation_parameter_d[1];///obs*obs
	c=correlation_parameter_d[2];///tmp*obs

	if((ix<receiver_num)&&(iz<lt))
	{
		in_idx=ix*lt+iz;
			
		adj_shot_x_d[in_idx]=(1.0/sqrt(a*b))*(1.0*c/a*tmp_shot_x_d[in_idx]-obs_shot_x_d[in_idx]);
	}

}

__global__ void cal_gradient_in_elastic_media(float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *vx_t_d,float *vz_t_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,float *rvx1_d,float *rvz1_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cal_gradient_in_elastic_media<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		float lame1;
		float lame2;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			lame1=s_density_d[in_idx1]*s_velocity_d[in_idx1]*s_velocity_d[in_idx1]-2.0*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];
			lame2=s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];

			grad_den_d[in_idx]=grad_den_d[in_idx]+s_density_d[in_idx1]*(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+lame1*(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])*(-1.0);

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+lame2*(2.0*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2.0*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]))*(-1.0);
		}
}

__global__ void cal_gradient_in_elastic_media_new(float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *vx_t_d,float *vz_t_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,float *rvx1_d,float *rvz1_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cal_gradient_in_elastic_media_new<<<dimGrid,dimBlock>>>(grad_lame11_d,grad_lame22_d,grad_den1_d,vx_t_d,vz_t_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,rvx2_d,rvz2_d,rtxx2_d,rtxz2_d,rtzz2_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			grad_den_d[in_idx]=grad_den_d[in_idx]+(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])*(-1.0);

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+(2.0*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2.0*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]))*(-1.0);
		}
}

__global__ void cal_gradient_for_den_old(float *grad_density_d,float *vx1_d,float *vx2_d,float *vz1_d,float *vz2_d,float *rvx1_d,float *rvz1_d,float dt,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		float dt_real;
		dt_real=dt/1000;


		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			grad_density_d[in_idx]+=(rvx1_d[in_idx1]*(vx2_d[in_idx1]-vx1_d[in_idx1])/dt_real+rvz1_d[in_idx1]*(vz2_d[in_idx1]-vz1_d[in_idx1])/dt_real)*(-1.0);
		}
}

///////////////////////the following  cal_gradient_for_den cal_gradient_for_lame1 cal_gradient_for_lame2 - + +////+ - -
__global__ void cal_gradient_for_den(float *grad_den_d,float *vx_t_d,float *vz_t_d,float *rvx1_d,float *rvz1_d,float dt,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_den<<<dimGrid,dimBlock>>>(grad_den_d,vx_t_d,vz_t_d,rvx1_d,rvz1_d,dt,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		//float dt_real;
		//dt_real=dt/1000;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			grad_den_d[in_idx]=grad_den_d[in_idx]+(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);
			
			//grad_den_d[in_idx]=grad_den_d[in_idx]+(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1])*(-1.0);
		}
}

__global__ void cal_gradient_for_lame1(float *grad_lame1_d,float *rtxx1_d,float *rtzz1_d,float *vx_x_d,float *vz_z_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_lame1<<<dimGrid,dimBlock>>>(grad_lame1_d,rtxx1_d,rtzz1_d,vx_x_d,vz_z_d,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])*(-1.0);

			//grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1]);
		}
}

__global__ void cal_gradient_for_lame2(float *grad_lame2_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_lame2<<<dimGrid,dimBlock>>>(grad_lame2_d,rtxx1_d,rtxz1_d,rtzz1_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;	

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+(2*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]))*(-1.0);
			//grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+(2*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]));
		}
}

/////
///////////////////////the following  cal_gradient_for_d_mul cal_gradient_for_lam_mul cal_gradient_for_lam_mul - + +////+ - -
__global__ void cal_gradient_for_den_mul(float *grad_den_d,float *vx_t_d,float *vz_t_d,float *rvx1_d,float *rvz1_d,float dt,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cal_gradient_for_den_mul<<<dimGrid,dimBlock>>>(grad_den1_d,vx_t_d,vz_t_d,rvx2_d,rvz2_d,dt,boundary_left,boundary_up,nx,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		//float dt_real;
		//dt_real=dt/1000;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			grad_den_d[in_idx]=grad_den_d[in_idx]+s_density_d[in_idx1]*(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);
			
			//grad_den_d[in_idx]=grad_den_d[in_idx]+s_density_d[in_idx1]*(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1])*(-1.0);
		}
}

__global__ void cal_gradient_for_lame1_mul(float *grad_lame1_d,float *rtxx1_d,float *rtzz1_d,float *vx_x_d,float *vz_z_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cal_gradient_for_lame1_mul<<<dimGrid,dimBlock>>>(grad_lame11_d,rtxx2_d,rtzz2_d,vx_x_d,vz_z_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		float lame1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;

			lame1=s_density_d[in_idx1]*s_velocity_d[in_idx1]*s_velocity_d[in_idx1]-2.0*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];

			grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+lame1*(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1])*(-1.0);

			//grad_lame1_d[in_idx]=grad_lame1_d[in_idx]+lame1*(rtxx1_d[in_idx1]+rtzz1_d[in_idx1])*(vx_x_d[in_idx1]+vz_z_d[in_idx1]);
		}
}

__global__ void cal_gradient_for_lame2_mul(float *grad_lame2_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,float *vx_x_d,float *vz_z_d,float *vx_z_d,float *vz_x_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d)
//cal_gradient_for_lame2_mul<<<dimGrid,dimBlock>>>(grad_lame22_d,rtxx2_d,rtxz2_d,rtzz2_d,vx_x_d,vz_z_d,vx_z_d,vz_x_d,boundary_left,boundary_up,nx_size,nz,nx_append,nz_append,s_velocity_d,s_velocity1_d,s_density_d);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		float lame2;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			lame2=s_density_d[in_idx1]*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1];	

			grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+lame2*(2.0*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2.0*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]))*(-1.0);
			//grad_lame2_d[in_idx]=grad_lame2_d[in_idx]+lame2*(2*rtxx1_d[in_idx1]*vx_x_d[in_idx1]+2*rtzz1_d[in_idx1]*vz_z_d[in_idx1]+rtxz1_d[in_idx1]*(vx_z_d[in_idx1]+vz_x_d[in_idx1]));
		}
}

///////////////////////the following  cal_gradient_for_den cal_gradient_for_lame1 cal_gradient_for_lame2 - + +////+ - -
__global__ void cal_gradient_for_den_new(float *grad_den_d,float *vx_t_d,float *vz_t_d,float *rvx1_d,float *rvz1_d,float dt,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_den<<<dimGrid,dimBlock>>>(grad_den_d,vx_t_d,vz_t_d,rvx1_d,rvz1_d,dt,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;
		//float dt_real;
		//dt_real=dt/1000;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			grad_den_d[in_idx]=grad_den_d[in_idx]+(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1]);
			
			//grad_den_d[in_idx]=grad_den_d[in_idx]+(rvx1_d[in_idx1]*vx_t_d[in_idx1]+rvz1_d[in_idx1]*vz_t_d[in_idx1])*(-1.0);
		}
}

__global__ void cal_gradient_for_lame1_new(float *grad_lame1_d,float *rtxx1_d,float *rtzz1_d,float *txx1_d,float *tzz1_d,float *velocity_d,float *velocity1_d,float *s_density_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_lame1_new<<<dimGrid,dimBlock>>>(grad_lame11_d,rtxx2_d,rtzz2_d,txx1_d,tzz1_d,s_velocity_d,s_velocity1_d,s_density_d,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int id;
		float lame1,lame2;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			id=ix*nz+iz;
			
			lame1=s_density_d[in_idx]*velocity_d[in_idx]*velocity_d[in_idx]-2*s_density_d[in_idx]*velocity1_d[in_idx]*velocity1_d[in_idx];
			lame2=s_density_d[in_idx]*velocity1_d[in_idx]*velocity1_d[in_idx];
			
			grad_lame1_d[id]=grad_lame1_d[id]+(txx1_d[in_idx]+tzz1_d[in_idx])*(rtxx1_d[in_idx]+rtzz1_d[in_idx])/(4*(lame1+lame2)*(lame1+lame2))*(-1.0);
		}
}

__global__ void cal_gradient_for_lame2_new(float *grad_lame2_d,float *rtxx1_d,float *rtxz1_d,float *rtzz1_d,float *txx1_d,float *tzz1_d,float *txz1_d,float *velocity_d,float *velocity1_d,float *s_density_d,int boundary_left,int boundary_up,int nx,int nz,int dimx,int dimz)
//cal_gradient_for_lame2_new<<<dimGrid,dimBlock>>>(grad_lame22_d,rtxx1_d,rtxz1_d,rtzz2_d,txx1_d,tzz1_d,txz1_d,s_velocity_d,s_velocity1_d,s_density_d,boundary_left,boundary_up,nx,nz,nx_append,nz_append);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int id;
		float lame1,lame2;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			id=ix*nz+iz;
			
			lame1=s_density_d[in_idx]*velocity_d[in_idx]*velocity_d[in_idx]-2*s_density_d[in_idx]*velocity1_d[in_idx]*velocity1_d[in_idx];
			lame2=s_density_d[in_idx]*velocity1_d[in_idx]*velocity1_d[in_idx];
			
			grad_lame2_d[id]=grad_lame2_d[id]+(1.0*txz1_d[in_idx]*rtxz1_d[in_idx]/(lame2*lame2)+(txx1_d[in_idx]+tzz1_d[in_idx])*(rtxx1_d[in_idx]+rtzz1_d[in_idx])/(4.0*(lame1+lame2)*(lame1+lame2))+(txx1_d[in_idx]-tzz1_d[in_idx])*(rtxx1_d[in_idx]-rtzz1_d[in_idx])/(4.0*lame2*lame2))*(-1.0);			
		}
}

__global__ void cal_gradient_for_vp(float *grad_vp_d,float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up)
//vp<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_lame1_d,grad_lame2_d,grad_den_d,s_velocity_d,s_velocity1_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
			
			 grad_vp_d[in_idx]=2*s_density_d[in_idx1]*s_velocity_d[in_idx1]*grad_lame1_d[in_idx];
		}
}

__global__ void cal_gradient_for_vs(float *grad_vs_d,float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up)
//cal_gradient_for_vs<<<dimGrid,dimBlock>>>(grad_vs1_d,grad_lame1_d,grad_lame2_d,grad_den_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
		
			grad_vs_d[in_idx]=(-4*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*grad_lame1_d[in_idx]+2*s_density_d[in_idx1]*s_velocity1_d[in_idx1]*grad_lame2_d[in_idx]);
		}
}

__global__ void cal_gradient_for_density(float *grad_density_d,float *grad_lame1_d,float *grad_lame2_d,float *grad_den_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up)
//cal_gradient_for_density<<<dimGrid,dimBlock>>>(grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		int in_idx1;

		if((ix<nx)&&(iz<nz))
		{
			in_idx=ix*nz+iz;//iz*nz+ix;
			in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;//iz*nz+ix;
		
			grad_density_d[in_idx]=((s_velocity_d[in_idx1]*s_velocity_d[in_idx1]-2*s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1])*grad_lame1_d[in_idx]+s_velocity1_d[in_idx1]*s_velocity1_d[in_idx1]*grad_lame2_d[in_idx]+grad_den_d[in_idx]);
		}
}

__global__ void invert_lame_to_vp(float *grad_vp1_d,float *grad_lame11_d,float *grad_lame22_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_vp<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_lame11_d,grad_lame22_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;
		
		grad_vp1_d[id]=2*s_density_d[id1]*s_velocity_d[id1]*grad_lame11_d[id];
	}

}

__global__ void invert_lame_to_vs(float *grad_vs1_d,float *grad_lame11_d,float *grad_lame22_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_vs<<<dimGrid,dimBlock>>>(grad_vs1_d,grad_lame11_d,grad_lame22_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;
		
		grad_vs1_d[id]=-4*s_density_d[id1]*s_velocity1_d[id1]*grad_lame11_d[id]+2*s_density_d[id1]*s_velocity1_d[id1]*grad_lame22_d[id];
	}

}

__global__ void invert_lame_to_density(float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_density<<<dimGrid,dimBlock>>>(grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;
		
		grad_density1_d[id]=((s_velocity_d[id1]*s_velocity_d[id1]-2*s_velocity1_d[id1]*s_velocity1_d[id1])*grad_lame11_d[id]+s_velocity1_d[id1]*s_velocity1_d[id1]*grad_lame22_d[id]+grad_den1_d[id]);
	}

}

__global__ void invert_lame_to_velocity_para_new(float *grad_vp1_d,float *grad_vs1_d,float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_velocity_para<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;


		grad_vp1_d[id]=2*s_velocity_d[id1]*s_density_d[id1]*grad_lame11_d[id];

		grad_vs1_d[id]=-4.0*s_velocity1_d[id1]*s_density_d[id1]*grad_lame11_d[id]+2.0*s_velocity1_d[id1]*s_density_d[id1]*grad_lame22_d[id];
		
		grad_density1_d[id]=(-2.0*s_velocity1_d[id1]*s_velocity1_d[id1]+s_velocity_d[id1]*s_velocity_d[id1])*grad_lame11_d[id]+s_velocity1_d[id1]*s_velocity1_d[id1]*grad_lame22_d[id]+grad_den1_d[id];
	}
}

__global__ void invert_lame_to_velocity_para(float *grad_vp1_d,float *grad_vs1_d,float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_velocity_para<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if(ix<nx&&iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		//grad_vp1_d[id]=2*(lame1+2*lame2)*1.0/lame1*grad_lame11_d[id];

		//grad_vs1_d[id]=-4*lame2*1.0/lame1*grad_lame11_d[id]+2*grad_lame22_d[id];

		//grad_density1_d[id]=-1.0*grad_lame11_d[id]-1.0*grad_lame22_d[id]+grad_den1_d[id];

		grad_vp1_d[id]=2.0*(lame1+2*lame2)*grad_lame11_d[id]*1.0/lame1;

		grad_vs1_d[id]=-4.0*lame2*grad_lame11_d[id]*1.0/lame1+2.0*grad_lame22_d[id];
		
		grad_density1_d[id]=grad_lame11_d[id]+grad_lame22_d[id]+grad_den1_d[id];
	}

}

__global__ void invert_lame_to_velocity_vp(float *grad_vp1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_velocity_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if(ix<nx&&iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		grad_vp1_d[id]=2.0*(lame1+2*lame2)*grad_lame11_d[id]*1.0/lame1;
	}

}

__global__ void invert_lame_to_velocity_vs(float *grad_vs1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_velocity_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if(ix<nx&&iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		grad_vs1_d[id]=-4.0*lame2*grad_lame11_d[id]*1.0/lame1+2.0*grad_lame22_d[id];
	}

}

__global__ void invert_lame_to_velocity_density(float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_velocity_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;
	//int id1;

	//float lame1,lame2;

	if(ix<nx&&iz<nz)
	{
		id=ix*nz+iz;
		//id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		//lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		//lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		
		grad_density1_d[id]=grad_lame11_d[id]+grad_lame22_d[id]+grad_den1_d[id];
	}

}

__global__ void invert_lame_to_impedance_para_new(float *grad_vp1_d,float *grad_vs1_d,float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
//invert_lame_to_impedance_para_new<<<dimGrid_new,dimBlock>>>(all_grad_vp1_d,all_grad_vs1_d,all_grad_density1_d,all_grad_lame11_d,all_grad_lame22_d,all_grad_den1_d,s_velocity_all_d,s_velocity1_all_d,s_density_all_d,nx,nz,nx_append_new,nz_append,boundary_left,boundary_up);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;


		grad_vp1_d[id]=2*s_velocity_d[id1]*grad_lame11_d[id];

		grad_vs1_d[id]=-4.0*s_velocity1_d[id1]*grad_lame11_d[id]+2.0*s_velocity1_d[id1]*grad_lame22_d[id];
		
		grad_density1_d[id]=(2.0*s_velocity1_d[id1]*s_velocity1_d[id1]-s_velocity_d[id1]*s_velocity_d[id1])*grad_lame11_d[id]-s_velocity1_d[id1]*s_velocity1_d[id1]*grad_lame22_d[id]+grad_den1_d[id];
	}
}


__global__ void invert_lame_to_impedance_para(float *grad_vp1_d,float *grad_vs1_d,float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_impedance_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left);;
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		grad_vp1_d[id]=2*(lame1+2*lame2)*1.0/lame1*grad_lame11_d[id];

		grad_vs1_d[id]=-4*lame2*1.0/lame1*grad_lame11_d[id]+2*grad_lame22_d[id];
		
		grad_density1_d[id]=-1.0*grad_lame11_d[id]-1.0*grad_lame22_d[id]+grad_den1_d[id];
	}
}

__global__ void invert_lame_to_impedance_vp(float *grad_vp1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_impedance_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left);;
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		grad_vp1_d[id]=2*(lame1+2*lame2)*1.0/lame1*grad_lame11_d[id];
	}
}

__global__ void invert_lame_to_impedance_vs(float *grad_vs1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_impedance_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left);;
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	float lame1,lame2;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];

		grad_vs1_d[id]=-4*lame2*1.0/lame1*grad_lame11_d[id]+2*grad_lame22_d[id];
	}
}

__global__ void invert_lame_to_impedance_density(float *grad_density1_d,float *grad_lame11_d,float *grad_lame22_d,float *grad_den1_d,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
///invert_lame_to_impedance_para<<<dimGrid,dimBlock>>>(grad_vp1_d,grad_vs1_d,grad_density1_d,grad_lame11_d,grad_lame22_d,grad_den1_d,s_velocity_d,s_velocity1_d,s_density_d,nx,nz,nx_append,nz_append,boundary_left);;
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;
	//int id1;

	//float lame1,lame2;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		//id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		//lame1=s_density_d[id1]*s_velocity_d[id1]*s_velocity_d[id1]-2*s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		//lame2=s_density_d[id1]*s_velocity1_d[id1]*s_velocity1_d[id1];
		
		grad_density1_d[id]=-1.0*grad_lame11_d[id]-1.0*grad_lame22_d[id]+grad_den1_d[id];
	}
}

__global__ void cuda_cal_residuals_new(float *res_shot_x_d,float *cal_shot_x_d,float *obs_shot_x_d,int receiver_num,int lt)
///cuda_cal_residuals_new<<<dimGrid_lt,dimBlock>>>(res_shot_x_d,cal_shot_x_d,obs_shot_x_d,receiver_num,lt);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<receiver_num)&&(iz<lt))
		{
			in_idx=ix*lt+iz;
			
			res_shot_x_d[in_idx]=cal_shot_x_d[in_idx]-obs_shot_x_d[in_idx];
		}
}

__global__ void scale_cal_shot(float *cal_shot_x_d,float *cal_max,float *obs_max,int receiver_num,int lt)
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;

		if((ix<receiver_num)&&(iz<lt))
		{
			in_idx=ix*lt+iz;

			if(cal_max[0]!=0)	cal_shot_x_d[in_idx]=cal_shot_x_d[in_idx]*obs_max[0]/cal_max[0]*1.0;
		}
}

__global__ void cuda_cal_beta(float *beta, float *g0, float *g1, float *cg, int N)
/*< calculate beta for nonlinear conjugate gradient algorithm 
configuration requirement: <<<1,Block_Size>>> >*/
{
    	__shared__ float sdata[Block_Size];
	__shared__ float tdata[Block_Size];
	__shared__ float rdata[Block_Size];
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
	tdata[tid] = 0.0f;
	rdata[tid] = 0.0f;
	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?g0[id]:0.0f;
		float b=(id<N)?g1[id]:0.0f;
		//float c=(id<N)?cg[id]:0.0f;

		/* HS: Hestenses-Stiefel NLCG algorithm */
/*
		sdata[tid] += b*(b-a);	// numerator of HS
		tdata[tid] += c*(b-a);	// denominator of HS,DY
		rdata[tid] += b*b;	// numerator of DY
*/
  	
		// PRP: Polark-Ribiere-Polyar NLCG algorithm 

		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += a*a;	// denominator

		// HS: Hestenses-Stiefel NLCG algorithm 
/*
		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += c*(b-a);	// denominator
*/
		// FR: Fletcher-Reeves NLCG algorithm 
/*
		sdata[tid] += b*b;	// numerator
		tdata[tid] += a*a;	// denominator
*/
		// PRP: Polark-Ribiere-Polyar NLCG algorithm 
/*
		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += a*a;	// denominator
*/
		// CD: Fletcher NLCG algorithm  
/*
		sdata[tid] += b*b;	// numerator
		tdata[tid] -= c*a;	// denominator
*/
		// DY: Dai-Yuan NLCG algorithm 
/*
		sdata[tid] += b*b;	// numerator
		tdata[tid] += c*(b-a);	// denominator
*/
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{ sdata[tid]+=sdata[tid+s]; tdata[tid]+=tdata[tid+s]; rdata[tid]+=rdata[tid+s];}
		__syncthreads();
    	}     
   	if (tid < 32)
   	{
		if (blockDim.x >=64) { sdata[tid]+=sdata[tid+32]; tdata[tid]+=tdata[tid+32]; rdata[tid]+=rdata[tid+32];}
		if (blockDim.x >=32) { sdata[tid]+=sdata[tid+16]; tdata[tid]+=tdata[tid+16]; rdata[tid]+=rdata[tid+16];}
		if (blockDim.x >=16) { sdata[tid]+=sdata[tid+ 8]; tdata[tid]+=tdata[tid+ 8]; rdata[tid]+=rdata[tid+ 8];}
		if (blockDim.x >= 8) { sdata[tid]+=sdata[tid+ 4]; tdata[tid]+=tdata[tid+ 4]; rdata[tid]+=rdata[tid+ 4];}
		if (blockDim.x >= 4) { sdata[tid]+=sdata[tid+ 2]; tdata[tid]+=tdata[tid+ 2]; rdata[tid]+=rdata[tid+ 2];}
		if (blockDim.x >= 2) { sdata[tid]+=sdata[tid+ 1]; tdata[tid]+=tdata[tid+ 1]; rdata[tid]+=rdata[tid+ 1];}
    	}
     
	if (tid == 0) 
	{ 
		//float beta_HS=0.0;
		//float beta_DY=0.0;
		float beta_PRP=0.0;
		if(fabsf(tdata[0])>EPS) 
		{
			//beta_HS=sdata[0]/tdata[0]; 
			//beta_DY=rdata[0]/tdata[0];

			beta_PRP=sdata[0]/tdata[0];
		} 
		//*beta=max(0.0, min(beta_HS, beta_DY));/* Hybrid HS-DY method combined with iteration restart */

		*beta=beta_PRP;/* Hybrid HS-DY method combined with iteration restart */
	}	
}

__global__ void cuda_cal_beta_new(float *beta, float *g0, float *g1, float *cg, int N,int mark)
///cuda_cal_beta_new<<<1, Block_Size>>>(beta_d, grad_lame1_d, grad_lame11_d, conj_lame1_d, nxnz,0);
/*< calculate beta for nonlinear conjugate gradient algorithm 
configuration requirement: <<<1,Block_Size>>> >*/
{
    	__shared__ float sdata[Block_Size];
	__shared__ float tdata[Block_Size];
	__shared__ float rdata[Block_Size];
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
	tdata[tid] = 0.0f;
	rdata[tid] = 0.0f;

	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?g0[id]:0.0f;
		float b=(id<N)?g1[id]:0.0f;
		float c=(id<N)?cg[id]:0.0f;

		/* HS: Hestenses-Stiefel NLCG algorithm */

		sdata[tid] += b*(b-a);	// numerator of HS
		tdata[tid] += c*(b-a);	// denominator of HS,DY
		rdata[tid] += b*b;	// numerator of DY

		
/*   	
		// PRP: Polark-Ribiere-Polyar NLCG algorithm 
		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += a*a;	// denominator
		// HS: Hestenses-Stiefel NLCG algorithm 
		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += c*(b-a);	// denominator
		// FR: Fletcher-Reeves NLCG algorithm 
		sdata[tid] += b*b;	// numerator
		tdata[tid] += a*a;	// denominator
*/

/*
		// PRP: Polark-Ribiere-Polyar NLCG algorithm 
		sdata[tid] += b*(b-a);	// numerator
		tdata[tid] += a*a;	// denominator
*/

/*
		// CD: Fletcher NLCG algorithm  
		sdata[tid] += b*b;	// numerator
		tdata[tid] -= c*a;	// denominator
*/
/*
		// DY: Dai-Yuan NLCG algorithm 
		sdata[tid] += b*b;	// numerator
		tdata[tid] += c*(b-a);	// denominator
*/
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{ sdata[tid]+=sdata[tid+s]; tdata[tid]+=tdata[tid+s]; rdata[tid]+=rdata[tid+s];}
		__syncthreads();
    	}     
   	if (tid < 32)
   	{
		if (blockDim.x >=64) { sdata[tid]+=sdata[tid+32]; tdata[tid]+=tdata[tid+32]; rdata[tid]+=rdata[tid+32];}
		if (blockDim.x >=32) { sdata[tid]+=sdata[tid+16]; tdata[tid]+=tdata[tid+16]; rdata[tid]+=rdata[tid+16];}
		if (blockDim.x >=16) { sdata[tid]+=sdata[tid+ 8]; tdata[tid]+=tdata[tid+ 8]; rdata[tid]+=rdata[tid+ 8];}
		if (blockDim.x >= 8) { sdata[tid]+=sdata[tid+ 4]; tdata[tid]+=tdata[tid+ 4]; rdata[tid]+=rdata[tid+ 4];}
		if (blockDim.x >= 4) { sdata[tid]+=sdata[tid+ 2]; tdata[tid]+=tdata[tid+ 2]; rdata[tid]+=rdata[tid+ 2];}
		if (blockDim.x >= 2) { sdata[tid]+=sdata[tid+ 1]; tdata[tid]+=tdata[tid+ 1]; rdata[tid]+=rdata[tid+ 1];}
    	}
     
	if (tid == 0) 
	{ 
		float beta_HS=0.0;
		float beta_DY=0.0;
		//float beta_PRP=0.0;
		if(fabsf(tdata[0])>EPS) 
		{
			beta_HS=sdata[0]/tdata[0]; 
			beta_DY=rdata[0]/tdata[0];
			//beta_PRP=sdata[0]/tdata[0];
		}
		//*beta=max(0.0, min(beta_HS, beta_DY));/* Hybrid HS-DY method combined with iteration restart */ 
		beta[mark]=max(0.0, min(beta_HS, beta_DY));/* Hybrid HS-DY method combined with iteration restart */

		//beta[1]=beta_PRP;/* PRP method combined with iteration restart */
		//beta[mark]=beta_PRP;
	}	
}

__global__ void cuda_cal_conjgrad(float *g1, float *cg, float beta, int nz, int nx)
/*< calculate nonlinear conjugate gradient >*/
{
	int i1=blockIdx.x*blockDim.x+threadIdx.x;
	int i2=blockIdx.y*blockDim.y+threadIdx.y;
	int id=i1+i2*nz;

	if (i1<nz && i2<nx) cg[id]=-g1[id]+beta*cg[id];
}

__global__ void cuda_cal_conjgrad_new(float *g1, float *cg, float *beta, int nx, int nz,int mark)
///cuda_cal_conjgrad_new<<<dimGrid,dimBlock>>>(grad_lame22_d, conj_lame2_d, beta_d, nx, nz,1);
//cuda_cal_conjgrad_new<<<dimGrid,dimBlock>>>(grad_vp1_d, conj_vp_d, beta_d, nx, nz,0);
/*< calculate nonlinear conjugate gradient >*/
{
	int ix=blockIdx.x*blockDim.x+threadIdx.x;
	int iz=blockIdx.y*blockDim.y+threadIdx.y;

	int id=ix*nz+iz;

	if (ix<nx && iz<nz) cg[id]=-1.0*g1[id]+beta[mark]*cg[id];
}

__global__ void cuda_cal_window(float *s_velocity_d,float *v_window_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
////cuda_cal_window<<<dimGrid,dimBlock>>>(expand_perturb_lame1_d,v_window_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
{
	int ix=blockIdx.x*blockDim.x+threadIdx.x;
	int iz=blockIdx.y*blockDim.y+threadIdx.y;

	int id=ix*nz+iz;
	int id1=(ix+boundary_left)*nz_append+iz+boundary_up;

	if (ix<nx && iz<nz) 	v_window_d[id]=s_velocity_d[id1];
}

//cuda_cal_expand<<<dimGrid,dimBlock>>>(s_velocity_d,v_window_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up);
__global__ void cuda_cal_expand(float *s_velocity_d,float *v_window_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
{
	int ix=blockIdx.x*blockDim.x+threadIdx.x;
	int iz=blockIdx.y*blockDim.y+threadIdx.y;

	//int id=ix*nz+iz;

	int id0=(ix-boundary_left)*nz+iz-boundary_up;

	//int id1=(ix+boundary_left)*nz_append+iz+boundary_up;

	int id2=ix*nz_append+iz;

	if (ix<nx_append && iz<nz_append)
		{
			if(ix>=boundary_left&&ix<boundary_left+nx&&iz>=boundary_up&&iz<boundary_up+nz)	s_velocity_d[id2]=v_window_d[id0];
///up		
			if(ix>=boundary_left&&ix<boundary_left+nx&&iz<boundary_up)			s_velocity_d[id2]=v_window_d[(ix-boundary_left)*nz];
///down
			if(ix>=boundary_left&&ix<boundary_left+nx&&iz>=nz+boundary_up)			s_velocity_d[id2]=v_window_d[(ix-boundary_left)*nz+nz-1];
//left
			if(ix<boundary_left&&iz>=boundary_up&&iz<boundary_up+nz)				s_velocity_d[id2]=v_window_d[0*nz+iz-boundary_up];
//right
			if(ix>=nx+boundary_left&&iz>=boundary_up&&iz<boundary_up+nz)			s_velocity_d[id2]=v_window_d[(nx-1)*nz+iz-boundary_up];
//up left
			if(ix<boundary_left&&iz<boundary_up)						s_velocity_d[id2]=v_window_d[0];
//up right
			if(ix>=nx+boundary_left&&iz<boundary_up)						s_velocity_d[id2]=v_window_d[(nx-1)*nz];
//down left
			if(ix<boundary_left&&iz>=nz+boundary_up)						s_velocity_d[id2]=v_window_d[nz-1];
//down right
			if(ix>=nx+boundary_left&&iz>=nz+boundary_up)					s_velocity_d[id2]=v_window_d[(nx-1)*nz+nz-1];
		}
}

__global__ void cuda_cal_epsilon(float *vv, float *cg, float *epsil, int N)
/*< calculate estimated stepsize (epsil) according to Taratola's method
configuration requirement: <<<1, Block_Size>>> >*/ 
{
    	__shared__ float sdata[Block_Size];/* find max(|vv(:)|) */
	__shared__ float tdata[Block_Size];/* find max(|cg(:)|) */
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
    	tdata[tid] = 0.0f;
	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?fabsf(vv[id]):0.0f;
		float b=(id<N)?fabsf(cg[id]):0.0f;
		sdata[tid]= max(sdata[tid], a);
		tdata[tid]= max(tdata[tid], b);
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{sdata[tid]=max(sdata[tid], sdata[tid+s]);tdata[tid]=max(tdata[tid], tdata[tid+s]);} 
		__syncthreads();
    	}  
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] =max(sdata[tid],sdata[tid + 32]);tdata[tid]=max(tdata[tid], tdata[tid+32]);}
		if (blockDim.x >=  32) { sdata[tid] =max(sdata[tid],sdata[tid + 16]);tdata[tid]=max(tdata[tid], tdata[tid+16]);}
		if (blockDim.x >=  16) { sdata[tid] =max(sdata[tid],sdata[tid + 8]);tdata[tid]=max(tdata[tid], tdata[tid+8]);}
		if (blockDim.x >=   8) { sdata[tid] =max(sdata[tid],sdata[tid + 4]);tdata[tid]=max(tdata[tid], tdata[tid+4]);}
		if (blockDim.x >=   4) { sdata[tid] =max(sdata[tid],sdata[tid + 2]);tdata[tid]=max(tdata[tid], tdata[tid+2]);}
		if (blockDim.x >=   2) { sdata[tid] =max(sdata[tid],sdata[tid + 1]);tdata[tid]=max(tdata[tid], tdata[tid+1]);}
    	}

    	if (tid == 0) { if(tdata[0]>EPS) *epsil=0.01*sdata[0]/tdata[0]; else *epsil=0.0;}
}


__global__ void cuda_cal_epsilon_new(float *vv, float *cg, float *epsil, int N,int mark)
/*< calculate estimated stepsize (epsil) according to Taratola's method
configuration requirement: <<<1, Block_Size>>> >*/ 
{
    	__shared__ float sdata[Block_Size];/* find max(|vv(:)|) */
	__shared__ float tdata[Block_Size];/* find max(|cg(:)|) */
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
    	tdata[tid] = 0.0f;
	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?fabsf(vv[id]):0.0f;
		float b=(id<N)?fabsf(cg[id]):0.0f;
		sdata[tid]= max(sdata[tid], a);
		tdata[tid]= max(tdata[tid], b);
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{sdata[tid]=max(sdata[tid], sdata[tid+s]);tdata[tid]=max(tdata[tid], tdata[tid+s]);} 
		__syncthreads();
    	}  
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] =max(sdata[tid],sdata[tid + 32]);tdata[tid]=max(tdata[tid], tdata[tid+32]);}
		if (blockDim.x >=  32) { sdata[tid] =max(sdata[tid],sdata[tid + 16]);tdata[tid]=max(tdata[tid], tdata[tid+16]);}
		if (blockDim.x >=  16) { sdata[tid] =max(sdata[tid],sdata[tid + 8]);tdata[tid]=max(tdata[tid], tdata[tid+8]);}
		if (blockDim.x >=   8) { sdata[tid] =max(sdata[tid],sdata[tid + 4]);tdata[tid]=max(tdata[tid], tdata[tid+4]);}
		if (blockDim.x >=   4) { sdata[tid] =max(sdata[tid],sdata[tid + 2]);tdata[tid]=max(tdata[tid], tdata[tid+2]);}
		if (blockDim.x >=   2) { sdata[tid] =max(sdata[tid],sdata[tid + 1]);tdata[tid]=max(tdata[tid], tdata[tid+1]);}
    	}

    	if (tid == 0) { if(tdata[0]>EPS) epsil[mark]=0.01*sdata[0]/tdata[0]; else epsil[mark]=0.0;}

	//if (tid == 0) { if(tdata[0]>EPS) epsil[mark]=0.0; else epsil[mark]=0.0;}
}

__global__ void cuda_cal_vtmp(float *vtmp, float *vv, float *cg, float epsil, int nz, int nx, int window_vel)
/*< calculate temporary velocity >*/ 
{
	int i1=threadIdx.x+blockIdx.x*blockDim.x;
	int i2=threadIdx.y+blockIdx.y*blockDim.x;
	int id=i1+i2*nz;

	//if (i1<nz && i1>=window_vel && i2<nx)	vtmp[id]=vv[id]+epsil*cg[id];

	if (i1<nz && i2<nx)	vtmp[id]=vv[id]+epsil*cg[id];
}

__global__ void cuda_cal_vtmp_new(float *vtmp, float *vv, float *cg, float *epsil, int nx, int nz,int nx_append,int nz_append,int boundary_left,int boundary_up,int mark)
//cuda_cal_vtmp_new<<<dimGrid,dimBlock>>>(v_window_d,s_velocity_d, conj_vp_d, epsil_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up,0);
//cuda_cal_vtmp_new<<<dimGrid,dimBlock>>>(v_window_d,s_density_d,conj_density_d,epsil_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up,1);
/*< calculate temporary velocity >*/ 
{
	//int i1=threadIdx.x+blockIdx.x*blockDim.x;
	//int i2=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		vtmp[id]=vv[id1]+epsil[mark]*cg[id];

		if(iz<1)	vtmp[id]=vv[id1];
	}
}

__global__ void cuda_cal_vtmp_new_new(float *vtmp, float *vv, float *cg, float *epsil, int nx, int nz,int nx_append,int nz_append,int boundary_left,int boundary_up)
/*< calculate temporary velocity >*/ 
{
	//int i1=threadIdx.x+blockIdx.x*blockDim.x;
	//int i2=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		if(epsil[1]>epsil[0])	vtmp[id]=vv[id1]+cg[id];

		if(iz<1)	vtmp[id]=vv[id1];
	}
}

__global__ void cuda_cal_vtmp_fixed(float *vtmp, float *vv, float *cg, int nx, int nz,int nx_append,int nz_append,int boundary_left,int boundary_up,float mark)
//cuda_cal_vtmp_fixed<<<dimGrid,dimBlock>>>(v_window_d,s_velocity_d, conj_vp_d, epsil_d,nx,nz,nx_append,nz_append,boundary_left,boundary_up,0);
/*< calculate temporary velocity >*/ 
{
	//int i1=threadIdx.x+blockIdx.x*blockDim.x;
	//int i2=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id,id1;

	if (ix<nx && iz<nz)
	{
		id=ix*nz+iz;
		id1=(ix+boundary_left)*nz_append+iz+boundary_up;

		vtmp[id]=vv[id1]+mark*cg[id];

		if(iz<1)	vtmp[id]=vv[id1];
	}
}


__global__ void cuda_update_vel_new(float *vv, float *cg, float *alpha, int nx, int nz, int mark)
/*< update velocity model with obtained stepsize (alpha) >*/
{
	//int ix=threadIdx.x+blockIdx.x*blockDim.x;
	//int iz=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	int id=iz+ix*nz;

	//if (i1<nz && i1>=window_vel && i2<nx) vv[id]=vv[id]+alpha*cg[id];
	if (ix<nx && iz<nz&&iz>1) vv[id]=vv[id]+alpha[mark]*cg[id];
}

__global__ void cuda_sum_alpha12(float *alpha1, float *alpha2, float *dcaltmp, float *dobs, float *derr, int ng)
//cuda_sum_alpha12<<<(ng+511)/512, 512>>>(d_alpha1, d_alpha2, d_dcal, &d_dobs[it*ng], &d_derr[it*ng], ng);
/*< calculate the numerator and denominator of alpha
	alpha1: numerator; length=ng
	alpha2: denominator; length=ng >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	if(id<ng) { 
		float c=derr[id];
		float a=dobs[id]+c;/* since f(mk)-dobs[id]=derr[id], thus f(mk)=b+c; */
		float b=dcaltmp[id]-a;/* f(mk+epsil*cg)-f(mk) */
		alpha1[id]-=b*c; alpha2[id]+=b*b; 
	}
}

__global__ void cuda_sum_alpha12_new(float *alpha1, float *alpha2, float *dcaltmp, float *dobs, float *derr, int ng,int lt)
//cuda_sum_alpha12_new<<<dimGrid_lt,dimBlock>>>(d_alpha1,d_alpha2,cal_shot_d,obs_shot_d,res_shot_d,receiver_num,lt);
/*< calculate the numerator and denominator of alpha
	alpha1: numerator; length=ng
	alpha2: denominator; length=ng >*/
{
	int id;
	//int id=threadIdx.x+blockDim.x*blockIdx.x;
	//int ix=threadIdx.x+blockIdx.x*blockDim.x;
	//int iz=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	if(ix<ng&&iz<lt) 
	{ 
		id=ix*lt+iz;		

		float c=derr[id];
		float a=dobs[id]+c;/* since f(mk)-dobs[id]=derr[id], thus f(mk)=b+c; */
		float b=dcaltmp[id]-a;/* f(mk+epsil*cg)-f(mk) */
		alpha1[id]+=b*c; alpha2[id]+=b*b; 
	}
}

__global__ void cuda_sum_alpha12_new_for_lsrtm(float *alpha1, float *alpha2, float *dcaltmp, float *dobs, float *derr, int ng,int lt)
/*< calculate the numerator and denominator of alpha
	alpha1: numerator; length=ng
	alpha2: denominator; length=ng >*/
{
	int id;
	//int id=threadIdx.x+blockDim.x*blockIdx.x;
	//int ix=threadIdx.x+blockIdx.x*blockDim.x;
	//int iz=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	if(ix<ng&&iz<lt) 
	{ 
		id=ix*lt+iz;		

		float c=derr[id];
		//float a=dobs[id]+c;/* since f(mk)-dobs[id]=derr[id], thus f(mk)=b+c; */
		float b=dcaltmp[id];/* f(mk+epsil*cg)-f(mk) */
		alpha1[id]-=b*c; alpha2[id]+=b*b; 
	}
}

__global__ void cuda_cal_alpha(float *alpha, float *alpha1, float *alpha2, float epsil, int ng)
/*< calculate searched stepsize (alpha) according to Taratola's method
configuration requirement: <<<1, Block_Size>>> >*/ 
{
  	__shared__ float sdata[Block_Size];
	__shared__ float tdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	tdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?alpha1[id]:0.0f;
		float b=(id<ng)?alpha2[id]:0.0f;
		sdata[tid] +=a;	
		tdata[tid] +=b;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) { sdata[tid] += sdata[tid + s];tdata[tid] += tdata[tid + s]; } __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; tdata[tid] += tdata[tid + 32];}
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; tdata[tid] += tdata[tid + 16];}
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; tdata[tid] += tdata[tid +  8];}
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; tdata[tid] += tdata[tid +  4];}
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; tdata[tid] += tdata[tid +  2];}
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; tdata[tid] += tdata[tid +  1];}
    	}
     
    	if (tid == 0) { if(tdata[0]>EPS) *alpha=epsil*sdata[0]/(tdata[0]+EPS); else *alpha=0.0;}
}

__global__ void cuda_cal_alpha_new(float *alpha, float *alpha1, float *alpha2, float *epsil, int ng,int mark)
/*< calculate searched stepsize (alpha) according to Taratola's method
configuration requirement: <<<1, Block_Size>>> >*/ 
{
  	__shared__ float sdata[Block_Size];
	__shared__ float tdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	tdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?alpha1[id]:0.0f;
		float b=(id<ng)?alpha2[id]:0.0f;
		sdata[tid] +=a;	
		tdata[tid] +=b;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) { sdata[tid] += sdata[tid + s];tdata[tid] += tdata[tid + s]; } __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; tdata[tid] += tdata[tid + 32];}
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; tdata[tid] += tdata[tid + 16];}
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; tdata[tid] += tdata[tid +  8];}
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; tdata[tid] += tdata[tid +  4];}
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; tdata[tid] += tdata[tid +  2];}
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; tdata[tid] += tdata[tid +  1];}
    	}
     
    	if (tid == 0) { if(tdata[0]>EPS) alpha[mark]=-1.0*epsil[mark]*sdata[0]/(tdata[0]+EPS); else alpha[mark]=0.0;}
}

__global__ void cuda_cal_alpha_new_for_lsrtm(float *alpha, float *alpha1, float *alpha2, float *epsil, int ng,int mark)
/*< calculate searched stepsize (alpha) according to Taratola's method
configuration requirement: <<<1, Block_Size>>> >*/ 
{
  	__shared__ float sdata[Block_Size];
	__shared__ float tdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	tdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?alpha1[id]:0.0f;
		float b=(id<ng)?alpha2[id]:0.0f;
		sdata[tid] +=a;	
		tdata[tid] +=b;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) { sdata[tid] += sdata[tid + s];tdata[tid] += tdata[tid + s]; } __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; tdata[tid] += tdata[tid + 32];}
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; tdata[tid] += tdata[tid + 16];}
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; tdata[tid] += tdata[tid +  8];}
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; tdata[tid] += tdata[tid +  4];}
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; tdata[tid] += tdata[tid +  2];}
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; tdata[tid] += tdata[tid +  1];}
    	}
     
    	//if (tid == 0) { if(tdata[0]>EPS) alpha[mark]=sdata[0]/(tdata[0]+EPS); else *alpha=0.0;}
	
	if (tid == 0) {alpha[mark]=1.0*sdata[0]/(tdata[0]);}
}

__global__ void cuda_cal_alpha_new_for_correlation_lsrtm(float *alpha,float *correlation_parameter_d,int mark)
{
	float a,b;
	float tmp_tmp,obs_obs,cal_cal,tmp_obs,tmp_cal,cal_obs;

	tmp_tmp=correlation_parameter_d[0];///////////////#p*#p

	obs_obs=correlation_parameter_d[1];///////////////p_obs*p_obs

	cal_cal=correlation_parameter_d[2];///////////////p_p*p_p

	tmp_obs=correlation_parameter_d[3];///////////////#p*p_obs

	tmp_cal=correlation_parameter_d[4];///////////////#p*p_p

	cal_obs=correlation_parameter_d[5];///////////////p_p*_obs

	
	//a=1.0/obs_obs/tmp_tmp*(	(2.0*cal_obs*tmp_cal+tmp_obs*cal_cal)/tmp_tmp-3.0*tmp_obs*tmp_cal*tmp_cal/tmp_tmp/tmp_tmp	);

	//b=1.0/obs_obs/tmp_tmp*(tmp_obs*tmp_cal/tmp_tmp-cal_obs);

	a=1.0/sqrt(1.0*obs_obs*tmp_tmp)*(	(2.0*cal_obs*tmp_cal+tmp_obs*cal_cal)/tmp_tmp-3.0*tmp_obs*tmp_cal*tmp_cal/tmp_tmp/tmp_tmp	);

	b=1.0/sqrt(1.0*obs_obs*tmp_tmp)*(tmp_obs*tmp_cal/tmp_tmp-cal_obs);


	alpha[mark]=-1.0*b/a;
}

__global__ void cal_hydrid_conj(float *hydrid_conj_d,float *conj_vp_d,float *conj_vs_d,float *conj_density_d,float *beta_step,int nx,int nz)
{
	int id;
	//int id=threadIdx.x+blockDim.x*blockIdx.x;
	//int ix=threadIdx.x+blockIdx.x*blockDim.x;
	//int iz=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	if(ix<nx&&iz<nz) 
	{ 
		id=ix*nz+iz;	
		hydrid_conj_d[id]=beta_step[0]*conj_vp_d[id]+beta_step[1]*conj_vs_d[id]+beta_step[2]*conj_density_d[id];
	}
}

__global__ void cuda_update_vel(float *vv, float *cg, float alpha, int nx, int nz)
/*< update velocity model with obtained stepsize (alpha) >*/
{
	//int ix=threadIdx.x+blockIdx.x*blockDim.x;
	//int iz=threadIdx.y+blockIdx.y*blockDim.x;
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	int id=iz+ix*nz;

	//if (i1<nz && i1>=window_vel && i2<nx) vv[id]=vv[id]+alpha*cg[id];
	if (ix<nx && iz<nz) vv[id]=vv[id]+alpha*cg[id];
}

__global__ void cuda_update_shots(float *res_shot_x_d,float *cal_shot_x_d,float *beta_step_d,int receiver_num,int lt,int mark)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;

	if (ix<receiver_num && iz<lt)
	{
		id=ix*lt+iz;
		
		res_shot_x_d[id]=res_shot_x_d[id]+beta_step_d[mark]*cal_shot_x_d[id];
	}
}

__global__ void cuda_update_shots_new(float *res_shot_x_d,float *cal_shot_x_d,float *beta_step_d,int receiver_num,int lt,int mark,int precon_z2)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;

	if (ix<receiver_num && iz<lt)
	{
		id=ix*lt+iz;
		if(iz>precon_z2)	res_shot_x_d[id]=1.0*res_shot_x_d[id]+beta_step_d[mark]*cal_shot_x_d[id];

		else			res_shot_x_d[id]=0.01*res_shot_x_d[id]+beta_step_d[mark]*cal_shot_x_d[id];
	}
}

__global__ void cuda_update_tmp_shots(float *tmp_shot_x_d,float *cal_shot_x_d,float *beta_step_d,int receiver_num,int lt,int mark)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;

	if (ix<receiver_num && iz<lt)
	{
		id=ix*lt+iz;
		
		tmp_shot_x_d[id]=tmp_shot_x_d[id]+beta_step_d[mark]*cal_shot_x_d[id];
	}
}

__global__ void cuda_update_shots_and_image(float *cal_shot_x_d,float *res_shot_x1_d,float *res_shot_x2_d,float *cg_parameter_d,int receiver_num,int lt,int mark)
//cuda_update_shots_and_image<<<dimGrid_lt,dimBlock>>>(cal_shot_x_d,res_shot_x1_d,res_shot_x2_d,cg_parameter_d,receiver_num,lt);
//cuda_update_shots_and_image<<<dimGrid_lt,dimBlock>>>(cal_shot_z_d,res_shot_z1_d,res_shot_z2_d,cg_parameter_d,receiver_num,lt);
//cuda_update_shots_and_image<<<dimGrid,dimBlock>>>(grad_lame11_d,perturb_lame1_d,perturb_lame11_d,cg_parameter_d,nx,nz);						
//cuda_update_shots_and_image<<<dimGrid,dimBlock>>>(grad_lame22_d,perturb_lame2_d,perturb_lame22_d,cg_parameter_d,nx,nz);

{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  	
	int id;

	if (ix<receiver_num && iz<lt)
	{
		id=ix*lt+iz;
		if(mark==0)
		{
			res_shot_x2_d[id]=(cg_parameter_d[5]*cal_shot_x_d[id]+cg_parameter_d[6]*res_shot_x1_d[id]);
		}
		if(mark==1)
		{
			res_shot_x2_d[id]=-1.0*(cg_parameter_d[5]*cal_shot_x_d[id]+cg_parameter_d[6]*res_shot_x1_d[id]);
		}
		
	}
}
__global__ void cuda_scale_gradient_new(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;
	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrt(d_illum[in_idx1]+EPS));

		grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(d_illum[in_idx1]+EPS);
	}
}

__global__ void cuda_scale_gradient_new_1(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;

	//float m=0.0;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrtf(d_illum[in_idx1]+EPS));

		grad_vp1_d[in_idx]=scale*grad_vp1_d[in_idx]/(d_illum[in_idx1]+EPS);

		/*if(z1!=z2)		
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(iz>=z1&&iz<=z2)
			{		
				m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m));
			}
		}*/
	}
}

__global__ void cuda_attenuation_after_lap(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int z1,int z2)
{
/////////////////////2017年03月25日 星期六 21时32分15秒 仔细想一想，应该先laplace  再衰减
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	//int in_idx1;

	float m=0.0;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		//in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(z1!=z2&&z2!=0)		
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(iz>=z1&&iz<=z2)
			{		
				m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m));
			}
		}
	}
}

__global__ void cuda_attenuation_after_lap_new(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int z1,int z2)
{
/////////////////////2017年03月25日 星期六 21时32分15秒 仔细想一想，应该先laplace  再衰减
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	//int in_idx1;

	float m=0.0;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		//in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(z1!=z2&&z2!=0)		
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(iz>=z1&&iz<=z2)
			{		
				m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m));
			}
		}

		if(ix==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(ix==nx-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;
		
		if(iz==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(iz==nz-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;	
	}
}

__global__ void cuda_attenuation_after_lap_new1(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int z1,int z2)
{
/////////////////////2017年03月25日 星期六 21时32分15秒 仔细想一想，应该先laplace  再衰减
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	//int in_idx1;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		//in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		
		if(iz<=z1)
		{
			grad_vp1_d[in_idx]=0.0;
		}

		if(iz>=z1&&iz<=z2)
		{		
			m=1.0*(z2-iz)/(z2-z1);
			change=pow(cos(pai/2*m),3);
				
			grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change*1.0;
		}
		

		if(ix==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(ix==nx-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;
		
		if(iz==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(iz==nz-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;	
	}
}

__global__ void cuda_attenuation_after_lap_new2(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int z1,int z2)
{
/////////////////////2017年03月25日 星期六 21时32分15秒 仔细想一想，应该先laplace  再衰减
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	//int in_idx1;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;
		//in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(z1==0)
		{
			if(iz<=z1)
			{
				grad_vp1_d[in_idx]=0.0;
			}

			if(iz>=z1&&iz<=z2)
			{		
				m=1.0*(z2-iz)/(z2-z1);
				change=pow(cos(pai/2*m),3);
				
				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change*1.0;
			}
		}

		else
		{
			if(z1!=z2&&z2!=0)		
			{
				if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
				if(iz>=z1&&iz<=z2)
				{		
					m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m));
				}
			}
		}

		if(ix==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(ix==nx-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;
		
		if(iz==0)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;

		if(iz==nz-1)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*0.6666;	
	}
}

__global__ void cuda_attenuation_adj(float *adj_shot_x_d,int receiver_num,int lt,int offset_left,int offset_right,int receiver_offset)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	float m0=0.0;
	
	double change;

	int distance_left;
	int distance_right;

	/*float p0=0.1,p1=0.9;

	float angle0,angle1;

	angle0=float(acos(sqrt(1.0*p0)));
	angle1=float(acos(sqrt(1.0*p1)));*/

	float pd;

	int off0,off1,off2,off3;

	if (ix<receiver_num && iz<lt)
	{
		in_idx=ix*lt+iz;
		
		if((offset_left=receiver_offset)||(offset_right=receiver_offset))
		{
				off0=0;
				off1=int(offset_left/3);
				off2=int(receiver_num-offset_right/3);
				off3=receiver_num;
				
				if(ix>=off0&&ix<off1)
				{
					//m0=-1.0*angle0+(ix-off0)*(angle0/(off1-off0));

					m0=1.0*(off1-ix)/(off1-off0)*pai/2;

					pd=float(pow(cos(m0*1.0),3.0));

					adj_shot_x_d[in_idx]=adj_shot_x_d[in_idx]*pd*1.0;
				}

				if(ix>=off2&&ix<off3)
				{
					//m0=1.0*(ix-off2)*(angle1/(off3-off2));

					m0=1.0*(ix-off2)/(off3-off2)*pai/2;

					pd=float(pow(cos(m0*1.0),3.0));
					adj_shot_x_d[in_idx]=adj_shot_x_d[in_idx]*pd*1.0;
				}			
		}

		if(offset_left>receiver_offset)
		{
			if(ix>=0&&ix<(offset_left-receiver_offset))
				adj_shot_x_d[in_idx]=0;

			distance_left=int(receiver_offset/3);

			if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0)
			{
				m0=1.0*(offset_left-receiver_offset+distance_left-ix);

				change=pow(cos(pai/2*m0/distance_left),3);				

				adj_shot_x_d[in_idx]=adj_shot_x_d[in_idx]*change;
			}
		}

		if(offset_right>receiver_offset)
		{
			if(ix>=(offset_left+receiver_offset)&&ix<receiver_num)
				adj_shot_x_d[in_idx]=0;
			
			distance_right=int(receiver_offset/3);

			if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0)
			{
				m0=1.0*(ix-(offset_left+receiver_offset-distance_right));

				change=pow(cos(pai/2*m0/distance_right),3);

				adj_shot_x_d[in_idx]=adj_shot_x_d[in_idx]*change;
			}
		}

	}
}

__global__ void cuda_scale_gradient_2(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	
	float m=0.0;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
		if(z1!=z2&&z2!=0)		
		{
			if(iz>=z1&&iz<=z2)
			{		
				m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m))*scale;
			}

			if(iz>z2)	grad_vp1_d[in_idx]=(iz*1.0/z2)*grad_vp1_d[in_idx]*scale;
		}
	}
}

__global__ void cuda_scale_gradient_acqusition(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;

	float m=0.0;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(precon==1)
		{
			//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrtf(d_illum[in_idx1]+EPS));
	
			grad_vp1_d[in_idx]=1.0*scale*grad_vp1_d[in_idx]/(d_illum[in_idx1]+EPS);
		}

		if(precon==2)
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(z1!=z2&&z2!=0)		
			{
				if(iz>=z1&&iz<=z2)
				{		
					m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m))*scale;
				}

				if(iz>z2)	grad_vp1_d[in_idx]=(iz*1.0/z2)*grad_vp1_d[in_idx]*scale;
			}
		}
		
	}
}


__global__ void cuda_scale_gradient_acqusition_only_RTM(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale,int offset_left,int offset_right,int receiver_offset,int offset_attenuation)
///cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultppz_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	int distance_left;
	int distance_right;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		if(precon==1)
		{
			//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrtf(d_illum[in_idx]+EPS));
			//grad_vp1_d[in_idx]=1.0*scale*grad_vp1_d[in_idx]/(d_illum[in_idx]+EPS);

			if(d_illum[in_idx]!=0)	grad_vp1_d[in_idx]=1.0*scale*grad_vp1_d[in_idx]/(d_illum[in_idx]);
		}

		if(precon==2)
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(z1!=z2&&z2!=0)		
			{
				if(iz>=z1&&iz<=z2)
				{		
					m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m))*scale;
				}

				if(iz>z2)	grad_vp1_d[in_idx]=(iz*1.0/z2)*grad_vp1_d[in_idx]*scale;
			}
		}

		if(receiver_offset!=0&&offset_attenuation!=0)
		{

			if(offset_left<=receiver_offset||offset_right<=receiver_offset)
			{
				distance_left=int(offset_left/offset_attenuation);
				distance_right=int(offset_right/offset_attenuation);

				//if(distance_left<=2)	distance_left=3;
				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix<=distance_left&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}

				if(ix>=nx-distance_right&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-nx+distance_right);

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_left>receiver_offset)
			{
				if(ix>=0&&ix<(offset_left-receiver_offset))
					grad_vp1_d[in_idx]=0;

				distance_left=int(receiver_offset/offset_attenuation);

				//if(distance_left<=2)		distance_left=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(offset_left-receiver_offset+distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);				

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_right>receiver_offset)
			{
				if(ix>=(offset_left+receiver_offset)&&ix<nx)
					grad_vp1_d[in_idx]=0;
				
				distance_right=int(receiver_offset/offset_attenuation);

				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-(offset_left+receiver_offset-distance_right));

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}	
		}	
	}
}

__global__ void cuda_scale_gradient_acqusition_only_RTM_ex_amp(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale,int offset_left,int offset_right,int receiver_offset,int offset_attenuation)
///cuda_scale_gradient_acqusition_only_RTM<<<dimGrid,dimBlock>>>(vresultppz_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	int distance_left;
	int distance_right;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		if(receiver_offset!=0&&offset_attenuation!=0)
		{

			if(offset_left<=receiver_offset||offset_right<=receiver_offset)
			{
				distance_left=int(offset_left/offset_attenuation);
				distance_right=int(offset_right/offset_attenuation);

				//if(distance_left<=2)	distance_left=3;
				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix<=distance_left&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}

				if(ix>=nx-distance_right&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-nx+distance_right);

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_left>receiver_offset)
			{
				if(ix>=0&&ix<(offset_left-receiver_offset))
					grad_vp1_d[in_idx]=0;

				distance_left=int(receiver_offset/offset_attenuation);

				//if(distance_left<=2)		distance_left=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(offset_left-receiver_offset+distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);				

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_right>receiver_offset)
			{
				if(ix>=(offset_left+receiver_offset)&&ix<nx)
					grad_vp1_d[in_idx]=0;
				
				distance_right=int(receiver_offset/offset_attenuation);

				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-(offset_left+receiver_offset-distance_right));

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}	
		}	
	}
}

///////////////attenuation two point
__global__ void cuda_scale_gradient_acqusition_new(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale,int offset_left,int offset_right,int receiver_offset,int offset_attenuation)
///cuda_scale_gradient_acqusition_new<<<dimGrid,dimBlock>>>(vresultppz_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;

	int distance_left;
	int distance_right;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(precon==1)
		{
			//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrtf(d_illum[in_idx1]+EPS));
			grad_vp1_d[in_idx]=scale*grad_vp1_d[in_idx]/(d_illum[in_idx1]+EPS);
		}

		if(precon==2)
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(z1!=z2&&z2!=0)		
			{
				if(iz>=z1&&iz<=z2)
				{		
					m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m))*scale;
				}

				if(iz>z2)	grad_vp1_d[in_idx]=(iz*1.0/z2)*grad_vp1_d[in_idx]*scale;
			}
		}

		if(receiver_offset!=0&&offset_attenuation!=0)
		{

			if(offset_left<=receiver_offset||offset_right<=receiver_offset)
			{
				distance_left=int(offset_left/offset_attenuation);
				distance_right=int(offset_right/offset_attenuation);

				//if(distance_left<=2)	distance_left=3;
				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix<=distance_left&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}

				if(ix>=nx-distance_right&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-nx+distance_right);

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_left>receiver_offset)
			{
				if(ix>=0&&ix<(offset_left-receiver_offset))
					grad_vp1_d[in_idx]=0;

				distance_left=int(receiver_offset/offset_attenuation);

				//if(distance_left<=2)		distance_left=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left-receiver_offset)&&ix<(offset_left-receiver_offset+distance_left)&&offset_left!=0&&distance_left!=0)
				{
					m=1.0*(offset_left-receiver_offset+distance_left-ix);

					change=pow(cos(pai/2*m/distance_left),3);				

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}

			if(offset_right>receiver_offset)
			{
				if(ix>=(offset_left+receiver_offset)&&ix<nx)
					grad_vp1_d[in_idx]=0;
				
				distance_right=int(receiver_offset/offset_attenuation);

				//if(distance_right<=2)	distance_right=3;///2017年09月05日 星期二 08时44分25秒  it is important,when I join offset_attenuation

				if(ix>=(offset_left+receiver_offset-distance_right)&&ix<(offset_left+receiver_offset)&&offset_right!=0&&distance_right!=0)
				{
					m=1.0*(ix-(offset_left+receiver_offset-distance_right));

					change=pow(cos(pai/2*m/distance_right),3);

					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
				}
			}	
		}	
	}
}


__global__ void cuda_scale_gradient_acqusition_new_old(float *grad_vp1_d,float *d_illum,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up,int precon,int z1,int z2,float scale,int offset_left,int offset_right,int receiver_offset)
///cuda_scale_gradient_acqusition_new<<<dimGrid,dimBlock>>>(vresultppz_d,d_illum,nx_size,nz,nx_append,nz_append,boundary_left,boundary_up,precon,precon_z1,precon_z2,0.0000001,offset_left[ishot],offset_right[ishot],receiver_offset);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;

	int distance_left;
	int distance_right;

	float m=0.0;
	double change;

	if (ix<nx && iz<nz)
	{
		in_idx=ix*nz+iz;

		in_idx1=(ix+boundary_left)*dimz+iz+boundary_up;

		if(precon==1)
		{
			//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]/(sqrtf(d_illum[in_idx1]+EPS));
			grad_vp1_d[in_idx]=scale*grad_vp1_d[in_idx]/(d_illum[in_idx1]+EPS);
		}

		if(precon==2)
		{
			if(iz<z1)	grad_vp1_d[in_idx]=0.0;
////Ren 2016 Kohn 2012
			if(z1!=z2&&z2!=0)		
			{
				if(iz>=z1&&iz<=z2)
				{		
					m=2.0*3.0*(iz-z1-(z2-z1))/(z2-z1);			
					grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.5*m*m))*scale;
				}

				if(iz>z2)	grad_vp1_d[in_idx]=(iz*1.0/z2)*grad_vp1_d[in_idx]*scale;
			}
		}

		distance_left=int(offset_left/3);
		distance_right=int(offset_right/3);

		if(receiver_offset!=0)
		{
			if(ix<=distance_left&&offset_left!=0)
			{
				m=1.0*(distance_left-ix);

				change=pow(cos(pai/2*m/distance_left),3);

				//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.1*m*m));

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}

			if(ix>=nx-distance_right&&offset_right!=0)
			{
				m=1.0*(ix-nx+distance_right);

				change=pow(cos(pai/2*m/distance_right),3);

				//grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*float(exp(-0.1*m*m));

				grad_vp1_d[in_idx]=grad_vp1_d[in_idx]*change;
			}
		}	
	}
}

__global__ void cuda_vsp_precondition(float *grad_vp1_d,int nx,int nz,int nx_append,int nz_append,int boundary_left,int boundary_up,int receiver_num,int receiver_x_cord,int receiver_interval,int receiver_z_cord,int receiver_z_interval)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	//int id;

	if (ix<nx&&iz<nz)
	{
		in_idx=ix*nz+iz;
		if(iz>=receiver_z_cord&&iz<(receiver_z_cord+receiver_num*receiver_z_interval))
		{
			if(receiver_x_cord<3)
			{
				if(ix<receiver_x_cord+2)				grad_vp1_d[in_idx]=0.0;
			}

			if(receiver_x_cord>=3&&receiver_x_cord<=nx-3)
			{
				if(ix>=receiver_x_cord-2&&ix<=receiver_x_cord+2)	grad_vp1_d[in_idx]=0.0;
			}	

			if(receiver_x_cord>nx-3)
			{
				if(ix>receiver_x_cord)				grad_vp1_d[in_idx]=0.0;
			}
		}
	}
}

/////////////////////////precondition:cut the top artifact
__global__ void cuda_precon_cut(float *grad_vp1_d,int nx,int nz,int dimx,int dimz,int boundary_left,int boundary_up)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;
	int in_idx1;
	int zzzz=40;
	float change;
	if(ix<nx&&iz<nz)
	{
		in_idx=ix*nz+iz;
		in_idx1=ix*nz+zzzz;
		change=cos((zzzz-iz)/zzzz*pai/2);

		//change=exp(-1.0*(zzzz-iz));

		//if(iz<zzzz)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx1];
		
		if(iz<zzzz)	grad_vp1_d[in_idx]=grad_vp1_d[in_idx1]*change*change*change;
	}
}

__global__ void cuda_bell_smoothz(float *g, float *smg, int rbell, int nz, int nx)
/*< smoothing with gaussian function >*/
{
	int i;
	int i1=threadIdx.x+blockIdx.x*blockDim.x;
	int i2=threadIdx.y+blockIdx.y*blockDim.y;
	int id=i1+i2*nz;
	if(i1<nz && i2<nx)
	{
		float s=0;
		for(i=-rbell; i<=rbell; i++) if(i1+i>=0 && i1+i<nz) s+=expf(-(2.0*i*i)/rbell)*g[id+i];
		smg[id]=s;
	}
}

__global__ void cuda_bell_smoothx(float *g, float *smg, int rbell, int nz, int nx)
/*< smoothing with gaussian function >*/
{
	int i;
	int i1=threadIdx.x+blockIdx.x*blockDim.x;
	int i2=threadIdx.y+blockIdx.y*blockDim.y;
	int id=i1+i2*nz;
	if(i1<nz && i2<nx)
	{
		float s=0;
		for(i=-rbell; i<=rbell; i++) if(i2+i>=0 && i2+i<nx) s+=expf(-(2.0*i*i)/rbell)*g[id+nz*i];
		smg[id]=s;
	}
}

__global__ void cuda_cal_max(float *obs_max,float *obs_shot_x_d,int N)
////configuration requirement: <<<1, Block_Size>>> >*/ 
{
    	__shared__ float sdata[Block_Size];/* find max(|vv(:)|) */
	
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
    
	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?fabsf(obs_shot_x_d[id]):0.0f;
		//float b=(id<N)?fabsf(cg[id]):0.0f;
		sdata[tid]= max(sdata[tid], a);
		//tdata[tid]= max(tdata[tid], b);
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{sdata[tid]=max(sdata[tid], sdata[tid+s]);} 
		__syncthreads();
    	}  
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] =max(sdata[tid],sdata[tid + 32]);}
		if (blockDim.x >=  32) { sdata[tid] =max(sdata[tid],sdata[tid + 16]);}
		if (blockDim.x >=  16) { sdata[tid] =max(sdata[tid],sdata[tid + 8]);}
		if (blockDim.x >=   8) { sdata[tid] =max(sdata[tid],sdata[tid + 4]);}
		if (blockDim.x >=   4) { sdata[tid] =max(sdata[tid],sdata[tid + 2]);}
		if (blockDim.x >=   2) { sdata[tid] =max(sdata[tid],sdata[tid + 1]);}
    	}

    	if (tid == 0) {*obs_max=sdata[0]; }
}

__global__ void cuda_dot(float *matrix1,float *matrix2,int ng,float *dot)
{
	__shared__ float  sdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?matrix1[id]:0.0f;
		float b=(id<ng)?matrix2[id]:0.0f;
		sdata[tid] += a*b;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) sdata[tid] += sdata[tid + s]; __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; }
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; }
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; }
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; }
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; }
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; }
    	}
     
    	if (tid == 0) { *dot=sdata[0]; }
}

__global__ void cuda_dot_sum(float *matrix1,float *matrix2,int ng,float *dot_sum )
{
	__shared__ float  sdata[Block_Size];
    	int tid=threadIdx.x;
    	sdata[tid]=0.0f;
	for(int s=0; s<(ng+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<ng)?matrix1[id]:0.0f;
		float b=(id<ng)?matrix2[id]:0.0f;
		sdata[tid] += a*b;	
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s) sdata[tid] += sdata[tid + s]; __syncthreads();
    	}
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] += sdata[tid + 32]; }
		if (blockDim.x >=  32) { sdata[tid] += sdata[tid + 16]; }
		if (blockDim.x >=  16) { sdata[tid] += sdata[tid +  8]; }
		if (blockDim.x >=   8) { sdata[tid] += sdata[tid +  4]; }
		if (blockDim.x >=   4) { sdata[tid] += sdata[tid +  2]; }
		if (blockDim.x >=   2) { sdata[tid] += sdata[tid +  1]; }
    	}
     
    	if (tid == 0) { *dot_sum+=sdata[0]; }
}

__global__ void cuda_cal_alpha_and_beta(float *cg_parameter_d)
{
		
	float gg=cg_parameter_d[0];
	float ss=cg_parameter_d[1];
	float gr=-1.0*cg_parameter_d[2];
	float gs=cg_parameter_d[3];
	float sr=-1.0*cg_parameter_d[4];

	float denominator=gg*ss*max(1.0-(gs/gg)*(gs/ss),EPS);
	//float denominator=gg*ss*(1.0-(gs/gg)*(gs/ss));

	cg_parameter_d[5]=(ss*gr-gs*sr)/denominator;

	cg_parameter_d[6]=(gg*sr-gs*gr)/denominator;

	if(ss==0)//////////the steepest decline	
	{
		cg_parameter_d[5]=gr/gg;
		cg_parameter_d[6]=0;
	}
}

__global__ void cuda_cal_alpha_and_beta_old(float *cg_parameter_d)
{
		
	float gg=cg_parameter_d[0];
	float ss=cg_parameter_d[1];
	float gr=cg_parameter_d[2];
	float gs=cg_parameter_d[3];
	float sr=cg_parameter_d[4];

	float denominator=gg*ss*max(1.0-(gs/gg)*(gs/ss),EPS);
	//float denominator=gg*ss*(1.0-(gs/gg)*(gs/ss));

	cg_parameter_d[5]=(gs*sr-ss*gr)/denominator;

	cg_parameter_d[6]=(gs*gr-gg*sr)/denominator;

	if(ss==0)//////////the steepest decline	
	{
		cg_parameter_d[5]=-1.0*gr/gg;
		cg_parameter_d[6]=0;
	}
}

__global__ void cuda_cal_lame_to_velocity(float *tmp_lame1_d,float *tmp_lame2_d,float *tmp_density_d,float *tmp_velocity_d,float *tmp_velocity1_d,int dimx,int dimz)
//cuda_cal_lame_to_velocity<<<dimGrid,dimBlock>>>(expand_perturb_lame1_d,expand_perturb_lame2_d,s_density_d,expand_perturb_vp_d,expand_perturb_vs_d,nx_append,nz_append);
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	int id=iz+ix*dimz;

	if(ix<dimx&&iz<dimz)
	{
///////////note:::sqrt(A)          A must >0!!!!
		tmp_velocity_d[id]=(tmp_lame1_d[id]+2*tmp_lame2_d[id])/tmp_density_d[id]*1.0;	
		tmp_velocity1_d[id]=tmp_lame2_d[id]/tmp_density_d[id];

		//tmp_velocity_d[id]=float(sqrt((tmp_lame1_d[id]+2*tmp_lame2_d[id])/tmp_density_d[id]*1.0));
		//tmp_velocity1_d[id]=float(sqrt(tmp_lame2_d[id]/tmp_density_d[id]));	
	}
}

__global__ void cuda_cal_velocity_to_lame(float *tmp_lame1_d,float *tmp_lame2_d,float *tmp_density_d,float *tmp_velocity_d,float *tmp_velocity1_d,int dimx,int dimz)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;  
	int id=iz+ix*dimz;

	if(ix<dimx&&iz<dimz)
	{
		tmp_lame2_d[id]=tmp_density_d[id]*tmp_velocity1_d[id]*tmp_velocity1_d[id];

		tmp_lame1_d[id]=tmp_density_d[id]*tmp_velocity_d[id]*tmp_velocity_d[id]-2*tmp_lame2_d[id];		
	}
}

__global__ void sum_integral(float *vxp_integral_d,float *vxp_d,int dimx,int dimz)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	if((ix<dimx)&&(iz<dimz))
	{
		dimx=dimx+2*radius;dimz=dimz+2*radius;
		ix=ix+radius;iz=iz+radius;
		in_idx = ix*dimz+iz;//iz*dimx+ix;

		vxp_integral_d[in_idx]+=vxp_d[in_idx];/////////////vxp_integral_d  :every shot for zero
	}
}


__global__ void calcualte_hydrid_grad(float *hydrid_grad_d,float *grad_lame11_d,int nx,int nz,int mark)
{
	int ix = blockIdx.x*blockDim.x+threadIdx.x;
	int iz = blockIdx.y*blockDim.y+threadIdx.y;
	int in_idx;

	if((ix<nx)&&(iz<nz))
	{
		in_idx=ix*nz+iz;//iz*dimx+ix;

		hydrid_grad_d[mark*nx*nz+in_idx]=grad_lame11_d[in_idx];/////////////vxp_integral_d  :every shot for zero
	}
}



__global__ void cuda_cal_dem_parameter_elastic_media(float *dem_p1_d,float *dem_p2_d,float *dem_p3_d,float *dem_p4_d,float *dem_p5_d,float *vx_x_d,float *vx_z_d,float *vz_x_d,float *vz_z_d,float *vx_t_d,float *vz_t_d,float *tmp_perturb_lame1_d,float *tmp_perturb_lame2_d,float *tmp_perturb_den_d,float *tmp_perturb_vp_d,float *tmp_perturb_vs_d,float *tmp_perturb_density_d,int dimx,int dimz,float *s_velocity_d,float *s_velocity1_d,float *s_density_d,int inversion_para)
//cuda_cal_dem_parameter_elastic_media<<<dimGrid,dimBlock>>>(dem_p1_d,dem_p2_d,dem_p3_d,dem_p4_d,dem_p5_d,vx_x_d,vx_z_d,vz_x_d,vz_z_d,vx_t_d,vz_t_d,tmp_perturb_lame1_d,tmp_perturb_lame2_d,tmp_perturb_den_d,tmp_perturb_vp_d,tmp_perturb_vs_d,tmp_perturb_density_d,nx_append_radius,nz_append_radius,s_velocity_d,s_velocity1_d,s_density_d,inversion_para);
{
		int ix = blockIdx.x*blockDim.x+threadIdx.x;
		int iz = blockIdx.y*blockDim.y+threadIdx.y;
		int in_idx;
		
		//float dt_real=dt/1000;
		float lame1,lame2;

		float p1,p2,p3;
		float p4,p5,p6;

		if((ix<dimx)&&(iz<dimz))
		{
				dimx=dimx+2*radius2;dimz=dimz+2*radius2;
				ix=ix+radius2;iz=iz+radius2;
				in_idx = ix*dimz+iz;//iz*dimx+ix;			

				lame1=s_density_d[in_idx]*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				lame2=s_density_d[in_idx]*s_velocity1_d[in_idx]*s_velocity1_d[in_idx];
				
				if(inversion_para==1)
				{
					p1=tmp_perturb_lame1_d[in_idx]*lame1*1.0;

					p2=tmp_perturb_lame2_d[in_idx]*lame2*1.0;

					p3=tmp_perturb_den_d[in_idx]*s_density_d[in_idx]*1.0;				
				}
				
				if(inversion_para==2)
				{
					p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*1.0;

					p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*1.0;

					p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;
				}

				if(inversion_para==3)
				{
					p4=tmp_perturb_vp_d[in_idx]*s_velocity_d[in_idx]*s_density_d[in_idx]*1.0;

					p5=tmp_perturb_vs_d[in_idx]*s_velocity1_d[in_idx]*s_density_d[in_idx]*1.0;

					p6=tmp_perturb_density_d[in_idx]*s_density_d[in_idx]*1.0;
				}

				if(inversion_para==1)
				{
					p1=1.0*p1;

					p2=1.0*p2;

					p3=1.0*p3;
				}

				if(inversion_para==2)
				{
					p1=2.0*s_density_d[in_idx]*s_velocity_d[in_idx]*p4-4.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*(1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]-2.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;

					//p2=-2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;
					p2=2.0*s_density_d[in_idx]*s_velocity1_d[in_idx]*p5+1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

				if(inversion_para==3)
				{
					p1=2*s_velocity_d[in_idx]*p4-4*s_velocity1_d[in_idx]*p5+(-1.0*s_velocity_d[in_idx]*s_velocity_d[in_idx]+2*s_velocity1_d[in_idx]*s_velocity1_d[in_idx])*p6;

					p2=2.0*s_velocity1_d[in_idx]*p5-1.0*s_velocity1_d[in_idx]*s_velocity1_d[in_idx]*p6;

					p3=p6;
				}

					dem_p1_d[in_idx]=(-1.0)*p3*vx_t_d[in_idx];
					dem_p2_d[in_idx]=(-1.0)*p3*vz_t_d[in_idx];

					dem_p3_d[in_idx]=(p1+2*p2)*vx_x_d[in_idx]+p1*vz_z_d[in_idx];
		
					dem_p4_d[in_idx]=(p1+2*p2)*vz_z_d[in_idx]+p1*vx_x_d[in_idx];

					dem_p5_d[in_idx]=p2*(vx_z_d[in_idx]+vz_x_d[in_idx]);				
		}
}
