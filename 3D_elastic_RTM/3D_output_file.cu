
void output_3d_wavefiled_tao(GPUdevice *mgdevice,int it)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].tp1_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			sprintf(filename,"./wavefield/tp-%d-%d",i,it);
			output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}

			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./wavefield/tp-%d",it);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./wavefield/cut-tp-%d",it);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}

void output_3d_wavefiled_vx(GPUdevice *mgdevice,int it)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vx1_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			sprintf(filename,"./wavefield/vx-%d-%d",i,it);
			output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}

			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./wavefield/vx-%d",it);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./wavefield/cut-vx-%d",it);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}

void output_3d_wavefiled_vz(GPUdevice *mgdevice,int it)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vz1_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			sprintf(filename,"./wavefield/vz-%d-%d",i,it);
			output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}

			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./wavefield/vz-%d",it);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./wavefield/cut-vz-%d",it);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}


void output_3d_wavefiled_vzp(GPUdevice *mgdevice,int it)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vzp1_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			sprintf(filename,"./wavefield/vzp-%d-%d",i,it);
			output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}

			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./wavefield/vzp-%d",it);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./wavefield/cut-vzp-%d",it);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}

void output_3d_wavefiled_vzs(GPUdevice *mgdevice,int it)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vzs1_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			sprintf(filename,"./wavefield/vzs-%d-%d",i,it);
			output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}

			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./wavefield/vzs-%d",it);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./wavefield/cut-vzs-%d",it);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}



void output_3d_wavefiled_excitation_amp_time(GPUdevice *mgdevice,int isx,int isy,int isz)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].ex_time_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf0_h,mgdevice[i].ex_amp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf1_h,mgdevice[i].ex_tp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].ex_vxp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf3_h,mgdevice[i].ex_vyp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf4_h,mgdevice[i].ex_vzp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
			//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}
			checkCudaErrors(hipDeviceSynchronize());
		
			
						
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());

		sprintf(filename,"./someoutput/ex-time-%d-%d",sy_real,sx_real);
		output_file_xyz(filename,wf_3d,nnx,nny,nnz);
		sprintf(filename,"./someoutput/cut-ex-time-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////amp=vx*vx+vy*vy+vz*vz
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf0_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-ex-amp-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////tp
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf1_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-ex-tp-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////vx
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-ex-vxp-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);


//////////////////////////////////vy
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf3_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-ex-vyp-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);


//////////////////////////////////vz
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf4_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-ex-vzp-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}

void output_3d_poyn_p(GPUdevice *mgdevice,int isx,int isy,int isz)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].poyn_px_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf3_h,mgdevice[i].poyn_py_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[i].wf4_h,mgdevice[i].poyn_pz_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

			//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
			//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
		}
			checkCudaErrors(hipDeviceSynchronize());
		
			
//////////////////////////////////vx
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-px-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);


//////////////////////////////////vy
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf3_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-py-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);


//////////////////////////////////vz
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));

			seperate_or_togather_vel_att2(mgdevice[i].wf4_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
		}

			checkCudaErrors(hipDeviceSynchronize());
		sprintf(filename,"./someoutput/cut-pz-%d-%d",sy_real,sx_real);
		output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
}



void output_3d_result(GPUdevice *mgdevice,int isx,int isy,int isz)
{
		if(vsp==0)
		{
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vresult_tp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf1_h,mgdevice[i].vresult_pp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].vresult_ps_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				
				//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
				//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
			}
				checkCudaErrors(hipDeviceSynchronize());
			
				
							
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			//sprintf(filename,"./result/result-tp-%d-%d",sy_real,sx_real);
			//output_file_xyz(filename,wf_3d,nnx,nny,nnz);
			sprintf(filename,"./result/result-tp-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

	//////////////////////////////////tp
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf1_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());
			
			sprintf(filename,"./result/result-pp-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

	//////////////////////////////////vx
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			sprintf(filename,"./result/result-ps-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
		}

		else
		{
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vresult_tp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf1_h,mgdevice[i].vresult_pp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].vresult_ps_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				
				//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
				//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
			}
				checkCudaErrors(hipDeviceSynchronize());
			
				
							
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			//sprintf(filename,"./result/result-tp-%d",sz_real);
			//output_file_xyz(filename,wf_3d,nnx,nny,nnz);
			sprintf(filename,"./result/result-tp-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

	//////////////////////////////////tp
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf1_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());
			
			sprintf(filename,"./result/result-pp-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

	//////////////////////////////////vx
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			sprintf(filename,"./result/result-ps-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
		}


}

void output_3d_result_compensate(GPUdevice *mgdevice,int isx,int isy,int isz)
{
		if(vsp==0)
		{
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vresult_tp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf1_h,mgdevice[i].vresult_pp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].vresult_ps_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				
				//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
				//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
			}
				checkCudaErrors(hipDeviceSynchronize());
			
				
							
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			//sprintf(filename,"./result/result-tp-%d-%d",sy_real,sx_real);
			//output_file_xyz(filename,wf_3d,nnx,nny,nnz);
			sprintf(filename,"./result/compensate-result-tp-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////pp
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf1_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());
			
			sprintf(filename,"./result/compensate-result-pp-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////ps
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			sprintf(filename,"./result/compensate-result-ps-%d-%d",sy_real,sx_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
		}

		else
		{
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].wf_h,mgdevice[i].vresult_tp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf1_h,mgdevice[i].vresult_pp_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].wf2_h,mgdevice[i].vresult_ps_d,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				
				//sprintf(filename,"./wavefield/ex-time-%d-%d",i,it);
				//output_file_xyz(filename,mgdevice[i].wf_h,nnx,nny,nnz_device_append);
			}
				checkCudaErrors(hipDeviceSynchronize());
			
				
							
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			//sprintf(filename,"./result/result-tp-%d",sz_real);
			//output_file_xyz(filename,wf_3d,nnx,nny,nnz);
			sprintf(filename,"./result/compensate-result-tp-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////pp
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf1_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());
			
			sprintf(filename,"./result/compensate-result-pp-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);

//////////////////////////////////ps
			for(int i=0;i<GPU_N;i++)
			{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].wf2_h,wf_3d,nnx,nny,nnz,nnz_device,i,radius,1);
			}

				checkCudaErrors(hipDeviceSynchronize());

			sprintf(filename,"./result/compensate-result-ps-%d",sz_real);
			output_file_xyz_boundary(filename,wf_3d,nx,ny,nz,bl,bf,bu,nnx,nny,nnz);
		}


}
