#include "hip/hip_runtime.h"




///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
void forward_together_using_real_model(GPUdevice *mgdevice)
{


		dim3 dimBlock(32,16);

		dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间
		
		dim3 dimGridwf_radius((nnx_radius+dimBlock.x-1)/dimBlock.x,(nny_radius+dimBlock.y-1)/dimBlock.y,nnz_radius);///单块卡的整个空间减去半径
		
		dim3 dimGrid_rec_lt_x_y((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_y+dimBlock.y-1)/dimBlock.y);

		dim3 dimGrid_rec_lt_x_z((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_z+dimBlock.y-1)/dimBlock.y);///seismic process and receive



					for(it=0;it<lt;it++)
						{
							if(fmod(it+1.0,1000.0)==1)	
							{
								warn("forward for modeling,isx=%d,isy=%d,isz=%d,it=%d",isx+1,isy+1,isz+1,it);
							}

							if(it<wavelet_length)
							{
								checkCudaErrors(hipSetDevice(gpuid[choose_ns]));

								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].txx1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].tyy1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].tzz1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("add_source_3D is passing");

							exchange_device_nz_kernel_txxyyzz(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_txxyyzz is passing");


							if((fmod(it+1.0,wavefield_interval)==0)&&join_wavefield!=0)
							//if((it==500)&&join_wavefield!=0)
							{
								system("mkdir wavefield");
								output_3d_wavefiled_tao(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vx(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vz(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzp(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzs(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								if(((it+1.0)/wavefield_interval)==(lt/wavefield_interval))
								{
									system("rm -r wavefield1");
									system("mv wavefield wavefield1");
								}
							}
					
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("output_3d_wavefiled is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_txxzzxzpp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp2_d,mgdevice[i].tp1_d,mgdevice[i].txx2_d,mgdevice[i].txx1_d,mgdevice[i].tyy2_d,mgdevice[i].tyy1_d,mgdevice[i].tzz2_d,mgdevice[i].tzz1_d,mgdevice[i].txy2_d,mgdevice[i].txy1_d,mgdevice[i].txz2_d,mgdevice[i].txz1_d,mgdevice[i].tyz2_d,mgdevice[i].tyz1_d,mgdevice[i].vx1_d,mgdevice[i].vy1_d,mgdevice[i].vz1_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);							
							}
					
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_txxzzxzpp_3D is passing");

							exchange_device_nz_kernel_tao2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///		

							exchange_device_nz_kernel_taop2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_tao2 taop2 is passing");

							for(int i=0;i<GPU_N;i++)
							{
								hipSetDevice(gpuid[i]);

								fwd_vx_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx2_d,mgdevice[i].vx1_d,mgdevice[i].txx2_d,mgdevice[i].txy2_d,mgdevice[i].txz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vy_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy2_d,mgdevice[i].vy1_d,mgdevice[i].txy2_d,mgdevice[i].tyy2_d,mgdevice[i].tyz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vz_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz2_d,mgdevice[i].vz1_d,mgdevice[i].txz2_d,mgdevice[i].tyz2_d,mgdevice[i].tzz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vx_3D fwd_vy_3D fwd_vz_3D is passing");
					
							exchange_device_nz_kernel_vx_vy_vz2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_vx_vy_vz2 is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_vxp_vzp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxp2_d,mgdevice[i].vxp1_d,mgdevice[i].vyp2_d,mgdevice[i].vyp1_d,mgdevice[i].vzp2_d,mgdevice[i].vzp1_d,mgdevice[i].tp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].vx2_d,mgdevice[i].vy2_d,mgdevice[i].vz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);	
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vxp_vzp_3D is passing");
																		
							{
								checkCudaErrors(hipSetDevice(gpuid[choose_re]));

								write_or_add_shot_3D_surface_three<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_re].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].vx2_d,mgdevice[choose_re].vy2_d,mgdevice[choose_re].vz2_d,nnx,nny,nnz_device_append,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,0);

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("write_or_add_shot_3D_surface_three is passing");
	
								if(vsp==0)
								{
									//cut_direct_shot_3D_surface_three<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_re].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].velocity_d,nnx,nny,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,sx_real,sy_real,sz_real,it,wavelet_length,dx,dy,dz,dt);
									cut_direct_shot_3D_surface_three_new<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_re].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].velocity_d,nnx,nny,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,sx_real,sy_real,sz_real,it,wavelet_length,dx,dy,dz,dt,cut_direct_wave);												
								}
								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("cut_direct_shot_3D_surface is passing");
															
								transfer_gpu_to_cpu_multicomponent_seismic(mgdevice,it,0);

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("transfer_gpu_to_cpu_multicomponent_seismic is passing");	
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("write_shot is passing");

							exchange_wavefiled_new(mgdevice);//////////////////change wavefield_new
							//exchange_wavefiled_old(mgdevice);//////////////////change wavefield_old	

							checkCudaErrors(hipDeviceSynchronize());		
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_wavefiled is passing");
						}
}

///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
void forward_together_using_smoothed_model(GPUdevice *mgdevice)
{
		dim3 dimBlock(32,16);

		dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间
		
		dim3 dimGridwf_radius((nnx_radius+dimBlock.x-1)/dimBlock.x,(nny_radius+dimBlock.y-1)/dimBlock.y,nnz_radius);///单块卡的整个空间减去半径
		
		dim3 dimGrid_rec_lt_x_y((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_y+dimBlock.y-1)/dimBlock.y);

		dim3 dimGrid_rec_lt_x_z((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_z+dimBlock.y-1)/dimBlock.y);///seismic process and receive

					for(it=0;it<2*lt/3;it++)
						{
							if(fmod(it+1.0,1000.0)==1)	
							{
								warn("forward for Elastic RTM,isx=%d,isy=%d,isz=%d,it=%d",isx+1,isy+1,isz+1,it);
							}

							if(it<wavelet_length)
							{
								checkCudaErrors(hipSetDevice(gpuid[choose_ns]));

								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].txx1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].tyy1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
								add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].tzz1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("add_source_3D is passing");

							exchange_device_nz_kernel_txxyyzz(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_txxyyzz is passing");


							if((fmod(it+1.0,wavefield_interval)==0)&&join_wavefield!=0)
							//if((it==500)&&join_wavefield!=0)
							{
								system("mkdir wavefield");
								output_3d_wavefiled_tao(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vx(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vz(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzp(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzs(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								if(((it+1.0)/wavefield_interval)==((2*lt/3)/wavefield_interval))
								{
									system("rm -r wavefield2");
									system("mv wavefield wavefield2");
								}
							}
					
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("output_3d_wavefiled is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_txxzzxzpp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp2_d,mgdevice[i].tp1_d,mgdevice[i].txx2_d,mgdevice[i].txx1_d,mgdevice[i].tyy2_d,mgdevice[i].tyy1_d,mgdevice[i].tzz2_d,mgdevice[i].tzz1_d,mgdevice[i].txy2_d,mgdevice[i].txy1_d,mgdevice[i].txz2_d,mgdevice[i].txz1_d,mgdevice[i].tyz2_d,mgdevice[i].tyz1_d,mgdevice[i].vx1_d,mgdevice[i].vy1_d,mgdevice[i].vz1_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);							
							}
					
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_txxzzxzpp_3D is passing");

							exchange_device_nz_kernel_tao2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///		

							exchange_device_nz_kernel_taop2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							
							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_tao2 taop2 is passing");

							for(int i=0;i<GPU_N;i++)
							{
								hipSetDevice(gpuid[i]);

								fwd_vx_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx2_d,mgdevice[i].vx1_d,mgdevice[i].txx2_d,mgdevice[i].txy2_d,mgdevice[i].txz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vy_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy2_d,mgdevice[i].vy1_d,mgdevice[i].txy2_d,mgdevice[i].tyy2_d,mgdevice[i].tyz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vz_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz2_d,mgdevice[i].vz1_d,mgdevice[i].txz2_d,mgdevice[i].tyz2_d,mgdevice[i].tzz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vx_3D fwd_vy_3D fwd_vz_3D is passing");
					
							exchange_device_nz_kernel_vx_vy_vz2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_vx_vy_vz2 is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_vxp_vzp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxp2_d,mgdevice[i].vxp1_d,mgdevice[i].vyp2_d,mgdevice[i].vyp1_d,mgdevice[i].vzp2_d,mgdevice[i].vzp1_d,mgdevice[i].tp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].vx2_d,mgdevice[i].vy2_d,mgdevice[i].vz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);	
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vxp_vzp_3D is passing");							

							exchange_wavefiled_new(mgdevice);//////////////////change wavefield_new
							//exchange_wavefiled_old(mgdevice);//////////////////change wavefield_old	

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("exchange_wavefiled_new is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								//cuda_cal_excitation_amp_time<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].ex_vxp_d,mgdevice[i].vxp2_d,mgdevice[i].ex_vyp_d,mgdevice[i].vyp2_d,mgdevice[i].ex_vzp_d,mgdevice[i].vzp2_d,nnx,nny,nnz_device_append,it);
								cuda_cal_excitation_amp_time_new<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_amp_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].ex_vxp_d,mgdevice[i].vxp2_d,mgdevice[i].ex_vyp_d,mgdevice[i].vyp2_d,mgdevice[i].ex_vzp_d,mgdevice[i].vzp2_d,nnx,nny,nnz_device_append,it);
								//cuda_cal_source_poyn_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].poyn_px_d,mgdevice[i].poyn_py_d,mgdevice[i].poyn_pz_d,mgdevice[i].ex_time_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].tp2_d,nnx,nny,nnz_device_append,it);
							}

							checkCudaErrors(hipDeviceSynchronize());
							//if(fmod(it+1.0,1000.0)==1)	warn("cuda_cal_excitation_amp_time_new is passing");
						
						}
}

///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************///////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////**********************************************//////////////////////////////////////////////////////
void backward_together_using_smoothed_model(GPUdevice *mgdevice)
{

		dim3 dimBlock(32,16);

		dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间
		
		dim3 dimGridwf_radius((nnx_radius+dimBlock.x-1)/dimBlock.x,(nny_radius+dimBlock.y-1)/dimBlock.y,nnz_radius);///单块卡的整个空间减去半径
		
		dim3 dimGrid_rec_lt_x_y((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_y+dimBlock.y-1)/dimBlock.y);

		dim3 dimGrid_rec_lt_x_z((receiver_num_x+dimBlock.x-1)/dimBlock.x,(receiver_num_z+dimBlock.y-1)/dimBlock.y);///seismic process and receive


						for(it=lt-1;it>=0;it--)
						{
							if(fmod(it+1.0,1000.0)==1)	
							{
								warn("backward for Elastic RTM,isx=%d,isy=%d,isz=%d,it=%d",isx+1,isy+1,isz+1,it);
							}
							
							{
								transfer_gpu_to_cpu_multicomponent_seismic(mgdevice,it,1);

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("transfer_gpu_to_cpu_multicomponent_seismic is passing");			
							}


							{
								checkCudaErrors(hipSetDevice(gpuid[choose_re]));

								write_or_add_shot_3D_surface_three<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_re].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].vx1_d,mgdevice[choose_re].vy1_d,mgdevice[choose_re].vz1_d,nnx,nny,nnz_device_append,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,add_receiver_bool);	

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("write_or_add_shot_3D_surface_three is passing");
							}
								
							{
								exchange_device_nz_kernel_vx_vy_vz1(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_vx_vy_vz1 is passing");
							}

							if((fmod(it+1.0,wavefield_interval)==0)&&join_wavefield!=0)
							//if((it==500)&&join_wavefield!=0)
							{
								system("mkdir wavefield");
								output_3d_wavefiled_tao(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vx(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vz(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzp(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								output_3d_wavefiled_vzs(mgdevice,it+1);
								checkCudaErrors(hipDeviceSynchronize());

								if(((it+1.0)/wavefield_interval)==(lt/wavefield_interval))
								{
									system("rm -r wavefield3");
									system("mv wavefield wavefield3");
								}
							}
					
								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("output_3d_wavefiled is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_txxzzxzpp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp2_d,mgdevice[i].tp1_d,mgdevice[i].txx2_d,mgdevice[i].txx1_d,mgdevice[i].tyy2_d,mgdevice[i].tyy1_d,mgdevice[i].tzz2_d,mgdevice[i].tzz1_d,mgdevice[i].txy2_d,mgdevice[i].txy1_d,mgdevice[i].txz2_d,mgdevice[i].txz1_d,mgdevice[i].tyz2_d,mgdevice[i].tyz1_d,mgdevice[i].vx1_d,mgdevice[i].vy1_d,mgdevice[i].vz1_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);							
							}
					
								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("fwd_txxzzxzpp_3D is passing");

							{
								exchange_device_nz_kernel_tao2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///

								exchange_device_nz_kernel_taop2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///
								
								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_tao2 taop2 is passing");
							}

							for(int i=0;i<GPU_N;i++)
							{
								hipSetDevice(gpuid[i]);

								fwd_vx_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx2_d,mgdevice[i].vx1_d,mgdevice[i].txx2_d,mgdevice[i].txy2_d,mgdevice[i].txz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vy_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy2_d,mgdevice[i].vy1_d,mgdevice[i].txy2_d,mgdevice[i].tyy2_d,mgdevice[i].tyz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);

								fwd_vz_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz2_d,mgdevice[i].vz1_d,mgdevice[i].txz2_d,mgdevice[i].tyz2_d,mgdevice[i].tzz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
							}

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vx_3D fwd_vy_3D fwd_vz_3D is passing");
					
							{
								exchange_device_nz_kernel_vx_vy_vz2(mgdevice,exchange_device_bool);//////////////gpu_i exchange  gpu_i+1;///

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("exchange_device_nz_kernel_vx_vy_vz2 is passing");
							}

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								fwd_vxp_vzp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxp2_d,mgdevice[i].vxp1_d,mgdevice[i].vyp2_d,mgdevice[i].vyp1_d,mgdevice[i].vzp2_d,mgdevice[i].vzp1_d,mgdevice[i].tp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].vx2_d,mgdevice[i].vy2_d,mgdevice[i].vz2_d,mgdevice[i].s_velocity_d,mgdevice[i].s_velocity1_d,mgdevice[i].s_density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);	
							}

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("fwd_vxp_vzp_3D is passing");							

							{
								exchange_wavefiled_new(mgdevice);//////////////////change wavefield_new
								//exchange_wavefiled_old(mgdevice);//////////////////change wavefield_old	

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("exchange_wavefiled_new is passing");	
							}

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								//cuda_cal_receiver_poyn_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].poyn_rpx_d,mgdevice[i].poyn_rpy_d,mgdevice[i].poyn_rpz_d,mgdevice[i].poyn_rsx_d,mgdevice[i].poyn_rsy_d,mgdevice[i].poyn_rsz_d,mgdevice[i].ex_time_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].tp2_d,mgdevice[i].txx2_d,mgdevice[i].tyy2_d,mgdevice[i].tzz2_d,mgdevice[i].txy2_d,mgdevice[i].txz2_d,mgdevice[i].tyz2_d,nnx,nny,nnz_device_append,it);
							}	
					
								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("cuda_cal_receiver_poyn_3D is passing");

							for(int i=0;i<GPU_N;i++)
							{
								checkCudaErrors(hipSetDevice(gpuid[i]));

								imaging_correlation_ex<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].vresult_tp_d,nnx,nny,nnz_device_append,it,amp_max,precon_z1);

								imaging_inner_product_ex<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_vxp_d,mgdevice[i].ex_vyp_d,mgdevice[i].ex_vzp_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].vresult_pp_d,nnx,nny,nnz_device_append,it,tp_max,precon_z1);

								imaging_inner_product_ex<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_vxp_d,mgdevice[i].ex_vyp_d,mgdevice[i].ex_vzp_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].vresult_ps_d,nnx,nny,nnz_device_append,it,tp_max,precon_z1);
							}

								checkCudaErrors(hipDeviceSynchronize());
								//if(fmod(it+1.0,1000.0)==1)	warn("imaging_inner_product_ex is passing");

						}
}
