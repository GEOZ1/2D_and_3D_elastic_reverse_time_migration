#include "hip/hip_runtime.h"

void seperate_or_togather_vel_att2(float *data_1d,float *data_3d,int nxb,int nyb,int nzb,int nzb_aver,int i,int orders,int mark)
{
	int ix,iy,iz;
	//ASSIGN THE BIG 3D ARRAY TO THE LITTLE 1D ARRAY ON HOST FOR EACH GPU
	for(iz=0;iz<nzb_aver;iz++)
	{
		for(iy=0;iy<nyb;iy++)
		{
			for(ix=0;ix<nxb;ix++)
			{
				if(i*nzb_aver+iz<nzb)
				{
					if(mark==0)	data_1d[(iz+orders)*nxb*nyb+iy*nxb+ix]=data_3d[(i*nzb_aver+iz)*nxb*nyb+iy*nxb+ix];

					else		data_3d[(i*nzb_aver+iz)*nxb*nyb+iy*nxb+ix]=data_1d[(iz+orders)*nxb*nyb+iy*nxb+ix];
				}
			}
		}
	}
}

void seperate_vel_att1(GPUdevice *mgdevice)
{
		for(int i=0;i<GPU_N;i++)
		{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				seperate_or_togather_vel_att2(mgdevice[i].velocity_h,velocity_pml,nnx,nny,nnz,nnz_device,i,radius,0);

				seperate_or_togather_vel_att2(mgdevice[i].velocity1_h,velocity1_pml,nnx,nny,nnz,nnz_device,i,radius,0);

				seperate_or_togather_vel_att2(mgdevice[i].density_h,density_pml,nnx,nny,nnz,nnz_device,i,radius,0);

				seperate_or_togather_vel_att2(mgdevice[i].att_h,att_pml,nnx,nny,nnz,nnz_device,i,radius,0);

			
				//sprintf(filename,"./someoutput/velocity_%d.bin",i);
				//output_3d(filename,mgdevice[i].velocity_h,nnx,nny,nnz_device_append);

				//sprintf(filename,"./someoutput/velocity1_%d.bin",i);
				//output_3d(filename,mgdevice[i].velocity1_h,nnx,nny,nnz_device_append);

				//sprintf(filename,"./someoutput/density1_%d.bin",i);
				//output_3d(filename,mgdevice[i].density1_h,nnx,nny,nnz_device_append);

				//sprintf(filename,"./someoutput/att_%d.bin",i);
				//output_3d(filename,mgdevice[i].att_h,nnx,nny,nnz_device_append);
		}

		hipDeviceSynchronize();
}

void expand_nnz_residual(float *data_1d,int nnx,int nny,int nnz_device_append,int nnz_residual)
//expand_nnz_residual(mgdevice[GPU_N-1].density_h,nnx,nny,nz_device_append,nnz_residual);
{
	int ix,iy,iz;
	//ASSIGN THE BIG 3D ARRAY TO THE LITTLE 1D ARRAY ON HOST FOR EACH GPU
	for(iz=0;iz<nnz_residual;iz++)
	{
		for(iy=0;iy<nny;iy++)
		{
			for(ix=0;ix<nnx;ix++)
			{
				data_1d[(nnz_device_append-radius-nnz_residual+iz)*nnx*nny+iy*nnx+ix]=data_1d[(nnz_device_append-radius-nnz_residual-1)*nnx*nny+iy*nnx+ix];
			}
		}
	}
}

void elastic_modeling_parameter_cpu_to_gpu(GPUdevice *mgdevice)
{
	
		for(int i=0;i<GPU_N;i++)
		{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].velocity_d,mgdevice[i].velocity_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].velocity1_d,mgdevice[i].velocity1_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].density_d,mgdevice[i].density_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].att_d,mgdevice[i].att_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].coe_d,coe_opt,(radius+1)*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].wavelet_d,wavelet,wavelet_length*sizeof(float),hipMemcpyDefault));
		}

		checkCudaErrors(hipDeviceSynchronize());

}


void elastic_RTM_parameter_cpu_to_gpu(GPUdevice *mgdevice)
{
	
		for(int i=0;i<GPU_N;i++)
		{
				checkCudaErrors(hipSetDevice(gpuid[i]));

				checkCudaErrors(hipMemcpy(mgdevice[i].s_velocity_d,mgdevice[i].velocity_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].s_velocity1_d,mgdevice[i].velocity1_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));

				checkCudaErrors(hipMemcpy(mgdevice[i].s_density_d,mgdevice[i].density_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
		}

		checkCudaErrors(hipDeviceSynchronize());

}

void get_real_model_parameter()
{
			input_file_xyz_boundary(velocity_name,velocity_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			//output_file_xyz("./someoutput/velocity_all1.bin",velocity_pml,nnx,nny,nnz);
			//output_file_xyz_boundary("./someoutput/velocity_cut.bin",velocity_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);

			add_pml_layers_v_h(velocity_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);	
			output_file_xyz("./someoutput/vp_all.bin",velocity_pml,nnx,nny,nnz);
					

			input_file_xyz_boundary(velocity1_name,velocity1_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);	
			add_pml_layers_v_h(velocity1_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			output_file_xyz("./someoutput/vs_all.bin",velocity1_pml,nnx,nny,nnz);


			input_file_xyz_boundary(density_name,density_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);	
			add_pml_layers_v_h(density_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			output_file_xyz("./someoutput/den_all.bin",density_pml,nnx,nny,nnz);
}

void get_smoothed_model_parameter()
{

////////////////////////read smoothed vp
			if(smooth_time_vp!=0)	
			{	
				openfile=fopen("smooth_3d","wb+");//////cal_shot_*_iter_1  res_shot_*_iter_1
				fprintf(openfile,"#!/bin/sh\n");

				fprintf(openfile,"smooth3d< ./someoutput/vp_all.bin n1=%d n2=%d n3=%d r1=%f r2=%f r3=%f >./someoutput/s_vp.bin \n",nnz,nny,nnx,smooth_time_vp,smooth_time_vp,smooth_time_vp);					
				fclose(openfile);
				system("sh smooth_3d");
				input_file_xyz("./someoutput/s_vp.bin",velocity_pml,nnx,nny,nnz);

				output_file_xyz_boundary("./someoutput/cut_s_vp.bin",velocity_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			}



////////////////////////read smoothed vs
			if(smooth_time_vs!=0)	
			{	
				openfile=fopen("smooth_3d","wb+");//////cal_shot_*_iter_1  res_shot_*_iter_1
				fprintf(openfile,"#!/bin/sh\n");

				fprintf(openfile,"smooth3d< ./someoutput/vs_all.bin n1=%d n2=%d n3=%d r1=%f r2=%f r3=%f >./someoutput/s_vs.bin \n",nnz,nny,nnx,smooth_time_vs,smooth_time_vs,smooth_time_vs);					
				fclose(openfile);
				system("sh smooth_3d");
				input_file_xyz("./someoutput/s_vs.bin",velocity1_pml,nnx,nny,nnz);

				output_file_xyz_boundary("./someoutput/cut_s_vs.bin",velocity1_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			}


		
////////////////////////read smoothed den			
			if(smooth_time_density!=0)	
			{	
				openfile=fopen("smooth_3d","wb+");//////cal_shot_*_iter_1  res_shot_*_iter_1
				fprintf(openfile,"#!/bin/sh\n");

				fprintf(openfile,"smooth3d< ./someoutput/den_all.bin n1=%d n2=%d n3=%d r1=%f r2=%f r3=%f >./someoutput/s_den.bin \n",nnz,nny,nnx,smooth_time_density,smooth_time_density,smooth_time_density);					
				fclose(openfile);
				system("sh smooth_3d");
				input_file_xyz("./someoutput/s_den.bin",density_pml,nnx,nny,nnz);

				output_file_xyz_boundary("./someoutput/cut_s_den.bin",density_pml,nx,ny,nz,bl,bb,bu,nnx,nny,nnz);
			}

}




void exchange_wavefiled_new(GPUdevice *mgdevice)
{
		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
				
			mgdevice[i].rep=mgdevice[i].vx1_d;	mgdevice[i].vx1_d=mgdevice[i].vx2_d;	mgdevice[i].vx2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vy1_d;	mgdevice[i].vy1_d=mgdevice[i].vy2_d;	mgdevice[i].vy2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vz1_d;	mgdevice[i].vz1_d=mgdevice[i].vz2_d;	mgdevice[i].vz2_d=mgdevice[i].rep;
			
			mgdevice[i].rep=mgdevice[i].txx1_d;	mgdevice[i].txx1_d=mgdevice[i].txx2_d;	mgdevice[i].txx2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].tyy1_d;	mgdevice[i].tyy1_d=mgdevice[i].tyy2_d;	mgdevice[i].tyy2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].tzz1_d;	mgdevice[i].tzz1_d=mgdevice[i].tzz2_d;	mgdevice[i].tzz2_d=mgdevice[i].rep;

			mgdevice[i].rep=mgdevice[i].txy1_d;	mgdevice[i].txy1_d=mgdevice[i].txy2_d;	mgdevice[i].txy2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].txz1_d;	mgdevice[i].txz1_d=mgdevice[i].txz2_d;	mgdevice[i].txz2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].tyz1_d;	mgdevice[i].tyz1_d=mgdevice[i].tyz2_d;	mgdevice[i].tyz2_d=mgdevice[i].rep;
			
			
			mgdevice[i].rep=mgdevice[i].tp1_d;	mgdevice[i].tp1_d=mgdevice[i].tp2_d;	mgdevice[i].tp2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vxp1_d;	mgdevice[i].vxp1_d=mgdevice[i].vxp2_d;	mgdevice[i].vxp2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vyp1_d;	mgdevice[i].vyp1_d=mgdevice[i].vyp2_d;	mgdevice[i].vyp2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vzp1_d;	mgdevice[i].vzp1_d=mgdevice[i].vzp2_d;	mgdevice[i].vzp2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vxs1_d;	mgdevice[i].vxs1_d=mgdevice[i].vxs2_d;	mgdevice[i].vxs2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vys1_d;	mgdevice[i].vys1_d=mgdevice[i].vys2_d;	mgdevice[i].vys2_d=mgdevice[i].rep;
			mgdevice[i].rep=mgdevice[i].vzs1_d;	mgdevice[i].vzs1_d=mgdevice[i].vzs2_d;	mgdevice[i].vzs2_d=mgdevice[i].rep;
		}

		checkCudaErrors(hipDeviceSynchronize());

}

__global__ void exchange_wf(float *wf1,float *wf2,int nnx,int nny,int nnz)
{
		//int ix=blockIdx.x*blockDim.x+threadIdx.x;
		//int iy=blockIdx.y*blockDim.y+threadIdx.y;
		//int iz=blockIdx.z*blockDim.z+threadIdx.z;;

		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;
		int iz=blockIdx.z;

		int indx;
		float change;

		if(ix<nnx&&iy<nny&&iz<nnz)
		{
				indx=iz*nnx*nny+iy*nnx+ix;
				change=wf1[indx];
				wf1[indx]=wf2[indx];
				wf2[indx]=change;
		}
}

void exchange_wavefiled_old(GPUdevice *mgdevice)
{
		dim3 dimBlock(32,16);

		dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

		for(int i=0;i<GPU_N;i++)
		{
			checkCudaErrors(hipSetDevice(gpuid[i]));
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy1_d,mgdevice[i].vy2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz1_d,mgdevice[i].vz2_d,nnx,nny,nnz_device_append);

			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,nnx,nny,nnz_device_append);

			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txy1_d,mgdevice[i].txy2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txz1_d,mgdevice[i].txz2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyz1_d,mgdevice[i].tyz2_d,nnx,nny,nnz_device_append);

			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp1_d,mgdevice[i].tp2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxp1_d,mgdevice[i].vxp2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vyp1_d,mgdevice[i].vyp2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vzp1_d,mgdevice[i].vzp2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxs1_d,mgdevice[i].vxs2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vys1_d,mgdevice[i].vys2_d,nnx,nny,nnz_device_append);
			exchange_wf<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vzs1_d,mgdevice[i].vzs2_d,nnx,nny,nnz_device_append);
		}
			
		checkCudaErrors(hipDeviceSynchronize());
}



//////////////////////////////////////////////////////in GPU
__global__ void add_source_3D(float *wf_d,float *wavelet_d,int nnx,int nny,int nnz_device_append,int nnz_device,int it,int sx,int sy,int sz,int bl,int bb,int bu)
//add_source_3D<<<1,1,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_ns].txx1_d,mgdevice[choose_ns].wavelet_d,nnx,nny,nnz_device_append,nnz_device,it,sx_real,sy_real,sz_real,bl,bb,bu);
{
		int sx_real=sx+bl;
	
		int sy_real=sy+bb;

		//int choose_ns=(sz+bu)/nnz_device;
		//int sz_real=sz+bu-choose_ns*nnz_device+radius;	

		int sz_real=(sz+bu)%nnz_device+radius;
	
		int id=sz_real*(nnx*nny)+sy_real*nnx+sx_real;

		wf_d[id]+=wavelet_d[it];

		//wf_d[id]+=1.0;///test!!!!
}

__global__ void exchange_device(float *wf1_device1,float *wf2_device1,float *wf1_device2,float *wf2_device2,int nnx,int nny,int nnz)
//exchange_device<<<dimGridwf_append,dimBlock>>>(mgdevice[i].wf1_d,mgdevice[i].wf2_d,mgdevice[i+1].wf1_d,mgdevice[i+1].wf2_d,nnx_device_append,nny,nnz);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;
		int iz=blockIdx.z;

		if(ix<radius&&iy<nny&&iz<nnz)
		{
				wf1_device1[iz*nnx*nny+iy*nnx+nnx-radius+ix]=wf1_device2[iz*nnx*nny+iy*nnx+radius+ix];
				wf2_device1[iz*nnx*nny+iy*nnx+nnx-radius+ix]=wf2_device2[iz*nnx*nny+iy*nnx+radius+ix];

				wf1_device2[iz*nnx*nny+iy*nnx+ix]=wf1_device1[iz*nnx*nny+iy*nnx+nnx-2*radius+ix];
				wf2_device2[iz*nnx*nny+iy*nnx+ix]=wf2_device1[iz*nnx*nny+iy*nnx+nnx-2*radius+ix];

				//wf1_device2[iz*nnx*nny+iy*nnx+nnx-radius+ix]=wf1_device3[iz*nnx*nny+iy*nnx+radius+ix];
				//wf2_device2[iz*nnx*nny+iy*nnx+nnx-radius+ix]=wf2_device3[iz*nnx*nny+iy*nnx+radius+ix];

				//wf1_device3[iz*nnx*nny+iy*nnx+ix]=wf1_device2[iz*nnx*nny+iy*nnx+nnx-2*radius+ix];
				//wf2_device3[iz*nnx*nny+iy*nnx+ix]=wf2_device2[iz*nnx*nny+iy*nnx+nnx-2*radius+ix];
		}
}

__global__ void exchange_device_new(float *wf1_device1,float *wf2_device1,float *wf1_device2,float *wf2_device2,int nx,int ny,int nz1,int nz2,int orders)
//exchange_device_new<<<dimGrid3D,dimBlock2D,0,plan[i].stream>>>(plan[i-1].wf1_d,plan[i-1].wf2_d,plan[i].wf1_d,plan[i].wf2_d,plan[i].nxb,plan[i].nyb,plan[i-1].nzb,plan[i].nzb,orders);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		if(tx<nx&&ty<ny&&tz<orders)
		{

				wf1_device2[tz*nx*ny+ty*nx+tx]=wf1_device1[(nz1-2*orders+tz)*nx*ny+ty*nx+tx];
				wf2_device2[tz*nx*ny+ty*nx+tx]=wf2_device1[(nz1-2*orders+tz)*nx*ny+ty*nx+tx];

				wf1_device1[(nz1-orders+tz)*nx*ny+ty*nx+tx]=wf1_device2[(tz+orders)*nx*ny+ty*nx+tx];
				wf2_device1[(nz1-orders+tz)*nx*ny+ty*nx+tx]=wf2_device2[(tz+orders)*nx*ny+ty*nx+tx];

		}
}

__global__ void exchange_device_nz(float *wf1_device1,float *wf2_device1,float *wf1_device2,float *wf2_device2,int nnx,int nny,int nnz)
//exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;
		int iz=blockIdx.z;

		if(ix<nnx&&iy<nny&&iz<radius)
		{
				wf1_device2[iz*nnx*nny+iy*nnx+ix]=wf1_device1[(nnz+iz-2*radius)*nnx*nny+iy*nnx+ix];
				wf2_device2[iz*nnx*nny+iy*nnx+ix]=wf2_device1[(nnz+iz-2*radius)*nnx*nny+iy*nnx+ix];//////important


				wf1_device1[(nnz-radius+iz)*nnx*nny+iy*nnx+ix]=wf1_device2[(iz+radius)*nnx*nny+iy*nnx+ix];
				wf2_device1[(nnz-radius+iz)*nnx*nny+iy*nnx+ix]=wf2_device2[(iz+radius)*nnx*nny+iy*nnx+ix];
		}
}

__global__ void exchange_device_nz_one(float *wf1_device1,float *wf1_device2,int nnx,int nny,int nnz)
//exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;
		int iz=blockIdx.z;

		if(ix<nnx&&iy<nny&&iz<radius)
		{
				wf1_device2[iz*nnx*nny+iy*nnx+ix]=wf1_device1[(nnz+iz-2*radius)*nnx*nny+iy*nnx+ix];//////important


				wf1_device1[(nnz-radius+iz)*nnx*nny+iy*nnx+ix]=wf1_device2[(iz+radius)*nnx*nny+iy*nnx+ix];
		}
}

void exchange_device_nz_kernel_txxyyzz(GPUdevice *mgdevice,int mark)
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
				}


				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx1_d,mgdevice[i+1].txx1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy1_d,mgdevice[i+1].tyy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz1_d,mgdevice[i+1].tzz1_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i+1].txx1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i+1].tyy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i+1].tzz1_d,nnx,nny,nnz_device_append);
				}
			}

			checkCudaErrors(hipDeviceSynchronize());
}

void exchange_device_nz_kernel_tao1(GPUdevice *mgdevice,int mark)
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txy1_d,mgdevice[i].txy2_d,mgdevice[i+1].txy1_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txz1_d,mgdevice[i].txz2_d,mgdevice[i+1].txz1_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);					
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyz1_d,mgdevice[i].tyz2_d,mgdevice[i+1].tyz1_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}


				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx1_d,mgdevice[i+1].txx1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy1_d,mgdevice[i+1].tyy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz1_d,mgdevice[i+1].tzz1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txy1_d,mgdevice[i+1].txy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txz1_d,mgdevice[i+1].txz1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyz1_d,mgdevice[i+1].tyz1_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txy1_d,mgdevice[i].txy2_d,mgdevice[i+1].txy1_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txz1_d,mgdevice[i].txz2_d,mgdevice[i+1].txz1_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyz1_d,mgdevice[i].tyz2_d,mgdevice[i+1].tyz1_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i+1].txx1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i+1].tyy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i+1].tzz1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txy1_d,mgdevice[i+1].txy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txz1_d,mgdevice[i+1].txz1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyz1_d,mgdevice[i+1].tyz1_d,nnx,nny,nnz_device_append);
				}
			}

			checkCudaErrors(hipDeviceSynchronize());
}

void exchange_device_nz_kernel_tao2(GPUdevice *mgdevice,int mark)
///exchange_device_nz_kernel_tao(mgdevice);
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txy1_d,mgdevice[i].txy2_d,mgdevice[i+1].txy1_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txz1_d,mgdevice[i].txz2_d,mgdevice[i+1].txz1_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);					
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyz1_d,mgdevice[i].tyz2_d,mgdevice[i+1].tyz1_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}


				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txx2_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyy2_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tzz2_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txy2_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].txz2_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tyz2_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx1_d,mgdevice[i].txx2_d,mgdevice[i+1].txx1_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy1_d,mgdevice[i].tyy2_d,mgdevice[i+1].tyy1_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz1_d,mgdevice[i].tzz2_d,mgdevice[i+1].tzz1_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txy1_d,mgdevice[i].txy2_d,mgdevice[i+1].txy1_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txz1_d,mgdevice[i].txz2_d,mgdevice[i+1].txz1_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyz1_d,mgdevice[i].tyz2_d,mgdevice[i+1].tyz1_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txx2_d,mgdevice[i+1].txx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyy2_d,mgdevice[i+1].tyy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tzz2_d,mgdevice[i+1].tzz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txy2_d,mgdevice[i+1].txy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].txz2_d,mgdevice[i+1].txz2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tyz2_d,mgdevice[i+1].tyz2_d,nnx,nny,nnz_device_append);
				}
			}

			checkCudaErrors(hipDeviceSynchronize());
}

void exchange_device_nz_kernel_vx_vy_vz1(GPUdevice *mgdevice,int mark)
///exchange_device_nz_kernel(mgdevice);
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vy1_d,mgdevice[i].vy2_d,mgdevice[i+1].vy1_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vz1_d,mgdevice[i].vz2_d,mgdevice[i+1].vz1_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx1_d,mgdevice[i+1].vx1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vy1_d,mgdevice[i+1].vy1_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vz1_d,mgdevice[i+1].vz1_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{				
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy1_d,mgdevice[i].vy2_d,mgdevice[i+1].vy1_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz1_d,mgdevice[i].vz2_d,mgdevice[i+1].vz1_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{	
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx1_d,mgdevice[i+1].vx1_d,nnx,nny,nnz_device_append);

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy1_d,mgdevice[i+1].vy1_d,nnx,nny,nnz_device_append);

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz1_d,mgdevice[i+1].vz1_d,nnx,nny,nnz_device_append);
				}

			}

			checkCudaErrors(hipDeviceSynchronize());
}

void exchange_device_nz_kernel_vx_vy_vz2(GPUdevice *mgdevice,int mark)
///exchange_device_nz_kernel(mgdevice);
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vy1_d,mgdevice[i].vy2_d,mgdevice[i+1].vy1_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vz1_d,mgdevice[i].vz2_d,mgdevice[i+1].vz1_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vx2_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vy2_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);
					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].vz2_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{				
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx1_d,mgdevice[i].vx2_d,mgdevice[i+1].vx1_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy1_d,mgdevice[i].vy2_d,mgdevice[i+1].vy1_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz1_d,mgdevice[i].vz2_d,mgdevice[i+1].vz1_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{	
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx2_d,mgdevice[i+1].vx2_d,nnx,nny,nnz_device_append);

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy2_d,mgdevice[i+1].vy2_d,nnx,nny,nnz_device_append);

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz2_d,mgdevice[i+1].vz2_d,nnx,nny,nnz_device_append);
				}

			}

			checkCudaErrors(hipDeviceSynchronize());
}



void exchange_device_nz_kernel_taop1(GPUdevice *mgdevice,int mark)
///exchange_device_nz_kernel(mgdevice);
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tp1_d,mgdevice[i].tp2_d,mgdevice[i+1].tp1_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tp1_d,mgdevice[i+1].tp1_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{				
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp1_d,mgdevice[i].tp2_d,mgdevice[i+1].tp1_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{	
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp1_d,mgdevice[i+1].tp1_d,nnx,nny,nnz_device_append);
				}

			}

			checkCudaErrors(hipDeviceSynchronize());
}


void exchange_device_nz_kernel_taop2(GPUdevice *mgdevice,int mark)
///exchange_device_nz_kernel(mgdevice);
{

			dim3 dimBlock(32,16);

			dim3 dimGridwf_append((nnx+dimBlock.x-1)/dimBlock.x,(nny+dimBlock.y-1)/dimBlock.y,nnz_device_append);//////单块卡的整个空间

			for(int i=0;i<GPU_N-1;i++)
			{
				if(mark==0)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tp1_d,mgdevice[i].tp2_d,mgdevice[i+1].tp1_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

				if(mark==1)
				{
					checkCudaErrors(hipSetDevice(gpuid[0]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock>>>(mgdevice[i].tp2_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

				if(mark==2)
				{				
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp1_d,mgdevice[i].tp2_d,mgdevice[i+1].tp1_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

				if(mark==3)
				{	
					checkCudaErrors(hipSetDevice(gpuid[i]));

					exchange_device_nz_one<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp2_d,mgdevice[i+1].tp2_d,nnx,nny,nnz_device_append);
				}

			}

			checkCudaErrors(hipDeviceSynchronize());
}













void test_exchange_device(GPUdevice *mgdevice)
{
	for(int i=0;i<GPU_N;i++)
		{
				checkCudaErrors(hipSetDevice(gpuid[i]));
				
				for(int ix=0;ix<nnx;ix++)
					for(int iy=0;iy<nny;iy++)
						for(int iz=0;iz<nnz_device_append;iz++)
							mgdevice[i].wf_h[iz*nnx*nny+iy*nnx+ix]=1.0*i+1;

				checkCudaErrors(hipMemcpy(mgdevice[i].vx1_d,mgdevice[i].wf_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
				checkCudaErrors(hipMemcpy(mgdevice[i].txx1_d,mgdevice[i].wf_h,nnx*nny*nnz_device_append*sizeof(float),hipMemcpyDefault));
								
		}

		hipDeviceSynchronize();
}



void transfer_gpu_to_cpu_multicomponent_seismic(GPUdevice *mgdevice,int it,int mark)
{
		
		if(mark==0)
		{
			checkCudaErrors(hipSetDevice(gpuid[choose_re]));

			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_x_h,mgdevice[choose_re].obs_shot_x_d,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_y_h,mgdevice[choose_re].obs_shot_y_d,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_z_h,mgdevice[choose_re].obs_shot_z_d,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));

			hipDeviceSynchronize();

			checkCudaErrors(hipSetDevice(gpuid[choose_re]));


			for(int ix=0;ix<receiver_num_x;ix++)
				for(int iy=0;iy<receiver_num_y;iy++)
					{
						obs_shot_x_all[it][iy][ix]=mgdevice[choose_re].obs_shot_x_h[iy*receiver_num_x+ix];
						obs_shot_y_all[it][iy][ix]=mgdevice[choose_re].obs_shot_y_h[iy*receiver_num_x+ix];
						obs_shot_z_all[it][iy][ix]=mgdevice[choose_re].obs_shot_z_h[iy*receiver_num_x+ix];
					}

			hipDeviceSynchronize();
		}

		else
		{
			checkCudaErrors(hipSetDevice(gpuid[choose_re]));

			for(int ix=0;ix<receiver_num_x;ix++)
				for(int iy=0;iy<receiver_num_y;iy++)
					{
						mgdevice[choose_re].obs_shot_x_h[iy*receiver_num_x+ix]=obs_shot_x_all[it][iy][ix];
						mgdevice[choose_re].obs_shot_y_h[iy*receiver_num_x+ix]=obs_shot_y_all[it][iy][ix];
						mgdevice[choose_re].obs_shot_z_h[iy*receiver_num_x+ix]=obs_shot_z_all[it][iy][ix];
					}
			hipDeviceSynchronize();

			checkCudaErrors(hipSetDevice(gpuid[choose_re]));
		
			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_x_h,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_y_h,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));
			checkCudaErrors(hipMemcpy(mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].obs_shot_z_h,receiver_num_x*receiver_num_y*sizeof(float),hipMemcpyDefault));

			hipDeviceSynchronize();

		}
}

void output_or_input_multicomponent_seismic(int mark)
{
		//system("mkdir shotgather");
		if(mark==0)
		{
			sprintf(filename,"./shotgather/obs_x_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			write_file_3d(obs_shot_x_all,receiver_num_x,receiver_num_y,lt,filename);

			sprintf(filename,"./shotgather/obs_y_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			write_file_3d(obs_shot_y_all,receiver_num_x,receiver_num_y,lt,filename);

			sprintf(filename,"./shotgather/obs_z_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			write_file_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt,filename);
		}

		else
		{
			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_x_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_x_all,receiver_num_x,receiver_num_y,lt,filename);


			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_y_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_y_all,receiver_num_x,receiver_num_y,lt,filename);


			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_z_shot_%d_%d",sy_real*int(dy),sx_real*int(dx));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt,filename);
		}
}

void output_or_input_multicomponent_seismic_vsp(int mark)
{
		//system("mkdir shotgather");
		if(mark==0)
		{
			sprintf(filename,"./shotgather/obs_x_shot_%d",sz_real*int(dz));
			write_file_3d(obs_shot_x_all,receiver_num_x,receiver_num_y,lt,filename);

			sprintf(filename,"./shotgather/obs_y_shot_%d",sz_real*int(dz));
			write_file_3d(obs_shot_y_all,receiver_num_x,receiver_num_y,lt,filename);

			sprintf(filename,"./shotgather/obs_z_shot_%d",sz_real*int(dz));
			write_file_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt,filename);
		}

		if(mark==3)
		{
			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_x_shot_%d",sz_real*int(dz));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_x_all,receiver_num_x,receiver_num_y,lt,filename);


			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_y_shot_%d",sz_real*int(dz));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_y_all,receiver_num_x,receiver_num_y,lt,filename);


			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_z_shot_%d",sz_real*int(dz));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt,filename);
		}

		if(mark==1)
		{
			set_zero_3d(obs_shot_x_all,receiver_num_x,receiver_num_y,lt);
			set_zero_3d(obs_shot_y_all,receiver_num_x,receiver_num_y,lt);
			set_zero_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt);


			sprintf(filename,infile_shot_name);
			sprintf(filename1,"_%d",sz_real*int(dz));
			strcat(filename,filename1);
			fread_file_3d(obs_shot_z_all,receiver_num_x,receiver_num_y,lt,filename);
		}
}
//sprintf(filename1,"/nvresultppx-shot_%d",ishot);
//sprintf(filename,infile_shot_name);
//strcat(filename,filename1);
//write_file_1d(nvresultppx,nx*nz,filename);























//////////////2018年01月11日 星期四 20时32分14秒 3D elastic modeling
__global__ void fwd_vx_3D(float *vx2_d,float *vx1_d,float *txx1_d,float *txy1_d,float *txz1_d,float *velocity_d,float *velocity1_d,float *density_d,float *att_d,float *coe_d,int nx_pml,int ny_pml,int nz_pml,float dt,float coe_x,float coe_y,float coe_z)
//fwd_vx_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vx2_d,mgdevice[i].vx1_d,mgdevice[i].txx1_d,mgdevice[i].txy1_d,mgdevice[i].txz1_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
{
		float dt_real;
		dt_real=dt/1000;

		float s_att;
		//float vp,vs;
		float den;

		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

					
				s_att=att_d[idx];
				//vp=velocity_d[idx];
				//vs=velocity1_d[idx];
				den=density_d[idx];


				float txx_x=			coe_d[1]*coe_x*(txx1_d[idx+1]-txx1_d[idx]);
								txx_x+=coe_d[2]*coe_x*(txx1_d[idx+2]-txx1_d[idx-1]);
								txx_x+=coe_d[3]*coe_x*(txx1_d[idx+3]-txx1_d[idx-2]);
								txx_x+=coe_d[4]*coe_x*(txx1_d[idx+4]-txx1_d[idx-3]);
								txx_x+=coe_d[5]*coe_x*(txx1_d[idx+5]-txx1_d[idx-4]);
								txx_x+=coe_d[6]*coe_x*(txx1_d[idx+6]-txx1_d[idx-5]);

				float txy_y=			coe_d[1]*coe_y*(txy1_d[idx]-txy1_d[idx-1*nx_pml]);
								txy_y+=coe_d[2]*coe_y*(txy1_d[idx+1*nx_pml]-txy1_d[idx-2*nx_pml]);
								txy_y+=coe_d[3]*coe_y*(txy1_d[idx+2*nx_pml]-txy1_d[idx-3*nx_pml]);
								txy_y+=coe_d[4]*coe_y*(txy1_d[idx+3*nx_pml]-txy1_d[idx-4*nx_pml]);
								txy_y+=coe_d[5]*coe_y*(txy1_d[idx+4*nx_pml]-txy1_d[idx-5*nx_pml]);
								txy_y+=coe_d[6]*coe_y*(txy1_d[idx+5*nx_pml]-txy1_d[idx-6*nx_pml]);

				float txz_z=			coe_d[1]*coe_z*(txz1_d[idx]-txz1_d[idx-1*nx_pml*ny_pml]);
								txz_z+=coe_d[2]*coe_z*(txz1_d[idx+1*nx_pml*ny_pml]-txz1_d[idx-2*nx_pml*ny_pml]);
								txz_z+=coe_d[3]*coe_z*(txz1_d[idx+2*nx_pml*ny_pml]-txz1_d[idx-3*nx_pml*ny_pml]);
							 	txz_z+=coe_d[4]*coe_z*(txz1_d[idx+3*nx_pml*ny_pml]-txz1_d[idx-4*nx_pml*ny_pml]);
							 	txz_z+=coe_d[5]*coe_z*(txz1_d[idx+4*nx_pml*ny_pml]-txz1_d[idx-5*nx_pml*ny_pml]);
							 	txz_z+=coe_d[6]*coe_z*(txz1_d[idx+5*nx_pml*ny_pml]-txz1_d[idx-6*nx_pml*ny_pml]);

				//if(den!=0)	vx2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vx1_d[idx]+1.0/den*(txx_x+txy_y+txz_z));

						vx2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vx1_d[idx]+1.0/den*(txx_x+txy_y+txz_z));

		}
}

__global__ void fwd_vy_3D(float *vy2_d,float *vy1_d,float *txy1_d,float *tyy1_d,float *tyz1_d,float *velocity_d,float *velocity1_d,float *density_d,float *att_d,float *coe_d,int nx_pml,int ny_pml,int nz_pml,float dt,float coe_x,float coe_y,float coe_z)
//fwd_vy_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vy2_d,mgdevice[i].vy1_d,mgdevice[i].txy1_d,mgdevice[i].tyy1_d,mgdevice[i].tyz1_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
{
		float dt_real;
		dt_real=dt/1000;

		float s_att;
		//float vp,vs;
		float den;

		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

					
				s_att=att_d[idx];
				//vp=velocity_d[idx];
				//vs=velocity1_d[idx];
				den=density_d[idx];


				float txy_x=			coe_d[1]*coe_x*(txy1_d[idx]-txy1_d[idx-1]);
								txy_x+=coe_d[2]*coe_x*(txy1_d[idx+1]-txy1_d[idx-2]);
								txy_x+=coe_d[3]*coe_x*(txy1_d[idx+2]-txy1_d[idx-3]);
								txy_x+=coe_d[4]*coe_x*(txy1_d[idx+3]-txy1_d[idx-4]);
								txy_x+=coe_d[5]*coe_x*(txy1_d[idx+4]-txy1_d[idx-5]);
								txy_x+=coe_d[6]*coe_x*(txy1_d[idx+5]-txy1_d[idx-6]);

				float tyy_y=			coe_d[1]*coe_y*(tyy1_d[idx+1*nx_pml]-tyy1_d[idx]);
								tyy_y+=coe_d[2]*coe_y*(tyy1_d[idx+2*nx_pml]-tyy1_d[idx-1*nx_pml]);
								tyy_y+=coe_d[3]*coe_y*(tyy1_d[idx+3*nx_pml]-tyy1_d[idx-2*nx_pml]);
								tyy_y+=coe_d[4]*coe_y*(tyy1_d[idx+4*nx_pml]-tyy1_d[idx-3*nx_pml]);
								tyy_y+=coe_d[5]*coe_y*(tyy1_d[idx+5*nx_pml]-tyy1_d[idx-4*nx_pml]);
								tyy_y+=coe_d[6]*coe_y*(tyy1_d[idx+6*nx_pml]-tyy1_d[idx-5*nx_pml]);

				float tyz_z=			coe_d[1]*coe_z*(tyz1_d[idx]-tyz1_d[idx-1*nx_pml*ny_pml]);
								tyz_z+=coe_d[2]*coe_z*(tyz1_d[idx+1*nx_pml*ny_pml]-tyz1_d[idx-2*nx_pml*ny_pml]);
								tyz_z+=coe_d[3]*coe_z*(tyz1_d[idx+2*nx_pml*ny_pml]-tyz1_d[idx-3*nx_pml*ny_pml]);
							 	tyz_z+=coe_d[4]*coe_z*(tyz1_d[idx+3*nx_pml*ny_pml]-tyz1_d[idx-4*nx_pml*ny_pml]);
							 	tyz_z+=coe_d[5]*coe_z*(tyz1_d[idx+4*nx_pml*ny_pml]-tyz1_d[idx-5*nx_pml*ny_pml]);
							 	tyz_z+=coe_d[6]*coe_z*(tyz1_d[idx+5*nx_pml*ny_pml]-tyz1_d[idx-6*nx_pml*ny_pml]);

				//if(den!=0)	vy2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vy1_d[idx]+1.0/den*(txy_x+tyy_y+tyz_z));
			
						vy2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vy1_d[idx]+1.0/den*(txy_x+tyy_y+tyz_z));

		}
}

__global__ void fwd_vz_3D(float *vz2_d,float *vz1_d,float *txz1_d,float *tyz1_d,float *tzz1_d,float *velocity_d,float *velocity1_d,float *density_d,float *att_d,float *coe_d,int nx_pml,int ny_pml,int nz_pml,float dt,float coe_x,float coe_y,float coe_z)
//fwd_vz_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vz2_d,mgdevice[i].vz1_d,mgdevice[i].txz1_d,mgdevice[i].tyz1_d,mgdevice[i].tzz1_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
{
		float dt_real;
		dt_real=dt/1000;

		float s_att;
		//float vp,vs;
		float den;

		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

					
				s_att=att_d[idx];
				//vp=velocity_d[idx];
				//vs=velocity1_d[idx];
				den=density_d[idx];


				float txz_x=			coe_d[1]*coe_x*(txz1_d[idx]-txz1_d[idx-1]);
								txz_x+=coe_d[2]*coe_x*(txz1_d[idx+1]-txz1_d[idx-2]);
								txz_x+=coe_d[3]*coe_x*(txz1_d[idx+2]-txz1_d[idx-3]);
								txz_x+=coe_d[4]*coe_x*(txz1_d[idx+3]-txz1_d[idx-4]);
								txz_x+=coe_d[5]*coe_x*(txz1_d[idx+4]-txz1_d[idx-5]);
								txz_x+=coe_d[6]*coe_x*(txz1_d[idx+5]-txz1_d[idx-6]);

				float tyz_y=			coe_d[1]*coe_y*(tyz1_d[idx]-tyz1_d[idx-1*nx_pml]);
								tyz_y+=coe_d[2]*coe_y*(tyz1_d[idx+1*nx_pml]-tyz1_d[idx-2*nx_pml]);
								tyz_y+=coe_d[3]*coe_y*(tyz1_d[idx+2*nx_pml]-tyz1_d[idx-3*nx_pml]);
								tyz_y+=coe_d[4]*coe_y*(tyz1_d[idx+3*nx_pml]-tyz1_d[idx-4*nx_pml]);
								tyz_y+=coe_d[5]*coe_y*(tyz1_d[idx+4*nx_pml]-tyz1_d[idx-5*nx_pml]);
								tyz_y+=coe_d[6]*coe_y*(tyz1_d[idx+5*nx_pml]-tyz1_d[idx-6*nx_pml]);

				float tzz_z=			coe_d[1]*coe_z*(tzz1_d[idx+1*nx_pml*ny_pml]-tzz1_d[idx]);
								tzz_z+=coe_d[2]*coe_z*(tzz1_d[idx+2*nx_pml*ny_pml]-tzz1_d[idx-1*nx_pml*ny_pml]);
								tzz_z+=coe_d[3]*coe_z*(tzz1_d[idx+3*nx_pml*ny_pml]-tzz1_d[idx-2*nx_pml*ny_pml]);
							 	tzz_z+=coe_d[4]*coe_z*(tzz1_d[idx+4*nx_pml*ny_pml]-tzz1_d[idx-3*nx_pml*ny_pml]);
							 	tzz_z+=coe_d[5]*coe_z*(tzz1_d[idx+5*nx_pml*ny_pml]-tzz1_d[idx-4*nx_pml*ny_pml]);
							 	tzz_z+=coe_d[6]*coe_z*(tzz1_d[idx+6*nx_pml*ny_pml]-tzz1_d[idx-5*nx_pml*ny_pml]);

				//if(den!=0)	vz2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vz1_d[idx]+1.0/den*(txz_x+tyz_y+tzz_z));

						vz2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vz1_d[idx]+1.0/den*(txz_x+tyz_y+tzz_z));

		}
}


__global__ void fwd_txxzzxzpp_3D(float *tp2_d,float *tp1_d,float *txx2_d,float *txx1_d,float *tyy2_d,float *tyy1_d,float *tzz2_d,float *tzz1_d,float *txy2_d,float *txy1_d,float *txz2_d,float *txz1_d,float *tyz2_d,float *tyz1_d,float *vx2_d,float *vy2_d,float *vz2_d,float *velocity_d,float *velocity1_d,float *density_d,float *att_d,float *coe_d,int nx_pml,int ny_pml,int nz_pml,float dt,float coe_x,float coe_y,float coe_z)
//fwd_txxzzxzpp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].tp2_d,mgdevice[i].tp1_d,mgdevice[i].txx2_d,mgdevice[i].txx1_d,mgdevice[i].tyy2_d,mgdevice[i].tyy1_d,mgdevice[i].tzz2_d,mgdevice[i].tzz1_d,mgdevice[i].txy2_d,mgdevice[i].txy1_d,mgdevice[i].txz2_d,mgdevice[i].txz1_d,mgdevice[i].tyz2_d,mgdevice[i].tyz1_d,mgdevice[i].vx2_d,mgdevice[i].vy2_d,mgdevice[i].vz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);
{
		float dt_real;
		dt_real=dt/1000;

		float s_att;
		float vp,vs;
		float den;

		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

					
				s_att=att_d[idx];
				vp=velocity_d[idx];
				vs=velocity1_d[idx];
				den=density_d[idx];


				float vx_x=			coe_d[1]*coe_x*(vx2_d[idx]-vx2_d[idx-1]);
								vx_x+=coe_d[2]*coe_x*(vx2_d[idx+1]-vx2_d[idx-2]);
								vx_x+=coe_d[3]*coe_x*(vx2_d[idx+2]-vx2_d[idx-3]);
								vx_x+=coe_d[4]*coe_x*(vx2_d[idx+3]-vx2_d[idx-4]);
								vx_x+=coe_d[5]*coe_x*(vx2_d[idx+4]-vx2_d[idx-5]);
								vx_x+=coe_d[6]*coe_x*(vx2_d[idx+5]-vx2_d[idx-6]);

				float vy_y=			coe_d[1]*coe_y*(vy2_d[idx]-vy2_d[idx-1*nx_pml]);
								vy_y+=coe_d[2]*coe_y*(vy2_d[idx+1*nx_pml]-vy2_d[idx-2*nx_pml]);
								vy_y+=coe_d[3]*coe_y*(vy2_d[idx+2*nx_pml]-vy2_d[idx-3*nx_pml]);
								vy_y+=coe_d[4]*coe_y*(vy2_d[idx+3*nx_pml]-vy2_d[idx-4*nx_pml]);
								vy_y+=coe_d[5]*coe_y*(vy2_d[idx+4*nx_pml]-vy2_d[idx-5*nx_pml]);
								vy_y+=coe_d[6]*coe_y*(vy2_d[idx+5*nx_pml]-vy2_d[idx-6*nx_pml]);

				float vz_z=			coe_d[1]*coe_z*(vz2_d[idx]-vz2_d[idx-1*nx_pml*ny_pml]);
								vz_z+=coe_d[2]*coe_z*(vz2_d[idx+1*nx_pml*ny_pml]-vz2_d[idx-2*nx_pml*ny_pml]);
								vz_z+=coe_d[3]*coe_z*(vz2_d[idx+2*nx_pml*ny_pml]-vz2_d[idx-3*nx_pml*ny_pml]);
							 	vz_z+=coe_d[4]*coe_z*(vz2_d[idx+3*nx_pml*ny_pml]-vz2_d[idx-4*nx_pml*ny_pml]);
							 	vz_z+=coe_d[5]*coe_z*(vz2_d[idx+4*nx_pml*ny_pml]-vz2_d[idx-5*nx_pml*ny_pml]);
							 	vz_z+=coe_d[6]*coe_z*(vz2_d[idx+5*nx_pml*ny_pml]-vz2_d[idx-6*nx_pml*ny_pml]);

				txx2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*txx1_d[idx]+
									vp*vp*den*(vx_x+vy_y+vz_z)-2.0*vs*vs*den*(vy_y+vz_z));

				
				tyy2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*tyy1_d[idx]+
									vp*vp*den*(vx_x+vy_y+vz_z)-2.0*vs*vs*den*(vx_x+vz_z));
				 

				tzz2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*tzz1_d[idx]+
									vp*vp*den*(vx_x+vy_y+vz_z)-2.0*vs*vs*den*(vx_x+vy_y));


				tp2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*tp1_d[idx]+vp*vp*den*(vx_x+vy_y+vz_z));


				float vx_y=			coe_d[1]*coe_y*(vx2_d[idx+1*nx_pml]-vx2_d[idx]);
								vx_y+=coe_d[2]*coe_y*(vx2_d[idx+2*nx_pml]-vx2_d[idx-1*nx_pml]);
								vx_y+=coe_d[3]*coe_y*(vx2_d[idx+3*nx_pml]-vx2_d[idx-2*nx_pml]);
								vx_y+=coe_d[4]*coe_y*(vx2_d[idx+4*nx_pml]-vx2_d[idx-3*nx_pml]);
								vx_y+=coe_d[5]*coe_y*(vx2_d[idx+5*nx_pml]-vx2_d[idx-4*nx_pml]);
								vx_y+=coe_d[6]*coe_y*(vx2_d[idx+6*nx_pml]-vx2_d[idx-5*nx_pml]);

				float vx_z=			coe_d[1]*coe_z*(vx2_d[idx+1*nx_pml*ny_pml]-vx2_d[idx]);
								vx_z+=coe_d[2]*coe_z*(vx2_d[idx+2*nx_pml*ny_pml]-vx2_d[idx-1*nx_pml*ny_pml]);
								vx_z+=coe_d[3]*coe_z*(vx2_d[idx+3*nx_pml*ny_pml]-vx2_d[idx-2*nx_pml*ny_pml]);
							 	vx_z+=coe_d[4]*coe_z*(vx2_d[idx+4*nx_pml*ny_pml]-vx2_d[idx-3*nx_pml*ny_pml]);
							 	vx_z+=coe_d[5]*coe_z*(vx2_d[idx+5*nx_pml*ny_pml]-vx2_d[idx-4*nx_pml*ny_pml]);
							 	vx_z+=coe_d[6]*coe_z*(vx2_d[idx+6*nx_pml*ny_pml]-vx2_d[idx-5*nx_pml*ny_pml]);;

				float vy_x=			coe_d[1]*coe_x*(vy2_d[idx+1]-vy2_d[idx]);
								vy_x+=coe_d[2]*coe_x*(vy2_d[idx+2]-vy2_d[idx-1]);
								vy_x+=coe_d[3]*coe_x*(vy2_d[idx+3]-vy2_d[idx-2]);
								vy_x+=coe_d[4]*coe_x*(vy2_d[idx+4]-vy2_d[idx-3]);
								vy_x+=coe_d[5]*coe_x*(vy2_d[idx+5]-vy2_d[idx-4]);
								vy_x+=coe_d[6]*coe_x*(vy2_d[idx+6]-vy2_d[idx-5]);

				float vy_z=			coe_d[1]*coe_z*(vy2_d[idx+1*nx_pml*ny_pml]-vy2_d[idx]);
								vy_z+=coe_d[2]*coe_z*(vy2_d[idx+2*nx_pml*ny_pml]-vy2_d[idx-1*nx_pml*ny_pml]);
								vy_z+=coe_d[3]*coe_z*(vy2_d[idx+3*nx_pml*ny_pml]-vy2_d[idx-2*nx_pml*ny_pml]);
							 	vy_z+=coe_d[4]*coe_z*(vy2_d[idx+4*nx_pml*ny_pml]-vy2_d[idx-3*nx_pml*ny_pml]);
							 	vy_z+=coe_d[5]*coe_z*(vy2_d[idx+5*nx_pml*ny_pml]-vy2_d[idx-4*nx_pml*ny_pml]);
							 	vy_z+=coe_d[6]*coe_z*(vy2_d[idx+6*nx_pml*ny_pml]-vy2_d[idx-5*nx_pml*ny_pml]);


				float vz_x=			coe_d[1]*coe_x*(vz2_d[idx+1]-vz2_d[idx]);
								vz_x+=coe_d[2]*coe_x*(vz2_d[idx+2]-vz2_d[idx-1]);
								vz_x+=coe_d[3]*coe_x*(vz2_d[idx+3]-vz2_d[idx-2]);
								vz_x+=coe_d[4]*coe_x*(vz2_d[idx+4]-vz2_d[idx-3]);
								vz_x+=coe_d[5]*coe_x*(vz2_d[idx+5]-vz2_d[idx-4]);
								vz_x+=coe_d[6]*coe_x*(vz2_d[idx+6]-vz2_d[idx-5]);

				float vz_y=			coe_d[1]*coe_y*(vz2_d[idx+1*nx_pml]-vz2_d[idx]);
								vz_y+=coe_d[2]*coe_y*(vz2_d[idx+2*nx_pml]-vz2_d[idx-1*nx_pml]);
								vz_y+=coe_d[3]*coe_y*(vz2_d[idx+3*nx_pml]-vz2_d[idx-2*nx_pml]);
								vz_y+=coe_d[4]*coe_y*(vz2_d[idx+4*nx_pml]-vz2_d[idx-3*nx_pml]);
								vz_y+=coe_d[5]*coe_y*(vz2_d[idx+5*nx_pml]-vz2_d[idx-4*nx_pml]);
								vz_y+=coe_d[6]*coe_y*(vz2_d[idx+6*nx_pml]-vz2_d[idx-5*nx_pml]);


					txy2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*txy1_d[idx]+vs*vs*den*(vx_y+vy_x));

					txz2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*txz1_d[idx]+vs*vs*den*(vx_z+vz_x));
										
					tyz2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*tyz1_d[idx]+vs*vs*den*(vy_z+vz_y));	

		}
}
__global__ void fwd_vxp_vzp_3D(float *vxp2_d,float *vxp1_d,float *vyp2_d,float *vyp1_d,float *vzp2_d,float *vzp1_d,float *tp2_d,float *vxs2_d,float *vys2_d,float *vzs2_d,float *vx2_d,float *vy2_d,float *vz2_d,float *velocity_d,float *velocity1_d,float *density_d,float *att_d,float *coe_d,int nx_pml,int ny_pml,int nz_pml,float dt,float coe_x,float coe_y,float coe_z)
//fwd_vxp_vzp_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].vxp2_d,mgdevice[i].vxp1_d,mgdevice[i].vyp2_d,mgdevice[i].vyp1_d,mgdevice[i].vzp2_d,mgdevice[i].vzp1_d,mgdevice[i].tp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].vx2_d,mgdevice[i].vy2_d,mgdevice[i].vz2_d,mgdevice[i].velocity_d,mgdevice[i].velocity1_d,mgdevice[i].density_d,mgdevice[i].att_d,mgdevice[i].coe_d,nnx,nny,nnz_device_append,dt,coe_x,coe_y,coe_z);	
{
		float dt_real;
		dt_real=dt/1000;

		float s_att;
		//float vp,vs;
		float den;

		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

					
				s_att=att_d[idx];
				//vp=velocity_d[idx];
				//vs=velocity1_d[idx];
				den=density_d[idx];


				float tp_x=			coe_d[1]*coe_x*(tp2_d[idx+1]-tp2_d[idx]);
								tp_x+=coe_d[2]*coe_x*(tp2_d[idx+2]-tp2_d[idx-1]);
								tp_x+=coe_d[3]*coe_x*(tp2_d[idx+3]-tp2_d[idx-2]);
								tp_x+=coe_d[4]*coe_x*(tp2_d[idx+4]-tp2_d[idx-3]);
								tp_x+=coe_d[5]*coe_x*(tp2_d[idx+5]-tp2_d[idx-4]);
								tp_x+=coe_d[6]*coe_x*(tp2_d[idx+6]-tp2_d[idx-5]);

				float tp_y=			coe_d[1]*coe_y*(tp2_d[idx+1*nx_pml]-tp2_d[idx]);
								tp_y+=coe_d[2]*coe_y*(tp2_d[idx+2*nx_pml]-tp2_d[idx-1*nx_pml]);
								tp_y+=coe_d[3]*coe_y*(tp2_d[idx+3*nx_pml]-tp2_d[idx-2*nx_pml]);
								tp_y+=coe_d[4]*coe_y*(tp2_d[idx+4*nx_pml]-tp2_d[idx-3*nx_pml]);
								tp_y+=coe_d[5]*coe_y*(tp2_d[idx+5*nx_pml]-tp2_d[idx-4*nx_pml]);
								tp_y+=coe_d[6]*coe_y*(tp2_d[idx+6*nx_pml]-tp2_d[idx-5*nx_pml]);

				float tp_z=			coe_d[1]*coe_z*(tp2_d[idx+1*nx_pml*ny_pml]-tp2_d[idx]);
								tp_z+=coe_d[2]*coe_z*(tp2_d[idx+2*nx_pml*ny_pml]-tp2_d[idx-1*nx_pml*ny_pml]);
								tp_z+=coe_d[3]*coe_z*(tp2_d[idx+3*nx_pml*ny_pml]-tp2_d[idx-2*nx_pml*ny_pml]);
							 	tp_z+=coe_d[4]*coe_z*(tp2_d[idx+4*nx_pml*ny_pml]-tp2_d[idx-3*nx_pml*ny_pml]);
							 	tp_z+=coe_d[5]*coe_z*(tp2_d[idx+5*nx_pml*ny_pml]-tp2_d[idx-4*nx_pml*ny_pml]);
							 	tp_z+=coe_d[6]*coe_z*(tp2_d[idx+6*nx_pml*ny_pml]-tp2_d[idx-5*nx_pml*ny_pml]);


				vxp2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vxp1_d[idx]+1.0/den*tp_x);

				vyp2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vyp1_d[idx]+1.0/den*tp_y);

				vzp2_d[idx]=1.0/(1.0+s_att*dt_real/2.0)*((1.0-s_att*dt_real/2.0)*vzp1_d[idx]+1.0/den*tp_z);

				vxs2_d[idx]=vx2_d[idx]-vxp2_d[idx];

				vys2_d[idx]=vy2_d[idx]-vyp2_d[idx];

				vzs2_d[idx]=vz2_d[idx]-vzp2_d[idx];

		}
}	
__global__ void vp_vs_3D(float *vx2_d,float *vy2_d,float *vz2_d,float *vxp2_d,float *vyp2_d,float *vzp2_d,float *vxs2_d,float *vys2_d,float *vzs2_d,int nx_pml,int ny_pml,int nz_pml)
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

				vxs2_d[idx]=vx2_d[idx]-vxp2_d[idx];
				vys2_d[idx]=vy2_d[idx]-vyp2_d[idx];
				vzs2_d[idx]=vz2_d[idx]-vzp2_d[idx];
		}

}


__global__ void write_or_add_shot_3D_surface(float *obs_shot_x_d,float *vx2_d,int nnx,int nny,int nnz_device_append,int nnz_device,int bl,int bb,int bu,int receiver_start_x,int receiver_num_x,int receiver_interval_x,int receiver_start_y,int receiver_num_y,int receiver_interval_y,int receiver_start_z,int receiver_num_z,int receiver_interval_z,int mark)
//write_or_add_shot_3D_surface(obs_shot_x_d,vx2_d,nnx,nny,nnz_device_append,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,0);
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;

		int id,id1;

		int remain=(receiver_start_z+bu)%nnz_device+radius;

		if(ix<receiver_num_x&&iy<receiver_num_y)
		{
			id=iy*receiver_num_x+ix;

			id1=remain*nnx*nny+(receiver_start_y+iy*receiver_interval_y+bb)*nnx+(receiver_start_x+ix*receiver_interval_x+bl);

			if(mark==0)	obs_shot_x_d[id]=vx2_d[id1];

			if(mark==1)	vx2_d[id1]+=obs_shot_x_d[id];
		}
}

__global__ void write_or_add_shot_3D_surface_three(float *obs_shot_x_d,float *obs_shot_y_d,float *obs_shot_z_d,float *vx2_d,float *vy2_d,float *vz2_d,int nnx,int nny,int nnz_device_append,int nnz_device,int bl,int bb,int bu,int receiver_start_x,int receiver_num_x,int receiver_interval_x,int receiver_start_y,int receiver_num_y,int receiver_interval_y,int receiver_start_z,int receiver_num_z,int receiver_interval_z,int mark)
//write_or_add_shot_3D_surface_three<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_re].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].obs_shot_y_d,mgdevice[choose_re].obs_shot_z_d,mgdevice[choose_re].vx2_d,mgdevice[choose_re].vy2_d,mgdevice[choose_re].vz2_d,nnx,nny,nnz_device_append,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,0);	
{
		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;

		int id,id1;

		int remain=(receiver_start_z+bu)%nnz_device+radius;

		if(ix<receiver_num_x&&iy<receiver_num_y)
		{
			id=iy*receiver_num_x+ix;

			id1=remain*nnx*nny+(receiver_start_y+iy*receiver_interval_y+bb)*nnx+(receiver_start_x+ix*receiver_interval_x+bl);

			if(mark==0)	
			{
				obs_shot_x_d[id]=vx2_d[id1];
				obs_shot_y_d[id]=vy2_d[id1];
				obs_shot_z_d[id]=vz2_d[id1];
			}

			if(mark==1)	
			{
				vx2_d[id1]=obs_shot_x_d[id];
				vy2_d[id1]=obs_shot_y_d[id];
				vz2_d[id1]=obs_shot_z_d[id];
			}

			if(mark==2)	
			{
				vx2_d[id1]+=obs_shot_x_d[id];
				vy2_d[id1]+=obs_shot_y_d[id];
				vz2_d[id1]+=obs_shot_z_d[id];
			}
		}
}


__global__ void cut_direct_shot_3D_surface_three(float *obs_shot_x_d,float *obs_shot_y_d,float *obs_shot_z_d,float *velocity_d,int nnx,int nny,int nnz_device,int bl,int bb,int bu,int receiver_start_x,int receiver_num_x,int receiver_interval_x,int receiver_start_y,int receiver_num_y,int receiver_interval_y,int receiver_start_z,int receiver_num_z,int receiver_interval_z,int sx,int sy,int sz,int it,int wavelet_length,float dx,float dy,float dz,float dt)
//cut_direct_shot_3D_surface<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].velocity_d,nnx,nny,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,sx_real,sy_real,sz_real,it,wavelet_length,dx,dy,dz,dt);
{

		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;

		int id,id1;

		int sx_real=sx+bl;
	
		int sy_real=sy+bb;

		int sz_real=(sz+bu)%nnz_device+radius;

		int rx=receiver_start_x+ix*receiver_interval_x;
		int ry=receiver_start_y+iy*receiver_interval_y;
		int rz=receiver_start_z;

		float distance;

		float d_x=(sx-rx)*dx;
		float d_y=(sy-ry)*dy;
		float d_z=(sz-rz)*dz;
	
		
		distance=sqrt(d_x*d_x*1.0+d_y*d_y*1.0+d_z*d_z*1.0);

		int time;

		if(ix<receiver_num_x&&iy<receiver_num_y)
		{
			id=iy*receiver_num_x+ix;

			id1=sz_real*(nnx*nny)+sy_real*nnx+sx_real;

			time=(distance/velocity_d[id1])*1000/dt;

			if(it<time+wavelet_length+50)	
			{
				obs_shot_x_d[id]=0.0;
				obs_shot_y_d[id]=0.0;
				obs_shot_z_d[id]=0.0;
			}
		}
}

__global__ void cut_direct_shot_3D_surface_three_new(float *obs_shot_x_d,float *obs_shot_y_d,float *obs_shot_z_d,float *velocity_d,int nnx,int nny,int nnz_device,int bl,int bb,int bu,int receiver_start_x,int receiver_num_x,int receiver_interval_x,int receiver_start_y,int receiver_num_y,int receiver_interval_y,int receiver_start_z,int receiver_num_z,int receiver_interval_z,int sx,int sy,int sz,int it,int wavelet_length,float dx,float dy,float dz,float dt,int cut_direct_wave)
//cut_direct_shot_3D_surface<<<dimGrid_rec_lt_x_y,dimBlock,0,mgdevice[choose_ns].stream>>>(mgdevice[choose_re].obs_shot_x_d,mgdevice[choose_re].velocity_d,nnx,nny,nnz_device,bl,bb,bu,receiver_start_x,receiver_num_x,receiver_interval_x,receiver_start_y,receiver_num_y,receiver_interval_y,receiver_start_z,receiver_num_z,receiver_interval_z,sx_real,sy_real,sz_real,it,wavelet_length,dx,dy,dz,dt);
{

		int ix=blockIdx.x*blockDim.x+threadIdx.x;
		int iy=blockIdx.y*blockDim.y+threadIdx.y;

		int id,id1;

		int sx_real=sx+bl;
	
		int sy_real=sy+bb;

		int sz_real=(sz+bu)%nnz_device+radius;

		int rx=receiver_start_x+ix*receiver_interval_x;
		int ry=receiver_start_y+iy*receiver_interval_y;
		int rz=receiver_start_z;

		float distance;

		float d_x=(sx-rx)*dx;
		float d_y=(sy-ry)*dy;
		float d_z=(sz-rz)*dz;
	
		
		distance=sqrt(d_x*d_x*1.0+d_y*d_y*1.0+d_z*d_z*1.0);

		int time;

		if(ix<receiver_num_x&&iy<receiver_num_y)
		{
			id=iy*receiver_num_x+ix;

			id1=sz_real*(nnx*nny)+sy_real*nnx+sx_real;

			time=(distance/velocity_d[id1])*1000/dt;

			if(it<time+cut_direct_wave)	
			{
				obs_shot_x_d[id]=0.0;
				obs_shot_y_d[id]=0.0;
				obs_shot_z_d[id]=0.0;
			}
		}
}
















/////////////////////////For Elastic RTM2018年01月17日 星期三 16时29分25秒
__global__ void cuda_cal_excitation_amp_time(float *ex_time_d,float *ex_tp_d,float *tp2_d,float *ex_vxp_d,float *vxp2_d,float *ex_vyp_d,float *vyp2_d,float *ex_vzp_d,float *vzp2_d,int nx_pml,int ny_pml,int nz_pml,int it)
//cuda_cal_excitation_amp_time<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].ex_vxp_d,mgdevice[i].vxp2_d,mgdevice[i].ex_vyp_d,mgdevice[i].vyp2_d,mgdevice[i].ex_vzp_d,mgdevice[i].vzp2_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

				if(fabs(tp2_d[idx])>fabs(ex_tp_d[idx]))
				{
					ex_time_d[idx]=it;

					ex_tp_d[idx]=tp2_d[idx];

					ex_vxp_d[idx]=vxp2_d[idx];

					ex_vyp_d[idx]=vyp2_d[idx];

					ex_vzp_d[idx]=vzp2_d[idx];
				}
		}
}


__global__ void cuda_cal_excitation_amp_time_new(float *ex_time_d,float *ex_amp_d,float *ex_tp_d,float *tp2_d,float *ex_vxp_d,float *vxp2_d,float *ex_vyp_d,float *vyp2_d,float *ex_vzp_d,float *vzp2_d,int nx_pml,int ny_pml,int nz_pml,int it)
//cuda_cal_excitation_amp_time_new<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_amp_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].ex_vxp_d,mgdevice[i].vxp2_d,mgdevice[i].ex_vyp_d,mgdevice[i].vyp2_d,mgdevice[i].ex_vzp_d,mgdevice[i].vzp2_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;
		float change;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;
			
				change=sqrt(vxp2_d[idx]*vxp2_d[idx]+vyp2_d[idx]*vyp2_d[idx]+vzp2_d[idx]*vzp2_d[idx]);

				if(ex_amp_d[idx]<change)
				{
					ex_time_d[idx]=it;
					
					ex_amp_d[idx]=change;

					ex_tp_d[idx]=tp2_d[idx];

					ex_vxp_d[idx]=vxp2_d[idx];

					ex_vyp_d[idx]=vyp2_d[idx];

					ex_vzp_d[idx]=vzp2_d[idx];
				}
		}
}

__global__ void cuda_cal_source_poyn_3D(float *poyn_px_d,float *poyn_py_d,float *poyn_pz_d,float *ex_time_d,float *vxp2_d,float *vyp2_d,float *vzp2_d,float *tp2_d,int nx_pml,int ny_pml,int nz_pml,int it)
//cuda_cal_source_poyn_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].poyn_px_d,mgdevice[i].poyn_py_d,mgdevice[i].poyn_pz_d,mgdevice[i].ex_time_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].tp2_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;
			
				if(ex_time_d[idx]==it)
				{
					poyn_px_d[idx]=-1.0*vxp2_d[idx]*tp2_d[idx];

					poyn_py_d[idx]=-1.0*vyp2_d[idx]*tp2_d[idx];

					poyn_pz_d[idx]=-1.0*vzp2_d[idx]*tp2_d[idx];
				}
		}
}
__global__ void cuda_cal_receiver_poyn_3D(float *poyn_px_d,float *poyn_py_d,float *poyn_pz_d,float *poyn_sx_d,float *poyn_sy_d,float *poyn_sz_d,float *ex_time_d,float *vxp2_d,float *vyp2_d,float *vzp2_d,float *vxs2_d,float *vys2_d,float *vzs2_d,float *tp2_d,float *txx2_d,float *tyy2_d,float *tzz2_d,float *txy2_d,float *txz2_d,float *tyz2_d,int nx_pml,int ny_pml,int nz_pml,int it)
//cuda_cal_receiver_poyn_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].poyn_rpx_d,mgdevice[i].poyn_rpy_d,mgdevice[i].poyn_rpz_d,mgdevice[i].poyn_rsx_d,mgdevice[i].poyn_rsy_d,mgdevice[i].poyn_rsz_d,mgdevice[i].ex_time_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].vxs2_d,mgdevice[i].vys2_d,mgdevice[i].vzs2_d,mgdevice[i].tp2_d,mgdevice[i].txx2_d,mgdevice[i].tyy2_d,mgdevice[i].tzz2_d,mgdevice[i].txy2_d,mgdevice[i].txz2_d,mgdevice[i].tyz2_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;
			
				if(ex_time_d[idx]==it)
				{
					poyn_px_d[idx]=-1.0*vxp2_d[idx]*tp2_d[idx];

					poyn_py_d[idx]=-1.0*vyp2_d[idx]*tp2_d[idx];

					poyn_pz_d[idx]=-1.0*vzp2_d[idx]*tp2_d[idx];


					poyn_sx_d[idx]=-1.0*((txx2_d[idx]-tp2_d[idx])*vxs2_d[idx]+txy2_d[idx]*vys2_d[idx]+txz2_d[idx]*vzs2_d[idx]);

					poyn_sy_d[idx]=-1.0*(txy2_d[idx]*vxs2_d[idx]+(tyy2_d[idx]-tp2_d[idx])*vys2_d[idx]+tyz2_d[idx]*vzs2_d[idx]);

					poyn_sz_d[idx]=-1.0*(txz2_d[idx]*vxs2_d[idx]+tyz2_d[idx]*vys2_d[idx]+(tzz2_d[idx]-tp2_d[idx])*vzs2_d[idx]);
				}
		}
}

__global__ void cuda_cal_angle_3D(float *angle_pp_d,float *angle_ps_d,float *poyn_px_d,float *poyn_py_d,float *poyn_pz_d,float *poyn_rpx_d,float *poyn_rpy_d,float *poyn_rpz_d,float *poyn_rsx_d,float *poyn_rsy_d,float *poyn_rsz_d,int nx_pml,int ny_pml,int nz_pml)
//cuda_cal_angle_3D<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].angle_pp_d,mgdevice[i].angle_ps_d,mgdevice[i].poyn_px_d,mgdevice[i].poyn_py_d,mgdevice[i].poyn_pz_d,mgdevice[i].poyn_rpx_d,mgdevice[i].poyn_rpy_d,mgdevice[i].poyn_rpz_d,mgdevice[i].poyn_rsx_d,mgdevice[i].poyn_rsy_d,mgdevice[i].poyn_rsz_d,nnx,nny,nnz_device_append);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		float magntiude1;
		float magntiude2;
		float magntiude3;

		float radian_pp;
		float radian_ps;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

				magntiude1=float(sqrt(1.0*poyn_px_d[idx]*poyn_px_d[idx]+1.0*poyn_py_d[idx]*poyn_py_d[idx]+1.0*poyn_py_d[idx]*poyn_py_d[idx]));
				
				magntiude2=float(sqrt(1.0*poyn_rpx_d[idx]*poyn_rpx_d[idx]+1.0*poyn_rpy_d[idx]*poyn_rpy_d[idx]+1.0*poyn_rpy_d[idx]*poyn_rpy_d[idx]));

				magntiude3=float(sqrt(1.0*poyn_rsx_d[idx]*poyn_rsx_d[idx]+1.0*poyn_rsy_d[idx]*poyn_rsy_d[idx]+1.0*poyn_rsy_d[idx]*poyn_rsy_d[idx]));
				
				if(magntiude1!=0&&magntiude2!=0)
						radian_pp=1.0*(poyn_px_d[idx]*poyn_rpx_d[idx]+poyn_py_d[idx]*poyn_rpy_d[idx]+poyn_pz_d[idx]*poyn_rpz_d[idx])/magntiude1/magntiude2*1.0;

				if(magntiude1!=0&&magntiude3!=0)
						radian_ps=1.0*(poyn_px_d[idx]*poyn_rsx_d[idx]+poyn_py_d[idx]*poyn_rsy_d[idx]+poyn_pz_d[idx]*poyn_rsz_d[idx])/magntiude1/magntiude3*1.0;

				angle_pp_d[idx]=acos(1.0*radian_pp);
				angle_ps_d[idx]=acos(1.0*radian_ps);
		}
}

__global__ void imaging_correlation_ex(float *ex_time_d,float *ex_tp_d,float *tp2_d,float *vresult_tp_d,int nx_pml,int ny_pml,int nz_pml,int it,float max,int precon_z1)
//imaging_correlation_ex<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_tp_d,mgdevice[i].tp2_d,mgdevice[i].vresult_tp_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

				if(it==ex_time_d[idx])
				{
					//vresult_tp_d[idx]=tp2_d[idx]*1.0/(ex_tp_d[idx]+0.00001*max);
					
					vresult_tp_d[idx]=tp2_d[idx]*1.0/(ex_tp_d[idx]);
				}
		}
}

__global__ void imaging_inner_product_ex(float *ex_time_d,float *ex_vxp_d,float *ex_vyp_d,float *ex_vzp_d,float *vxp2_d,float *vyp2_d,float *vzp2_d,float *vresult_pp_d,int nx_pml,int ny_pml,int nz_pml,int it,float max,int precon_z1)
//imaging_inner_product_ex<<<dimGridwf_append,dimBlock,0,mgdevice[i].stream>>>(mgdevice[i].ex_time_d,mgdevice[i].ex_vxp_d,mgdevice[i].ex_vyp_d,mgdevice[i].ex_vzp_d,mgdevice[i].vxp2_d,mgdevice[i].vyp2_d,mgdevice[i].vzp2_d,mgdevice[i].vresult_pp_d,nnx,nny,nnz_device_append,it);
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;

				if(it==ex_time_d[idx])
				{
					float molecular=ex_vxp_d[idx]*vxp2_d[idx]+ex_vyp_d[idx]*vyp2_d[idx]+ex_vzp_d[idx]*vzp2_d[idx];

					float denominator=ex_vxp_d[idx]*ex_vxp_d[idx]+ex_vyp_d[idx]*ex_vyp_d[idx]+ex_vzp_d[idx]*ex_vzp_d[idx];

					//vresult_pp_d[idx]=molecular*1.0/(denominator+0.00001*max*max);

					vresult_pp_d[idx]=molecular*1.0/(denominator);
				}
		}
}

__global__ void imaging_compensate_dependent_angle(float *vresult_pp_d,float *vresult_ps_d,float *angle_pp_d,float *angle_ps_d,int nx_pml,int ny_pml,int nz_pml)
{
		int tx=blockIdx.x*blockDim.x+threadIdx.x;
		int ty=blockIdx.y*blockDim.y+threadIdx.y;
		int tz=blockIdx.z;

		int idx;

		if((tx<nx_pml-2*radius)&&(ty<ny_pml-2*radius)&&(tz<nz_pml-2*radius))
		{
				tx=tx+radius;ty=ty+radius;tz=tz+radius;

				idx=tz*nx_pml*ny_pml+ty*nx_pml+tx;


				if(fabs(cos(1.0*angle_pp_d[idx]))>0.001)
				{
					vresult_pp_d[idx]=1.0*vresult_pp_d[idx]/cos(1.0*angle_pp_d[idx]);
				}

				else 	vresult_pp_d[idx]=1.0*vresult_pp_d[idx]/(cos(1.0*angle_pp_d[idx])+0.001);
		
				if(fabs(sin(1.0*angle_ps_d[idx]))<0.001)
				{
					vresult_ps_d[idx]=1.0*vresult_ps_d[idx]/sin(1.0*angle_ps_d[idx]);
				}

				else 	vresult_ps_d[idx]=1.0*vresult_ps_d[idx]/(sin(1.0*angle_ps_d[idx])+0.001);
		}
}

__global__ void cuda_cal_max(float *obs_max,float *obs_shot_x_d,int N)
////configuration requirement: <<<1, Block_Size>>> >*/ 
{
    	__shared__ float sdata[Block_Size];/* find max(|vv(:)|) */
	
    	int tid = threadIdx.x;
    	sdata[tid] = 0.0f;
    
	for(int s=0; s<(N+Block_Size-1)/Block_Size; s++)
	{
		int id=s*blockDim.x+threadIdx.x;
		float a=(id<N)?fabsf(obs_shot_x_d[id]):0.0f;
		//float b=(id<N)?fabsf(cg[id]):0.0f;
		sdata[tid]= max(sdata[tid], a);
		//tdata[tid]= max(tdata[tid], b);
	} 
    	__syncthreads();

    	/* do reduction in shared mem */
    	for(int s=blockDim.x/2; s>32; s>>=1) 
    	{
		if (threadIdx.x < s)	{sdata[tid]=max(sdata[tid], sdata[tid+s]);} 
		__syncthreads();
    	}  
   	if (tid < 32)
   	{
		if (blockDim.x >=  64) { sdata[tid] =max(sdata[tid],sdata[tid + 32]);}
		if (blockDim.x >=  32) { sdata[tid] =max(sdata[tid],sdata[tid + 16]);}
		if (blockDim.x >=  16) { sdata[tid] =max(sdata[tid],sdata[tid + 8]);}
		if (blockDim.x >=   8) { sdata[tid] =max(sdata[tid],sdata[tid + 4]);}
		if (blockDim.x >=   4) { sdata[tid] =max(sdata[tid],sdata[tid + 2]);}
		if (blockDim.x >=   2) { sdata[tid] =max(sdata[tid],sdata[tid + 1]);}
    	}

    	if (tid == 0) {*obs_max=sdata[0]; }
}
